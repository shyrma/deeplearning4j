/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by raver119 on 30.11.17.
//

#include <execution/LaunchContext.h>
#include <logger.h>
#include <exceptions/cuda_exception.h>
#include <helpers/cublasHelper.h>
#include <thread>
#include <execution/AffinityManager.h>

thread_local nd4j::ContextBuffers contextBuffers = nd4j::ContextBuffers();

namespace nd4j {

    std::vector<std::shared_ptr<LaunchContext>> LaunchContext::_contexts = std::vector<std::shared_ptr<LaunchContext>>();
    std::mutex LaunchContext::_mutex;

////////////////////////////////////////////////////////////////////////
LaunchContext::LaunchContext(hipStream_t *cudaStream, hipStream_t& specialCudaStream, void* reductionPointer, void* scalarPointer, int* allocationPointer)  {

	//_cudaStream 	   = cudaStream;
	//_cudaSpecialStream = &specialCudaStream; // ideal is = new hipStream_t; *_cudaSpecialStream = specialCudaStream;
	//_reductionPointer  = reductionPointer;
	//_scalarPointer     = scalarPointer;
	//_allocationPointer = allocationPointer;
	_workspace = nullptr;
	_isAllocated = false;
}

LaunchContext::~LaunchContext() {
    if (_isAllocated) {

    }
}

////////////////////////////////////////////////////////////////////////
LaunchContext::LaunchContext() {
            // default constructor, just to make clang/ranlib happy
    _workspace = nullptr;
    _deviceID = 0;

    _isAllocated = true;
}

    LaunchContext::LaunchContext(Nd4jPointer cudaStream, Nd4jPointer reductionPointer, Nd4jPointer scalarPointer, Nd4jPointer allocationPointer) {
        _isAllocated = false;
        //_cudaStream = reinterpret_cast<hipStream_t*>(cudaStream);
       // _cudaSpecialStream = reinterpret_cast<hipStream_t*>(cudaStream);
        //_reductionPointer = reductionPointer;
        //_scalarPointer = scalarPointer;
        //_allocationPointer = reinterpret_cast<int *>(allocationPointer);
    }

    LaunchContext* LaunchContext::defaultContext() {
        /**
        * This method returns LaunchContext, that has multiple entities within:
        * 1) temporary buffers. they must be per-thread
        * 2) CUDA stream. it must be either per-thread or per-device
        * 3) cuBLAS handle. it must be per-device
        */
        auto deviceId = AffinityManager::currentDeviceId();

        // we need this block synchronous, to avoid double initialization etc
        _mutex.lock();
        if (LaunchContext::_contexts.empty()) {
            // create one context per device
            auto numDevices = AffinityManager::numberOfDevices();

            _contexts.resize(numDevices);
            for (int e = 0; e < numDevices; e++) {
                AffinityManager::setCurrentNativeDevice(e);

                LaunchContext::_contexts[e] = std::make_shared<LaunchContext>();
            }

            // don't forget to restore device back again
            AffinityManager::setCurrentNativeDevice(deviceId);
        }
        _mutex.unlock();

        // return context for current device
        return LaunchContext::_contexts[deviceId].get();
    }


    void* LaunchContext::getReductionPointer () const {
        return contextBuffers.reductionBuffer();
    };

    void* LaunchContext::getScalarPointer() const {
        return contextBuffers.scalarBuffer();
    };

    int* LaunchContext::getAllocationPointer() const {
        return reinterpret_cast<int*>(contextBuffers.allocationBuffer());
    };

    void* LaunchContext::getCublasHandle() const {
        return CublasHelper::getInstance()->handle();
    };

    void* LaunchContext::getCusolverHandle() const {
        return CublasHelper::getInstance()->solver();
    };

    hipStream_t* LaunchContext::getCudaStream() const {
        return reinterpret_cast<hipStream_t*>(contextBuffers.execStream());
    };

    hipStream_t* LaunchContext::getCudaSpecialStream() const {
        return reinterpret_cast<hipStream_t*>(contextBuffers.specialStream());;
    };


    void LaunchContext::setReductionPointer (void* reductionPointer) {
        contextBuffers.setReductionBuffer(reductionPointer);
    };

    void LaunchContext::setScalarPointer(void* scalarPointer) {
        contextBuffers.setScalarBuffer(scalarPointer);
    };

    void LaunchContext::setAllocationPointer(int* allocationPointer) {
        contextBuffers.setAllocationBuffer(allocationPointer);
    };

    void LaunchContext::setCudaStream(hipStream_t* cudaStream)  {
        //_cudaStream = cudaStream;
    };

    void LaunchContext::setCudaSpecialStream(hipStream_t* cudaStream)  {
        //_cudaSpecialStream = cudaStream;
    };

    void LaunchContext::setCublasHandle(void *handle) {
        _cublasHandle = handle;
    };

    void LaunchContext::swapContextBuffers(ContextBuffers &buffers) {
        contextBuffers = buffers;
    };

    void LaunchContext::releaseBuffers() {
        //nd4j_printf("LaunchContext::releaseBuffers() was invoked\n", "");
        contextBuffers.release();
    }

    bool LaunchContext::isInitialized() {
        return contextBuffers.isInitialized();
    }

    sd::ErrorReference* LaunchContext::errorReference() {
        return contextBuffers.errorReference();
    }

    void* LaunchContext::engine() {
        return _engine;
    }
}