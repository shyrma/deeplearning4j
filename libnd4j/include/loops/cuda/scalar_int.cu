#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 08.11.2018
// @author raver119@gmail.com
//

#include "../scalar_int.h"
#include <op_boilerplate.h>
#include <types/types.h>

#include "../legacy_ops.h"

using namespace simdOps;

////////////////////////////////////////////////////////////////////////
template <typename X, typename OpType>
__global__ void scalarAlongDimension(void *x, Nd4jLong *xShapeInfo,
                                    void *extraParams,
                                    void *z, Nd4jLong *zShapeInfo,
                                    void *scalars,
                                    int *dimension, int dimensionLength,
                                    Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                                    Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {

    functions::scalar::ScalarIntTransform<X>::template transformCuda<OpType>(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ);
}


////////////////////////////////////////////////////////////////////////
template <typename X, typename OpType>
__global__ void scalarSimpleShaped(void* x, void *y, Nd4jLong *xShapeInfo, void *params, void *z, Nd4jLong *zShapeInfo, int *allocationBuffer) {

    functions::scalar::ScalarIntTransform<X>::template transformCuda<OpType>(y, x, xShapeInfo, params, z, zShapeInfo, allocationBuffer);
}





// *********************************************************************//
// *********************************************************************//
namespace functions {
namespace scalar    {

////////////////////////////////////////////////////////////////////////
template<typename X>
template<typename OpType>
__device__ void  ScalarIntTransform<X>::transformCuda(void* vscalar,
                                                        void *vy, Nd4jLong *yShapeInfo,
                                                        void *vparams,
                                                        void *vz, Nd4jLong *zShapeInfo,
                                                        int *allocationBuffer) {
    auto scalar = reinterpret_cast<X*>(vscalar)[0];
    auto y      = reinterpret_cast<X*>(vy);
    auto params = reinterpret_cast<X*>(vparams);
    auto z      = reinterpret_cast<X*>(vz);

    auto yRank   = shape::rank(yShapeInfo);
    auto yEWS    = shape::elementWiseStride(yShapeInfo);
    auto yShape  = shape::shapeOf(yShapeInfo);
    auto yStride = shape::stride(yShapeInfo);

    auto zRank   = shape::rank(zShapeInfo);
    auto zEWS    = shape::elementWiseStride(zShapeInfo);
    auto zShape  = shape::shapeOf(zShapeInfo);
    auto zStride = shape::stride(zShapeInfo);

    int totalThreads = gridDim.x * blockDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int len;
    if(threadIdx.x == 0)
        len = shape::length(yShapeInfo);
    __syncthreads();

    if(yEWS >= 1 && zEWS >= 1 && shape::order(yShapeInfo) == shape::order(zShapeInfo)) {
            transformCuda<OpType>(len, vscalar, vy, yEWS, vparams, vz, zEWS, allocationBuffer);
    }
    else {
        for (Nd4jLong i = tid; i < len; i+= totalThreads)
            z[shape::getIndexOffset(i, zShapeInfo)] = OpType::op(y[shape::getIndexOffset(i, yShapeInfo)], scalar, params);
    }
}

////////////////////////////////////////////////////////////////////////
template<typename X>
template<typename OpType>
__device__ void  ScalarIntTransform<X>::transformCuda(Nd4jLong len,
                                                          void* vx,
                                                          void *vy, Nd4jLong yEWS,
                                                          void *vparams,
                                                          void *vz, Nd4jLong zEWS,
                                                          int *allocationBuffer) {

    auto x = reinterpret_cast<X*>(vx)[0];
    auto y = reinterpret_cast<X*>(vy);
    auto z = reinterpret_cast<X*>(vz);
    auto params = reinterpret_cast<X*>(vparams);

    int totalThreads = gridDim.x * blockDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    Nd4jLong i = tid;
    if(yEWS == 1 && zEWS == 1) {
        for (; i < len; i += totalThreads)
            z[i] = OpType::op(y[i], x, params);
    }
    else {
        for (; i < len; i += totalThreads)
            z[i * zEWS] = OpType::op(y[i * yEWS], x, params);
    }
}


////////////////////////////////////////////////////////////////////////
template<typename X>
template<typename OpType>
__device__ void  ScalarIntTransform<X>::transformCuda(void *vx, Nd4jLong *xShapeInfo,
                                                        void *vextraParams,
                                                        void *vz, Nd4jLong *zShapeInfo,
                                                        void *vscalars,
                                                        int *dimension, int dimensionLength,
                                                        Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                                                        Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {
    auto x = reinterpret_cast<X*>(vx);
    auto scalars = reinterpret_cast<X*>(vscalars);
    auto z = reinterpret_cast<X*>(vz);
    auto extraParams = reinterpret_cast<X*>(vextraParams);

    if (tadShapeInfoZ == nullptr) {
        tadShapeInfoZ = tadShapeInfo;
        tadOffsetsZ = tadOffsets;
    }

    // tad preparation
    auto tadEws = shape::elementWiseStride(tadShapeInfo);
    auto zEws = shape::elementWiseStride(tadShapeInfoZ);
    auto tadLength = shape::length(tadShapeInfo);//shape::tadLength(xShapeInfo, dimension, dimensionLength);
    auto numTads =shape::length(xShapeInfo) / tadLength;

    if (tadEws > 0 && zEws > 0 && shape::order(tadShapeInfo) == shape::order(zShapeInfo)) {

        // main loop, rolling over tads
        for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
            X *oZ = z + tadOffsetsZ[r];
            X *oX = x + tadOffsets[r];

            auto s = scalars[r];

            for (int f = threadIdx.x; f < tadLength; f += blockDim.x)
                oZ[f * zEws] = OpType::op(oX[f * tadEws], s, extraParams);
        }
    } else {
        // main loop, rolling over tads
        for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
            X *oZ = z + tadOffsetsZ[r];
            X *oX = x + tadOffsets[r];

            auto s = scalars[r];

            for (int f = threadIdx.x; f < tadLength; f += blockDim.x)
                oZ[shape::getIndexOffset(f, tadShapeInfoZ)] = OpType::op(oX[shape::getIndexOffset(f, tadShapeInfo)], s, extraParams);
        }
    }
}


////////////////////////////////////////////////////////////////////////
template<typename X>
template <typename OpType>
_CUDA_H void ScalarIntTransform<X>::intermediateAlongDimension(dim3& launchDims, hipStream_t *stream,
                                                                void *x, Nd4jLong *xShapeInfo,
                                                                void *z, Nd4jLong *zShapeInfo,
                                                                void *scalars,
                                                                void *extraParams,
                                                                int *dimension, int dimensionLength,
                                                                Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets,
                                                                Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {

    scalarAlongDimension<X, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ);
}

////////////////////////////////////////////////////////////////////////
template<typename X>
template<typename OpType>
void _CUDA_H ScalarIntTransform<X>::intermediateShaped(dim3& launchDims, hipStream_t *stream,
                                                            void *vx, Nd4jLong *xShapeInfo,
                                                            void *vz, Nd4jLong *zShapeInfo,
                                                            void* vscalar,
                                                            void *vextraParams, int *allocPointer){

    scalarSimpleShaped<X, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, vscalar, xShapeInfo, vextraParams, vz, zShapeInfo, allocPointer);
}

////////////////////////////////////////////////////////////////////////
template<typename X>
void ScalarIntTransform<X>::executeCudaShaped(dim3& launchDims, hipStream_t *stream,
                                                int opNum,
                                                void *vx, Nd4jLong *xShapeInfo,
                                                void *vz, Nd4jLong *zShapeInfo,
                                                void* vscalar,
                                                void *vextraParams) {

    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
        printf("H14 opNum:[%i]\n", opNum);

    DISPATCH_BY_OPNUM_T(intermediateShaped, PARAMS(launchDims, stream, vx, xShapeInfo, vz, zShapeInfo, vscalar, vextraParams, nullptr), SCALAR_INT_OPS);
}

////////////////////////////////////////////////////////////////////////
template<typename X>
void ScalarIntTransform<X>::executeCudaAlongDimension(dim3& launchDims, hipStream_t *stream, int opNum, void *vx, Nd4jLong *xShapeInfo, void *vz, Nd4jLong *zShapeInfo, void *vscalars, void *vextraParams, int *dimension, int dimensionLength, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {
    DISPATCH_BY_OPNUM_T(intermediateAlongDimension, PARAMS(launchDims, stream, vx, xShapeInfo, vz, zShapeInfo, vscalars, vextraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), SCALAR_INT_OPS);
}

    BUILD_SINGLE_TEMPLATE(template class ND4J_EXPORT ScalarIntTransform, , INTEGER_TYPES);


    template<typename X>
    template <typename OpType>
    void ScalarIntTransform<X,>::transform(void *x, Nd4jLong *xShapeInfo, void *extraParams, void *z, Nd4jLong *zShapeInfo, void *scalars, int *dimension, int dimensionLength, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {

    }

    template<typename X>
    void ScalarIntTransform<X>::transform(int opNum, void *x, Nd4jLong *xShapeInfo, void *extraParams, void *z, Nd4jLong *zShapeInfo, void *scalars, int *dimension, int dimensionLength, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ) {

    }

    template<typename X>
    void ScalarIntTransform<X>::transform(const int opNum, void *x, Nd4jLong *xShapeInfo, void *result, Nd4jLong *resultShapeInfo,  void *scalar,  void *extraParams) {

    }

    template<typename X>
    void ScalarIntTransform<X>::transform(const int opNum, void *x, Nd4jLong xStride, void *result, Nd4jLong resultStride, void *scalar, void *extraParams, const Nd4jLong n) {

    }

    template<typename X>
    template<typename OpType>
    void ScalarIntTransform<X>::transform(void *x, Nd4jLong *xShapeInfo, void *result, Nd4jLong *resultShapeInfo, void *scalar, void *extraParams) {

    }


    template<typename X>
    template<typename OpType>
    void ScalarIntTransform<X>::transform(void *x, Nd4jLong xStride, void *result, Nd4jLong resultStride, void *scalar, void *extraParams, const Nd4jLong n) {

    }
}
}

