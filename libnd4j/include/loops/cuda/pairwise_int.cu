#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//  @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com), created on 08.11.2018

#ifndef PAIRWISE_INT_CU
#define PAIRWISE_INT_CU


#include "../pairwise_int.h"


using namespace simdOps;

////////////////////////////////////////////////////////////////////////////////
template <typename X, typename OpType>
__global__ static void pairwiseSimpleShaped(void* vx, Nd4jLong *xShapeInfo,
											void *vy, Nd4jLong *yShapeInfo,
											void *vz, Nd4jLong *zShapeInfo,
											void *vextraParams) {

	auto x = reinterpret_cast<X*>(vx);
	auto y = reinterpret_cast<X*>(vy);
	auto z = reinterpret_cast<X*>(vz);
	auto extraParams = reinterpret_cast<X*>(vextraParams);

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int xEws;
	__shared__ int yEws;
	__shared__ int zEws;
	__shared__ char xOrder;
	__shared__ char yOrder;
	__shared__ char zOrder;
	__shared__ Nd4jLong len;

	if (threadIdx.x == 0) {
		xEws = shape::elementWiseStride(xShapeInfo);
		yEws = shape::elementWiseStride(yShapeInfo);
    	zEws = shape::elementWiseStride(zShapeInfo);
		xOrder = shape::order(xShapeInfo);
		yOrder = shape::order(yShapeInfo);
		zOrder = shape::order(zShapeInfo);
		len = shape::length(xShapeInfo);
	}
	__syncthreads();


	if (xEws >= 1 && yEws >= 1 && zEws >= 1 && xOrder == yOrder && xOrder == zOrder) {
		for (Nd4jLong i = tid; i < len; i += gridDim.x * blockDim.x) {
			z[i * zEws] = OpType::op(x[i * xEws], y[i * yEws], extraParams);
		}
	}
	else if (vx == vz) {
		for (Nd4jLong i = tid; i < len; i += gridDim.x * blockDim.x) {
			auto xOffset = shape::getIndexOffset(i, xShapeInfo);
			auto yOffset = shape::getIndexOffset(i, yShapeInfo);

			z[xOffset] = OpType::op(x[xOffset], y[yOffset], extraParams);
		}
	}
	else {
		for (Nd4jLong i = tid; i < len; i += gridDim.x * blockDim.x) {
			auto xOffset = shape::getIndexOffset(i, xShapeInfo);
			auto yOffset = shape::getIndexOffset(i, yShapeInfo);
			auto zOffset = shape::getIndexOffset(i, zShapeInfo);

			z[zOffset] = OpType::op(x[xOffset], y[yOffset], extraParams);
		}
	}
}


namespace functions           {
namespace pairwise_transforms {

////////////////////////////////////////////////////////////////////////////////
template<typename X>
template<typename OpType>
void _CUDA_H PairWiseIntTransform<X>::intermediateShaped(dim3& launchDims, hipStream_t *stream,
														void *vx, Nd4jLong *xShapeInfo,
														void *vy, Nd4jLong *yShapeInfo,
														void *vz, Nd4jLong *zShapeInfo,
														void *vextraParams){

	pairwiseSimpleShaped<X, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, vextraParams);
}


////////////////////////////////////////////////////////////////////////////////
template<typename X>
void PairWiseIntTransform<X>::executeCudaShaped(dim3& launchDims, hipStream_t *stream, int opNum, void *vx, Nd4jLong *xShapeInfo, void *vy, Nd4jLong *yShapeInfo, void *vz, Nd4jLong *zShapeInfo, void *vextraParams) {
    auto xType = nd4j::DataTypeUtils::fromT<X>();

	DISPATCH_BY_OPNUM_T(intermediateShaped, PARAMS(launchDims, stream, vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, vextraParams), PAIRWISE_INT_OPS);
}


    template<typename X>
    void PairWiseIntTransform<X>::exec(
            const int opNum,
            void *dx,
            Nd4jLong *xShapeBuffer,
            void *y,
            Nd4jLong *yShapeBuffer,
            void *result,
            Nd4jLong *resultShapeBuffer,
            void *extraParams) {

    }

    template<typename X>
    void PairWiseIntTransform<X>::exec(
            const int opNum,
            void *dx,
            Nd4jLong xStride,
            void *y,
            Nd4jLong yStride,
            void *result,
            Nd4jLong resultStride,
            void *extraParams,
            Nd4jLong n) {

    }


    template<typename X>
    template<typename OpType>
    void PairWiseIntTransform<X>::exec(
            void *vx,
            Nd4jLong* xShapeBuffer,
            void *vy,
            Nd4jLong* yShapeBuffer,
            void *vresult,
            Nd4jLong* resultShapeBuffer,
            void *vextraParams) {

    }

    template<typename X>
    template<typename OpType>
    void PairWiseIntTransform<X>::exec(void *vx,
                     Nd4jLong xStride,
                     void *vy,
                     Nd4jLong yStride,
                     void *vresult,
                     Nd4jLong resultStride,
                     void *vextraParams,
                     const Nd4jLong n) {

    }



    BUILD_SINGLE_TEMPLATE(template class ND4J_EXPORT PairWiseIntTransform, , INTEGER_TYPES);
}
}

#endif // PAIRWISE_INT_CU