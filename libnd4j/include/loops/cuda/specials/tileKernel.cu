#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author GS <sgazeos@gmail.com>, created on 16.01.2019
//

#include <loops/special_kernels.h>

namespace nd4j {
    static Nd4jLong __device__ __noinline__ _getIndexOffset(Nd4jLong index, Nd4jLong *shapeInfo) {
        return shape::getIndexOffset(index, shapeInfo);
    }

    static Nd4jLong __device__ __noinline__ _subArrayOffset(Nd4jLong index, Nd4jLong *shapeInfoA, Nd4jLong *shapeInfoB) {
        return shape::subArrayOffset(index, shapeInfoA, shapeInfoB);
    }

    static Nd4jLong __device__ __noinline__ _length(Nd4jLong *shapeInfo) {
        return shape::length(shapeInfo);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//  tileKernel:
//  input: (inputBuffer and inputShape) - NDArray buffer and shape to tile
//  output: (outputBuffer and outputShape) - NDArray to tile input
//  resultLength - length for output array
    template<typename T>
    static __global__ void
    tileKernel(void const *inputBuffer, Nd4jLong *inputShape, void *outputBuffer, Nd4jLong *outputShape,
               Nd4jLong resultLength) {
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//        Original code to transform in cuda-based
        auto tid = blockIdx.x * blockDim.x + threadIdx.x; // copy linear sequence of elements, so one-level threading
        int totalThreads = gridDim.x * blockDim.x;
        if (shape::order(outputShape) == 'c') {           //  ews == 1 always here
            for (int i = tid; i < resultLength; i += totalThreads) {
                auto yOffset = _subArrayOffset(i, outputShape, inputShape);
                *(reinterpret_cast<T *>(outputBuffer) + i) = *(reinterpret_cast<T const *>(inputBuffer) + yOffset);
            }
        } else {
            for (int i = tid; i < resultLength; i += totalThreads) {
                auto xOffset = _getIndexOffset(i, outputShape);
                auto yOffset = _subArrayOffset(i, outputShape, inputShape);
                *(reinterpret_cast<T *>(outputBuffer) + xOffset) = *(reinterpret_cast<T const *>(inputBuffer) + yOffset);
            }
        }

    }

    BUILD_SINGLE_TEMPLATE(template __global__ void tileKernel,(void const* inputBuffer, Nd4jLong* inputShape, void* outputBuffer, Nd4jLong* outputShape, Nd4jLong resultLength), LIBND4J_TYPES);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template<typename T>
    void tileKernelH(void const *inputBuffer, Nd4jLong *inputShape, void *outputBuffer, Nd4jLong *outputShape, Nd4jLong resultLength, hipStream_t *stream) {
        dim3 launchDims(256, 512, 8192);
        tileKernel<T> << < launchDims.x, launchDims.y, launchDims.z, *stream>>>(inputBuffer, inputShape, outputBuffer, outputShape, resultLength);
    }

    BUILD_SINGLE_TEMPLATE(template void tileKernelH, (void const* inputBuffer, Nd4jLong* inputShape, void* outputBuffer, Nd4jLong* outputShape, Nd4jLong resultLength, hipStream_t *stream), LIBND4J_TYPES);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// enhancement for tileKernel to different input and output data types: X - output type, Y - input type
    template<typename X, typename Y>
    static __global__ void
    tileKernelDouble(void const *inputBuffer, Nd4jLong *inputShape, void *outputBuffer, Nd4jLong *outputShape, Nd4jLong resultLength, Nd4jLong ews) {
        char ordering = shape::order(outputShape);
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        int totalThreads = gridDim.x * blockDim.x;

        if (ordering == 'c' && ews == 1) {           //  ews == 1 always here
            for (int i = tid; i < resultLength; i += totalThreads) {
                auto yOffset = _subArrayOffset(i, outputShape, inputShape);
                *(reinterpret_cast<X *>(outputBuffer) + i) = static_cast<X>(*(reinterpret_cast<Y const *>(inputBuffer) + yOffset));
            }
        } else if (ordering == 'c' && ews > 1) {
            for (int i = tid; i < resultLength; i += totalThreads) {
                auto yOffset = _subArrayOffset(i, outputShape, inputShape);
                *(reinterpret_cast<X *>(outputBuffer) + i * ews) = static_cast<X>(*(reinterpret_cast<Y const *>(inputBuffer) + yOffset));
            }
        } else {

            for (int i = tid; i < resultLength; i += totalThreads) {

                auto xOffset = _getIndexOffset(i, outputShape);
                auto yOffset = _subArrayOffset(i, outputShape, inputShape);
                *(reinterpret_cast<X *>(outputBuffer) + xOffset) = static_cast<X>(*(reinterpret_cast<Y const *>(inputBuffer) + yOffset));
            }
        }
    }

    BUILD_SINGLE_TEMPLATE_TWICE(template __global__ void tileKernelDouble, (void const* inputBuffer, Nd4jLong* inputShape, void* outputBuffer, Nd4jLong* outputShape, Nd4jLong resultLength, Nd4jLong ews), LIBND4J_TYPES);

    template<typename X, typename Y>
    void tileKernelHH(void const *inputBuffer, Nd4jLong *inputShape, void *outputBuffer, Nd4jLong *outputShape, Nd4jLong resultLength, Nd4jLong ews, hipStream_t *stream) {
        dim3 launchDims(256, 512, 8192);
        tileKernelDouble<X, Y><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(inputBuffer, inputShape, outputBuffer, outputShape, resultLength, ews);
    }

    BUILD_SINGLE_TEMPLATE_TWICE(template void tileKernelHH, (void const* inputBuffer, Nd4jLong* inputShape, void* outputBuffer, Nd4jLong* outputShape, Nd4jLong resultLength, Nd4jLong ews, hipStream_t *stream),LIBND4J_TYPES);
}