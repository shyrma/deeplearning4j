#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 28.11.2018
//

#include <ops/specials_cuda.h>

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
__global__ void bitonicArbitraryStepKernelKey(void *vx, Nd4jLong *xShapeInfo, void *vy, Nd4jLong *yShapeInfo, int window, int length,  int reverse, bool descending) {
    auto x = static_cast<X*>(vx);
    auto y = static_cast<Y*>(vy);

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int half = window>>1;

    __shared__ Nd4jLong xLength;
    if (threadIdx.x == 0) {
        xLength = shape::length(xShapeInfo);
    }
    __syncthreads();

    //for (int i = 0; i < length; i+= window)
    /*
        if window == 4;
        iterations will be: 0; 4; 8; 12; 16; 20
        if gridDim = 3;
        on first iteration we'll have: 0; 4; 8;
        on second iteration we'll have: 0 + (3 * 4) = 12;  4 + (3 * 4) = 16; 8 + (3 * 4) = 20
    */
    int firstPosition;
    int firstStep;
    int secondPosition;
    int secondStep;

    int WARP_SIZE = 32;
    int numWarps = (gridDim.x * blockDim.x) / 32;
    int warpId = tid / WARP_SIZE;
    int warpIdx = tid % WARP_SIZE;

    if (half >= 128) {
        firstPosition = blockIdx.x * window;
        firstStep = gridDim.x * window;

        secondPosition = threadIdx.x;
        secondStep = blockDim.x;
    } else if (half >= 32) {
        firstPosition = warpId * window;
        firstStep = numWarps * window;

        secondPosition = warpIdx;
        secondStep = WARP_SIZE;
    } else {
        firstPosition = tid * window;
        firstStep = blockDim.x * gridDim.x * window;

        secondPosition = 0;
        secondStep = 1;
    }


    for (int i = firstPosition; i < length; i += firstStep) {
        for (int j = secondPosition; j < half; j += secondStep) {
            int it = (reverse) ? i + j + half : i + window - j - 1;
            int ij = i+j;
            if (it < length && ij < length ) {
                int posIT = shape::getIndexOffset(it, xShapeInfo);
                int posIJ = shape::getIndexOffset(ij, xShapeInfo);

                X v0 = x[posIJ];
                X v1 = x[posIT];

                if(!descending == (v0 > v1)) {
                    x[posIJ] = v1;
                    x[posIT] = v0;

                    Y ytemp = y[posIJ];
                    y[posIJ] = y[posIT];
                    y[posIT] = ytemp;
                }
            }
        }
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ void execBitonicArbitraryStepKernel(void *vx, Nd4jLong *xShapeInfo, int window, int length,  int reverse, bool descending) {
    auto x = static_cast<T*>(vx);

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int half = window>>1;

    __shared__ T *shmem;
    __shared__ Nd4jLong xLength;
    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shrd[];
        shmem = (T *) shrd;
        xLength = shape::length(xShapeInfo);
    }
    __syncthreads();

    //for (int i = 0; i < length; i+= window)
    /*
        if window == 4;
        iterations will be: 0; 4; 8; 12; 16; 20
        if gridDim = 3;
        on first iteration we'll have: 0; 4; 8;
        on second iteration we'll have: 0 + (3 * 4) = 12;  4 + (3 * 4) = 16; 8 + (3 * 4) = 20
    */
    int firstPosition;
    int firstStep;
    int secondPosition;
    int secondStep;

    int WARP_SIZE = 32;
    int numWarps = (gridDim.x * blockDim.x) / 32;
    int warpId = tid / WARP_SIZE;
    int warpIdx = tid % WARP_SIZE;

    if (half >= 128) {
        firstPosition = blockIdx.x * window;
        firstStep = gridDim.x * window;

        secondPosition = threadIdx.x;
        secondStep = blockDim.x;
    } else if (half >= 32) {
        firstPosition = warpId * window;
        firstStep = numWarps * window;

        secondPosition = warpIdx;
        secondStep = WARP_SIZE;
    } else {
        firstPosition = tid * window;
        firstStep = blockDim.x * gridDim.x * window;

        secondPosition = 0;
        secondStep = 1;
    }


    for (int i = firstPosition; i < length; i += firstStep) {
        for (int j = secondPosition; j < half; j += secondStep) {
            int it = (reverse) ? i + j + half : i + window - j - 1;
            int ij = i+j;
            if (it < length && ij < length ) {
                int posIT = shape::getIndexOffset(it, xShapeInfo);
                int posIJ = shape::getIndexOffset(ij, xShapeInfo);

                shmem[threadIdx.x] = x[posIJ];
                shmem[threadIdx.x + blockDim.x] = x[posIT];

                if(!descending == (shmem[threadIdx.x] > shmem[threadIdx.x + blockDim.x])) {
                    x[posIJ] = shmem[threadIdx.x + blockDim.x];
                    x[posIT] = shmem[threadIdx.x];
                }
            }
        }
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__host__ void bitonicArbitraryStepGeneric(dim3 &launchDims, hipStream_t *stream, void *vx, Nd4jLong *xShapeInfo, int window, int length,  int reverse, bool descending) {
    execBitonicArbitraryStepKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xShapeInfo, window, length, reverse, descending);
}

template <typename X, typename Y>
__host__ void bitonicArbitraryStepGenericKey(dim3 &launchDims, hipStream_t *stream, void *vx, Nd4jLong *xShapeInfo, void *vy, Nd4jLong *yShapeInfo, int window, int length,  int reverse, bool descending) {
    bitonicArbitraryStepKernelKey<X,Y><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, window, length, reverse, descending);
}

BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT bitonicArbitraryStepGeneric, (dim3 &launchDims, hipStream_t *stream, void *vx, Nd4jLong *xShapeInfo, int window, int length,  int reverse, bool descending), LIBND4J_TYPES);
BUILD_DOUBLE_TEMPLATE(template void ND4J_EXPORT bitonicArbitraryStepGenericKey, (dim3 &launchDims, hipStream_t *stream, void *vx, Nd4jLong *xShapeInfo, void *vy, Nd4jLong *yShapeInfo, int window, int length,  int reverse, bool descending), LIBND4J_TYPES, LIBND4J_TYPES);
