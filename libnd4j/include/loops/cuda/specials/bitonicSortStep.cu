#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 28.11.2018
//

#include <ops/specials_cuda.h>


//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
__global__ void bitonicSortStepKernelKey(void *vx, Nd4jLong *xShapeInfo, void *vy, Nd4jLong *yShapeInfo, int j, int k, int length, bool descending) {

    auto x = static_cast<X*>(vx);
    auto y = static_cast<Y*>(vy);

    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;

    __shared__ Nd4jLong xLength;
    if (threadIdx.x == 0)
        xLength = shape::length(xShapeInfo);

    __syncthreads();


    if (i >= length)
        return;

    ixj = i^j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj)>i) {
        int posI = shape::getIndexOffset(i, xShapeInfo);
        int posIXJ = shape::getIndexOffset(ixj, xShapeInfo);

        if ((i&k)==0) {
            /* Sort ascending */
            if (!descending == (x[posI]>x[posIXJ])) {
                /* exchange(i,ixj); */
                X temp = x[posI];
                x[posI] = x[posIXJ];
                x[posIXJ] = temp;

                Y ytemp = y[posI];
                y[posI] = y[posIXJ];
                y[posIXJ] = ytemp;
            }
        } else if ((i&k)!=0) {
            /* Sort descending */
            if (!descending == (x[posI]<x[posIXJ])) {
                /* exchange(i,ixj); */
                X temp = x[posI];
                x[posI] = x[posIXJ];
                x[posIXJ] = temp;

                Y ytemp = y[posI];
                y[posI] = y[posIXJ];
                y[posIXJ] = ytemp;
            }
        }
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ void bitonicSortStepKernel(void *vx, Nd4jLong *xShapeInfo, int j, int k, int length, bool descending) {

    auto x = static_cast<T*>(vx);

    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;

    __shared__ Nd4jLong xLength;
    if (threadIdx.x == 0)
        xLength = shape::length(xShapeInfo);

    __syncthreads();


    if (i >= length)
        return;

    ixj = i^j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj)>i) {
        int posI = shape::getIndexOffset(i, xShapeInfo);
        int posIXJ = shape::getIndexOffset(ixj, xShapeInfo);

        if ((i&k)==0) {
            /* Sort ascending */
            if (!descending == (x[posI]>x[posIXJ])) {
                /* exchange(i,ixj); */
                T temp = x[posI];
                x[posI] = x[posIXJ];
                x[posIXJ] = temp;
            }
        } else if ((i&k)!=0) {
            /* Sort descending */
            if (!descending == (x[posI]<x[posIXJ])) {
                /* exchange(i,ixj); */
                T temp = x[posI];
                x[posI] = x[posIXJ];
                x[posIXJ] = temp;
            }
        }
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__host__ void bitonicSortStepGeneric(dim3 &launchDims, hipStream_t *stream, void *vx, Nd4jLong *xShapeInfo, int j, int k, int length, bool descending) {
    bitonicSortStepKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xShapeInfo, j, k, length, descending);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
__host__ void bitonicSortStepGenericKey(dim3 &launchDims, hipStream_t *stream, void *vx, Nd4jLong *xShapeInfo, void *vy, Nd4jLong *yShapeInfo, int j, int k, int length, bool descending) {
    bitonicSortStepKernelKey<X,Y><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, j, k, length, descending);
}


BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT bitonicSortStepGeneric, (dim3 &launchDims, hipStream_t *stream, void *vx, Nd4jLong *xShapeInfo, int j, int k, int length, bool descending), LIBND4J_TYPES);
BUILD_DOUBLE_TEMPLATE(template void ND4J_EXPORT bitonicSortStepGenericKey, (dim3 &launchDims, hipStream_t *stream, void *vx, Nd4jLong *xShapeInfo, void *vy, Nd4jLong *yShapeInfo, int j, int k, int length, bool descending), LIBND4J_TYPES, LIBND4J_TYPES);
