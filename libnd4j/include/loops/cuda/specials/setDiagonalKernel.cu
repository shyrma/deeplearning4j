#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author GS <sgazeos@gmail.com>, created on 21.01.2019
//

#include <loops/special_kernels.h>
#include <NDArray.h>
namespace nd4j {

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// set up given value to upper diagonal given
// buffer - input buffer
// shape - input shape
// value - given value
// diagonal - given upper diagonal (acceptable negative values also, 0 - the main diagonal)
// row, cols - height and width of given matrix (MxN, rows = M, cols = N)
//
    template <typename T>
    static __global__ void setDiagValueUpperKernel(void* buffer, Nd4jLong* shape, T value, int diagonal, Nd4jLong rows,
            Nd4jLong cols) {

        __shared__ Nd4jLong  rank;
        __shared__ T* array;

        if (0 == threadIdx.x) {
            rank = shape::rank(shape);
            array = reinterpret_cast<T *>(buffer);
        }
        __syncthreads();

        for (Nd4jLong i = blockIdx.x; i < rows; i += gridDim.x) {
            for (int j = threadIdx.x; j < cols; j += blockDim.x) {
                Nd4jLong coords[2] = {i, j};
                Nd4jLong xOffset = shape::getOffset(shape, coords);
                if (i + diagonal <= j)
                    array[xOffset] = value;
            }
        }
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// set up given value to lower given diagonal
// buffer - input buffer
// shape - input shape
// value - given value
// diagonal - given lower diagonal (acceptable negative values also, 0 - the main diagonal)
// row, cols - height and width of given matrix (MxN, rows = M, cols = N)
//

    template <typename T>
    static __global__ void setDiagValueLowerKernel(void* buffer, Nd4jLong* shape, T value, int diagonal, Nd4jLong rows, Nd4jLong cols) {
        Nd4jLong  rank = shape::rank(shape);
        int totalThreads = blockDim.x;
        for (Nd4jLong i = blockIdx.x; i < rows; i += gridDim.x) {
            for (int j = threadIdx.x; j < cols; j += totalThreads) {
                Nd4jLong coords[2] = {i, j};
                auto xOffset = shape::getOffset(shape, coords);
                if (i + diagonal >= j)
                    *(reinterpret_cast<T*>(buffer) + xOffset) = value;
            }
        }
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template __global__ void setDiagValueLowerKernel(void* buffer, Nd4jLong* shape, double value,   int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueUpperKernel(void* buffer, Nd4jLong* shape, double value,   int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueLowerKernel(void* buffer, Nd4jLong* shape, float value,    int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueUpperKernel(void* buffer, Nd4jLong* shape, float value,    int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueLowerKernel(void* buffer, Nd4jLong* shape, int value,      int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueUpperKernel(void* buffer, Nd4jLong* shape, int value,      int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueLowerKernel(void* buffer, Nd4jLong* shape, float16 value,  int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueUpperKernel(void* buffer, Nd4jLong* shape, float16 value,  int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueLowerKernel(void* buffer, Nd4jLong* shape, bfloat16 value, int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueUpperKernel(void* buffer, Nd4jLong* shape, bfloat16 value, int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueLowerKernel(void* buffer, Nd4jLong* shape, Nd4jLong value, int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueUpperKernel(void* buffer, Nd4jLong* shape, Nd4jLong value, int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueLowerKernel(void* buffer, Nd4jLong* shape, int16_t value,  int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueUpperKernel(void* buffer, Nd4jLong* shape, int16_t value,  int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueLowerKernel(void* buffer, Nd4jLong* shape, uint8_t value,  int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueUpperKernel(void* buffer, Nd4jLong* shape, uint8_t value,  int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueLowerKernel(void* buffer, Nd4jLong* shape, int8_t value,   int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueUpperKernel(void* buffer, Nd4jLong* shape, int8_t value,   int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueLowerKernel(void* buffer, Nd4jLong* shape, bool value,     int diagonal, Nd4jLong rows, Nd4jLong cols);
    template __global__ void setDiagValueUpperKernel(void* buffer, Nd4jLong* shape, bool value,     int diagonal, Nd4jLong rows, Nd4jLong cols);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    static void setDiagonalValueUpper(void* buffer, Nd4jLong* shape, NDArray const& value, int diagonal, Nd4jLong rows, Nd4jLong cols, hipStream_t& stream) {
        dim3 launchDims(256, 512, 8192);
        setDiagValueUpperKernel<T><<<launchDims.x, launchDims.y, launchDims.z, stream>>>(buffer, shape, value.e<T>(0), diagonal, rows, cols);
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    template <typename T>
    static void setDiagonalValueLower(void* buffer, Nd4jLong* shape, NDArray const& value, int diagonal, Nd4jLong rows, Nd4jLong cols, hipStream_t& stream) {
        dim3 launchDims(256, 512, 8192);
        setDiagValueLowerKernel<T><<<launchDims.x, launchDims.y, launchDims.z, stream>>>(buffer, shape, value.e<T>(0), diagonal, rows, cols);
    }

    BUILD_SINGLE_TEMPLATE(template void setDiagonalValueUpper, (void* buffer, Nd4jLong* shape, NDArray const& value,
            int diagonal, Nd4jLong rows, Nd4jLong cols, hipStream_t& stream), LIBND4J_TYPES);
    BUILD_SINGLE_TEMPLATE(template void setDiagonalValueLower, (void* buffer, Nd4jLong* shape, NDArray const& value,
            int diagonal, Nd4jLong rows, Nd4jLong cols, hipStream_t& stream), LIBND4J_TYPES);
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

}