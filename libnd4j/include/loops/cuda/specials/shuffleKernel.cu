#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//

#include <loops/special_kernels.h>

namespace nd4j {

////////////////////////////////////////////////////////////////////////
    template<typename T>
    __global__ void execShuffleKernel(void **vdX, Nd4jLong **dxShapeInfo,
                                      void **vdZ,
                                      int N,
                                      int *shuffleMap,
                                      Nd4jLong **tadOnlyShapeInfo, Nd4jLong **tadOffsets) {

        // we assume that shuffle map for each X contains pair TAD Y
        auto dX = reinterpret_cast<T **>(vdX);
        auto dZ = reinterpret_cast<T **>(vdZ);

        __shared__ int tadLength;
        __shared__ int xRank;
        __shared__ int tadEWS;
        __shared__ int numTads;
        __shared__ Nd4jLong* xShapeInfo;
        __shared__ Nd4jLong xLength;

        for (int f = 0; f < N; f++) {
            auto x = reinterpret_cast<T *>(dX[f]);
            auto z = reinterpret_cast<T *>(dZ[f]);

            if (threadIdx.x == 0) {
                tadLength = shape::length(tadOnlyShapeInfo[f]);
                tadEWS = shape::elementWiseStride(tadOnlyShapeInfo[f]);
                xShapeInfo = dxShapeInfo[f];
                xRank = shape::rank(xShapeInfo);
                xLength = shape::length(xShapeInfo);
                numTads = xLength / tadLength;
            }
            __syncthreads();

            if (xRank == 1) {
                int tid = threadIdx.x + blockIdx.x * blockDim.x;
                for (int r = tid; r < xLength; r += gridDim.x * blockDim.x) {
                    auto swapIndex = shuffleMap[r];
                    if (swapIndex >= 0 && swapIndex < xLength) {
                        int idx = r * tadEWS;
                        int swap = swapIndex * tadEWS;
                        T oldX = x[idx];
                        x[idx] = x[swap];
                        x[swap] = oldX;
                    }
                }
            } else {
                // we roll over the pairs of TADs, thus limit is numTads / 2
                for (uint r = blockIdx.x; r < numTads; r += gridDim.x) {
                    if (shuffleMap[r] >= 0) {
                        auto oldOffset = tadOffsets[f][r];
                        auto newOffset = tadOffsets[f][shuffleMap[r]];

                        auto rX = x + oldOffset;
                        auto rY = x + newOffset;

                        auto zX = z + oldOffset;
                        auto zY = z + newOffset;

                        // so we're going to change TAD[oldOffset] with TAD[newOffset]
                        if (tadEWS == 1) {
                            for (Nd4jLong i = threadIdx.x; i < tadLength; i += blockDim.x) {
                                T oldX = rX[i];
                                rX[i] = rY[i];
                                zY[i] = oldX;
                            }

                        } else {
                            for (Nd4jLong i = threadIdx.x; i < tadLength; i += blockDim.x) {

                                auto xOffset = shape::getIndexOffset(i, tadOnlyShapeInfo[f]);
                                auto yOffset = newOffset + xOffset;
                                xOffset += oldOffset;

                                T oldX = x[xOffset];
                                z[xOffset] = x[yOffset];
                                z[yOffset] = oldX;
                            }
                        }
                    }
                }
            }
            __syncthreads();
        }
    }

////////////////////////////////////////////////////////////////////////
    template<typename T>
    __host__ void shuffleKernelGeneric(dim3 &launchDims, hipStream_t *stream,
                                       void **vdX, Nd4jLong **xShapeInfo,
                                       void **vdZ,
                                       int N,
                                       int *shuffleMap,
                                       Nd4jLong **tadOnlyShapeInfo, Nd4jLong **tadOffsets) {

        execShuffleKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vdX, xShapeInfo, vdZ, N, shuffleMap, tadOnlyShapeInfo, tadOffsets);
        nd4j::DebugHelper::checkErrorCode(stream, "shuffleGeneric(...) failed");
    }

    BUILD_SINGLE_TEMPLATE(template void ND4J_EXPORT shuffleKernelGeneric, (dim3 & launchDims, hipStream_t * stream, void * *vdX, Nd4jLong * *xShapeInfo, void **vdZ, int N, int * shuffleMap, Nd4jLong * *tadOnlyShapeInfo, Nd4jLong * *tadOffsets), LIBND4J_TYPES);
}