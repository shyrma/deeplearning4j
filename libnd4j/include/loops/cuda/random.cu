#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <op_boilerplate.h>
#include <loops/random.h>
#include <dll.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <helpers/DebugHelper.h>
#include <specials_cuda.h>

using namespace randomOps;

template <typename T, typename OpClass>
static inline __device__ void randomSingleGeneric(
        Nd4jPointer state,
        void *z,
        Nd4jLong *zShapeBuffer,
        void *extraArguments) {


    functions::random::RandomFunction<T>::template execTransformCuda<OpClass>(
            state,
            z,
            zShapeBuffer,
            extraArguments);
}

template <typename T, typename OpClass>
static inline __device__ void randomDoubleGeneric(
        Nd4jPointer state,
        void *x,
        Nd4jLong *xShapeBuffer,
        void *z,
        Nd4jLong *zShapeBuffer,
        void *extraArguments) {


    functions::random::RandomFunction<T>::template execTransformCuda<OpClass>(
            state,
            x,
            xShapeBuffer,
            z,
            zShapeBuffer,
            extraArguments);
}


template <typename T, typename OpClass>
static inline __device__ void randomTripleGeneric(
        Nd4jPointer state,
        void *x,
        Nd4jLong *xShapeBuffer,
        void *y,
        Nd4jLong *yShapeBuffer,
        void *z,
        Nd4jLong *zShapeBuffer,
        void *extraArguments) {


    functions::random::RandomFunction<T>::template execTransformCuda<OpClass>(
            state,
            x,
            xShapeBuffer,
            y,
            yShapeBuffer,
            z,
            zShapeBuffer,
            extraArguments);
}


#ifndef __CLION_IDE__
// here we generate kernels for target operations
DISPATCH_KERNEL_SIMPLE(randomSingle_, randomSingleGeneric, float, INPUT(Nd4jPointer state, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomSingle_, randomSingleGeneric, double, INPUT(Nd4jPointer state, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomSingle_, randomSingleGeneric, float16, INPUT(Nd4jPointer state, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomSingle_, randomSingleGeneric, bfloat16, INPUT(Nd4jPointer state, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

DISPATCH_KERNEL_SIMPLE(randomDouble_, randomDoubleGeneric, float, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomDouble_, randomDoubleGeneric, double, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomDouble_, randomDoubleGeneric, float16, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomDouble_, randomDoubleGeneric, bfloat16, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

DISPATCH_KERNEL_SIMPLE(randomTriple_, randomTripleGeneric, float, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *y, Nd4jLong *yShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomTriple_, randomTripleGeneric, double, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *y, Nd4jLong *yShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomTriple_, randomTripleGeneric, float16, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *y, Nd4jLong *yShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomTriple_, randomTripleGeneric, bfloat16, INPUT(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *y, Nd4jLong *yShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments), PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

#endif

namespace functions {
    namespace random {
            template<typename T>
            template<typename OpClass>
            void _CUDA_D RandomFunction<T>::execTransformCuda(Nd4jPointer state, void *vx, Nd4jLong *xShapeBuffer, void *vy, Nd4jLong *yShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

                auto x = reinterpret_cast<T*>(vx);
                auto y = reinterpret_cast<T*>(vy);
                auto z = reinterpret_cast<T*>(vz);
                auto extraArguments = reinterpret_cast<T*>(vextraArguments);

                if (OpClass::requiresSpecial) {
                    OpClass::specialOpCuda(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments);
                    return;
                } else {

                __shared__ Nd4jLong length;
                __shared__ int xEWS;
                __shared__ int yEWS;
                __shared__ int zEWS;
                __shared__ char xOrder;
                __shared__ char yOrder;
                __shared__ char zOrder;

                __shared__ nd4j::graph::RandomGenerator *buffer;
                __shared__ unsigned char *cB;
                __shared__ unsigned char *dB;
                nd4j::graph::RandomGenerator *devBuffer;
                if (threadIdx.x == 0) {
                    length = shape::length(zShapeBuffer);
                    xEWS = shape::elementWiseStride(xShapeBuffer);
                    yEWS = shape::elementWiseStride(yShapeBuffer);
                    zEWS = shape::elementWiseStride(zShapeBuffer);
                    xOrder = shape::order(xShapeBuffer);
                    yOrder = shape::order(yShapeBuffer);
                    zOrder = shape::order(zShapeBuffer);

                    extern __shared__ unsigned char shmem[];
                    buffer = (nd4j::graph::RandomGenerator *) shmem;
                    cB = shmem;
                    devBuffer = reinterpret_cast<nd4j::graph::RandomGenerator *> (state);
                    dB = reinterpret_cast<unsigned char *> (state);
                }
                __syncthreads();

                // using this loop instead of memcpy
                for (int e = threadIdx.x; e < sizeof(nd4j::graph::RandomGenerator); e+= blockDim.x)
                    cB[e] = dB[e];

                __syncthreads();


                int tid = blockIdx.x * blockDim.x + threadIdx.x;

                if (xEWS >= 1 && yEWS >= 1 && zEWS >= 1 && xOrder == yOrder && xOrder == zOrder) {
                    for (Nd4jLong e = tid; e < length; e += blockDim.x * gridDim.x) {
                        z[e * zEWS] = OpClass::op(x[e * xEWS], y[e * yEWS], e, length, buffer, extraArguments);
                    }
                } else {
                    for (Nd4jLong i = tid; i < length; i += blockDim.x * gridDim.x) {

                        auto xOffset2 = shape::getIndexOffset(i, xShapeBuffer);
                        auto yOffset2 = shape::getIndexOffset(i, yShapeBuffer);
                        auto zOffset2 = shape::getIndexOffset(i, zShapeBuffer);

                            z[zOffset2] = OpClass::op(x[xOffset2], y[yOffset2], i, length, buffer, extraArguments);
                        }
                    }
                }
            };


            template<typename T>
            template<typename OpClass>
            void _CUDA_D RandomFunction<T>::execTransformCuda(Nd4jPointer state, void *vx, Nd4jLong *xShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

                auto x = reinterpret_cast<T*>(vx);
                auto z = reinterpret_cast<T*>(vz);
                auto extraArguments = reinterpret_cast<T*>(vextraArguments);

                __shared__ Nd4jLong length;
                __shared__ int xEWS;
                __shared__ int zEWS;
                __shared__ char xOrder;
                __shared__ char zOrder;

                __shared__ nd4j::graph::RandomGenerator *buffer;
                __shared__ unsigned char *cB;
                __shared__ unsigned char *dB;
                __shared__ nd4j::graph::RandomGenerator *devBuffer;

                if (threadIdx.x == 0) {
                    extern __shared__ unsigned char shmem[];
                    buffer = (nd4j::graph::RandomGenerator *) shmem;
                    cB = shmem;
                    devBuffer = reinterpret_cast<nd4j::graph::RandomGenerator *> (state);
                    dB = reinterpret_cast<unsigned char *> (state);

                    length = shape::length(zShapeBuffer);
                    xEWS = shape::elementWiseStride(xShapeBuffer);
                    zEWS = shape::elementWiseStride(zShapeBuffer);
                    xOrder = shape::order(xShapeBuffer);
                    zOrder = shape::order(zShapeBuffer);
                }
                __syncthreads();

                // using this loop instead of memcpy
                for (int e = threadIdx.x; e < sizeof(nd4j::graph::RandomGenerator); e+= blockDim.x)
                    cB[e] = dB[e];

                __syncthreads();


                if (xEWS >= 1 && zEWS >= 1 && xOrder == zOrder) {
                    for (Nd4jLong e = blockIdx.x * blockDim.x + threadIdx.x; e < length; e += blockDim.x * gridDim.x) {
                        z[e * zEWS] = OpClass::op(x[e * xEWS], e, length, buffer, extraArguments);
                    }
                } else {

                    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < length; i += blockDim.x * gridDim.x) {

                        auto xOffset2 = shape::getIndexOffset(i, xShapeBuffer);
                        auto zOffset2 = shape::getIndexOffset(i, zShapeBuffer);

                        z[zOffset2] = OpClass::op(x[xOffset2], i, length, buffer, extraArguments);
                    }
                }
            }


            template<typename T>
            template<typename OpClass>
            void _CUDA_D RandomFunction<T>::execTransformCuda(Nd4jPointer state, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

                auto z = reinterpret_cast<T*>(vz);
                auto extraArguments = reinterpret_cast<T*>(vextraArguments);

                __shared__ Nd4jLong length;
                __shared__ Nd4jLong ews;
                __shared__ nd4j::graph::RandomGenerator *buffer;
                __shared__ unsigned char *cB;
                __shared__ unsigned char *dB;
                __shared__ nd4j::graph::RandomGenerator *devBuffer;

                if (threadIdx.x == 0) {
                    extern __shared__ unsigned char shmem[];
                    buffer = (nd4j::graph::RandomGenerator *) shmem;
                    cB = shmem;
                    devBuffer = reinterpret_cast<nd4j::graph::RandomGenerator *> (state);
                    dB = reinterpret_cast<unsigned char *> (state);
                    length = shape::length(zShapeBuffer);
                    ews = shape::elementWiseStride(zShapeBuffer);
                }
                __syncthreads();

                // using this loop instead of memcpy
                for (int e = threadIdx.x; e < sizeof(nd4j::graph::RandomGenerator); e+= blockDim.x)
                    cB[e] = dB[e];

                __syncthreads();

                int tid = blockIdx.x * blockDim.x + threadIdx.x;

                if (ews > 0) {
                    for (Nd4jLong i = tid; i < length; i += blockDim.x * gridDim.x) {
                        z[i * ews] = OpClass::op(i, length, buffer, extraArguments);
                    }
                } else {

                    for (Nd4jLong i = tid; i < length; i += blockDim.x * gridDim.x) {
                        auto zOffset2 = shape::getIndexOffset(i, zShapeBuffer);
                        z[zOffset2] = OpClass::op(i, length, buffer, extraArguments);
                    }
                }
            }

        template <>
        _CUDA_H void RandomFunction<float>::executeCudaSingle(dim3& launchDims, hipStream_t* stream, int opNum, Nd4jPointer stateHost, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto z = reinterpret_cast<float*>(vz);
            auto extraArguments = reinterpret_cast<float*>(vextraArguments);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomSingle, float, PARAMS(stateHost, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<float16>::executeCudaSingle(dim3& launchDims, hipStream_t* stream, int opNum, Nd4jPointer stateHost, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto z = reinterpret_cast<float16*>(vz);
            auto extraArguments = reinterpret_cast<float16*>(vextraArguments);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomSingle, float16, PARAMS(stateHost, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<bfloat16>::executeCudaSingle(dim3& launchDims, hipStream_t* stream, int opNum, Nd4jPointer stateHost, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto z = reinterpret_cast<bfloat16*>(vz);
            auto extraArguments = reinterpret_cast<bfloat16*>(vextraArguments);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomSingle, bfloat16, PARAMS(stateHost, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<double>::executeCudaSingle(dim3& launchDims, hipStream_t *stream, int opNum, Nd4jPointer stateHost, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto z = reinterpret_cast<double*>(vz);
            auto extraArguments = reinterpret_cast<double*>(vextraArguments);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomSingle, double, PARAMS(stateHost, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<float>::executeCudaDouble(dim3& launchDims, hipStream_t* stream, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto x = reinterpret_cast<float*>(vx);
            auto z = reinterpret_cast<float*>(vz);
            auto extraArguments = reinterpret_cast<float*>(vextraArguments);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomDouble, float, PARAMS(stateHost, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }


        template <>
        _CUDA_H void RandomFunction<float16>::executeCudaDouble(dim3& launchDims, hipStream_t* stream, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto x = reinterpret_cast<float16*>(vx);
            auto z = reinterpret_cast<float16*>(vz);
            auto extraArguments = reinterpret_cast<float16*>(vextraArguments);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomDouble, float16, PARAMS(stateHost, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<bfloat16>::executeCudaDouble(dim3& launchDims, hipStream_t* stream, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto x = reinterpret_cast<bfloat16*>(vx);
            auto z = reinterpret_cast<bfloat16*>(vz);
            auto extraArguments = reinterpret_cast<bfloat16*>(vextraArguments);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomDouble, bfloat16, PARAMS(stateHost, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<double>::executeCudaDouble(dim3& launchDims, hipStream_t* stream, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto x = reinterpret_cast<double*>(vx);
            auto z = reinterpret_cast<double*>(vz);
            auto extraArguments = reinterpret_cast<double*>(vextraArguments);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomDouble, double, PARAMS(stateHost, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<float>::executeCudaTriple(dim3& launchDims, hipStream_t* stream, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vy, Nd4jLong *yShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {


            auto x = reinterpret_cast<float*>(vx);
            auto y = reinterpret_cast<float*>(vy);
            auto z = reinterpret_cast<float*>(vz);
            auto extraArguments = reinterpret_cast<float*>(vextraArguments);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomTriple, float, PARAMS(stateHost, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<float16>::executeCudaTriple(dim3& launchDims, hipStream_t* stream, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vy, Nd4jLong *yShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto x = reinterpret_cast<float16*>(vx);
            auto y = reinterpret_cast<float16*>(vy);
            auto z = reinterpret_cast<float16*>(vz);
            auto extraArguments = reinterpret_cast<float16*>(vextraArguments);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomTriple, float16, PARAMS(stateHost, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<bfloat16>::executeCudaTriple(dim3& launchDims, hipStream_t* stream, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vy, Nd4jLong *yShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto x = reinterpret_cast<bfloat16*>(vx);
            auto y = reinterpret_cast<bfloat16*>(vy);
            auto z = reinterpret_cast<bfloat16*>(vz);
            auto extraArguments = reinterpret_cast<bfloat16*>(vextraArguments);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomTriple, bfloat16, PARAMS(stateHost, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }



        template <>
        _CUDA_H void RandomFunction<double>::executeCudaTriple(dim3& launchDims, hipStream_t* stream, int opNum, Nd4jPointer stateHost, void *vx, Nd4jLong *xShapeBuffer, void *vy, Nd4jLong *yShapeBuffer, void *vz, Nd4jLong *zShapeBuffer, void *vextraArguments) {

            auto x = reinterpret_cast<double*>(vx);
            auto y = reinterpret_cast<double*>(vy);
            auto z = reinterpret_cast<double*>(vz);
            auto extraArguments = reinterpret_cast<double*>(vextraArguments);

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomTriple, double, PARAMS(stateHost, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template<typename T>
        template<typename OpClass>
        void RandomFunction<T>::execTransform(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *y, Nd4jLong *yShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments) {

        }

        template<typename T>
        template<typename OpClass>
        void RandomFunction<T>::execTransform(Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments) {

        }

        template<typename T>
        template<typename OpClass>
        void RandomFunction<T>::execTransform(Nd4jPointer state, void *z, Nd4jLong *zShapeBuffer, void *extraArguments) {

        }

        template<typename T>
        void RandomFunction<T>::execTransform(int opNum, Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments) {

        }

        template<typename T>
        void RandomFunction<T>::execTransform(int opNum, Nd4jPointer state, void *x, Nd4jLong *xShapeBuffer, void *y, Nd4jLong *yShapeBuffer, void *z, Nd4jLong *zShapeBuffer, void *extraArguments) {

        }

        template<typename T>
        void RandomFunction<T>::execTransform(int opNum, Nd4jPointer state, void *z, Nd4jLong *zShapeBuffer, void *extraArguments) {

        }

        BUILD_SINGLE_TEMPLATE(template class ND4J_EXPORT RandomFunction, , FLOAT_TYPES);
    }
}
