#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include <Environment.h>
#include <loops/transform_strict.h>
#include <types/types.h>
#include <op_boilerplate.h>

#include <loops/legacy_ops.h>
#include <helpers/DebugHelper.h>

using namespace simdOps;

template <typename X, typename OpType>
__global__ void transformStrictSimple(void *x, Nd4jLong *xShapeInfo, int xRank,
								void *params,
								void *z, Nd4jLong *zShapeInfo, int zRank,
								int *allocationPointer,
								void *reductionPointer,
								Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

	functions::transform::TransformStrict<X>::template transformCuda<OpType>(x,xShapeInfo,params,z,zShapeInfo,allocationPointer,reductionPointer,tadShapeInfo, tadOffsets);
}


namespace functions {
    namespace transform {

        template<typename X>
        _CUDA_H void TransformStrict<X>::executeTransformShaped(dim3 launchDims, hipStream_t *stream, int opNum, void *x, Nd4jLong *xShape, int xRank, void *extraParams, void *z, Nd4jLong *zShape, int zRank, int *allocationPointer, void *reductionPointer,  Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {
			DISPATCH_BY_OPNUM_T(intermediateShaped, PARAMS(launchDims, stream, x, xShape, xRank, extraParams, z, zShape, zRank, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets), TRANSFORM_STRICT_OPS);

            DEBUG_KERNEL(stream, opNum);
        }


        template<typename X>
        template <typename OpType>
        __device__ void TransformStrict<X>::transformCuda(void *vx, Nd4jLong *xShapeInfo,
        												void *vparams,
        												void *vz, Nd4jLong *zShapeInfo,
        												int *allocationPointer, void *vreductionPointer,
        												Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

        	auto x = static_cast<X*>(vx);
		    auto z = static_cast<X*>(vz);
		    auto params = static_cast<X*>(vparams);
		    auto reductionPointer = static_cast<X*>(vreductionPointer);


		    if(OpType::requiresSpecial) {
			    OpType::execSpecialCuda(x,xShapeInfo,z,zShapeInfo,params, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets);
			    return;
		    }
		    else {
		    	__shared__ Nd4jLong xEws;
    	        __shared__ Nd4jLong zEws;
        	    __shared__ char xOrder;
            	__shared__ char zOrder;
            	__shared__ Nd4jLong length;

	            if (threadIdx.x == 0) {

        	        xEws = shape::elementWiseStride(xShapeInfo);
            	    zEws = shape::elementWiseStride(zShapeInfo);
                	xOrder = shape::order(xShapeInfo);
					zOrder = shape::order(zShapeInfo);
					length = shape::length(xShapeInfo);
            	}
            	__syncthreads();

	    	    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
				int totalThreads = gridDim.x * blockDim.x;

		        if(xEws > 0 && zEws > 0 && xOrder == zOrder) {

					for (int i = tid; i < length; i += totalThreads)
						z[i * zEws] = OpType::op(x[i * xEws], params);
		        }
		        else {
					if(vx == vz) {
						for (Nd4jLong i = tid; i < length; i+= totalThreads) {
							auto xOffset = shape::getIndexOffset(i, xShapeInfo);
	    			    	z[xOffset] = OpType::op(x[xOffset], params);
		    	    	}
					}
					else {
		    	    	for (Nd4jLong i = tid; i < length; i+= totalThreads) {
							auto xOffset = shape::getIndexOffset(i, xShapeInfo);
							auto zOffset = shape::getIndexOffset(i, zShapeInfo);
	    			    	z[zOffset] = OpType::op(x[xOffset], params);
		    	    	}
		    		}
		        }
	  		}
	    };

		template<typename X>
		template <typename OpType>
		_CUDA_H void TransformStrict<X>::intermediateShaped(dim3 launchDims, hipStream_t *stream, void *x, Nd4jLong *xShape, int xRank, void *extraParams, void *z, Nd4jLong *zShape, int zRank, int *allocationPointer, void *reductionPointer,  Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {
			transformStrictSimple<X, OpType><<<launchDims.x, launchDims.x, launchDims.z, *stream>>>(x, xShape, xRank, extraParams, z, zShape, zRank, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets);
            nd4j::DebugHelper::checkErrorCode(stream, "transformStrict(...) failed");
		}

        template<typename X>
        void TransformStrict<X>::exec(int opNum, void *dx, Nd4jLong *xShapeInfo, void *result, Nd4jLong *resultShapeInfo, void *extraParams, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

        }

        template<typename X>
        template <typename OpType>
        void TransformStrict<X>::exec(void *dx, Nd4jLong *xShapeInfo, void *result, Nd4jLong *resultShapeInfo, void *extraParams, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

        }

        BUILD_SINGLE_TEMPLATE(template class ND4J_EXPORT TransformStrict, , FLOAT_TYPES);
    }
}
