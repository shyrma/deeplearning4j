#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include <Environment.h>
#include <loops/transform_any.h>
#include <types/types.h>
#include <op_boilerplate.h>

#include <loops/legacy_ops.h>
#include <helpers/DebugHelper.h>

using namespace simdOps;


template <typename X, typename Z, typename OpType>
__global__ void transformAnySimple(void *x, Nd4jLong *xShapeInfo, int xRank,
								void *params,
								void *z, Nd4jLong *zShapeInfo, int zRank,
								int *allocationPointer,
								void *reductionPointer,
								Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

	functions::transform::TransformAny<X,Z>::template transformCuda<OpType>(x,xShapeInfo,params,z,zShapeInfo,allocationPointer,reductionPointer,tadShapeInfo, tadOffsets);
}


namespace functions {
    namespace transform {

        template<typename X, typename Y>
        _CUDA_H void TransformAny<X,Y>::executeTransformShaped(dim3 launchDims, hipStream_t *stream, int opNum, void *x, Nd4jLong *xShape, int xRank, void *extraParams, void *z, Nd4jLong *zShape, int zRank, int *allocationPointer, void *reductionPointer,  Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {
			DISPATCH_BY_OPNUM_TT(intermediateShaped, PARAMS(launchDims, stream, x, xShape, xRank, extraParams, z, zShape, zRank, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets), TRANSFORM_ANY_OPS);

            DEBUG_KERNEL(stream, opNum);
        }


        template<typename X, typename Z>
        template <typename OpType>
        __device__ void TransformAny<X,Z>::transformCuda(void *vx, Nd4jLong *xShapeInfo,
        												void *vparams,
        												void *vz, Nd4jLong *zShapeInfo,
        												int *allocationPointer, void *vreductionPointer,
        												Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {

        	auto x = reinterpret_cast<X*>(vx);
		    auto z = reinterpret_cast<Z*>(vz);
		    auto params = reinterpret_cast<X*>(vparams);
		    auto reductionPointer = reinterpret_cast<Z*>(vreductionPointer);

		    __shared__ Nd4jLong xEws;
    	    __shared__ Nd4jLong zEws;
        	__shared__ char xOrder;
            __shared__ char zOrder;
            __shared__ Nd4jLong length;

	        if (threadIdx.x == 0) {

        		xEws = shape::elementWiseStride(xShapeInfo);
            	zEws = shape::elementWiseStride(zShapeInfo);
                xOrder = shape::order(xShapeInfo);
				zOrder = shape::order(zShapeInfo);
				length = shape::length(xShapeInfo);
            }
            __syncthreads();

	    	auto tid = blockIdx.x * blockDim.x + threadIdx.x;
			int totalThreads = gridDim.x * blockDim.x;

		    if(xEws > 0 && zEws > 0 && xOrder == zOrder) {

				for (int i = tid; i < length; i += totalThreads)
					z[i * zEws] = OpType::op(x[i * xEws], params);
		    }
		    else {
				if(vx == vz) {
					for (Nd4jLong i = tid; i < length; i+= totalThreads) {
						auto xOffset = shape::getIndexOffset(i, xShapeInfo);
	    		    	z[xOffset] = OpType::op(x[xOffset], params);
		    	   	}
				}
				else {
		    	   	for (Nd4jLong i = tid; i < length; i+= totalThreads) {
						auto xOffset = shape::getIndexOffset(i, xShapeInfo);
						auto zOffset = shape::getIndexOffset(i, zShapeInfo);
	    		    	z[zOffset] = OpType::op(x[xOffset], params);
		    	   	}
		    	}
		    }
	    };


		template<typename X, typename Z>
		template <typename OpType>
		_CUDA_H void TransformAny<X,Z>::intermediateShaped(dim3 launchDims, hipStream_t *stream, void *x, Nd4jLong *xShape, int xRank, void *extraParams, void *z, Nd4jLong *zShape, int zRank, int *allocationPointer, void *reductionPointer,  Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets) {
			transformAnySimple<X, Z, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShape, xRank, extraParams, z, zShape, zRank, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets);
            nd4j::DebugHelper::checkErrorCode(stream, "transformAny(...) failed");
		}

        template<typename X, typename Z>
        void TransformAny<X,Z>::exec(int opNum, void *dx, Nd4jLong *xShapeInfo, void *vz, Nd4jLong *zShapeInfo, void *extraParams, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, bool allowParallelism) {

        }

        template<typename X, typename Z>
        template <typename OpType>
        void TransformAny<X,Z>::exec(void *dx, Nd4jLong *xShapeInfo, void *vz, Nd4jLong *zShapeInfo, void *extraParams, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, bool allowParallelism) {

        }

        BUILD_DOUBLE_TEMPLATE(template class ND4J_EXPORT TransformAny, , LIBND4J_TYPES, LIBND4J_TYPES);
    }
}
