#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <op_boilerplate.h>
#include <loops/broadcasting_int.h>
#include <loops/legacy_ops.h>
#include <types/types.h>
#include <Environment.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdexcept>
#include <StringUtils.h>

using namespace simdOps;

//////////////////////////////////////////////////////////////////////////
template<typename X, typename OpClass>
static __global__ void broadcastIntSimple(
        void *x,
        Nd4jLong *xShapeInfo,
        void *y,
        Nd4jLong *yShapeInfo,
        void *z,
        Nd4jLong *zShapeInfo,
        int *dimension,
        int dimensionLength, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ) {

    functions::broadcast::BroadcastInt<X>::template transformCuda<OpClass>(x,xShapeInfo,y,yShapeInfo,z,zShapeInfo,dimension,dimensionLength,tadOnlyShapeInfo,tadOffsets,tadOnlyShapeInfoZ,tadOffsetsZ);
}

//////////////////////////////////////////////////////////////////////////
template<typename X, typename OpClass>
static __global__ void broadcastBoolInverseSimple(
        void *x,
        Nd4jLong *xShapeInfo,
        void *y,
        Nd4jLong *yShapeInfo,
        void *z,
        Nd4jLong *zShapeInfo,
        int *dimension,
        int dimensionLength, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ) {

    functions::broadcast::BroadcastInt<X>::template transformInverseCuda<OpClass>(x,xShapeInfo,y,yShapeInfo,z,zShapeInfo,dimension,dimensionLength,tadOnlyShapeInfo,tadOffsets,tadOnlyShapeInfoZ,tadOffsetsZ);
}

namespace functions {
    namespace broadcast {
//////////////////////////////////////////////////////////////////////////
        template<typename X>
        template <typename OpClass>
        __host__ void BroadcastInt<X>::intermediateBroadcast(dim3 launchDims, hipStream_t *stream, void *x, Nd4jLong *xShapeInfo, void *y, Nd4jLong *yShapeInfo, void *z, Nd4jLong *zShapeInfo, int *dimension, int dimensionLength, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ) {
            broadcastIntSimple<X, OpClass><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ);
        }

//////////////////////////////////////////////////////////////////////////
        template<typename X>
        __host__ void BroadcastInt<X>::execBroadcast(dim3 launchDims, hipStream_t *stream, int opNum, void *x, Nd4jLong *xShapeInfo, void *y, Nd4jLong *yShapeInfo, void *z, Nd4jLong *zShapeInfo, int *dimension, int dimensionLength, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ) {
            DISPATCH_BY_OPNUM_T(intermediateBroadcast,  PARAMS(launchDims, stream, x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), OPS_A(BROADCAST_INT_OPS))
        }

//////////////////////////////////////////////////////////////////////////
        template<typename X>
        template <typename OpClass>
        __host__ void BroadcastInt<X>::intermediateInverseBroadcast(dim3 launchDims, hipStream_t *stream, void *x, Nd4jLong *xShapeInfo, void *y, Nd4jLong *yShapeInfo, void *z, Nd4jLong *zShapeInfo, int *dimension, int dimensionLength, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ) {
            broadcastBoolInverseSimple<X, OpClass><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ);
        }

//////////////////////////////////////////////////////////////////////////
        template<typename X>
        __host__ void BroadcastInt<X>::execInverseBroadcast(dim3 launchDims, hipStream_t *stream, int opNum, void *x, Nd4jLong *xShapeInfo, void *y, Nd4jLong *yShapeInfo, void *z, Nd4jLong *zShapeInfo, int *dimension, int dimensionLength, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ) {
            DISPATCH_BY_OPNUM_T(intermediateInverseBroadcast,  PARAMS(launchDims, stream, x, xShapeInfo, y, yShapeInfo, z, zShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), OPS_A(BROADCAST_INT_OPS))
        }

//////////////////////////////////////////////////////////////////////////
        template<typename X>
        template <typename OpType>
        __device__ void BroadcastInt<X>::transformInverseCuda(
                void *vx, Nd4jLong *xShapeInfo,
                void *vy, Nd4jLong *yShapeInfo,
                void *vz, Nd4jLong *zShapeInfo,
                int *dimension, int dimensionLength,
                Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ) {

            if (tadOnlyShapeInfoZ == nullptr) {
                tadOnlyShapeInfoZ = tadOnlyShapeInfo;
                tadOffsetsZ = tadOffsets;
            }

            auto x = reinterpret_cast<X*>(vx);
            auto y = reinterpret_cast<X*>(vy);
            auto z = reinterpret_cast<X*>(vz);

            //decompose in to several sub tads after
            //moving all dimensions (in sorted order)
            //to the back.
            //permuted version of the x shape info for setting up the tad problem
            __shared__ Nd4jLong tadLength;
            __shared__ Nd4jLong tadEWS;
            __shared__ int numTads;
            __shared__ Nd4jLong xEWS;
            __shared__ Nd4jLong zEWS;

            if (threadIdx.x == 0) {
                tadLength = shape::length(tadOnlyShapeInfo);//shape::tadLength(xShapeInfo, dimension, dimensionLength);
                tadEWS = shape::elementWiseStride(tadOnlyShapeInfo);
                numTads = shape::length(yShapeInfo) / tadLength;
                xEWS = shape::elementWiseStride(xShapeInfo);
                zEWS = shape::elementWiseStride(tadOnlyShapeInfoZ);
            }
            __syncthreads();

            for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
                auto rZ = z + tadOffsetsZ[r];
                auto rY = y + tadOffsets[r];

                if(tadEWS > 0 && zEWS > 0 && xEWS > 0 && dimensionLength == 1) {

                    for (int i = threadIdx.x; i < tadLength; i+= blockDim.x)
                        rZ[i * zEWS] = OpType::op(x[i * xEWS], rY[i * tadEWS]);
                }
                else {
                    // it is expected that x and z tads and y array all have the same length
                    for (Nd4jLong i = threadIdx.x; i < tadLength; i+= blockDim.x) {
                        auto xOffset = shape::getIndexOffset(i, xShapeInfo);
                        auto yOffset = shape::getIndexOffset(i, tadOnlyShapeInfo);
                        auto zOffset = shape::getIndexOffset(i, tadOnlyShapeInfoZ);

                        rZ[zOffset] = OpType::op(x[xOffset], rY[yOffset]);
                    }
                }
            }
        }

//////////////////////////////////////////////////////////////////////////
        template<typename X>
        template <typename OpType>
		__device__ void BroadcastInt<X>::transformCuda(
		                              void *vx, Nd4jLong *xShapeInfo,
		                              void *vy, Nd4jLong *yShapeInfo,
		                              void *vz, Nd4jLong *zShapeInfo,
		                              int *dimension, int dimensionLength,
                                      Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ) {

            if (tadOnlyShapeInfoZ == nullptr) {
                tadOnlyShapeInfoZ = tadOnlyShapeInfo;
                tadOffsetsZ = tadOffsets;
            }

            auto x = reinterpret_cast<X*>(vx);
            auto y = reinterpret_cast<X*>(vy);
            auto z = reinterpret_cast<X*>(vz);

            //decompose in to several sub tads after
            //moving all dimensions (in sorted order)
            //to the back.
            //permuted version of the x shape info for setting up the tad problem
            __shared__ Nd4jLong tadLength;
            __shared__ Nd4jLong tadEWS;
            __shared__ int numTads;
            __shared__ Nd4jLong yEWS;
            __shared__ Nd4jLong zEWS;

            if (threadIdx.x == 0) {
   	            tadLength = shape::length(tadOnlyShapeInfo);//shape::tadLength(xShapeInfo, dimension, dimensionLength);
                tadEWS = shape::elementWiseStride(tadOnlyShapeInfo);
                numTads = shape::length(xShapeInfo) / tadLength;
                yEWS = shape::elementWiseStride(yShapeInfo);
                zEWS = shape::elementWiseStride(tadOnlyShapeInfoZ);
            }
            __syncthreads();

            __shared__ X *rZ;
            __shared__ X *rX;

		for (int r = blockIdx.x; r < numTads; r += gridDim.x) {

            if (threadIdx.x == 0) {
                rZ = z + tadOffsetsZ[r];
                rX = x + tadOffsets[r];
            }
            __syncthreads();


            if(tadEWS > 0 && zEWS > 0 && yEWS > 0 && dimensionLength == 1) {

                for (int i = threadIdx.x; i < tadLength; i+= blockDim.x)
                    rZ[i * zEWS] = OpType::op(rX[i * tadEWS], y[i * yEWS]);
            }
            else {
                // it is expected that x and z tads and y array all have the same length
                for (Nd4jLong i = threadIdx.x; i < tadLength; i+= blockDim.x) {
                    auto xOffset = shape::getIndexOffset(i, tadOnlyShapeInfo);
                    auto yOffset = shape::getIndexOffset(i, yShapeInfo);
                    auto zOffset = shape::getIndexOffset(i, tadOnlyShapeInfoZ);

                    rZ[zOffset] = OpType::op(rX[xOffset], y[yOffset]);
                }
            }
		}
	}


        template<typename X>
        void BroadcastInt<X>::exec(int opNum,
                         void *x,
                         Nd4jLong *xShapeInfo,
                         void *y,
                         Nd4jLong *yShapeInfo,
                         void *result,
                         Nd4jLong *resultShapeInfo,
                         int *dimension,
                         int dimensionLength,
                         Nd4jLong *tadShapeInfo,
                         Nd4jLong *tadOffset,
                         Nd4jLong *tadShapeInfoZ,
                         Nd4jLong *tadOffsetZ) {

        }

        template<typename X>
        void BroadcastInt<X>::execInverse(int opNum,
                                void *x,
                                Nd4jLong *xShapeInfo,
                                void *y,
                                Nd4jLong *yShapeInfo,
                                void *result,
                                Nd4jLong *resultShapeInfo,
                                int *dimension,
                                int dimensionLength,
                                Nd4jLong *tadShapeInfo,
                                Nd4jLong *tadOffset,
                                Nd4jLong *tadShapeInfoZ,
                                Nd4jLong *tadOffsetZ) {

        }

        template<typename X>
        template<typename OpType>
        void BroadcastInt<X>::exec(void *x,
                         Nd4jLong *xShapeInfo,
                         void *y,
                         Nd4jLong *yShapeInfo,
                         void *result,
                         Nd4jLong *resultShapeInfo,
                         int *dimension,
                         int dimensionLength,
                         Nd4jLong *tadShapeInfo,
                         Nd4jLong *tadOffset,
                         Nd4jLong *tadShapeInfoZ,
                         Nd4jLong *tadOffsetZ) {

        }

        template<typename X>
        template<typename OpType>
        void BroadcastInt<X>::execInverse(void *x,
                                Nd4jLong *xShapeInfo,
                                void *y,
                                Nd4jLong *yShapeInfo,
                                void *result,
                                Nd4jLong *resultShapeInfo,
                                int *dimension,
                                int dimensionLength,
                                Nd4jLong *tadShapeInfo,
                                Nd4jLong *tadOffset,
                                Nd4jLong *tadShapeInfoZ,
                                Nd4jLong *tadOffsetZ) {

        }

        BUILD_SINGLE_TEMPLATE(template class ND4J_EXPORT BroadcastInt, , INTEGER_TYPES);
    }
}