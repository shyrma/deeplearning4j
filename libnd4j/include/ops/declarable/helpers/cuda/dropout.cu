#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/dropout.h>
#include <NativeOps.h>
#include <vector>
#include <memory>
#include <cuda_exception.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    static __global__ void dropoutSimpleKernel(void const* inputBuf, Nd4jLong const* inputShape, void* outputBuf, Nd4jLong* outputShape, double probVal, int inLen, nd4j::graph::RandomGenerator* nodeRng) {
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        T const* input = reinterpret_cast<T const*>(inputBuf);
        T* output = reinterpret_cast<T*>(outputBuf);

        // trivial idea: loop through all elements, get independent probability for each element to be nullified
        for (Nd4jLong e = 0; e < inLen; ++e) {
            T val = nodeRng->relativeT(e, T(0.f), T(1.f));

            // if probability is ok - we're saving scaled value
            if (double(val) < probVal)
                output[shape::getIndexOffset(e, outputShape)] = T(input[shape::getIndexOffset(e, inputShape)] / probVal);
        }
    }

    template <typename T>
    static void dropoutSimple(nd4j::LaunchContext* context, NDArray const* input, NDArray* output, double probValue, int seed) {
        nd4j::graph::RandomGenerator nodeRng(3019L, seed);
        int inLen = input->lengthOf();
        nd4j::graph::RandomGenerator* dRandom;
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input});

        auto err = hipMalloc(&dRandom, sizeof(nd4j::graph::RandomGenerator));
        if (err) {
            throw cuda_exception::build("helpers::dropoutSimple: Cannot allocate device memory for random generator.", err);
        }
        err = hipMemcpy(dRandom, &nodeRng, sizeof(nd4j::graph::RandomGenerator), hipMemcpyHostToDevice);
        if (err) {
            throw cuda_exception::build("helpers::dropoutSimple: Cannot set up device memory for random generator.", err);
        }

        dropoutSimpleKernel<T><<<128, 256, 1024, *stream>>>(input->getSpecialBuffer(), input->getSpecialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), probValue, inLen, dRandom);
        err = hipFree(dRandom);
        if (err) {
            throw cuda_exception::build("helpers::dropoutSimple: Cannot deallocate device memory for random generator.", err);
        }
        NDArray::registerSpecialUse({output}, {input});
    }

    template <typename T>
    int _dropOutFunctor(graph::Context& context, NDArray* input, NDArray* output, NDArray* reduceShape, int seed, double probValue) {

        if (reduceShape == nullptr){
            dropoutSimple<T>(context.launchContext(), input, output, probValue, seed);
        }
        else {
            REQUIRE_TRUE(reduceShape->lengthOf() <= input->rankOf(), 0, "dropout: Noise shape should be fittable to input");

            std::vector<Nd4jLong> dims(reduceShape->lengthOf());
            reduceShape->syncToHost(); // to ensure that follows are actual
            bool fit = true;

            for( int i = 0; i < dims.size(); i++ ) {
                if (fit) {
                    dims[i] = reduceShape->e<Nd4jLong>(i);
                    for (int e = 0; e < input->rankOf(); ++e)
                        if (fit)
                            if (input->sizeAt(e) % dims[i]) {
                                fit = false;
                            }
                }
            }

            // check dims to fit input
            REQUIRE_TRUE(fit, 0, "dropout: Noise shape should fit to input rank.");
            std::unique_ptr<NDArray> chunk(new NDArray('c', dims, output->dataType(), context.launchContext()));
            chunk->assign(1.f);

            dropoutSimple<T>(context.launchContext(), chunk.get(), chunk.get(), probValue, seed);
            // broadcast chunk to full matrix
            std::unique_ptr<NDArray> dropOutMultiplier(new NDArray(*input));
            dropOutMultiplier->assign(1.f);

            *dropOutMultiplier += *chunk;

            // FIXME: we could do this in one step, aren't we?
            output->assign(*input * *dropOutMultiplier); //input->applyPairwiseTransform(pairwise::Multiply, dropOutMultiplier.get(), output, nullptr);
        }

        return Status::OK();
    }

    int dropOutFunctor(graph::Context& context, NDArray* input, NDArray* output, NDArray* reduceShape, int seed, double probValue) {
        auto xType = input->dataType();
        NDArray::prepareSpecialUse({output}, {input});

        BUILD_SINGLE_SELECTOR(xType, return _dropOutFunctor, (context, input, output, reduceShape, seed, probValue), FLOAT_TYPES);

        NDArray::registerSpecialUse({output}, {input});
    }

/////////////////////////////////// backrpopagations ///////////////////////////////////////////////
    template <typename T>
    static __global__ void dropoutBPKernel(void* outputBuf, Nd4jLong* outputShape, void* gradOutBuf, Nd4jLong* gradOutShape, double probValue) {
        __shared__ T* output;
        __shared__ T* input;
        __shared__ int len;

        if (threadIdx.x == 0) {
            len = shape::length(outputShape);
            output = reinterpret_cast<T*>(outputBuf);
            input = reinterpret_cast<T*>(gradOutBuf);
        }
        __syncthreads();

        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int e = tid; e < len; e += step) {
            const auto zOffset = shape::getIndexOffset(e, outputShape);

            // if probability was non-zero on FF step, we'll scale grads back
            if (output[zOffset] != T(0.))
                output[zOffset] = T(input[shape::getIndexOffset(e, gradOutShape)] / probValue);

        }
    }
    template <typename T>
    static int dropOutFunctorBP_(graph::Context& context, NDArray* input, NDArray* gradOut, NDArray* output, NDArray* reduceShape, int seed, double probValue) {
        // we're making additional FF run to see how probabilities played out with given seeds
        int res = dropOutFunctor(context, input, output, reduceShape, seed, probValue);
        auto stream = context.launchContext()->getCudaStream();

        NDArray::prepareSpecialUse({output}, {input, gradOut});

        if (ND4J_STATUS_OK == res)
            dropoutBPKernel<T><<<128, 256, 1024, *stream>>>(output->specialBuffer(), output->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(), probValue);

        NDArray::registerSpecialUse({output}, {input, gradOut});

        return res;
    }

    template <typename T>
    static __global__ void alphaDropoutSimpleKernel(void const* inputBuf, Nd4jLong const* inputShape, void* outputBuf, Nd4jLong* outputShape, double probValue, double alpha, double alpha1, double beta, int inLen, nd4j::graph::RandomGenerator* nodeRng) {
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        T const* input = reinterpret_cast<T const*>(inputBuf);
        T* output = reinterpret_cast<T*>(outputBuf);

        for (auto e = tid; e < inLen; e += step) {
            T val = nodeRng->relativeT(e, T(0.f), T(1.f));
            T xVal = input[shape::getIndexOffset(e, inputShape)];
            output[shape::getIndexOffset(e, outputShape)] = (val >= T(probValue) ? T(alpha * beta + alpha1) : T(alpha * (double)xVal + alpha1));
        }
    }
    template <typename T>
    static void alphaDropoutSimple(nd4j::LaunchContext* context, NDArray const* input, NDArray* output, int seed, double probValue, double alpha, double alpha1, double beta) {
        nd4j::graph::RandomGenerator nodeRng(3019L, seed), *dRandom;
        auto stream = context->getCudaStream();
        auto err = hipMalloc(&dRandom, sizeof(nd4j::graph::RandomGenerator));
        NDArray::prepareSpecialUse({output}, {input});
        if (err) {
            throw cuda_exception::build("helpers::alphaDropoutSimple: Cannot allocate device memory for random generator.", err);
        }
        err = hipMemcpy(dRandom, &nodeRng, sizeof(nd4j::graph::RandomGenerator), hipMemcpyHostToDevice);
        if (err) {
            throw cuda_exception::build("helpers::alphaDropoutSimple: Cannot set up device memory for random generator.", err);
        }

        alphaDropoutSimpleKernel<T><<<128, 256, 1024, *stream>>>(input->getSpecialBuffer(), input->getSpecialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), probValue, alpha, alpha1, beta, output->lengthOf(), dRandom);

        err = hipFree(dRandom);
        if (err) {
            throw cuda_exception::build("helpers::alphaDropoutSimple: Cannot deallocate device memory for random generator.", err);
        }
        NDArray::registerSpecialUse({output}, {input});
    }

    template <typename T>
    static int alphaDropOutFunctor_(graph::Context& context, NDArray* input, NDArray* output,
                            NDArray* reduceShape, int seed, double probValue, double alpha, double alpha1, double beta) {

        if (reduceShape == nullptr){
            alphaDropoutSimple<T>(context.launchContext(), input, output, seed, probValue, alpha, alpha1, beta);
        }
        else {
            REQUIRE_TRUE(reduceShape->lengthOf() <= input->rankOf(), 0, "dropout: Noise shape should be fittable to input");

            std::vector<Nd4jLong> dims(reduceShape->lengthOf());
            reduceShape->syncToHost(); // to ensure that follows are actual
            bool fit = true;

            for( int i = 0; i < dims.size(); i++ ) {
                if (fit) {
                    dims[i] = reduceShape->e<Nd4jLong>(i);
                    for (int e = 0; e < input->rankOf(); ++e)
                        if (fit)
                            if (input->sizeAt(e) % dims[i]) {
                                fit = false;
                            }
                }
            }

            // check dims to fit input
            REQUIRE_TRUE(fit, 0, "alpha_dropout: Noise shape should fit to input rank.");
            std::unique_ptr<NDArray> chunk(new NDArray('c', dims, output->dataType(), context.launchContext()));
            chunk->assign(1.f);

            alphaDropoutSimple<T>(context.launchContext(), chunk.get(), chunk.get(), seed, probValue, alpha, alpha1, beta);

            // broadcast chunk to full matrix
            std::unique_ptr<NDArray> dropOutMultiplier(new NDArray(*input));
            dropOutMultiplier->assign(1.f);

            *dropOutMultiplier += *chunk;

            output->assign(*input * *dropOutMultiplier); //input->applyPairwiseTransform(pairwise::Multiply, dropOutMultiplier.get(), output, nullptr);
        }


        return Status::OK();
    }

    template <typename T>
    int alphaDropOutFunctorBP_(graph::Context& context, NDArray* input, NDArray* gradOut, NDArray* output,
                              NDArray* reduceShape, int seed, double probValue, double alpha, double alpha1, double beta) {

        int res = alphaDropOutFunctor(context, input, output, reduceShape, seed, probValue, alpha, alpha1, beta);
        if (res == ND4J_STATUS_OK) {
            // FIXME: can we make it single-loop?
            (*output) *= alpha;
            (*output) *= (*gradOut); //->applyPairwiseTransform<transform::Multiply>(gradOut, output, nullptr);
        }
        return res;
    }

    int dropOutFunctorBP(graph::Context& context, NDArray* input, NDArray* gradOut, NDArray* output, NDArray* reduceShape, int seed, double probValue) {
        BUILD_SINGLE_SELECTOR(context.dataType(), return dropOutFunctorBP_, (context, input, gradOut, output, reduceShape, seed, probValue), FLOAT_TYPES);
    }

    int alphaDropOutFunctor(graph::Context& context, NDArray* input, NDArray* output, NDArray* reduceShape, int seed, double probValue, double alpha, double alpha1, double beta) {
        BUILD_SINGLE_SELECTOR(context.dataType(), return alphaDropOutFunctor_, (context, input, output, reduceShape, seed, probValue, alpha, alpha1, beta), FLOAT_TYPES);
    }

    int alphaDropOutFunctorBP(graph::Context& context, NDArray* input, NDArray* gradOut, NDArray* output, NDArray* reduceShape, int seed, double probValue, double alpha, double alpha1, double beta) {
        BUILD_SINGLE_SELECTOR(context.dataType(), return alphaDropOutFunctorBP_, (context, input, gradOut, output, reduceShape, seed, probValue, alpha, alpha1, beta), FLOAT_TYPES);
    }

}
}
}