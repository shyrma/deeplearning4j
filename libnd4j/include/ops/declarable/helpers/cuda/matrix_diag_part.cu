#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by GS <sgazeos@gmail.com> on 3/21/2018.
//

#include "ResultSet.h"
#include <ops/declarable/helpers/matrix_diag_part.h>
#include <Status.h>
#include <ShapeUtils.h>
#include <ShapeUtils.h>
#include <TAD.h>
#include <cuda_exception.h>
#include <helpers/ConstantTadHelper.h>

namespace nd4j {
namespace ops {
namespace helpers {

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// put diagonals from input batched matricies to output batched vectors
    template <typename T>
    static __global__ void matrixDiagPartKernel(void const* inputBuffer, void* outputBuffer, Nd4jLong numTads, Nd4jLong inputLength,
                                            Nd4jLong* tadOnlyInputShapeInfo,  Nd4jLong *tadInputOffsets,
                                            Nd4jLong* tadOnlyOutputShapeInfo, Nd4jLong *tadOutputOffsets) {
        int totalThreads = blockDim.x;
        for (Nd4jLong i = blockIdx.x; i < numTads; i += gridDim.x) {
            auto yOffset = tadInputOffsets[i];
            auto xOffset = tadOutputOffsets[i];
            for (Nd4jLong j = threadIdx.x; j < inputLength; j += totalThreads) {
                Nd4jLong coords[2] = {j, j};
                Nd4jLong tadOffset = shape::getOffset(tadOnlyInputShapeInfo, coords);
                *(reinterpret_cast<T*>(outputBuffer) + xOffset + shape::getIndexOffset(j, tadOnlyOutputShapeInfo)) = *(reinterpret_cast<T const*>(inputBuffer) + yOffset + tadOffset);
            }
        }
    }

//////////////////////////////////////////////////////////////////////////
// Returns a batched matrix tensor with new batched diagonal values.
// for detailed explanations please take a look on web page: https://www.tensorflow.org/api_docs/python/tf/matrix_set_diag
//
    template <typename T>
    int _matrixDiagPart(nd4j::LaunchContext * context, const NDArray* input, NDArray* output) {
        auto stream = context->getCudaStream();
        auto listOut  = output->allTensorsAlongDimension({output->rankOf() - 1});
        auto listDiag = input->allTensorsAlongDimension({input->rankOf() - 2, input->rankOf() - 1});

        if (listOut->size() != listDiag->size()) {
            nd4j_printf("matrix_diag_part: Input matrix has wrong shape.", "");
            return ND4J_STATUS_VALIDATION;
        }
        Nd4jLong lastDimension = nd4j::math::nd4j_min(input->sizeAt(-2), input->sizeAt(-1));

        std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(output->rankOf(), {output->rankOf() - 1});
        const Nd4jLong numTads = ShapeUtils::getNumOfSubArrs(input->getShapeInfo(), dimsToExclude); //this->tensorsAlongDimension({dimension});
        //printf("Repeat delta %lld, numTads %lld\n", repeatDelta, numTads);
        //tadOnlyInputShapeInfo, tadInputOffsets, tadOnlyOutputShapeInfo, tadOutputOffsets;
        std::vector<int> outputDims({output->rankOf() - 1});
        std::vector<int> inputDims({input->rankOf() - 2, input->rankOf() - 1});
        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), inputDims);
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), outputDims);


        if (!output->isActualOnDeviceSide())
            input->syncToDevice();

        if (!input->isActualOnDeviceSide())
            input->syncToDevice();


        dim3 launchDims(256, 512, 8192);
        matrixDiagPartKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(input->getSpecialBuffer(), output->getSpecialBuffer(), numTads, lastDimension, packX.specialShapeInfo(), packX.specialOffsets(), packZ.specialShapeInfo(), packZ.specialOffsets());

        return Status::OK();
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// caller for _matrixDiagPart
//
    int matrixDiagPart(nd4j::LaunchContext * context, const NDArray* input, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return _matrixDiagPart, (context, input, output), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template int _matrixDiagPart, (nd4j::LaunchContext * context, const NDArray* input, NDArray* output), LIBND4J_TYPES);

}
}
}