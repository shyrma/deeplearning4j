#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/scatter.h>
#include <numeric>
#include <helpers/ShapeUtils.h>
#include <TAD.h>
#include <helpers/ConstantShapeHelper.h>
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>

namespace nd4j    {
namespace ops     {
namespace helpers {

            // template<typename T, bool locking>
            // __global__ static void scatterCuda(const int opCode, const int numOfSubArrs,
            //                                          void* vx, const Nd4jLong *xShapeInfo, const Nd4jLong *xOffsets,
            //                                          void* vy, const Nd4jLong *yShapeInfo, const Nd4jLong *yOffsets,
            //                                          const int* indexes, unsigned int arrLenX, unsigned int arrLenY) {

            //     __shared__ T *x, *y;

            //     if (locking) {

            //         for (int e = 0; e < numOfSubArrs; e++) {

            //             const auto xIndex = indexes[e];
            //             const bool isOwner = xIndex < gridDim.x ? blockIdx.x == xIndex : blockIdx.x == xIndex % gridDim.x;

            //             if (!isOwner)
            //                 continue;

            //             if (threadIdx.x == 0) {
            //                 x = reinterpret_cast<T *>(vx) + xOffsets[xIndex];
            //                 y = reinterpret_cast<T *>(vy) + yOffsets[e];
            //             }
            //             __syncthreads();

            //             for (Nd4jLong i = threadIdx.x; i < arrLenX; i += blockDim.x) {

            //                 const auto xOffset = shape::getIndexOffset(i, xShapeInfo);
            //                 const auto yOffset = shape::getIndexOffset(i, yShapeInfo);

            //                 switch (opCode) {
            //                     case pairwise::Add:
            //                         x[xOffset] += y[yOffset];
            //                         break;
            //                     case pairwise::Subtract:
            //                         x[xOffset] -= y[yOffset];
            //                         break;
            //                     case pairwise::Multiply:
            //                         x[xOffset] *= y[yOffset];
            //                         break;
            //                     case pairwise::Divide:
            //                         x[xOffset] /= y[yOffset];
            //                         break;
            //                     case pairwise::ReverseSubtract:
            //                         x[xOffset] = y[yOffset] - x[xOffset];
            //                         break;
            //                     case pairwise::ReverseDivide:
            //                         x[xOffset] = y[yOffset] / x[xOffset];
            //                         break;
            //                     case pairwise::CopyPws:
            //                         x[xOffset] = y[yOffset];
            //                         break;
            //                     default:
            //                         continue;
            //                 }
            //             }
            //             __syncthreads();
            //         }
            //     } else {
            //         for (int e = blockIdx.x; e < numOfSubArrs; e+= gridDim.x) {

            //             if (threadIdx.x == 0) {
            //                 const auto xIndex = indexes[e];
            //                 x = reinterpret_cast<T *>(vx) + xOffsets[xIndex];
            //                 y = reinterpret_cast<T *>(vy) + yOffsets[e];
            //             }
            //             __syncthreads();

            //             for (Nd4jLong i = threadIdx.x; i < arrLenX; i += blockDim.x) {
            //                 const auto xOffset = shape::getIndexOffset(i, xShapeInfo);
            //                 const auto yOffset = shape::getIndexOffset(i, yShapeInfo);

            //                 switch (opCode) {
            //                     case pairwise::Add:
            //                         x[xOffset] += y[yOffset];
            //                         break;
            //                     case pairwise::Subtract:
            //                         x[xOffset] -= y[yOffset];
            //                         break;
            //                     case pairwise::Multiply:
            //                         x[xOffset] *= y[yOffset];
            //                         break;
            //                     case pairwise::Divide:
            //                         x[xOffset] /= y[yOffset];
            //                         break;
            //                     case pairwise::ReverseSubtract:
            //                         x[xOffset] = y[yOffset] - x[xOffset];
            //                         break;
            //                     case pairwise::ReverseDivide:
            //                         x[xOffset] = y[yOffset] / x[xOffset];
            //                         break;
            //                     case pairwise::CopyPws:
            //                         x[xOffset] = y[yOffset];
            //                         break;
            //                     default:
            //                         continue;
            //                 }
            //             }
            //             __syncthreads();
            //         }
            //     }
            // }


            // template <typename T>
            // void scatter_(nd4j::LaunchContext  *context, pairwise::Ops op, const NDArray& indices, const NDArray& updates, NDArray& output, const bool lock) {
            //     std::vector<int> dims = {0};
            //     auto inverted = ShapeUtils::evalDimsToExclude(output.rankOf(), dims);

            //     auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output.getShapeInfo(), inverted);
            //     auto packY = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(updates.getShapeInfo(), inverted);

            //     auto psX = packX.specialShapeInfo();
            //     auto psY = packY.specialShapeInfo();

            //     PointersManager manager(context, "scatter");

            //     auto poX = packX.specialOffsets();
            //     auto poY = packY.specialOffsets();

            //     NDArray::prepareSpecialUse({&output}, {&updates, &indices});

            //     unsigned int tadLengthX = shape::length(packX.primaryShapeInfo());
            //     unsigned int tadLengthY = shape::length(packY.primaryShapeInfo());
            //     if (tadLengthX != tadLengthY)
            //         throw std::runtime_error("scatter: Lengths of TADs must be equal");

            //     auto blockSize = nd4j::math::nd4j_max<int>(32, nd4j::math::nd4j_min<int>(tadLengthX, 1024));

            //     if (lock)
            //         scatterCuda<T, true><<<512, blockSize, 1024, *context->getCudaStream()>>>(op, indices.lengthOf(), output.getSpecialBuffer(), psX, poX, updates.getSpecialBuffer(), psY, poY, reinterpret_cast<int *>(indices.getSpecialBuffer()), tadLengthX, tadLengthY);
            //     else
            //         scatterCuda<T, false><<<512, blockSize, 1024, *context->getCudaStream()>>>(op, indices.lengthOf(), output.getSpecialBuffer(), psX, poX, updates.getSpecialBuffer(), psY, poY, reinterpret_cast<int *>(indices.getSpecialBuffer()), tadLengthX, tadLengthY);

            //      NDArray::registerSpecialUse({&output}, {&updates, &indices});
            //     manager.synchronize();
            // }

///////////////////////////////////////////////////////////////////
// x - indices, y - updates, z - input/output
template<typename X, typename Y>
__global__ static void scatterLockCuda(const int opCode,
                                       const void* vx, const Nd4jLong *xShapeInfo,
                                       const void* vy, const Nd4jLong *yTadShapeInfo, const Nd4jLong *yOffsets,
                                             void* vz, const Nd4jLong *zTadShapeInfo, const Nd4jLong *zOffsets,
                                       const Nd4jLong xLen, const Nd4jLong yTadLen, const Nd4jLong zTadLen) {

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<Y*>(vz);

    __shared__ bool vectorCase;
    if(threadIdx.x == 0)
        vectorCase = yTadLen == xLen && shape::rank(xShapeInfo) <= 1;
    __syncthreads();

    for (int e = 0; e < xLen; e++) {

        const Nd4jLong zIndex = x[shape::getIndexOffset(e, xShapeInfo)];
        const bool isOwner = zIndex < gridDim.x ? blockIdx.x == zIndex : blockIdx.x == zIndex % gridDim.x;

        if (!isOwner)
            continue;

        if(vectorCase) { // means z_rank = 1 and might be yTadLen != zTadLen in this case

            if(threadIdx.x != 0)
                continue;

            const auto yOffset = shape::getIndexOffset(e,      yTadShapeInfo);
            const auto zOffset = shape::getIndexOffset(zIndex, zTadShapeInfo);

            switch (opCode) {
                case pairwise::Add:
                    z[zOffset] += y[yOffset];
                    break;
                case pairwise::Subtract:
                    z[zOffset] -= y[yOffset];
                    break;
                case pairwise::Multiply:
                    z[zOffset] *= y[yOffset];
                    break;
                case pairwise::Divide:
                    z[zOffset] /= y[yOffset];
                    break;
                case pairwise::ReverseSubtract:
                    z[zOffset] = y[yOffset] - z[zOffset];
                    break;
                case pairwise::ReverseDivide:
                    z[zOffset] = y[yOffset] / z[zOffset];
                    break;
                case pairwise::CopyPws:
                    z[zOffset] = y[yOffset];
                    break;
                case pairwise::MaxPairwise:
                    if(z[zOffset] < y[yOffset]) z[zOffset] = y[yOffset];
                    break;
                case pairwise::MinPairwise:
                    if(z[zOffset] > y[yOffset]) z[zOffset] = y[yOffset];
                    break;
                default:
                    continue;
            }
        }
        else {      // yTadLen == zTadLen in this case

            const Y* yTad = y + yOffsets[e];
                  Y* zTad = z + zOffsets[zIndex];

            for (Nd4jLong i = threadIdx.x; i < zTadLen; i += blockDim.x) {

                const auto yOffset = shape::getIndexOffset(i, yTadShapeInfo);
                const auto zOffset = shape::getIndexOffset(i, zTadShapeInfo);

                switch (opCode) {
                    case pairwise::Add:
                        zTad[zOffset] += yTad[yOffset];
                        break;
                    case pairwise::Subtract:
                        zTad[zOffset] -= yTad[yOffset];
                        break;
                    case pairwise::Multiply:
                        zTad[zOffset] *= yTad[yOffset];
                        break;
                    case pairwise::Divide:
                        zTad[zOffset] /= yTad[yOffset];
                        break;
                    case pairwise::ReverseSubtract:
                        zTad[zOffset] = yTad[yOffset] - zTad[zOffset];
                        break;
                    case pairwise::ReverseDivide:
                        zTad[zOffset] = yTad[yOffset] / zTad[zOffset];
                        break;
                    case pairwise::CopyPws:
                        zTad[zOffset] = yTad[yOffset];
                        break;
                    case pairwise::MaxPairwise:
                        if(zTad[zOffset] < yTad[yOffset]) zTad[zOffset] = yTad[yOffset];
                        break;
                    case pairwise::MinPairwise:
                        if(zTad[zOffset] > yTad[yOffset]) zTad[zOffset] = yTad[yOffset];
                        break;
                    default:
                        continue;
                }
            }
        }
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
static void scatterLockCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                    const int opCode,
                                    const void* vx, const Nd4jLong *xShapeInfo,
                                    const void* vy, const Nd4jLong *yTadShapeInfo, const Nd4jLong *yOffsets,
                                          void* vz, const Nd4jLong *zTadShapeInfo, const Nd4jLong *zOffsets,
                                    const Nd4jLong xLen, const Nd4jLong yTadLen, const Nd4jLong zTadLen) {

    scatterLockCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(opCode, vx, xShapeInfo, vy, yTadShapeInfo, yOffsets, vz, zTadShapeInfo, zOffsets, xLen, yTadLen, zTadLen);
}

///////////////////////////////////////////////////////////////////
// x - indices, y - updates, z - input/output
template<typename X, typename Y>
__global__ static void scatterCuda(const int opCode,
                                   const void *vx, const Nd4jLong *xShapeInfo,
                                   const void *vy, const Nd4jLong *yShapeInfo,
                                         void *vz, const Nd4jLong *zShapeInfo) {

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<Y*>(vz);

    __shared__ int xRank, yRank, zRank;
    __shared__ Nd4jLong yLen, totalThreads, *coord;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        coord = reinterpret_cast<Nd4jLong*>(shmem);
        yLen = shape::length(yShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
        xRank = shape::rank(xShapeInfo);
        yRank = shape::rank(yShapeInfo);
        zRank = shape::rank(zShapeInfo);
    }
    __syncthreads();

    auto xCoord = coord + threadIdx.x * (xRank + yRank + zRank);
    auto yCoord = xCoord + xRank;
    auto zCoord = yCoord + yRank;

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < yLen; i += totalThreads) {

        shape::index2coords(i, yShapeInfo, yCoord);

        for (uint j = 0; j < xRank; ++j)
            xCoord[j] = yCoord[j];

        const auto xOffset = shape::getOffset(xShapeInfo, xCoord);
        zCoord[0] = x[xOffset];

        for (uint j = 0; j < yRank - xRank; ++j)
            zCoord[j + 1] = yCoord[xRank + j];

        const auto yOffset = shape::getOffset(yShapeInfo, yCoord);
        const auto zOffset = shape::getOffset(zShapeInfo, zCoord);

        switch (opCode) {
            case pairwise::Add:
                z[zOffset] += y[yOffset];
                break;
            case pairwise::Subtract:
                z[zOffset] -= y[yOffset];
                break;
            case pairwise::Multiply:
                z[zOffset] *= y[yOffset];
                break;
            case pairwise::Divide:
                z[zOffset] /= y[yOffset];
                break;
            case pairwise::ReverseSubtract:
                z[zOffset] = y[yOffset] - z[zOffset];
                break;
            case pairwise::ReverseDivide:
                z[zOffset] = y[yOffset] / z[zOffset];
                break;
            case pairwise::CopyPws:
                z[zOffset] = y[yOffset];
                break;
            case pairwise::MaxPairwise:
                if(z[zOffset] < y[yOffset]) z[zOffset] = y[yOffset];
                break;
            case pairwise::MinPairwise:
                if(z[zOffset] > y[yOffset]) z[zOffset] = y[yOffset];
                break;
            default:
                continue;
        }
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
static void scatterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const int opCode,
                                const void *vx, const Nd4jLong *xShapeInfo,
                                const void *vy, const Nd4jLong *yShapeInfo,
                                      void *vz, const Nd4jLong *zShapeInfo) {

    scatterCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(opCode, vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo);
}


///////////////////////////////////////////////////////////////////
void scatter(nd4j::LaunchContext  *context, pairwise::Ops op, const NDArray& indices, const NDArray& updates, NDArray& output, const bool lock) {

    PointersManager manager(context, "scatter");

    NDArray::prepareSpecialUse({&output}, {&updates, &indices});

    if(lock) {

        const int xRank = indices.rankOf();

        std::vector<int> zTadDims = ShapeUtils::evalDimsToExclude(output.rankOf(), {0});

        int sizeOfUpdDims = xRank;
        if(output.rankOf() == updates.rankOf() && indices.isVector())
            sizeOfUpdDims = 1;

        std::vector<int> yTadDims(sizeOfUpdDims);
        std::iota(yTadDims.begin(), yTadDims.end(), 0);

        auto packY = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(updates.getShapeInfo(), ShapeUtils::evalDimsToExclude(updates.rankOf(), yTadDims));
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output.getShapeInfo(), zTadDims);

        const Nd4jLong zTadLen = shape::length(packZ.primaryShapeInfo());
        const Nd4jLong yTadLen = shape::length(packY.primaryShapeInfo());

        const auto threadsPerBlock = nd4j::math::nd4j_max<int>(32, nd4j::math::nd4j_min<int>(zTadLen, 1024));
        const auto blocksPerGrid = indices.lengthOf();

        const auto xType = indices.dataType();
        const auto yType = updates.dataType();

        BUILD_DOUBLE_SELECTOR(xType, yType, scatterLockCudaLauncher, (blocksPerGrid, threadsPerBlock, 1024, context->getCudaStream(), op, indices.getSpecialBuffer(), indices.getSpecialShapeInfo(), updates.getSpecialBuffer(), packY.specialShapeInfo(), packY.specialOffsets(), output.getSpecialBuffer(), packZ.specialShapeInfo(), packZ.specialOffsets(), indices.lengthOf(), yTadLen, zTadLen), INDEXING_TYPES, GENERIC_NUMERIC_TYPES);
    }
    else {

        const int threadsPerBlock = MAX_NUM_THREADS / 8;
        const int blocksPerGrid = (updates.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
        const int sharedMem = 8 * threadsPerBlock * (indices.rankOf() + updates.rankOf() + output.rankOf()) + 128;

        const auto xType = indices.dataType();
        const auto yType = updates.dataType();

        BUILD_DOUBLE_SELECTOR(xType, yType, scatterCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), op, indices.getSpecialBuffer(), indices.getSpecialShapeInfo(), updates.getSpecialBuffer(), updates.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo()), INDEXING_TYPES, GENERIC_NUMERIC_TYPES);
    }

    NDArray::registerSpecialUse({&output}, {&updates, &indices});
    manager.synchronize();
}


///////////////////////////////////////////////////////////////////
// x - indices, y - updates, z - output
template<typename X, typename Y>
__global__ static void scatterNDLockCuda(const int opCode,
                                         const void* vx, const Nd4jLong *xTadShapeInfo, const Nd4jLong *xOffsets,
                                         const void* vy, const Nd4jLong *yTadShapeInfo, const Nd4jLong *yOffsets,
                                               void* vz, const Nd4jLong *zTadShapeInfo, const Nd4jLong *zOffsets,
                                         const Nd4jLong *zShapeInfo,
                                         const Nd4jLong numOfXTads, const Nd4jLong numOfZTads, const Nd4jLong yTadLen) {

    // zTadLen == yTadLen if numOfZTads > 1, in opposite case z and y are vectors
    // numOfXTads == numOfYTads if numOfZTads > 1, in opposite case z and y are vectors

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<Y*>(vz);

    __shared__ Nd4jLong *zTadCoords;
    __shared__ int xLastDim;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        zTadCoords = reinterpret_cast<Nd4jLong*>(shmem);
        xLastDim = xTadShapeInfo[1];   // xTad has rank = 1 always
    }
    __syncthreads();

    Nd4jLong* zTadCoordsPerThread = zTadCoords + threadIdx.x * xLastDim;

    for (Nd4jLong i = 0; i < numOfXTads; ++i) {

        const X* xTad = x + xOffsets[i];

        for (uint k = 0; k < xLastDim; ++k)
            zTadCoordsPerThread[k] = xTad[shape::getIndexOffset(k, xTadShapeInfo)];

        const auto zTadIndex = shape::coords2index(xLastDim, zShapeInfo + 1, zTadCoordsPerThread);

        const bool isOwner = zTadIndex < gridDim.x ? blockIdx.x == zTadIndex : blockIdx.x == zTadIndex % gridDim.x;

        if(!isOwner)
            continue;

        if(numOfZTads == 1) {     // yTadLen == numOfXTads in this case

            if(threadIdx.x != 0)
                continue;

            const auto yOffset = shape::getIndexOffset(i,         yTadShapeInfo);
            const auto zOffset = shape::getIndexOffset(zTadIndex, zTadShapeInfo);

            switch (opCode) {
                case pairwise::Add:
                    z[zOffset] += y[yOffset];
                    break;
                case pairwise::Subtract:
                    z[zOffset] -= y[yOffset];
                    break;
                case pairwise::Multiply:
                    z[zOffset] *= y[yOffset];
                    break;
                case pairwise::Divide:
                    z[zOffset] /= y[yOffset];
                    break;
                case pairwise::ReverseSubtract:
                    z[zOffset] = y[yOffset] - z[zOffset];
                    break;
                case pairwise::ReverseDivide:
                    z[zOffset] = y[yOffset] / z[zOffset];
                    break;
                case pairwise::CopyPws:
                    z[zOffset] = y[yOffset];
                    break;
                case pairwise::MaxPairwise:
                    if(z[zOffset] < y[yOffset]) z[zOffset] = y[yOffset];
                    break;
                case pairwise::MinPairwise:
                    if(z[zOffset] > y[yOffset]) z[zOffset] = y[yOffset];
                    break;
                default:
                    continue;
            }
        }
        else {
            const auto yTad = y + yOffsets[i];
            const auto zTad = z + zOffsets[zTadIndex];

            for (Nd4jLong j = threadIdx.x; j < yTadLen; j += blockDim.x) {

                const auto yOffset = shape::getIndexOffset(j, yTadShapeInfo);
                const auto zOffset = shape::getIndexOffset(j, zTadShapeInfo);

                switch (opCode) {
                    case pairwise::Add:
                        zTad[zOffset] += yTad[yOffset];
                        break;
                    case pairwise::Subtract:
                        zTad[zOffset] -= yTad[yOffset];
                        break;
                    case pairwise::Multiply:
                        zTad[zOffset] *= yTad[yOffset];
                        break;
                    case pairwise::Divide:
                        zTad[zOffset] /= yTad[yOffset];
                        break;
                    case pairwise::ReverseSubtract:
                        zTad[zOffset] = yTad[yOffset] - zTad[zOffset];
                        break;
                    case pairwise::ReverseDivide:
                        zTad[zOffset] = yTad[yOffset] / zTad[zOffset];
                        break;
                    case pairwise::CopyPws:
                        zTad[zOffset] = yTad[yOffset];
                        break;
                    case pairwise::MaxPairwise:
                        if(zTad[zOffset] < yTad[yOffset]) zTad[zOffset] = yTad[yOffset];
                        break;
                    case pairwise::MinPairwise:
                        if(zTad[zOffset] > yTad[yOffset]) zTad[zOffset] = yTad[yOffset];
                        break;
                    default:
                        continue;
                }
            }
        }
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
static void scatterNDLockCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                      const int opCode,
                                      const void* vx, const Nd4jLong *xTadShapeInfo, const Nd4jLong *xOffsets,
                                      const void* vy, const Nd4jLong *yTadShapeInfo, const Nd4jLong *yOffsets,
                                            void* vz, const Nd4jLong *zTadShapeInfo, const Nd4jLong *zOffsets,
                                      const Nd4jLong *zShapeInfo,
                                      const Nd4jLong numOfXTads, const Nd4jLong numOfZTads, const Nd4jLong zTadLen) {

    scatterNDLockCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(opCode,
                                                                                   vx, xTadShapeInfo, xOffsets,
                                                                                   vy, yTadShapeInfo, yOffsets,
                                                                                   vz, zTadShapeInfo, zOffsets,
                                                                                   zShapeInfo,
                                                                                   numOfXTads, numOfZTads, zTadLen);
}

///////////////////////////////////////////////////////////////////
// x - indices, y - updates, z - output
template<typename X, typename Y>
__global__ static void scatterNDCuda(const int opCode,
                                     const void *vx, const Nd4jLong *xShapeInfo,
                                     const void *vy, const Nd4jLong *yShapeInfo,
                                           void *vz, const Nd4jLong *zShapeInfo) {

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<Y*>(vz);

    __shared__ int xRank, yRank, zRank, xLastDim;
    __shared__ Nd4jLong yLen, totalThreads, *coord;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        coord = reinterpret_cast<Nd4jLong*>(shmem);
        yLen = shape::length(yShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
        xRank = shape::rank(xShapeInfo);
        yRank = shape::rank(yShapeInfo);
        zRank = shape::rank(zShapeInfo);
        xLastDim = xShapeInfo[xRank];
    }
    __syncthreads();

    auto xCoord = coord + threadIdx.x * (xRank + yRank + zRank);
    auto yCoord = xCoord + xRank;
    auto zCoord = yCoord + yRank;

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < yLen; i += totalThreads) {

        shape::index2coords(i, yShapeInfo, yCoord);

        for (uint j = 0; j < xRank - 1; ++j)
            xCoord[j] = yCoord[j];

        for (uint j = 0; j < xLastDim; ++j) {
            xCoord[xRank - 1] = j;
            const auto xOffset = shape::getOffset(xShapeInfo, xCoord);
            zCoord[j] = x[xOffset];
        }

        for (uint j = xLastDim; j < zRank; ++j)
            zCoord[j] = yCoord[yRank - zRank + j];

        const auto yOffset = shape::getOffset(yShapeInfo, yCoord);
        const auto zOffset = shape::getOffset(zShapeInfo, zCoord);

        switch (opCode) {
            case pairwise::Add:
                z[zOffset] += y[yOffset];
                break;
            case pairwise::Subtract:
                z[zOffset] -= y[yOffset];
                break;
            case pairwise::Multiply:
                z[zOffset] *= y[yOffset];
                break;
            case pairwise::Divide:
                z[zOffset] /= y[yOffset];
                break;
            case pairwise::ReverseSubtract:
                z[zOffset] = y[yOffset] - z[zOffset];
                break;
            case pairwise::ReverseDivide:
                z[zOffset] = y[yOffset] / z[zOffset];
                break;
            case pairwise::CopyPws:
                z[zOffset] = y[yOffset];
                break;
            case pairwise::MaxPairwise:
                if(z[zOffset] < y[yOffset]) z[zOffset] = y[yOffset];
                break;
            case pairwise::MinPairwise:
                if(z[zOffset] > y[yOffset]) z[zOffset] = y[yOffset];
                break;
            default:
                continue;
        }
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
static void scatterNDCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                  const int opCode,
                                  const void *vx, const Nd4jLong *xShapeInfo,
                                  const void *vy, const Nd4jLong *yShapeInfo,
                                        void *vz, const Nd4jLong *zShapeInfo) {

    scatterNDCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(opCode, vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
void scatterND(nd4j::LaunchContext  *context, pairwise::Ops op, const NDArray& indices, const NDArray& updates, NDArray& output, const bool lock) {

    const int xRank = indices.rankOf();
    const int yRank = updates.rankOf();
    const int zRank = output.rankOf();

    PointersManager manager(context, "scatterND");

    NDArray::prepareSpecialUse({&output}, {&updates, &indices});

    if(lock) {

        const int xLastDim = indices.sizeAt(-1);

        // y_tad and z_tad have the same shape
        std::vector<int> yTadDims(zRank - xLastDim), zTadDims(zRank - xLastDim);
        for (int j = 0, i = zTadDims.size() - 1; i >=0 ; --i, ++j) {
            yTadDims[i] = yRank - 1 - j;
            zTadDims[i] = zRank - 1 - j;
        }

        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(indices.getShapeInfo(), {xRank - 1});
        auto packY = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(updates.getShapeInfo(), yTadDims);
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output.getShapeInfo(), zTadDims);

        const int threadsPerBlock = MAX_NUM_THREADS / 4;
        const int blocksPerGrid = packZ.numberOfTads();
        const int sharedMem = 8 * threadsPerBlock * xLastDim + 128;

        const auto xType = indices.dataType();
        const auto yType = updates.dataType();

        BUILD_DOUBLE_SELECTOR(xType, yType, scatterNDLockCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), op, indices.getSpecialBuffer(), packX.specialShapeInfo(), packX.specialOffsets(), updates.getSpecialBuffer(), packY.specialShapeInfo(), packY.specialOffsets(), output.getSpecialBuffer(), packZ.specialShapeInfo(), packZ.specialOffsets(), output.getSpecialShapeInfo(), packX.numberOfTads(), packZ.numberOfTads(), shape::length(packY.primaryShapeInfo())), INDEXING_TYPES, GENERIC_NUMERIC_TYPES);
    }
    else {

        const int threadsPerBlock = MAX_NUM_THREADS / 8;
        const int blocksPerGrid = (updates.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
        const int sharedMem = 8 * threadsPerBlock * (xRank + yRank + zRank) + 128;

        const auto xType = indices.dataType();
        const auto yType = updates.dataType();

        BUILD_DOUBLE_SELECTOR(xType, yType, scatterNDCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), op, indices.getSpecialBuffer(), indices.getSpecialShapeInfo(), updates.getSpecialBuffer(), updates.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo()), INDEXING_TYPES, GENERIC_NUMERIC_TYPES);
    }

    NDArray::registerSpecialUse({&output}, {&updates, &indices});
    manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Z>
__global__ void scatterForLossCuda(const void *vx, const Nd4jLong *xShapeInfo,
                                         void *vy, const Nd4jLong *yShapeInfo,
                                         void *vz, const Nd4jLong *zShapeInfo) {

    const auto x = reinterpret_cast<const X*>(vx);
          auto y = reinterpret_cast<Z*>(vy);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ Nd4jLong xLen, *sharedMem;
    __shared__ int xRank;   // xRank = zRank, yRank = xRank + 1

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        xLen  = shape::length(xShapeInfo);
        xRank = shape::rank(xShapeInfo);
    }
    __syncthreads();

    const auto xInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(xInd >= xLen)
        return;

    auto coords = sharedMem + threadIdx.x * (xRank + 1);

    shape::index2coords(xInd, xShapeInfo, coords);

    // y last coordinate
    coords[xRank] = x[shape::getOffset(xShapeInfo, coords)];

    const auto yOffset = shape::getOffset(yShapeInfo, coords);

    if(z == nullptr) { // gradient calculation
        y[yOffset] -= 1.f;
    }
    else {
        z[shape::getOffset(zShapeInfo, coords)] = y[yOffset];
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Z>
static void scatterForLossCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const void *vx, const Nd4jLong* xShapeInfo, void *vy, const Nd4jLong* yShapeInfo, void *vz, const Nd4jLong* zShapeInfo) {

    scatterForLossCuda<X, Z><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
void scatterForLoss(nd4j::LaunchContext* context, const NDArray& indices, NDArray& updates, NDArray& output, const bool calcGrad) {
    // shapes of indices and output must be the same
    // shape of indices should be the same as updates shape with last dimension excluded, for example if updates is {a,b,c} then indices should be {a,b}

    PointersManager manager(context, "scatterForLoss");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (indices.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = updates.rankOf() * sizeof(Nd4jLong) * threadsPerBlock  + 128;

    if(calcGrad) {
        NDArray::prepareSpecialUse({&updates}, {&indices});
        BUILD_DOUBLE_SELECTOR(indices.dataType(), updates.dataType(), scatterForLossCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), indices.getSpecialBuffer(), indices.getSpecialShapeInfo(), updates.specialBuffer(), updates.specialShapeInfo(), nullptr, nullptr), INDEXING_TYPES, FLOAT_TYPES);
        NDArray::registerSpecialUse({&updates}, {&indices});
    }
    else {
        NDArray::prepareSpecialUse({&output}, {&indices, &updates});
        BUILD_DOUBLE_SELECTOR(indices.dataType(), updates.dataType(), scatterForLossCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), indices.getSpecialBuffer(), indices.getSpecialShapeInfo(), updates.getSpecialBuffer(), updates.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo()), INDEXING_TYPES, FLOAT_TYPES);
        NDArray::registerSpecialUse({&output}, {&indices, &updates});
    }

    manager.synchronize();
}

}
}
}

        // PointersManager manager(&context, "NativeOps::concat");
        // PointersManager::printDevContentOnDev<int>(vx, 2);
        // PointersManager::printDevContentOnDev<Nd4jLong>(xShapeInfo, 8);
        // PointersManager::printDevContentOnDev<float>(vy, 8);
        // PointersManager::printDevContentOnDev<Nd4jLong>(yShapeInfo, 8);
        // PointersManager::printDevContentOnDev<Nd4jLong>(zShapeInfo, 8);

        // manager.printDevContentOnHost<int>(indices.getSpecialBuffer(), indices.lengthOf());
        // manager.printDevContentOnHost<Nd4jLong>(indices.getSpecialShapeInfo(), shape::shapeInfoLength(indices.rankOf()));
        // manager.printDevContentOnHost<float>(updates.getSpecialBuffer(), updates.lengthOf());
        // manager.printDevContentOnHost<Nd4jLong>(updates.getSpecialShapeInfo(), shape::shapeInfoLength(updates.rankOf()));
        // manager.printDevContentOnHost<Nd4jLong>(output.getSpecialShapeInfo(), shape::shapeInfoLength(output.rankOf()));
        // printf("!!!!!!!\n");
        // manager.printDevContentOnHost<Nd4jLong>(packX.specialShapeInfo(), 2*shape::rank(packX.primaryShapeInfo()) + 4);
        // manager.printDevContentOnHost<Nd4jLong>(packX.specialOffsets(), packX.numberOfTads());
        // manager.printDevContentOnHost<Nd4jLong>(packY.specialShapeInfo(), 2*shape::rank(packY.primaryShapeInfo()) + 4);
        // manager.printDevContentOnHost<Nd4jLong>(packY.specialOffsets(), packY.numberOfTads());
        // manager.printDevContentOnHost<Nd4jLong>(packZ.specialShapeInfo(), 2*shape::rank(packZ.primaryShapeInfo()) + 4);
        // manager.printDevContentOnHost<Nd4jLong>(packZ.specialOffsets(), packZ.numberOfTads());
        // printf("dddddddd\n");
        // shape::printShapeInfoLinear(packY.primaryShapeInfo());