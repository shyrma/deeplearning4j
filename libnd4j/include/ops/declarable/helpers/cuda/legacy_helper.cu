#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/legacy_helpers.h>
#include <NDArrayFactory.h>
#include <op_boilerplate.h>

namespace nd4j {
namespace ops {
namespace helpers {

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    linkage void cubeDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return y * (3 * x * x);
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void cubeDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), cubeDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //return (x >= X(0.f) ? y: -y);
    template <typename T>
    linkage void reduceNorm1_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return x > T(0.f)? y : -y;
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void reduceNorm1(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), reduceNorm1_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////
    template <typename T>
    linkage void sigmCrossEntropy_(NDArray* logits, NDArray* labels, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return nd4j::math::nd4j_max<T>(x, (T)0.f) - x * y + nd4j::math::nd4j_log<T,T>((T)1.f + nd4j::math::nd4j_exp<T,T>(-nd4j::math::nd4j_abs(x)));
        };

        logits->applyPairwiseLambda(labels, functor, output);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void sigmCrossEntropy(nd4j::LaunchContext * context, NDArray* logits, NDArray* labels, NDArray* output) {
        BUILD_SINGLE_SELECTOR(logits->dataType(), sigmCrossEntropy_, (logits, labels, output), FLOAT_TYPES);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////
    template <typename T>
    linkage void sigmCrossEntropyGrad_(NDArray* logits, NDArray* labels, NDArray* output) {
        // 1 - labels - 1 / (1 + exp(logits))
        auto functor = LAMBDA_TT(x, y) {
            if(x <= 0)
                return static_cast<T>(1.) - y - static_cast<T>(1.) / (static_cast<T>(1.) + nd4j::math::nd4j_exp<T,T>(x));
            auto e = nd4j::math::nd4j_exp<T,T>(-x);
            return static_cast<T>(1.) - y - e / (static_cast<T>(1.) + e);
        };

        logits->applyPairwiseLambda(labels, functor, output);
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void sigmCrossEntropyGrad(nd4j::LaunchContext * context, NDArray* logits, NDArray* labels, NDArray* output) {
        BUILD_SINGLE_SELECTOR(logits->dataType(), sigmCrossEntropyGrad_, (logits, labels, output), FLOAT_TYPES);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //            X f = (X) 1.0f + nd4j::math::nd4j_abs<X>(d1);
    //            return (X) d2 * ((X) 1.0f / (f * f));
    //
    template <typename T>
    linkage void softSignDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            T ss = (T)1.f + nd4j::math::nd4j_abs<T>(x);
            return y * ((T) 1.0f  / (ss * ss));
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void softSignDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), softSignDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    linkage void softPlusDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            T p = nd4j::math::nd4j_pow<T, T, T>(static_cast<T>(M_E), x);
            return y * (p / (p + 1.));
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    void softPlusDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), softPlusDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///
/// \param input
/// \param epsilon
/// \param output
    template <typename T>
    linkage void sigmoidDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            T s = nd4j::math::nd4j_sigmoid<T,T>(x);
            return y * (s * ((T) 1.0f - s));
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    void sigmoidDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), sigmoidDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

    template <typename T>
    linkage void hardSigmoidDerivative_(NDArray* input, NDArray* epsilon, NDArray* output) {
        auto functor = LAMBDA_TT(x, y){
            return y * simdOps::HardSigmoidDerivative<T>::op(x, nullptr);
        };

        input->applyPairwiseLambda(epsilon, functor, output);
    }

    void hardSigmoidDerivative(nd4j::LaunchContext * context, NDArray* theFirst, NDArray* theSecond, NDArray* theOutput) {
        BUILD_SINGLE_SELECTOR(theFirst->dataType(), hardSigmoidDerivative_, (theFirst, theSecond, theOutput), FLOAT_TYPES);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    linkage void logSumExp_(NDArray* input, NDArray* axis, NDArray* output) {
        // reduce along axis with
        std::unique_ptr<NDArray> tempInput(input->dup());
        input->applyTransform(transform::Exp, tempInput.get());
        std::vector<int> axisVector;
        if (axis != nullptr) {
            axisVector.resize(axis->lengthOf());
            for (size_t i = 0; i < axisVector.size(); ++i)
                axisVector[i] = axis->e<int>(i);
        }
        tempInput->reduceAlongDimension(reduce::Sum, output, axisVector);
        output->applyTransform(transform::Log, nullptr, nullptr);
    }

    template <typename T>
    linkage void logSumExp_(NDArray* input, NDArray* subtrah, NDArray* axis, NDArray* output) {
        // reduce along axis with
        std::unique_ptr<NDArray> tempInput(input->dup());
        input->applyPairwiseTransform(pairwise::Subtract, subtrah, tempInput.get());
        tempInput->applyTransform(transform::Exp, nullptr, nullptr);

        std::vector<int> axisVector;
        if (axis != nullptr) {
            axisVector.resize(axis->lengthOf());
            for (size_t i = 0; i < axisVector.size(); ++i)
                axisVector[i] = axis->e<int>(i);
        }
        tempInput->reduceAlongDimension(reduce::Sum, output, axisVector);
        output->applyTransform(transform::Log, nullptr, nullptr);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void logSumExp(nd4j::LaunchContext * context, NDArray* input, NDArray* axis, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), logSumExp_, (input, axis, output), FLOAT_TYPES);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void logSumExp(nd4j::LaunchContext * context, NDArray* input, NDArray* subtrah, NDArray* axis, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), logSumExp_, (input, subtrah, axis, output), FLOAT_TYPES);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    void weightedCrossEntropyWithLogitsFunctor_(NDArray const* targets, NDArray const* input, NDArray const* weights, NDArray* output) {

        T posWeight = weights->e<T>(0);

        auto mainRoutineT1 = LAMBDA_TT(_x, _z, posWeight) {
            T targetWeight = (1. + (posWeight - (T)1.f) * _z);
            return (1. - _z) * _x +
                   targetWeight * (nd4j::math::nd4j_log<T,T>((T)1.f + nd4j::math::nd4j_exp<T,T>(-nd4j::math::nd4j_abs(_x))) +
                                   nd4j::math::nd4j_max(-_x, T(0.f))
                   );
        };

        auto mainRoutineT2 = LAMBDA_TTT(_x, _z, _w) {
            return (((T)1.0 - _z) * _x) +
                   _w * (nd4j::math::nd4j_log<T,T>(T(1.) + nd4j::math::nd4j_exp<T,T>(-nd4j::math::nd4j_abs(_x))) +
                         nd4j::math::nd4j_max(-_x, T(0.f)));
        };


        if (weights->isScalar()) {
            const_cast<NDArray*>(input)->applyPairwiseLambda(const_cast<NDArray*>(targets), mainRoutineT1, output);
        }
        else
        {
            std::unique_ptr<NDArray> targetVector(new NDArray(*weights));
            targetVector->applyScalar(scalar::Add, -1.f);

            std::unique_ptr<NDArray> targetTensor(new NDArray(*targets));
            *targetTensor = (*targetVector * *targetTensor) + T(1.f);
            const_cast<NDArray*>(input)->applyTriplewiseLambda(const_cast<NDArray*>(targets), targetTensor.get(), mainRoutineT2, output);
        }
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void weightedCrossEntropyWithLogitsFunctor(nd4j::LaunchContext * context, NDArray const* targets, NDArray const* input, NDArray const* weights, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {targets, input, weights});

        BUILD_SINGLE_SELECTOR(targets->dataType(), weightedCrossEntropyWithLogitsFunctor_, (targets, input, weights, output), FLOAT_TYPES);

        NDArray::registerSpecialUse({output}, {targets, input, weights});
    }

}
}
}