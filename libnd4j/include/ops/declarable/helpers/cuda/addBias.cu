#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//


#include<ops/declarable/helpers/addBias.h>
#include <PointersManager.h>

namespace nd4j    {
namespace ops     {
namespace helpers {

//////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__global__ static void addBiasCuda( const void* vx, const Nd4jLong* xShapeInfo,
                                    const void* vy, const Nd4jLong* yShapeInfo,
                                          void* vz, const Nd4jLong* zShapeInfo,
                                    const bool isNCHW) {

    // bias [oC]

    // if(input_rank == 4)
        // input and output have same shapes: [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW)
    // if(input_rank == 5)
        // input and output have same shapes: [bS, oD, oH, oW, oC] (NHWC) or [bS, oD, oC, oH, oW] (NCHW)

    const X* x = reinterpret_cast<const X*>(vx);
    const Y* y = reinterpret_cast<const Y*>(vy);
          X* z = reinterpret_cast<X*>(vz);

    __shared__ int rank, channelPosition;
    __shared__ Nd4jLong *sharedMem, len;
    __shared__ bool xzSameOffsets, xzAreSame;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        rank = shape::rank(xShapeInfo);     // xRank == zRank
        xzSameOffsets = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        len = shape::length(xShapeInfo);
        channelPosition = isNCHW ? 1 : rank - 1;        // second or last
        xzAreSame = x == z;
    }
    __syncthreads();

    auto coords = sharedMem + threadIdx.x * rank;

    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {

        shape::index2coords(i, xShapeInfo, coords);

        const auto xOffsets = shape::getOffset(xShapeInfo, coords);
        const auto zOffsets = xzSameOffsets ? xOffsets : shape::getOffset(zShapeInfo, coords);
        const auto yOffsets = shape::getOffset(yShapeInfo, coords + channelPosition);

        if(xzAreSame)
            z[zOffsets] += static_cast<X>(y[yOffsets]);
        else
            z[zOffsets] = x[xOffsets] + static_cast<X>(y[yOffsets]);
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
static void addBiasCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                         const void* vx, const Nd4jLong* xShapeInfo,
                                         const void* vy, const Nd4jLong* yShapeInfo,
                                               void* vz, const Nd4jLong* zShapeInfo,
                                         const bool isNCHW) {

    addBiasCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, isNCHW);
}

//////////////////////////////////////////////////////////////////////////
void addBias(nd4j::graph::Context& block, const NDArray& input, const NDArray& bias, NDArray& output, const bool isNCHW) {

    PointersManager manager(block.launchContext(), "addBias");

    const int threadsPerBlock = MAX_NUM_THREADS;
    const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = input.rankOf() * sizeof(Nd4jLong) * threadsPerBlock  + 128;

    NDArray::prepareSpecialUse({&output}, {&input, &bias});
    BUILD_DOUBLE_SELECTOR(input.dataType(), bias.dataType(), addBiasCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), bias.getSpecialBuffer(), bias.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), isNCHW), FLOAT_TYPES, FLOAT_TYPES);
    NDArray::registerSpecialUse({&output}, {&input, &bias});

    manager.synchronize();
}

}
}
}