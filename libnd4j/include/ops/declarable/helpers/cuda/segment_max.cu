#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>

#include <NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j {
    namespace ops {
        namespace helpers {

            // -------------------------------------------------------------------------------------------------------------- //
            // Segment ops linear kernels
            // -------------------------------------------------------------------------------------------------------------- //

            template<typename T, typename I>
            static __global__ void
            segmentMaxLinearKernel(void *input, Nd4jLong *inputShape, int *starts, int *lengths, Nd4jLong numOfClasses,
                                   void *output, Nd4jLong *outputShape) {
                __shared__                 T *val;
                __shared__                Nd4jLong xLen, zLen, zIndex;
                __shared__                T *x;
                __shared__                T *z;
                __shared__ int threadsPerSegment, start, finish;

                auto segment = blockIdx.x;
                if (threadIdx.x == 0) {
//                    threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
//                    segment = blockIdx.x / threadsPerSegment;
                    x = reinterpret_cast<T *>(input);
                    z = reinterpret_cast<T *>(output);
                    extern __shared__ unsigned char shmem[];
                    val = reinterpret_cast<T *>(shmem);
                    xLen = shape::length(inputShape);
                    zLen = shape::length(outputShape);

                    if (segment < numOfClasses) {
                        zIndex = shape::getIndexOffset(segment, outputShape);
                        start = starts[segment];
                        finish = start + lengths[segment];
                        z[zIndex] = x[shape::getIndexOffset(start, inputShape)];
                        val[segment] = z[zIndex];
                    }

                }
                __syncthreads();

                for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputShape);
                    nd4j::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
                }
            }
            // -------------------------------------------------------------------------------------------------------------- //

            template<typename T, typename I>
            static __global__ void
            unsortedSegmentMaxLinearKernel(void *input, Nd4jLong *inputShape, void *indices, Nd4jLong *indicesShape,
                                           int *starts, int *lengths, Nd4jLong numOfClasses, void *output,
                                           Nd4jLong *outputShape) {
                __shared__                 T *val;
                __shared__                Nd4jLong xLen, zLen, zIndex;
                __shared__                T *x;
                __shared__                T *z;
                __shared__                I *y; //int threadsPerSegment, start, finish;
                auto segment = blockIdx.x;

                if (threadIdx.x == 0) {
                    x = reinterpret_cast<T *>(input);
                    z = reinterpret_cast<T *>(output);
                    y = reinterpret_cast<I *>(indices);
                    xLen = shape::length(inputShape);
                    zLen = shape::length(outputShape);

                    zIndex = shape::getIndexOffset(segment, outputShape);
                    //start = starts[segment];
                    //finish = start + lengths[segment];
                    if (lengths[segment] > 0)
                        z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape)];
                    else
                        z[zIndex] = -DataTypeUtils::max<T>();
                }
                __syncthreads();
                if (lengths[segment] > 0)
                    for (auto e = threadIdx.x + 1; e < xLen; e += blockDim.x) {
                        auto xIndex = shape::getIndexOffset(e, inputShape);
                        auto yIndex = shape::getIndexOffset(e, indicesShape);
                        if (y[yIndex] == segment) {
                            nd4j::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
                        }
                    }
            }
            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            static __global__ void segmentMaxTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads,
                                                       Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf,
                                                       Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets, T filler = 0) {

                __shared__ T* val;
                __shared__ Nd4jLong len, zIndex, total;
                __shared__ T* z;
                __shared__ int start, finish;
                __shared__ I segment;

                if (threadIdx.x == 0) {
                    segment = indices[blockIdx.x]; // / threadsPerSegment;
                    z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
                    len = shape::length(inputTads);

                    start = starts[segment];
                    finish = start + lengths[segment];
                    total = shape::sizeAt(inputShape, 0);
                }
                __syncthreads();

                auto idx = blockIdx.x;
                if (idx <= total) {
                    auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
                    if (blockIdx.x == start) {
                        for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                            auto xIndex = shape::getIndexOffset(e, inputTads);
                            auto zIndex = shape::getIndexOffset(e, outputTads);
                            nd4j::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
                            //z[zIndex] = x[xIndex];
                        }
                    }
                    else {
                        for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                            auto xIndex = shape::getIndexOffset(e, inputTads);
                            auto zIndex = shape::getIndexOffset(e, outputTads);
                            if (lengths[segment])
                                nd4j::math::atomics::nd4j_atomicMax(&z[zIndex], x[xIndex]);
                        }
                    }
                }
            }
            // -------------------------------------------------------------------------------------------------------------- //

            template <typename T, typename I>
            static void segmentMaxFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
                //int numClasses = output->sizeAt(0);
                // if input is a vector: (as if in doc sample)
                //Nd4jLong idx = indices->e<Nd4jLong>(0);
                output->assign(-DataTypeUtils::infOrMax<T>());
                auto stream = context->getCudaStream();
                indices->syncToHost();
                Nd4jLong numOfClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
                NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
                NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});

                classesRangesBegs.assign(indices->lengthOf());
                classesRangesLens.assign(0);
                dim3 dims(256, 512, 256);
                int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
                int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
                fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);

                NDArray::prepareSpecialUse({output}, {input, indices, &classesRangesBegs, &classesRangesLens});

                if (input->isVector()) {

                    segmentMaxLinearKernel<T,I><<<numOfClasses, input->lengthOf(), numOfClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
                }
                else {
                    std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
                    auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
                    auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
                    Nd4jLong* inputTads = packX.specialShapeInfo();
                    Nd4jLong* inputTadOffsets = packX.specialOffsets();
                    Nd4jLong* outputTads = packZ.specialShapeInfo();
                    Nd4jLong* outputTadOffsets = packZ.specialOffsets();
                    segmentMaxTadKernel<T,I><<<packX.numberOfTads(), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
                }
                NDArray::registerSpecialUse({output}, {input, indices, &classesRangesBegs, &classesRangesLens});
            }
            // -------------------------------------------------------------------------------------------------------------- //
            void segmentMaxFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {input, indices});
                BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentMaxFunctor_, (context, input, indices, output), NUMERIC_TYPES, INDEXING_TYPES);
                NDArray::registerSpecialUse({output}, {input, indices});
            }
            // -------------------------------------------------------------------------------------------------------------- //

            template <typename T, typename I>
            static void unsortedSegmentMaxFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
                auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
                output->assign(DataTypeUtils::infOrMax<T>());

                NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
                NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
                classesRangesBegs.assign(indices->lengthOf());
                classesRangesLens.assign(0);
                dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
                fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
                int* begins = reinterpret_cast<int*>(classesRangesBegs.getSpecialBuffer());
                int* lengths = reinterpret_cast<int*>(classesRangesLens.getSpecialBuffer());

                if (input->isVector()) {
                    unsortedSegmentMaxLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
                }
                else {
                    std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
                    auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
                    auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
                    Nd4jLong* inputTads = packX.specialShapeInfo();
                    Nd4jLong* inputTadOffsets = packX.specialOffsets();
                    Nd4jLong* outputTads = packZ.specialShapeInfo();
                    Nd4jLong* outputTadOffsets = packZ.specialOffsets();
                    dims.x = input->sizeAt(0);
                    output->assign(-DataTypeUtils::max<T>());
                    segmentMaxTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
                }

            }
            // -------------------------------------------------------------------------------------------------------------- //
            void unsortedSegmentMaxFunctor(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {input, indices});
                output->nullify();
                BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentMaxFunctor_, (context, input, indices, numOfClasses, output), NUMERIC_TYPES, INDEXING_TYPES);
                NDArray::registerSpecialUse({output}, {input, indices});
            }

            // -------------------------------------------------------------------------------------------------------------- //
            // segment max
            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            static __global__ void segmentMaxBPLinearKernel(void* inputBuf, Nd4jLong* inputShape, void* forwardOutput,
                                                            Nd4jLong* forwardShape, void* eps, Nd4jLong* epsShape, void* indicesBuf, Nd4jLong* indicesShape,
                                                            void* outputBuf, Nd4jLong* outputShape) {
                __shared__ T* x;
                __shared__ T* gradIn;
                __shared__ T* gradOut;
                __shared__ I* y;
                __shared__ T* z;
                __shared__ Nd4jLong xLen, gradLen;

                if (threadIdx.x == 0) {
                    xLen = shape::length(inputShape);
                    x = reinterpret_cast<T*>(inputBuf);
                    y = reinterpret_cast<I*>(indicesBuf);
                    z = reinterpret_cast<T*>(outputBuf);
                    gradIn = reinterpret_cast<T*>(forwardOutput);
                    gradOut = reinterpret_cast<T*>(eps);
                    gradLen = shape::length(epsShape);
                }
                __syncthreads();

                auto start = blockIdx.x * blockDim.x + threadIdx.x;
                auto step = gridDim.x * blockDim.x;

                for (auto e = start; e < xLen; e += step) {

                    auto zOffset = shape::getIndexOffset(e, outputShape);
                    auto xOffset = shape::getIndexOffset(e, inputShape);
                    auto yOffset = shape::getIndexOffset(e, indicesShape);
                    auto classIndex = y[yOffset];
                    auto gradOffsetI = shape::getIndexOffset(classIndex, forwardShape);
                    auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

                    if (nd4j::math::nd4j_abs(gradIn[gradOffsetI] - x[xOffset]) <= T(1.e-6)) {
                        z[zOffset] = gradOut[gradOffsetO];
                    }
                }
            }

            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            static __global__ void segmentMaxBPTadKernel(void* inputBuf, Nd4jLong* inputShape, void* forwardOutput,
                                                         Nd4jLong* forwardShape, void* eps, Nd4jLong* epsShape, void* indicesBuf, Nd4jLong* indicesShape,
                                                         void* outputBuf, Nd4jLong* outputShape,Nd4jLong* inputTad,
                                                         Nd4jLong* inputOffsets, Nd4jLong* gradInTad, Nd4jLong* gradInOffsets,
                                                         Nd4jLong* gradOutTad, Nd4jLong* gradOutOffsets, Nd4jLong* outTad,
                                                         Nd4jLong* outOffsets) {
                __shared__ T* x;
                __shared__ T* gradIn;
                __shared__ T* gradOut;
                __shared__ I* y;
                __shared__ T* z;
                __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

                if (threadIdx.x == 0) {
                    xLen = shape::length(inputShape);
                    x = reinterpret_cast<T*>(inputBuf);
                    y = reinterpret_cast<I*>(indicesBuf);
                    z = reinterpret_cast<T*>(outputBuf);
                    yLen = shape::length(indicesShape);
                    gradOut = reinterpret_cast<T*>(eps);
                    gradIn = reinterpret_cast<T*>(forwardOutput);
                    gradLen = shape::length(epsShape);
                    currentLen = shape::length(outTad);
                }
                __syncthreads();

                for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
                    auto yIndex = shape::getIndexOffset(i, indicesShape);
                    auto segment = y[yIndex];
                    T* current = x + inputOffsets[i];
                    T* currentOut = z + outOffsets[i];
                    T* in = gradIn + gradInOffsets[segment];
                    T* outGrad = gradOut + gradOutOffsets[segment];

                    for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                        if (nd4j::math::nd4j_abs(in[e] - current[e]) <= T(1.e-6))
                            currentOut[e] = outGrad[e];
                    }
                }
            }
            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            int segmentMaxFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
                //int numOfClasses = gradOut->sizeAt(0);
                // if input is a vector: (as if in doc sample)
                auto stream = context->getCudaStream();
                NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
                segmentMaxFunctor_<T, I>(context, input, indices, &tempRes);
                NDArray::prepareSpecialUse({output}, {input, indices, gradOut, &tempRes});
                if (input->isVector()) {
                    Nd4jLong loop_size = input->lengthOf();
                    auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
                    segmentMaxBPLinearKernel<T,I><<<1 + gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                            tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                            indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
                }
                else {
                    std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
                    auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
                    auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
                    auto packGradIn = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempRes.getShapeInfo(), dimensions);
                    auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
                    Nd4jLong* inputTads = packX.specialShapeInfo();
                    Nd4jLong* inputTadOffsets = packX.specialOffsets();
                    Nd4jLong* outputTads = packZ.specialShapeInfo();
                    Nd4jLong* outputTadOffsets = packZ.specialOffsets();
                    Nd4jLong* gradInTads = packGradIn.specialShapeInfo();
                    Nd4jLong* gradInTadOffsets = packGradIn.specialOffsets();
                    Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
                    Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

                    segmentMaxBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                            tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                            indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                            inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                            outputTads, outputTadOffsets);
                }
                NDArray::registerSpecialUse({output}, {input, indices, gradOut, &tempRes});
                return Status::OK();
            }
            // -------------------------------------------------------------------------------------------------------------- //
            int segmentMaxFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
                BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentMaxFunctorBP_, (context, input,
                        indices, gradOut, output), FLOAT_TYPES, INDEXING_TYPES);
                NDArray::registerSpecialUse({output}, {input, indices, gradOut});
            }

            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            static int unsortedSegmentMaxFunctorBP_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
                //int numOfClasses = gradOut->sizeAt(0);
                // if input is a vector: (as if in doc sample)
                auto stream = context->getCudaStream();
                NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
                unsortedSegmentMaxFunctor_<T, I>(context, input, indices, numOfClasses, &tempRes);
                NDArray::prepareSpecialUse({output}, {input, indices, gradOut, &tempRes});
                if (input->isVector()) {
                    Nd4jLong loop_size = input->lengthOf();
                    auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
                    segmentMaxBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                            tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                            indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
                }
                else {
                    std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
                    auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
                    auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
                    auto packGradIn = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempRes.getShapeInfo(), dimensions);
                    auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
                    Nd4jLong* inputTads = packX.specialShapeInfo();
                    Nd4jLong* inputTadOffsets = packX.specialOffsets();
                    Nd4jLong* outputTads = packZ.specialShapeInfo();
                    Nd4jLong* outputTadOffsets = packZ.specialOffsets();
                    Nd4jLong* gradInTads = packGradIn.specialShapeInfo();
                    Nd4jLong* gradInTadOffsets = packGradIn.specialOffsets();
                    Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
                    Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

                    segmentMaxBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                            tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                            indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                            inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                            outputTads, outputTadOffsets);
                }
                NDArray::registerSpecialUse({output}, {input, indices, gradOut, &tempRes});
                return Status::OK();
            }
            // -------------------------------------------------------------------------------------------------------------- //
            int unsortedSegmentMaxFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
                BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentMaxFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), FLOAT_TYPES, INDEXING_TYPES);
                NDArray::registerSpecialUse({output}, {input, indices, gradOut});
            }
        }
    }
}