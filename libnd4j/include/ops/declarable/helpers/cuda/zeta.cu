#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 26.04.2019
//

#include<ops/declarable/helpers/zeta.h>

namespace nd4j {
namespace ops {
namespace helpers {


///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void zetaCuda(const void *vx, const Nd4jLong *xShapeInfo,
                                const void *vq, const Nd4jLong *qShapeInfo,
                                      void *vz, const Nd4jLong *zShapeInfo) {

    const auto x = reinterpret_cast<const T*>(vx);
    const auto q = reinterpret_cast<const T*>(vq);
          auto z = reinterpret_cast<T*>(vz);

    __shared__ Nd4jLong len;

    if (threadIdx.x == 0)
        len = shape::length(xShapeInfo);
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    const auto totalThreads = gridDim.x * blockDim.x;

    for (int i = tid; i < len; i += totalThreads) {

        const auto xOffset = shape::getIndexOffset(i, xShapeInfo);
        const auto qOffset = shape::getIndexOffset(i, qShapeInfo);
        const auto zOffset = shape::getIndexOffset(i, zShapeInfo);

        z[zOffset] = zetaScalar<T>(x[xOffset], q[qOffset]);
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
static void zetaCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void *vx, const Nd4jLong *xShapeInfo, const void *vq, const Nd4jLong *qShapeInfo, void *vz, const Nd4jLong *zShapeInfo) {

    zetaCuda<T><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(vx, xShapeInfo, vq, qShapeInfo, vz, zShapeInfo);
}

void zeta(nd4j::LaunchContext * context, const NDArray& x, const NDArray& q, NDArray& z) {

    if(!x.isActualOnDeviceSide()) x.syncToDevice();
    if(!q.isActualOnDeviceSide()) q.syncToDevice();

    int threadsPerBlock = MAX_NUM_THREADS / 2;
    int blocksPerGrid = (z.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    BUILD_SINGLE_SELECTOR(x.dataType(), zetaCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), x.getSpecialBuffer(), x.getSpecialShapeInfo(), q.getSpecialBuffer(), q.getSpecialShapeInfo(), z.getSpecialBuffer(), z.getSpecialShapeInfo()), FLOAT_TYPES);

    x.tickReadHost();
    q.tickReadHost();
    z.tickWriteDevice();
}

BUILD_SINGLE_TEMPLATE(template void zetaCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void *vx, const Nd4jLong *xShapeInfo, const void *vq, const Nd4jLong *qShapeInfo, void *vz, const Nd4jLong *zShapeInfo), FLOAT_TYPES);


}
}
}

