#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/convolutions.h>
#include <ops/declarable/helpers/im2col.h>
#include <ops/declarable/helpers/col2im.h>
#include<ops/declarable/helpers/addBias.h>
#include <exceptions/cuda_exception.h>
#include <NDArrayFactory.h>
#include <MmulHelper.h>
#include <PointersManager.h>
#include <templatemath.h>

namespace nd4j {
namespace ops  {

//////////////////////////////////////////////////////////////////////////
// vol [bS, iC, iD, iH, iW] is convoluted to col [bS, iC, kD, kH, kW, oD, oH, oW]
template <typename T>
static __global__ void vol2colCuda(const void* volume, const Nd4jLong* volShapeInfo, void* columns, const Nd4jLong* colShapeInfo,  const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    const T* vol = reinterpret_cast<const T*>(volume);
          T* col = reinterpret_cast<T*>(columns);

    __shared__ int colRank, volRank;
    __shared__ Nd4jLong colLen, iD, iH, iW, *sharedMem;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        volRank = 5;
        colRank = 8;

        colLen = shape::length(colShapeInfo);

        iD = volShapeInfo[3];
        iH = volShapeInfo[4];
        iW = volShapeInfo[5];
    }
    __syncthreads();

    const auto colInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(colInd >= colLen)
        return;

    auto coords = sharedMem + threadIdx.x * colRank;

    shape::index2coords(colInd, colShapeInfo, coords);

    // const auto colW = coords[7];
    // const auto colH = coords[6];
    // const auto colD = coords[5];
    // const auto kCol = coords[4];
    // const auto kRow = coords[3];
    // const auto kDep = coords[2];
    // const auto c    = coords[1];
    // const auto b    = coords[0];

    const auto colOffset = shape::getOffset(colShapeInfo, coords);

    coords[2] = -pD + coords[2] * dD + coords[5] * sD;     // const auto volDep = (-pD + kDep * dD) + colD * sD;
    coords[3] = -pH + coords[3] * dH + coords[6] * sH;     // const auto volRow = (-pH + kRow * dH) + colH * sH;
    coords[4] = -pW + coords[4] * dW + coords[7] * sW;     // const auto volCol = (-pW + kCol * dW) + colW * sW;

    if (static_cast<unsigned>(coords[2]) >= static_cast<unsigned>(iD) || static_cast<unsigned>(coords[3]) >= static_cast<unsigned>(iH) || static_cast<unsigned>(coords[4]) >= static_cast<unsigned>(iW))
        col[colOffset] = static_cast<T>(0.);
    else
        col[colOffset] = vol[shape::getOffset(volShapeInfo, coords)];
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void vol2colCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const void* volume, const Nd4jLong* volShapeInfo,
                                      void* columns, const Nd4jLong* colShapeInfo,
                                const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    vol2colCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(volume, volShapeInfo, columns, colShapeInfo,  sD, sH, sW, pD, pH, pW, dD, dH, dW);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::vol2col(nd4j::graph::Context& block, const NDArray& vol, NDArray& col, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    PointersManager manager(block.launchContext(), "vol2col");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (col.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = col.rankOf() * sizeof(Nd4jLong) * threadsPerBlock  + 128;

    NDArray::prepareSpecialUse({&col}, {&vol});
    BUILD_SINGLE_SELECTOR(vol.dataType(), vol2colCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), vol.getSpecialBuffer(), vol.getSpecialShapeInfo(), col.specialBuffer(), col.specialShapeInfo(), sD, sH, sW, pD, pH, pW, dD, dH, dW), FLOAT_TYPES);
    NDArray::registerSpecialUse({&col}, {&vol});

    manager.synchronize();
}

//////////////////////////////////////////////////////////////////////////
// columns [bS, iC, kD, kH, kW, oD, oH, oW] to be de-convoluted to volume [bS, iC, iD, iH, iW]
template <typename T>
static __global__ void col2volCuda(const void* columns, const Nd4jLong* colShapeInfo, void* volume, const Nd4jLong* volShapeInfo,  const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    const T* col = reinterpret_cast<const T*>(columns);
          T* vol = reinterpret_cast<T*>(volume);

    __shared__ int colRank, volRank, kDeff, kHeff, kWeff, oD, oH, oW;
    __shared__ Nd4jLong *sharedMem, volLen;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        oD = colShapeInfo[6];
        oH = colShapeInfo[7];
        oW = colShapeInfo[8];

        kDeff = colShapeInfo[3] + (colShapeInfo[3] - 1) * (dD - 1);
        kHeff = colShapeInfo[4] + (colShapeInfo[4] - 1) * (dH - 1);
        kWeff = colShapeInfo[5] + (colShapeInfo[5] - 1) * (dW - 1);

        volRank = 5;
        colRank = 8;

        volLen = shape::length(volShapeInfo);
    }
    __syncthreads();

    const auto volInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(volInd >= volLen)
        return;

    auto coords = sharedMem + threadIdx.x * colRank;

    shape::index2coords(volInd, volShapeInfo, coords);

    const auto volOffset = shape::getOffset(volShapeInfo, coords);

    const int imD = coords[2] + pD;
    const int imH = coords[3] + pH;
    const int imW = coords[4] + pW;

    const int colDstart = (imD < kDeff) ? 0 : (imD - kDeff) / sD + 1;
    const int colHstart = (imH < kHeff) ? 0 : (imH - kHeff) / sH + 1;
    const int colWstart = (imW < kWeff) ? 0 : (imW - kWeff) / sW + 1;

    const int colDend = nd4j::math::nd4j_min<uint>(imD / sD + 1, oD);
    const int colHend = nd4j::math::nd4j_min<uint>(imH / sH + 1, oH);
    const int colWend = nd4j::math::nd4j_min<uint>(imW / sW + 1, oW);

    T val = 0;

    for(coords[5] = colDstart; coords[5] < colDend; ++coords[5]) {
        coords[2] = imD - coords[5] * sD;

        for(coords[6] = colHstart; coords[6] < colHend; ++coords[6]) {
            coords[3] = imH - coords[6] * sH;

            for(coords[7] = colWstart; coords[7] < colWend; ++coords[7]) {
                coords[4] = imW - coords[7] * sW;

                if(coords[2] % dD == 0 && coords[3] % dH == 0 && coords[4] % dW == 0) {
                    coords[2] /= dD;
                    coords[3] /= dH;
                    coords[4] /= dW;

                    val += col[shape::getOffset(colShapeInfo, coords)];
                }
            }
        }
    }

    vol[volOffset] = val;
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void col2volCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const void* columns, const Nd4jLong* colShapeInfo,
                                      void* volume, const Nd4jLong* volShapeInfo,
                                const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    col2volCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(columns, colShapeInfo, volume, volShapeInfo, sD, sH, sW, pD, pH, pW, dD, dH, dW);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::col2vol(nd4j::graph::Context& block, const NDArray& col, NDArray& vol, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    PointersManager manager(block.launchContext(), "col2vol");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (vol.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = col.rankOf() * sizeof(Nd4jLong) * threadsPerBlock  + 128;

    NDArray::prepareSpecialUse({&vol}, {&col});
    BUILD_SINGLE_SELECTOR(vol.dataType(), col2volCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), col.getSpecialBuffer(), col.getSpecialShapeInfo(), vol.specialBuffer(), vol.specialShapeInfo(), sD, sH, sW, pD, pH, pW, dD, dH, dW), FLOAT_TYPES);
    NDArray::registerSpecialUse({&vol}, {&col});

    manager.synchronize();
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
static void conv2d_(nd4j::graph::Context& block, const NDArray* input, const NDArray* weights, const NDArray* bias, NDArray* output, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

    // input   [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
    // weights [kH, kW, iC, oC] always
    // bias    [oC]
    // output  [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW)

    // kH  filter(kernel) height
    // kW  filter(kernel) width
    // sH  strides height
    // sW  strides width
    // pH  paddings height
    // pW  paddings width
    // dH  dilations height
    // dW  dilations width
    // isSameMode 0-VALID, 1-SAME
    // isNCHW     1-NCHW,  0-NHWC

    int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
    int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, *input, *output, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);

    if(isSameMode)                       // SAME
        ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW);

    std::vector<int> permutForOutput;

    if(isNCHW)
        permutForOutput = {0, 3, 1, 2};                                             // [bS, oH, oW, oC] -> [bS, oC, oH, oW]
    else
        input = new NDArray(input->permute({0, 3, 1, 2}));                         // [bS, iH, iW, iC] -> [bS, iC, iH, iW] if NHWC

    NDArray col('c', {bS, oH, oW, kH, kW, iC}, input->dataType(), input->getContext());
    NDArray colP = col.permute({0, 5, 3, 4, 1, 2});            // {bS, iC, kH, kW, oH, oW}
    NDArray mmulResult('f', {bS*oH*oW, oC}, output->dataType(), output->getContext());

    //----- calculation of output -----//
    auto ctx = block.launchContext();
    helpers::im2col(*ctx, *input, colP, kH, kW, sH, sW, pH, pW, dH, dW, NDArrayFactory::create(0.f, input->getContext()));  // [bS, iC, iH, iW] is convoluted to [bS, iC, kH, kW, oH, oW]
    MmulHelper::tensorDot(&col, weights, &mmulResult, {3,4,5}, {0,1,2}, {}); // [bS, oH, oW, kH, kW, iC] x [kH, kW, iC, oC] = [bS, oH, oW, oC]

    //----- assign outTemp to output  -----//
    if(isNCHW) {
        mmulResult.reshapei({bS, oH, oW, oC});
        mmulResult.permutei(permutForOutput);
    }
    output->assign(mmulResult);

    //----- add biases if required -----//
    if(bias)
        // output->applyBroadcast(broadcast::Add, {indIOioC}, bias);
        helpers::addBias(block, *output, *bias, *output, isNCHW);

    if(!isNCHW)
        delete input;

}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::conv2d(nd4j::graph::Context& block, const NDArray* input, const NDArray* weights, const NDArray* bias, NDArray* output, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {
    BUILD_SINGLE_SELECTOR_TWICE(input->dataType(), conv2d_, (block, input, weights, bias, output, kH, kW, sH, sW, pH, pW, dH, dW, isSameMode, isNCHW), FLOAT_TYPES);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
static void depthwiseConv2d_(nd4j::graph::Context& block, const NDArray* input, const NDArray* weights, const NDArray* bias, NDArray* output, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

    // input     [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
    // weights   [kH, kW, iC, mC] always
    // bias      [oC] = iC*mC
    // output    [bS, oH, oW, iC*mC] (NHWC) or [bS, iC*mC, oH, oW] (NCHW)

    // kH           filter(kernel) height
    // kW           filter(kernel) width
    // sH           strides height
    // sW           strides width
    // pH           paddings height
    // pW           paddings width
    // dH           dilations height
    // dW           dilations width
    // isSameMode   0-VALID, 1-SAME
    // isNCHW       0-NCHW,  1-NHWC

    int bS, iC, iH, iW, mC, oC, oH, oW;                     // batch size, input channels, input height/width, channels multiplier(oC = iC*mC), output channels, output height/width
    int indIOioC, indIiH, indWmC, indWiC, indWkH, indOoH;   // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, *input, *output, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWmC, indWkH, indOoH);
    mC = weights->sizeAt(indWmC);                           // channels multiplier

    std::vector<std::vector<Nd4jLong>> modifColumns = {{1,0,4,5,2,3}, {iC,bS*oH*oW,kH*kW}};  // [bS,iC,kH,kW,oH,oW] -> [iC,bS,oH,oW,kH,kW] -> [iC,bS*oH*oW,kH*kW]
    std::vector<std::vector<Nd4jLong>> modifOutput;
    std::vector<Nd4jLong> outReShape;

    if(!isNCHW) {
        outReShape = {bS, oH, oW, iC, mC};                                              // [bS,oH,oW,iC*mC] -> [bS,oH,oW,iC,mC]
        modifOutput = {{3,0,1,2,4},{iC, bS*oH*oW, mC}};                                 // [bS,oH,oW,iC,mC] -> [iC,bS,oH,oW,mC] -> [iC,bS*oH*oW,mC]
        input = new NDArray(input->permute({0, 3, 1, 2}));                             // [bS,iH,iW,iC]    -> [bS,iC,iH,iW]
    }
    else {
        outReShape = {bS, iC, mC, oH, oW};                                              // [bS,iC*mC,oH,oW] -> [bS,iC,mC,oH,oW]
        modifOutput = {{1,0,3,4,2},{iC, bS*oH*oW, mC}};                                 // [bS,iC,mC,oH,oW] -> [iC,bS,oH,oW,mC] -> [iC,bS*oH*oW,mC]
    }

    if(isSameMode)                       // SAME
        ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW);

    NDArray columns(input->ordering(), {bS, iC, kH, kW, oH, oW}, input->dataType(), input->getContext());
    NDArray outputReshaped = output->reshape(output->ordering(), outReShape);

    helpers::im2col(*output->getContext(), *input, columns, kH, kW, sH, sW, pH, pW, dH, dW, NDArrayFactory::create(0.f, input->getContext()));  // [bS, iC, iH, iW] is convoluted to [bS, iC, kH, kW, oH, oW]
    MmulHelper::tensorDot(&columns, weights, &outputReshaped, modifColumns, {{2,0,1,3},{iC,kH*kW,mC}}, modifOutput);              // [iC, bS*oH*oW, kW*kH] x [iC, kH*kW, mC] = [iC, bS*oH*oW, mC]

    if(bias)
        // output->applyBroadcast(broadcast::Add, {indIOioC}, bias);
        helpers::addBias(block, *output, *bias, *output, isNCHW);

    if(!isNCHW)
        delete input;
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::depthwiseConv2d(nd4j::graph::Context& block, const NDArray* input, const NDArray* weights, const NDArray* bias, NDArray* output, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {
    BUILD_SINGLE_SELECTOR_TWICE(input->dataType(), depthwiseConv2d_, (block, input, weights, bias, output, kH, kW, sH, sW, pH, pW, dH, dW, isSameMode, isNCHW), FLOAT_TYPES);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
static void sconv2d_(nd4j::graph::Context& block, const NDArray* input, const NDArray* weightsDepth, const NDArray* weightsPoint, const NDArray* bias,  NDArray* output, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

    // input         [bS, iH, iW, iC]  (NHWC) or [bS, iC, iH, iW]  (NCHW)
    // weightsDepth  [kH, kW, iC, mC]  always
    // weightsPoint  [1, 1, iC*mC, oC] always
    // bias          [oC], oC = iC*mC if weightsPoint=nullptr
    // output is     [bS, oH, oW, oC]  (NHWC) or [bS, oC, oH, oW]  (NCHW)

    //  kH         filter(kernel) height
    //  kW         filter(kernel) width
    //  sH         strides height
    //  sW         strides width
    //  pH         paddings height
    //  pW         paddings width
    //  dH         dilations height
    //  dW         dilations width
    //  isSameMode 0-VALID, 1-SAME
    //  isNCHW     1-NCHW,  0-NHWC

    int bS, iC, iH, iW, mC, oC, oH, oW;                     // batch size, input channels, input height/width, channels multiplier, output channels, output height/width
    int indIOioC, indIiH, indWmC, indWiC, indWkH, indOoH;   // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, *input, *output, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWmC, indWkH, indOoH);
    mC = weightsDepth->sizeAt(indWmC);                      // channels multiplier

    NDArray* outputDepth = output;
    if(weightsPoint)                        // if pointwise convolution is expected
        outputDepth = new NDArray(output->ordering(), !isNCHW ? std::vector<Nd4jLong>({bS, oH, oW, iC*mC}) : std::vector<Nd4jLong>({bS, iC*mC, oH, oW}), input->dataType(), input->getContext());

    // ----- perform depthwise convolution (if weightsPoint is absent then oC = iC*mC) ----- //
    ConvolutionUtils::depthwiseConv2d(block, input, weightsDepth, weightsPoint ? nullptr : bias, outputDepth, kH,kW, sH,sW, pH,pW, dH,dW, isSameMode, isNCHW);

    // ----- perform pointwise convolution (oH = iH, oW = iW) ----- //
    if (weightsPoint) {
        ConvolutionUtils::conv2d(block, outputDepth, weightsPoint, bias, output, 1,1, 1,1, 0,0, 1,1, isSameMode, isNCHW);             // in this case oH=iH, oW=iW
        delete outputDepth;
    }
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::sconv2d(nd4j::graph::Context& block, const NDArray* input, const NDArray* weightsDepth, const NDArray* weightsPoint, const NDArray* bias,  NDArray* output, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {
    BUILD_SINGLE_SELECTOR_TWICE(input->dataType(), sconv2d_, (block, input, weightsDepth, weightsPoint, bias, output, kH, kW, sH, sW, pH, pW, dH, dW, isSameMode, isNCHW), FLOAT_TYPES);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static __global__ void avgPooling2dCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {

    // input is  [bS, iC, iH, iW]
    // output is [bS, iC, oH, oW]

    const auto x = reinterpret_cast<const X*>(vx);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ int bS, iC, oH, oW, iH, iW, strideB, strideC, strideY, strideX, strideOB, strideOC, strideOY, strideOX, length, kHEff, kWEff;

    if (threadIdx.x == 0) {
        bS = shape::sizeAt(xShapeInfo, 0);
        iC = shape::sizeAt(xShapeInfo, 1);
        oH = shape::sizeAt(zShapeInfo, 2);
        oW = shape::sizeAt(zShapeInfo, 3);
        iH = shape::sizeAt(xShapeInfo, 2);
        iW = shape::sizeAt(xShapeInfo, 3);

        strideB = shape::stride(xShapeInfo)[0];
        strideC = shape::stride(xShapeInfo)[1];
        strideY = shape::stride(xShapeInfo)[2];
        strideX = shape::stride(xShapeInfo)[3];

        strideOB = shape::stride(zShapeInfo)[0];
        strideOC = shape::stride(zShapeInfo)[1];
        strideOY = shape::stride(zShapeInfo)[2];
        strideOX = shape::stride(zShapeInfo)[3];

        length = shape::length(zShapeInfo);

        //Replace kernel H/W with *effective* kernel H/W accounting for dilatyon
        kHEff = kH + (kH-1)*(dH-1);
        kWEff = kW + (kW-1)*(dW-1);
    }
    __syncthreads();

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int index = tid; index < length; index += blockDim.x * gridDim.x) {

        const int pw = index % oW;
        const int ph = (index / oW) % oH;
        const int c = (index / oW / oH) % iC;
        const int n = index / oW / oH / iC;

        int hstart = sH * ph - pH;
        int wstart = sW * pw - pW;
        int hend = hstart + kHEff;
        int wend = wstart + kWEff;

        if(hstart < 0){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) -hstart / (Z)dH);
            hstart += f * dH;
        }
        if(wstart < 0){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) -wstart / (Z) dW);
            wstart += f * dW;
        }
        if(hend > iH){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) (hend-iH) / (Z) dH);
            hend -= f * dH;
        }
        if(wend > iW){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) (wend-iW) / (Z) dW);
            wend -= f * dW;
        }

        //Accounts for dilation
        int pool_size = nd4j::math::nd4j_ceil<double,int>((double) (hend-hstart) / (double) dH) * nd4j::math::nd4j_ceil<double,int>((double) (wend-wstart) / (double) dW);

        Z sum = 0.0f;

        const X *inSlice = x + (n * strideB + c * strideC);

        for (int h = hstart; h < hend; h += dH)
            for (int w = wstart; w < wend; w += dW)
                sum += static_cast<Z>(inSlice[h * strideY + w * strideX]);

        int divide_factor = pool_size;  //Case 0: exclude padding
        if (extraParam0 == 1)     //Case 1: include padding
            divide_factor = kH * kW;

        z[n * strideOB + c * strideOC + pw * strideOX + ph * strideOY] = sum / static_cast<Z>(divide_factor);
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static void avgPooling2dCudaLauncher(nd4j::LaunchContext & block, void *vx, Nd4jLong *vxShapeInfo, void *vz, Nd4jLong *vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {
    avgPooling2dCuda<X, Z><<<512, 512, 4192, *block.getCudaStream()>>>(vx, vxShapeInfo, vz, vzShapeInfo, kH, kW, sH, sW, pH, pW, dH, dW, extraParam0);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static __global__ void pnormPooling2dCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {

    // input is  [bS, iC, iH, iW]
    // output is [bS, iC, oH, oW]

    const auto x = reinterpret_cast<const X*>(vx);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ int bS, iC, oH, oW, iH, iW, strideB, strideC, strideY, strideX, strideOB, strideOC, strideOY, strideOX, length, kHEff, kWEff;
    __shared__ bool fOrder;

    if (threadIdx.x == 0) {
        bS = shape::sizeAt(xShapeInfo, 0);
        iC = shape::sizeAt(xShapeInfo, 1);
        oH = shape::sizeAt(zShapeInfo, 2);
        oW = shape::sizeAt(zShapeInfo, 3);
        iH = shape::sizeAt(xShapeInfo, 2);
        iW = shape::sizeAt(xShapeInfo, 3);

        strideB = shape::stride(xShapeInfo)[0];
        strideC = shape::stride(xShapeInfo)[1];
        strideY = shape::stride(xShapeInfo)[2];
        strideX = shape::stride(xShapeInfo)[3];

        strideOB = shape::stride(zShapeInfo)[0];
        strideOC = shape::stride(zShapeInfo)[1];
        strideOY = shape::stride(zShapeInfo)[2];
        strideOX = shape::stride(zShapeInfo)[3];

        length = shape::length(zShapeInfo);

        //Replace kernel H/W with *effective* kernel H/W accounting for dilatyon
        kHEff = kH + (kH-1)*(dH-1);
        kWEff = kW + (kW-1)*(dW-1);
    }
    __syncthreads();

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int index = tid; index < length; index += blockDim.x * gridDim.x) {

        const int pw = index % oW;
        const int ph = (index / oW) % oH;
        const int c = (index / oW / oH) % iC;
        const int n = index / oW / oH / iC;

        int hstart = sH * ph - pH;
        int wstart = sW * pw - pW;
        int hend = hstart + kHEff;
        int wend = wstart + kWEff;

        if (hstart < 0) {
            int f = nd4j::math::nd4j_ceil<Z, int>((Z) -hstart / (Z) dH);
            hstart += f * dH;
        }
        if (wstart < 0) {
            int f = nd4j::math::nd4j_ceil<Z, int>((Z) -wstart / (Z) dW);
            wstart += f * dW;
        }
        if (hend > iH) {
            int f = nd4j::math::nd4j_ceil<Z, int>((Z) (hend - iH) / (Z) dH);
            hend -= f * dH;
        }
        if (wend > iW) {
            int f = nd4j::math::nd4j_ceil<Z, int>((Z) (wend - iW) / (Z) dW);
            wend -= f * dW;
        }
        //Accounts for dilation
        int pool_size = nd4j::math::nd4j_ceil<double, int>((double) (hend - hstart) / (double) dH) *
                        nd4j::math::nd4j_ceil<double, int>((double) (wend - wstart) / (double) dW);

        Z sum = 0.f;

        const X *inSlice = x + (n * strideB + c * strideC);

        for (int h = hstart; h < hend; h += dH)
            for (int w = wstart; w < wend; w += dW)
                sum += nd4j::math::nd4j_pow<Z, Z, Z>(static_cast<Z>(nd4j::math::nd4j_abs<X>(inSlice[h * strideY + w * strideX])), extraParam0);

        z[n * strideOB + c * strideOC + pw * strideOX + ph * strideOY] = nd4j::math::nd4j_pow<Z, Z, Z>(sum, (Z) 1.0f / extraParam0);
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static void pnormPooling2dCudaLauncher(nd4j::LaunchContext & block, void *vx, Nd4jLong *vxShapeInfo, void *vz, Nd4jLong *vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {
    pnormPooling2dCuda<X, Z><<<512, 512, 4192, *block.getCudaStream()>>>(vx, vxShapeInfo, vz, vzShapeInfo, kH, kW, sH, sW, pH, pW, dH, dW, extraParam0);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static __global__ void maxPooling2dCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {

    // input is  [bS, iC, iH, iW]
    // output is [bS, iC, oH, oW]

    const auto x = reinterpret_cast<const X*>(vx);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ int bS, iC, oH, oW, iH, iW, strideB, strideC, strideY, strideX, strideOB, strideOC, strideOY, strideOX, length, kHEff, kWEff;
    __shared__ bool fOrder;

    if (threadIdx.x == 0) {
        bS = shape::sizeAt(xShapeInfo, 0);
        iC = shape::sizeAt(xShapeInfo, 1);
        oH = shape::sizeAt(zShapeInfo, 2);
        oW = shape::sizeAt(zShapeInfo, 3);
        iH = shape::sizeAt(xShapeInfo, 2);
        iW = shape::sizeAt(xShapeInfo, 3);

        strideB = shape::stride(xShapeInfo)[0];
        strideC = shape::stride(xShapeInfo)[1];
        strideY = shape::stride(xShapeInfo)[2];
        strideX = shape::stride(xShapeInfo)[3];

        strideOB = shape::stride(zShapeInfo)[0];
        strideOC = shape::stride(zShapeInfo)[1];
        strideOY = shape::stride(zShapeInfo)[2];
        strideOX = shape::stride(zShapeInfo)[3];

        length = shape::length(zShapeInfo);

        //Replace kernel H/W with *effective* kernel H/W accounting for dilatyon
        kHEff = kH + (kH-1)*(dH-1);
        kWEff = kW + (kW-1)*(dW-1);
    }
    __syncthreads();

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int index = tid; index < length; index += blockDim.x * gridDim.x) {

        const int pw = index % oW;
        const int ph = (index / oW) % oH;
        const int c = (index / oW / oH) % iC;
        const int n = index / oW / oH / iC;

        int hstart = sH * ph - pH;
        int wstart = sW * pw - pW;
        int hend = hstart + kHEff;
        int wend = wstart + kWEff;

        if(hstart < 0){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) -hstart / (Z)dH);
            hstart += f * dH;
        }
        if(wstart < 0){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) -wstart / (Z) dW);
            wstart += f * dW;
        }
        if(hend > iH){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) (hend-iH) / (Z) dH);
            hend -= f * dH;
        }
        if(wend > iW){
            int f = nd4j::math::nd4j_ceil<Z,int>((Z) (wend-iW) / (Z) dW);
            wend -= f * dW;
        }
        //Accounts for dilation
        int pool_size = nd4j::math::nd4j_ceil<double,int>((double) (hend-hstart) / (double) dH) * nd4j::math::nd4j_ceil<double,int>((double) (wend-wstart) / (double) dW);

        Z max = -nd4j::DataTypeUtils::max<Z>();

        const X *inSlice = x + (n * strideB + c * strideC);

        for (int h = hstart; h < hend; h += dH) {
            for (int w = wstart; w < wend; w += dW) {
                Z v = static_cast<Z>(inSlice[h * strideY + w * strideX]);
                if (v > max)
                    max = v;
            }
        }

        z[n * strideOB + c * strideOC + pw * strideOX + ph * strideOY] = max;
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static void maxPooling2dCudaLauncher(nd4j::LaunchContext & block, void *vx, Nd4jLong *vxShapeInfo, void *vz, Nd4jLong *vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {
    maxPooling2dCuda<X,Z><<<512, 512, 4192, *block.getCudaStream()>>>(vx, vxShapeInfo, vz, vzShapeInfo, kH, kW, sH, sW, pH, pW, dH, dW, extraParam0);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::pooling2d(nd4j::graph::Context& block, const NDArray& input, NDArray& output, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const PoolingType poolingMode, const int extraParam0) {

    if(!input.isActualOnDeviceSide()) input.syncToDevice();

    switch (poolingMode) {

        case MAX_POOL: {
                BUILD_SINGLE_SELECTOR_TWICE(input.dataType(), maxPooling2dCudaLauncher, (*block.launchContext(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), kH, kW, sH, sW, pH, pW, dH, dW, extraParam0), FLOAT_TYPES);
            }
            break;
        case AVG_POOL: {
                BUILD_SINGLE_SELECTOR_TWICE(input.dataType(), avgPooling2dCudaLauncher, (*block.launchContext(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), kH, kW, sH, sW, pH, pW, dH, dW, extraParam0), FLOAT_TYPES);
            }
            break;
        case PNORM_POOL: {
                BUILD_SINGLE_SELECTOR_TWICE(input.dataType(), pnormPooling2dCudaLauncher, (*block.launchContext(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), kH, kW, sH, sW, pH, pW, dH, dW, extraParam0), FLOAT_TYPES);
            }
            break;
        default:
            throw std::runtime_error("Pooling2D: Unknown PoolingType used");
    }

    output.tickWriteDevice();
    input.tickReadDevice();

    auto result = hipStreamSynchronize(*block.launchContext()->getCudaStream());
    if (result != 0)
        throw cuda_exception::build("Pooling2D failed", result);
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void pooling3dCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int kD, const int kH, const int kW, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW, const int poolingMode, const int extraParam0) {

    // x input  is [bS, iC, iD, iH, iW]
    // z output is [bS, iC, oD, oH, oW]

    const T* x = reinterpret_cast<const T*>(vx);
          T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank, kDeff, kHeff, kWeff, iD, iH, iW, kProd;
    __shared__ Nd4jLong *sharedMem, zLen;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        zLen = shape::length(zShapeInfo);
        rank = 5;

        kDeff = kD + (kD - 1) * (dD - 1);
        kHeff = kH + (kH - 1) * (dH - 1);
        kWeff = kW + (kW - 1) * (dW - 1);

        iD = xShapeInfo[3];
        iH = xShapeInfo[4];
        iW = xShapeInfo[5];

        kProd = kD * kH * kW;
    }
    __syncthreads();

    const auto zInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(zInd >= zLen)
        return;

    auto coords = sharedMem + threadIdx.x * rank;

    shape::index2coords(zInd, zShapeInfo, coords);

    const auto zOffset = shape::getOffset(zShapeInfo, coords);

    int dstart = coords[2] * sD - pD;
    int hstart = coords[3] * sH - pH;
    int wstart = coords[4] * sW - pW;
    int dend = dstart + kDeff;
    int hend = hstart + kHeff;
    int wend = wstart + kWeff;

    if(dstart < 0)
        dstart += dD * ((-dstart + dD - 1) / dD);
    if(hstart < 0)
        hstart += dH * ((-hstart + dH - 1) / dH);
    if(wstart < 0)
        wstart += dW * ((-wstart + dW - 1) / dW);
    if(dend > iD)
        dend -= dD * ((dend - iD + dD - 1) / dD);
    if(hend > iH)
        hend -= dH * ((hend - iH + dH - 1) / dH);
    if(wend > iW)
        wend -= dW * ((wend - iW + dW - 1) / dW);


    switch (poolingMode) {

        /*** max ***/
        case 0: {
            T max = -DataTypeUtils::max<T>();
            for (coords[2] = dstart; coords[2] < dend; coords[2] += dD) {
                for (coords[3] = hstart; coords[3] < hend; coords[3] += dH){
                    for (coords[4] = wstart; coords[4] < wend; coords[4] += dW) {
                        T val = x[shape::getOffset(xShapeInfo, coords)];
                        if (val > max)
                            max = val;
                    }
                }
            }
            z[zOffset] = max;
        }
        break;

        /*** avg ***/
        case 1: {
            T sum = static_cast<T>(0.);
            for (coords[2] = dstart; coords[2] < dend; coords[2] += dD)
                for (coords[3] = hstart; coords[3] < hend; coords[3] += dH)
                    for (coords[4] = wstart; coords[4] < wend; coords[4] += dW)
                        sum += x[shape::getOffset(xShapeInfo, coords)];

            if (extraParam0 == 0) {         //Exclude padding
                uint a = (dend - dstart) / dD + ((dend - dstart) % dD == 0 ? 0 : 1);
                uint b = (hend - hstart) / dH + ((hend - hstart) % dH == 0 ? 0 : 1);
                uint c = (wend - wstart) / dW + ((wend - wstart) % dW == 0 ? 0 : 1);
                sum /=  static_cast<T>(a * b * c);                                       //  /= nd4j::math::nd4j_ceil<double,T>(static_cast<double>(dend - dstart) / static_cast<double>(dD)) * nd4j::math::nd4j_ceil<double,T>(static_cast<double>(hend - hstart) / static_cast<double>(dH)) * nd4j::math::nd4j_ceil<double,T>(static_cast<double>(wend - wstart) / static_cast<double>(dW));   //Accounts for dilation
            }
            else if (extraParam0 == 1)    //Include padding
                sum /= kProd;

            z[zOffset] = sum;
        }
        break;

        /*** pnorm ***/
        case 2: {
            T sum = static_cast<T>(0.);
            for (coords[2] = dstart; coords[2] < dend; coords[2] += dD)
                for (coords[3] = hstart; coords[3] < hend; coords[3] += dH)
                    for (coords[4] = wstart; coords[4] < wend; coords[4] += dW)
                        sum += nd4j::math::nd4j_pow<T,T,T>(nd4j::math::nd4j_abs<T>(x[shape::getOffset(xShapeInfo, coords)]), extraParam0);

            sum = nd4j::math::nd4j_pow<T,T,T>(sum, (T) 1.f / extraParam0);

            z[zOffset] = sum;
        }
        break;
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void pooling3dCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const void* vx, const Nd4jLong* xShapeInfo,
                                      void* vz, const Nd4jLong* zShapeInfo,
                                const int kD, const int kH, const int kW,
                                const int sD, const int sH, const int sW,
                                const int pD, const int pH, const int pW,
                                const int dD, const int dH, const int dW,
                                const int poolingMode, const int extraParam0) {

    pooling3dCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, kD, kH, kW, sD, sH, sW, pD, pH, pW, dD, dH, dW, poolingMode, extraParam0);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::pooling3d(nd4j::graph::Context& block, const NDArray& input, NDArray& output, const int kD, const int kH, const int kW, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW, const int poolingMode, const int extraParam0) {

    PointersManager manager(block.launchContext(), "pooling3d");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = output.rankOf() * sizeof(Nd4jLong) * threadsPerBlock  + 128;

    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_SINGLE_SELECTOR(input.dataType(), pooling3dCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), kD, kH, kW, sD, sH, sW, pD, pH, pW, dD, dH, dW, poolingMode, extraParam0), FLOAT_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

    manager.synchronize();
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void pooling2dBPCuda(const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int poolingMode, const int extraParam0) {

    // x: input [bS, iC, iH, iW]
    // y: gradO [bS, iC, oH, oW]
    // z: gradI [bS, iC, iH, iW] -> gradI is output in this function

    const T* x = reinterpret_cast<const T*>(vx);
    const T* y = reinterpret_cast<const T*>(vy);
          T* z = reinterpret_cast<T*>(vz);

    Nd4jLong coord2, coord3;
    __shared__ int rank, kHeff, kWeff, iH, iW, kProd;
    __shared__ Nd4jLong *sharedMem, yLen;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        yLen = shape::length(yShapeInfo);
        rank = 4;

        kHeff = kH + (kH - 1) * (dH - 1);
        kWeff = kW + (kW - 1) * (dW - 1);

        iH = xShapeInfo[3];
        iW = xShapeInfo[4];

        kProd = kH * kW;
    }
    __syncthreads();

    const auto yInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(yInd >= yLen)
        return;

    auto coords = sharedMem + threadIdx.x * rank;

    shape::index2coords(yInd, yShapeInfo, coords);

    const auto yOffset = shape::getOffset(yShapeInfo, coords);

    int hstart = coords[2] * sH - pH;
    int wstart = coords[3] * sW - pW;
    int hend = hstart + kHeff;
    int wend = wstart + kWeff;
    if(hstart < 0)
        hstart += dH * ((-hstart + dH - 1) / dH);
    if(wstart < 0)
        wstart += dW * ((-wstart + dW - 1) / dW);
    if(hend > iH)
        hend -= dH * ((hend - iH + dH - 1) / dH);
    if(wend > iW)
        wend -= dW * ((wend - iW + dW - 1) / dW);


    switch (poolingMode) {

        /*** max ***/
        case 0: {
            coord2 = hstart;
            coord3 = wstart;

            T max = -DataTypeUtils::max<T>();
            for (coords[2] = hstart; coords[2] < hend; coords[2] += dH) {
                for (coords[3] = wstart; coords[3] < wend; coords[3] += dW){
                    T val = x[shape::getOffset(xShapeInfo, coords)];
                    if (val > max) {
                        max = val;
                        coord2 = coords[2];
                        coord3 = coords[3];
                    }
                }
            }
            coords[2] = coord2;
            coords[3] = coord3;
            auto zOffset = shape::getOffset(zShapeInfo, coords);
            nd4j::math::atomics::nd4j_atomicAdd<T>(&z[zOffset], y[yOffset]);
            //z[zOffset] += y[yOffset];
        }
        break;

        /*** avg ***/
        case 1: {

            T val = y[yOffset];

            if (extraParam0 == 0)         //Exclude padding
                val /= nd4j::math::nd4j_ceil<double,T>(static_cast<double>(hend - hstart) / static_cast<double>(dH)) * nd4j::math::nd4j_ceil<double,T>(static_cast<double>(wend - wstart) / static_cast<double>(dW));   //Accounts for dilation
            else if (extraParam0 == 1)    //Include padding
                val /= kProd;

            for (coords[2] = hstart; coords[2] < hend; coords[2] += dH)
                for (coords[3] = wstart; coords[3] < wend; coords[3] += dW)
                    nd4j::math::atomics::nd4j_atomicAdd<T>(&z[shape::getOffset(zShapeInfo, coords)], val);
        }
        break;

        /*** pnorm ***/
        case 2: {

            T sum = static_cast<T>(0.);
            T val = y[yOffset];

            for (coords[2] = hstart; coords[2] < hend; coords[2] += dH)
                for (coords[3] = wstart; coords[3] < wend; coords[3] += dW)
                    sum += nd4j::math::nd4j_pow<T,T,T>(nd4j::math::nd4j_abs<T>(x[shape::getOffset(xShapeInfo, coords)]), extraParam0);

            val *= nd4j::math::nd4j_pow<T,T,T>(sum, ((T)1.f - extraParam0) / extraParam0);

            for (coords[2] = hstart; coords[2] < hend; coords[2] += dH) {
                for (coords[3] = wstart; coords[3] < wend; coords[3] += dW) {
                    const auto xOffset = shape::getOffset(xShapeInfo, coords);
                    const auto zOffset = shape::getOffset(zShapeInfo, coords);
                    nd4j::math::atomics::nd4j_atomicAdd<T>(&z[zOffset], val * nd4j::math::nd4j_pow<T,T,T>(nd4j::math::nd4j_abs<T>(x[xOffset]), extraParam0 - 1.f) * nd4j::math::nd4j_sgn<T,T>(x[xOffset]));
                }
            }
        }
        break;
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void pooling2dBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                    const void* vx, const Nd4jLong* xShapeInfo,
                                    const void* vy, const Nd4jLong* yShapeInfo,
                                          void* vz, const Nd4jLong* zShapeInfo,
                                    const int kH, const int kW,
                                    const int sH, const int sW,
                                    const int pH, const int pW,
                                    const int dH, const int dW,
                                    const int poolingMode, const int extraParam0) {

    pooling2dBPCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, kH, kW, sH, sW, pH, pW, dH, dW, poolingMode, extraParam0);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::pooling2dBP(nd4j::graph::Context& block, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int poolingMode, const int extraParam0) {

    // initial zeroing of gradI
    gradI.nullify();

    PointersManager manager(block.launchContext(), "pooling2dBP");

    const int threadsPerBlock = 256;
    const int blocksPerGrid = (gradO.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = gradO.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;

    NDArray::prepareSpecialUse({&gradI}, {&input, &gradO});
    BUILD_SINGLE_SELECTOR(input.dataType(), pooling2dBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), gradO.getSpecialBuffer(), gradO.getSpecialShapeInfo(), gradI.specialBuffer(), gradI.specialShapeInfo(), kH, kW, sH, sW, pH, pW, dH, dW, poolingMode, extraParam0), FLOAT_TYPES);
    NDArray::registerSpecialUse({&gradI}, {&input, &gradO});

    manager.synchronize();
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void pooling3dBPCuda(const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int kD, const int kH, const int kW, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW, const int poolingMode, const int extraParam0) {

    // x: input [bS, iC, iD, iH, iW]
    // y: gradO [bS, iC, oD, oH, oW]
    // z: gradI [bS, iC, iD, iH, iW] -> gradI is output in this function


    const T* x = reinterpret_cast<const T*>(vx);
    const T* y = reinterpret_cast<const T*>(vy);
          T* z = reinterpret_cast<T*>(vz);

    Nd4jLong coord2, coord3, coord4;
    __shared__ int rank, kDeff, kHeff, kWeff, iD, iH, iW, kProd;
    __shared__ Nd4jLong *sharedMem, yLen;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        yLen = shape::length(yShapeInfo);
        rank = 5;

        kDeff = kD + (kD - 1) * (dD - 1);
        kHeff = kH + (kH - 1) * (dH - 1);
        kWeff = kW + (kW - 1) * (dW - 1);

        iD = xShapeInfo[3];
        iH = xShapeInfo[4];
        iW = xShapeInfo[5];

        kProd = kD * kH * kW;
    }
    __syncthreads();

    const auto yInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(yInd >= yLen)
        return;

    auto coords = sharedMem + threadIdx.x * rank;

    shape::index2coords(yInd, yShapeInfo, coords);

    const auto yOffset = shape::getOffset(yShapeInfo, coords);

    int dstart = coords[2] * sD - pD;
    int hstart = coords[3] * sH - pH;
    int wstart = coords[4] * sW - pW;
    int dend = dstart + kDeff;
    int hend = hstart + kHeff;
    int wend = wstart + kWeff;

    if(dstart < 0)
        dstart += dD * ((-dstart + dD - 1) / dD);
    if(hstart < 0)
        hstart += dH * ((-hstart + dH - 1) / dH);
    if(wstart < 0)
        wstart += dW * ((-wstart + dW - 1) / dW);
    if(dend > iD)
        dend -= dD * ((dend - iD + dD - 1) / dD);
    if(hend > iH)
        hend -= dH * ((hend - iH + dH - 1) / dH);
    if(wend > iW)
        wend -= dW * ((wend - iW + dW - 1) / dW);


    switch (poolingMode) {

        /*** max ***/
        case 0: {

            T max = -DataTypeUtils::max<T>();
            for (coords[2] = dstart; coords[2] < dend; coords[2] += dD) {
                for (coords[3] = hstart; coords[3] < hend; coords[3] += dH){
                    for (coords[4] = wstart; coords[4] < wend; coords[4] += dW) {
                        T val = x[shape::getOffset(xShapeInfo, coords)];
                        if (val > max) {
                            max = val;
                            coord2 = coords[2];
                            coord3 = coords[3];
                            coord4 = coords[4];
                        }
                    }
                }
            }
            coords[2] = coord2;
            coords[3] = coord3;
            coords[4] = coord4;
            nd4j::math::atomics::nd4j_atomicAdd<T>(&z[shape::getOffset(zShapeInfo, coords)], y[yOffset]);
        }
        break;

        /*** avg ***/
        case 1: {

            T val = y[yOffset];

            if (extraParam0 == 0)         //Exclude padding
                val /= nd4j::math::nd4j_ceil<double,T>(static_cast<double>(dend - dstart) / static_cast<double>(dD))  * nd4j::math::nd4j_ceil<double,T>(static_cast<double>(hend - hstart) / static_cast<double>(dH))     * nd4j::math::nd4j_ceil<double,T>(static_cast<double>(wend - wstart)    / static_cast<double>(dW));   //Accounts for dilation
            else if (extraParam0 == 1)    //Include padding
                val /= kProd;

            for (coords[2] = dstart; coords[2] < dend; coords[2] += dD)
                for (coords[3] = hstart; coords[3] < hend; coords[3] += dH)
                    for (coords[4] = wstart; coords[4] < wend; coords[4] += dW)
                        nd4j::math::atomics::nd4j_atomicAdd<T>(&z[shape::getOffset(zShapeInfo, coords)], val);
        }
        break;

        /*** pnorm ***/
        case 2: {

            T sum = static_cast<T>(0.);
            T val = y[yOffset];

            for (coords[2] = dstart; coords[2] < dend; coords[2] += dD)
                for (coords[3] = hstart; coords[3] < hend; coords[3] += dH)
                    for (coords[4] = wstart; coords[4] < wend; coords[4] += dW)
                        sum += nd4j::math::nd4j_pow<T,T,T>(nd4j::math::nd4j_abs<T>(x[shape::getOffset(xShapeInfo, coords)]), extraParam0);

            val *= nd4j::math::nd4j_pow<T,T,T>(sum, ((T)1.f - extraParam0) / extraParam0);

            for (coords[2] = dstart; coords[2] < dend; coords[2] += dD) {
                for (coords[3] = hstart; coords[3] < hend; coords[3] += dH) {
                    for (coords[4] = wstart; coords[4] < wend; coords[4] += dW) {
                        const auto xOffset = shape::getOffset(xShapeInfo, coords);
                        const auto zOffset = shape::getOffset(zShapeInfo, coords);
                        nd4j::math::atomics::nd4j_atomicAdd<T>(&z[zOffset], val * nd4j::math::nd4j_pow<T,T,T>(nd4j::math::nd4j_abs<T>(x[xOffset]), extraParam0 - 1.f) * nd4j::math::nd4j_sgn<T,T>(x[xOffset]));
                    }
                }
            }
        }
        break;
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void pooling3dBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                    const void* vx, const Nd4jLong* xShapeInfo,
                                    const void* vy, const Nd4jLong* yShapeInfo,
                                          void* vz, const Nd4jLong* zShapeInfo,
                                    const int kD, const int kH, const int kW,
                                    const int sD, const int sH, const int sW,
                                    const int pD, const int pH, const int pW,
                                    const int dD, const int dH, const int dW,
                                    const int poolingMode, const int extraParam0) {

    pooling3dBPCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, kD, kH, kW, sD, sH, sW, pD, pH, pW, dD, dH, dW, poolingMode, extraParam0);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::pooling3dBP(nd4j::graph::Context& block, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int kD, const int kH, const int kW, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW, const int poolingMode, const int extraParam0) {

    // initial zeroing of gradI
    gradI.nullify();

    PointersManager manager(block.launchContext(), "pooling3dBP");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (gradO.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = gradO.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;

    NDArray::prepareSpecialUse({&gradI}, {&input, &gradO});
    BUILD_SINGLE_SELECTOR(input.dataType(), pooling3dBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), gradO.getSpecialBuffer(), gradO.getSpecialShapeInfo(), gradI.specialBuffer(), gradI.specialShapeInfo(), kD, kH, kW, sD, sH, sW, pD, pH, pW, dD, dH, dW, poolingMode, extraParam0), FLOAT_TYPES);
    NDArray::registerSpecialUse({&gradI}, {&input, &gradO});

    manager.synchronize();
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
static void conv2dBP_(nd4j::graph::Context& block, const NDArray* input, const NDArray* weights, const NDArray* bias, const NDArray* gradO, NDArray* gradI, NDArray* gradW, NDArray* gradB, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

    // input   [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW)
    // weights [kH, kW, iC, oC] always
    // bias    [oC]
    // gradO   [bS, oH, oW, oC] (NHWC) or [bS, oC, oH, oW] (NCHW), epsilon_next

    // gradI    [bS, iH, iW, iC] (NHWC) or [bS, iC, iH, iW] (NCHW), epsilon
    // gradW    [kH, kW, iC, oC] always
    // gradB    [oC]

    // kH         filter(kernel) height
    // kW         filter(kernel) width
    // sH         strides height
    // sW         strides width
    // pH         paddings height
    // pW         paddings width
    // dH         dilations height
    // dW         dilations width
    // isSameMode 0-VALID, 1-SAME
    // isNCHW     0-NHWC, 1-NCHW

    int bS, iC, iH, iW, oC, oH, oW;                             // batch size, input channels, input height/width, output channels, output height/width;
    int indIOioC, indIiH, indWoC, indWiC, indWkH, indOoH;       // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, *input, *gradO, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWoC, indWkH, indOoH);

    if(isSameMode)                       // SAME
        ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW);

    std::vector<int> gradOaxesForDot;

    if(!isNCHW) {
        gradOaxesForDot  = {0, 1, 2};                                           // bS, oH, oW
        input = new NDArray(input->permute({0, 3, 1, 2}));                      // [bS, iH, iW, iC] -> [bS, iC, iH, iW]
        gradI = new NDArray(gradI->permute({0, 3, 1, 2}));                      // [bS, iH, iW, iC] -> [bS, iC, iH, iW]
    } else {
        gradOaxesForDot  = {0, 2, 3};                                           // bS, oH, oW
    }

    NDArray columns(input->ordering(), {bS, iC, kH, kW, oH, oW}, input->dataType(), input->getContext());

    // ----- calculation of gradW ----- //
    if(gradW) {
        auto ctx = block.launchContext();
        helpers::im2col(*ctx, *input, columns, kH, kW, sH, sW, pH, pW, dH, dW, NDArrayFactory::create(0.f, input->getContext()));   // [bS, iC, iH, iW] is convoluted to [bS, iC, kH, kW, oH, oW]
        nd4j::MmulHelper::tensorDot(&columns, gradO, gradW, {0,4,5}, gradOaxesForDot, {2, 0, 1, 3});       // [bS, iC, kH, kW, oH, oW] x [bS, oH, oW, oC]/[bS, oC, oH, oW] = [iC, kH, kW, oC]
    }

    // ----- calculation of gradB ----- //
    if(gradB) {
        NDArray* gradBR = gradB;
        if(gradB->rankOf() == 2)
            gradBR = new NDArray(gradB->reshape(gradB->ordering(), {(int)gradB->lengthOf()}));
        gradO->reduceAlongDimension(reduce::Sum, gradBR, gradOaxesForDot);                          // sum over bS, oH, oW
        if(gradBR != gradB)
            delete gradBR;
    }

    //----- calculation of gradI -----//
    nd4j::MmulHelper::tensorDot(weights, gradO, &columns, {indWoC}, {indIOioC}, {2, 3, 1, 0, 4, 5});  // [kH, kW, iC, oC]/[oC, iC, kH, kW]] x [bS, oH, oW, oC]/[bS, oC, oH, oW] = [kH, kW, iC, bS, oH, oW]

    helpers::col2im(*block.launchContext(), columns, *gradI, sH, sW, pH, pW, iH, iW, dH, dW);                          // [bS, iC, kH, kW, oH, oW] is de-convoluted to [bS, iC, iH, iW]

    if(!isNCHW) {
        delete input;
        delete gradI;
    }
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::conv2dBP(nd4j::graph::Context& block, const NDArray* input, const NDArray* weights, const NDArray* bias, const NDArray* gradO, NDArray* gradI, NDArray* gradW, NDArray* gradB, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {
    BUILD_SINGLE_SELECTOR_TWICE(input->dataType(), conv2dBP_, (block, input, weights, bias, gradO, gradI, gradW, gradB, kH, kW, sH, sW, pH, pW, dH, dW, isSameMode, isNCHW), FLOAT_TYPES);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
static void depthwiseConv2dBP_(const NDArray* input, const NDArray* weights, const NDArray* bias, const NDArray* gradO, NDArray* gradI, NDArray* gradW, NDArray* gradB, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

    // input    [bS, iH, iW, iC] (NDHWC) or [bS, iC, iH, iW] (NCDHW)
    // weights  [kH, kW, iC, mC] always
    // bias     [oC] = [iC*mC]
    // gradO    [bS, oH, oW, oC] (NDHWC) or [bS, oC, oH, oW] (NCDHW), epsilon_next
    // gradI    [bS, iH, iW, iC] (NDHWC) or [bS, iC, iH, iW] (NCDHW), epsilon
    // gradW    [kH, kW, iC, mC] always
    // gradB    [oC]

    //  kH          filter(kernel) height
    //  kW          filter(kernel) width
    //  sH          strides height
    //  sW          strides width
    //  pH          paddings height
    //  pW          paddings width
    //  dH          dilations height
    //  dW          dilations width
    //  isSameMode  0-VALID, 1-SAME
    //  isNCHW      0-NHWC, 1-NCHW

    int bS, iC, iH, iW, mC, oC, oH, oW;                     // batch size, input channels, input height/width, channels multiplier(oC = iC*mC), output channels, output height/width
    int indIOioC, indIiH, indWmC, indWiC, indWkH, indOoH;   // corresponding indexes
    ConvolutionUtils::getSizesAndIndexesConv2d(isNCHW, *input, *gradO, bS, iC, iH, iW, oC, oH, oW, indIOioC, indIiH, indWiC, indWmC, indWkH, indOoH);
    mC = weights->sizeAt(indWmC);                           // channels multiplier

    std::vector<std::vector<Nd4jLong>> modifColumns = {{1,2,3,0,4,5}, {iC, kH*kW, bS*oH*oW}};      // [bS,iC,kH,kW,oH,oW] -> [iC, kH*kW, bS*oH*oW]
    std::vector<std::vector<Nd4jLong>> modifGradO1, modifGradO2;
    std::vector<Nd4jLong> gradOreShape;

    if(!isNCHW) {
        gradOreShape = {bS, oH, oW, iC, mC};                                            // [bS,oH,oW,iC*mC] -> [bS,oH,oW,iC,mC]
        modifGradO1 = {{3,0,1,2,4},{iC, bS*oH*oW, mC}};                                 // [bS,oH,oW,iC,mC] -> [iC,bS,oH,oW,mC] -> [iC,bS*oH*oW,mC]
        modifGradO2 = {{3,0,1,2},{iC, mC, bS*oH*oW}};                                   // [bS,oH,oW,iC*mC] -> [iC*mC,bS,oH,oW] -> [iC,mC,bS*oH*oW]
        input = new NDArray(input->permute({0, 3, 1, 2}));                             // [bS,iH,iW,iC]    -> [bS,iC,iH,iW]
        gradI = new NDArray(gradI->permute({0, 3, 1, 2}));                             // [bS,iH,iW,iC]    -> [bS,iC,iH,iW]
    }
    else {
        gradOreShape = {bS, iC, mC, oH, oW};                                            // [bS,iC*mC,oH,oW] -> [bS,iC,mC,oH,oW]
        modifGradO1 = {{1,0,3,4,2},{iC, bS*oH*oW, mC}};                                 // [bS,iC,mC,oH,oW] -> [iC,bS,oH,oW,mC] -> [iC,bS*oH*oW,mC]
        modifGradO2 = {{1,0,2,3},{iC, mC, bS*oH*oW}};                                   // [bS,iC*mC,oH,oW] -> [iC*mC,bS,oH,oW] -> [iC,mC,bS*oH*oW]
    }

    if(isSameMode)                       // SAME
        ConvolutionUtils::calcPadding2D(pH, pW, oH, oW, iH, iW, kH, kW, sH, sW, dH, dW);

    NDArray columns(input->ordering(), {bS, iC, kH, kW, oH, oW}, input->dataType(), input->getContext());
    NDArray gradOreshaped = gradO->reshape(gradO->ordering(), gradOreShape);

    // ----- calculation of gradW and gradB ----- //

    helpers::im2col(*input->getContext(), *input, columns, kH, kW, sH, sW, pH, pW, dH, dW, NDArrayFactory::create(0.f, input->getContext()));  // [bS, iC, iH, iW] is convoluted to [bS, iC, kH, kW, oH, oW]
    nd4j::MmulHelper::tensorDot(&columns, &gradOreshaped, gradW, modifColumns, modifGradO1, {{2,0,1,3},{iC,kH*kW,mC}});  // [iC, kW*kH, bS*oH*oW] x [iC, bS*oH*oW, mC] = [iC, kH*kW, mC]

    // ----- calculation of gradB ----- //
    if(gradB) {
        NDArray* gradBR = gradB;
        if(gradB->rankOf() == 2)
            gradBR = new NDArray(gradB->reshape(gradB->ordering(), {(int)gradB->lengthOf()}));
        gradO->reduceAlongDimension(reduce::Sum, gradBR, {0,indOoH,indOoH+1});                      // sum over bS, oH, oW
        if(gradBR != gradB)
            delete gradBR;
    }

    //----- calculation of gradI -----//
    nd4j::MmulHelper::tensorDot(weights, gradO, &columns, {{2,0,1,3},{iC,kH*kW,mC}}, modifGradO2, modifColumns); // [iC, kH*kW, mC] x [iC, mC, bS*oH*oW] = [iC, kW*kH, bS*oH*oW]
    helpers::col2im(*input->getContext(), columns, *gradI, sH, sW, pH, pW, iH, iW, dH, dW);                                       // [bS, iC, kH, kW, oH, oW] is de-convoluted to [bS, iC, iH, iW]

    if(!isNCHW) {
        delete input;
        delete gradI;
    }
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::depthwiseConv2dBP(nd4j::graph::Context& block, const NDArray* input, const NDArray* weights, const NDArray* bias, const NDArray* gradO, NDArray* gradI, NDArray* gradW, NDArray* gradB, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {
    BUILD_SINGLE_SELECTOR_TWICE(input->dataType(), depthwiseConv2dBP_, (input, weights, bias, gradO, gradI, gradW, gradB, kH, kW, sH, sW, pH, pW, dH, dW, isSameMode, isNCHW), FLOAT_TYPES);
}


//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void upsampling2dCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int factorH, const int factorW, const bool isNCHW) {

    // x has shape [bS, iC, iH, iW] (NCHW) or [bS, iH, iW, iC] (NHWC)
    // z has shape [bS, iC, factorH*iH, factorW*iW ] (NCHW) or [bS, factorH*iH, factorW*iW, iC] (NHWC)

    const T* x = reinterpret_cast<const T*>(vx);
          T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank, dimIH;
    __shared__ Nd4jLong *sharedMem, zLen;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        dimIH  = isNCHW ? 2 : 1;
        zLen   = shape::length(zShapeInfo);
        rank   = 4;
    }
    __syncthreads();

    const auto zInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(zInd >= zLen)
        return;

    auto coords = sharedMem + threadIdx.x * rank;

    shape::index2coords(zInd, zShapeInfo, coords);

    const auto zOffset = shape::getOffset(zShapeInfo, coords);

    coords[dimIH]     /= factorH;
    coords[dimIH + 1] /= factorW;

    const auto xOffset = shape::getOffset(xShapeInfo, coords);

    z[zOffset] = x[xOffset];
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void upsampling2dCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                     const void* vx, const Nd4jLong* xShapeInfo,
                                           void* vz, const Nd4jLong* zShapeInfo,
                                     const int factorH, const int factorW, const bool isNCHW) {

    upsampling2dCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, factorH, factorW, isNCHW);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::upsampling2d(nd4j::graph::Context& block, const NDArray& input, NDArray& output, const int factorH, const int factorW, const bool isNCHW) {

    PointersManager manager(block.launchContext(), "upsampling2d");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = output.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;

    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_SINGLE_SELECTOR(input.dataType(), upsampling2dCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), factorH, factorW, isNCHW), FLOAT_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

    manager.synchronize();
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void upsampling3dCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int factorD, const int factorH, const int factorW, const bool isNCDHW) {

    // x has shape [bS, iC, iD, iH, iW] (NCDHW) or [bS, iD, iH, iW, iC] (NDHWC)
    // z has shape [bS, iC, factorD*iD, factorH*iH, factorW*iW ] (NCDHW) or [bS, factorD*iD, factorH*iH, factorW*iW, iC] (NDHWC)

    const T* x = reinterpret_cast<const T*>(vx);
          T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank, dimID;
    __shared__ Nd4jLong *sharedMem, zLen;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        dimID = isNCDHW ? 2 : 1;
        zLen  = shape::length(zShapeInfo);
        rank  = 5;
    }
    __syncthreads();

    const auto zInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(zInd >= zLen)
        return;

    auto coords = sharedMem + threadIdx.x * rank;

    shape::index2coords(zInd, zShapeInfo, coords);

    const auto zOffset = shape::getOffset(zShapeInfo, coords);

    coords[dimID]     /= factorD;
    coords[dimID + 1] /= factorH;
    coords[dimID + 2] /= factorW;

    const auto xOffset = shape::getOffset(xShapeInfo, coords);

    z[zOffset] = x[xOffset];
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void upsampling3dCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                     const void* vx, const Nd4jLong* xShapeInfo,
                                           void* vz, const Nd4jLong* zShapeInfo,
                                     const int factorD, const int factorH, const int factorW, const bool isNCDHW) {

    upsampling3dCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, factorD, factorH, factorW, isNCDHW);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::upsampling3d(nd4j::graph::Context& block, const NDArray& input, NDArray& output, const int factorD, const int factorH, const int factorW, const bool isNCDHW) {

    PointersManager manager(block.launchContext(), "upsampling3d");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = output.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;

    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_SINGLE_SELECTOR(input.dataType(), upsampling3dCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), factorD, factorH, factorW, isNCDHW), FLOAT_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

    manager.synchronize();
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void upsampling2dBPCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const bool isNCHW) {

    // x (gradO) has shape [bS, iC, factorH*iH, factorW*iW ] (NCHW) or [bS, factorH*iH, factorW*iW, iC] (NHWC)
    // z (gradI) has shape [bS, iC, iH, iW] (NCHW) or [bS, iH, iW, iC] (NHWC)

    const T* x = reinterpret_cast<const T*>(vx);
          T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank, dimIH;
    __shared__ uint factorH, factorW;
    __shared__ Nd4jLong *sharedMem, zLen;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        dimIH = isNCHW ? 2 : 1;
        zLen  = shape::length(zShapeInfo);
        rank  = 4;

        factorH = xShapeInfo[dimIH + 1] / zShapeInfo[dimIH + 1];
        factorW = xShapeInfo[dimIH + 2] / zShapeInfo[dimIH + 2];
    }
    __syncthreads();

    const auto zInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(zInd >= zLen)
        return;

    auto coords = sharedMem + threadIdx.x * rank;

    shape::index2coords(zInd, zShapeInfo, coords);

    const auto zOffset = shape::getOffset(zShapeInfo, coords);

    z[zOffset] = 0;

    const Nd4jLong zCoord2 = coords[dimIH]     * factorH;
    const Nd4jLong zCoord3 = coords[dimIH + 1] * factorW;

    for(coords[dimIH] = zCoord2; coords[dimIH] < zCoord2 + factorH; ++coords[dimIH])
        for(coords[dimIH + 1] = zCoord3; coords[dimIH + 1] < zCoord3 + factorW; ++coords[dimIH + 1])
            z[zOffset] += x[shape::getOffset(xShapeInfo, coords)];
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void upsampling2dBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                       const void* vx, const Nd4jLong* xShapeInfo,
                                             void* vz, const Nd4jLong* zShapeInfo,
                                       const bool isNCHW) {

    upsampling2dBPCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, isNCHW);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::upsampling2dBP(nd4j::graph::Context& block, const NDArray& gradO, NDArray& gradI, const bool isNCHW) {

    PointersManager manager(block.launchContext(), "upsampling2d_bp");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (gradI.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = gradI.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;

    NDArray::prepareSpecialUse({&gradI}, {&gradO});
    BUILD_SINGLE_SELECTOR(gradI.dataType(), upsampling2dBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), gradO.getSpecialBuffer(), gradO.getSpecialShapeInfo(), gradI.specialBuffer(), gradI.specialShapeInfo(), isNCHW), FLOAT_TYPES);
    NDArray::registerSpecialUse({&gradI}, {&gradO});

    manager.synchronize();
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void upsampling3dBPCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const bool isNCDHW) {

    // x (gradO) has shape [bS, iC, iD, iH, iW] (NCDHW) or [bS, iD, iH, iW, iC] (NDHWC)
    // z (gradI) has shape [bS, iC, factorD*iD, factorH*iH, factorW*iW ] (NCDHW) or [bS, factorD*iD, factorH*iH, factorW*iW, iC] (NDHWC)

    const T* x = reinterpret_cast<const T*>(vx);
          T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank, dimID;
    __shared__ uint factorD, factorH, factorW;
    __shared__ Nd4jLong *sharedMem, zLen;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        dimID = isNCDHW ? 2 : 1;
        zLen  = shape::length(zShapeInfo);
        rank  = 5;

        factorD = xShapeInfo[dimID + 1] / zShapeInfo[dimID + 1];
        factorH = xShapeInfo[dimID + 2] / zShapeInfo[dimID + 2];
        factorW = xShapeInfo[dimID + 3] / zShapeInfo[dimID + 3];
    }
    __syncthreads();

    const auto zInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(zInd >= zLen)
        return;

    auto coords = sharedMem + threadIdx.x * rank;

    shape::index2coords(zInd, zShapeInfo, coords);

    const auto zOffset = shape::getOffset(zShapeInfo, coords);

    z[zOffset] = 0;

    const Nd4jLong zCoord2 = coords[dimID]     * factorD;
    const Nd4jLong zCoord3 = coords[dimID + 1] * factorH;
    const Nd4jLong zCoord4 = coords[dimID + 2] * factorW;

    for(coords[dimID] = zCoord2; coords[dimID] < zCoord2 + factorD; ++coords[dimID])
        for(coords[dimID + 1] = zCoord3; coords[dimID + 1] < zCoord3 + factorH; ++coords[dimID + 1])
            for(coords[dimID + 2] = zCoord4; coords[dimID + 2] < zCoord4 + factorW; ++coords[dimID + 2])
                z[zOffset] += x[shape::getOffset(xShapeInfo, coords)];
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void upsampling3dBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                       const void* vx, const Nd4jLong* xShapeInfo,
                                             void* vz, const Nd4jLong* zShapeInfo,
                                       const bool isNCDHW) {

    upsampling3dBPCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, isNCDHW);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::upsampling3dBP(nd4j::graph::Context& block, const NDArray& gradO, NDArray& gradI, const bool isNCDHW) {

    PointersManager manager(block.launchContext(), "upsampling3d_bp");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (gradI.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = gradI.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;

    NDArray::prepareSpecialUse({&gradI}, {&gradO});
    BUILD_SINGLE_SELECTOR(gradI.dataType(), upsampling3dBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), gradO.getSpecialBuffer(), gradO.getSpecialShapeInfo(), gradI.specialBuffer(), gradI.specialShapeInfo(), isNCDHW), FLOAT_TYPES);
    NDArray::registerSpecialUse({&gradI}, {&gradO});

    manager.synchronize();
}









}
}