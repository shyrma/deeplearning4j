#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//
#include <ops/declarable/helpers/dynamic.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace nd4j {
    namespace ops {
        namespace helpers {


            template <typename X, typename Y>
            static _CUDA_G void dynamicPartitionScalarKernel(void *vx, Nd4jLong *xShapeInfo, void *vi, Nd4jLong *iShapeInfo, void **vz, Nd4jLong **zShapeInfos, const Nd4jLong numOutputs) {
                auto x = reinterpret_cast<X*>(vx);
                auto i = reinterpret_cast<Y*>(vi);
                auto xLength = shape::length(xShapeInfo);
                auto iLength = shape::length(iShapeInfo);

                extern __shared__ char shmem[];
                __shared__ Y *rawIndices;
                __shared__ Y *trueIndices;

                if (threadIdx.x == 0) {
                    rawIndices = reinterpret_cast<Y*>(shmem);
                    trueIndices = rawIndices + blockDim.x;
                }
                __syncthreads();

                // we run things in blocks, 1 partition per block of threads
                for (Nd4jLong o = blockIdx.x; o < numOutputs; o += gridDim.x) {
                    auto z = reinterpret_cast<X*>(vz[o]);

                    auto zShapeInfo = zShapeInfos[o];
                    auto zLength = shape::length(zShapeInfo);

                    // iLimit should be multiple of blockDim.x
                    auto iLimit = iLength <= blockDim.x ? blockDim.x : (iLength + (blockDim.x - (iLength % blockDim.x)));
                    int cnt = 0;

                    for (Nd4jLong e = threadIdx.x; e < iLimit; e += blockDim.x) {
                        // load set of indices into shared memory
                        if (e < iLength)
                            rawIndices[threadIdx.x] = i[shape::getIndexOffset(e, iShapeInfo)];
                        __syncthreads();

                        // now we need to find out where our actual updates will be mapped
                        // TODO: this can be improved obviously, by using prefix-sum like approach
                        if (threadIdx.x == 0) {
                            for (int f = 0; f < blockDim.x; f++) {
                                if (rawIndices[f] == static_cast<Y>(o))
                                    trueIndices[f] = cnt++;
                                else
                                    trueIndices[f] = -1;
                            }
                        }
                        __syncthreads();


                        // doing actual update
                        if (e < iLength)
                            if (trueIndices[threadIdx.x] >= 0) {
                                z[trueIndices[threadIdx.x]] = x[shape::getIndexOffset(e, xShapeInfo)];
                            }

                        __syncthreads();
                    }
                }
            }

            template <typename X, typename Y>
            static _CUDA_G void dynamicPartitionTadKernel(void *vx, Nd4jLong *xTadShapeInfo, Nd4jLong *xTadOffsets, Nd4jLong xLength, void *vindices, Nd4jLong *iShapeInfo, Nd4jLong iLength, void **vz, Nd4jLong **zTadShapeInfos, Nd4jLong **zTadOffsets, Nd4jLong numOutputs) {
                auto x = reinterpret_cast<X*>(vx);
                auto indices = reinterpret_cast<Y*>(vindices);

                // we run things in blocks, 1 partition per block of threads
                for (int i = blockIdx.x; i < numOutputs; i += gridDim.x) {
                    auto z = reinterpret_cast<X*>(vz[i]);

                    // each thread has own counter for partitions
                    int outCnt = 0;

                    for (Nd4jLong e = 0; e < iLength; e++) {
                        if (indices[shape::getIndexOffset(e, iShapeInfo)] == i) {
                            auto dx = x + xTadOffsets[e];
                            auto dz = z + zTadOffsets[i][outCnt++];

                            for (int f = threadIdx.x; f < xLength; f += blockDim.x) {
                                dz[shape::getIndexOffset(f, zTadShapeInfos[i])] = dx[shape::getIndexOffset(f, xTadShapeInfo)];
                            }
                        }
                    }
                }
            }

            template <typename X, typename Y>
            static void _dynamicPartitionFunctor(nd4j::LaunchContext * context, NDArray const* input, NDArray const* indices, std::vector<NDArray*>& outputList) {
                std::vector<std::pair<NDArray *, int>> outputs(outputList.size());
                int sourceDimsLen = input->rankOf() - indices->rankOf();

                unsigned int outSize = outputList.size();

                PointersManager pm(context, "dynamicPartition");

                if (sourceDimsLen) { // non-linear case
                    std::vector<int> sourceDims(sourceDimsLen);

                    for (int i = sourceDimsLen; i > 0; i--)
                        sourceDims[sourceDimsLen - i] = input->rankOf() - i;
                    //compute tad array for given dimensions
                    auto packX = ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), sourceDims);

                    std::vector<void *> outBuffers(outSize);
                    std::vector<Nd4jLong *> tadShapes(outSize);
                    std::vector<Nd4jLong *> tadOffsets(outSize);
                    std::vector<Nd4jLong> numTads(outSize);
                    // fill up dimensions array for before kernel
                    for (unsigned int i = 0; i < outSize; i++) {
                        outputs[i].first = outputList[i];
                        std::vector<int> outDims(outputs[i].first->rankOf() - 1);

                        int r = outputs[i].first->rankOf();

                        for (int k = 1; k < r; k++)
                            outDims[k - 1] = k;

                        auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(outputList.at(i)->getShapeInfo(), outDims);

                        outBuffers[i] = outputList.at(i)->getSpecialBuffer();
                        tadShapes[i] = packZ.platformShapeInfo();
                        tadOffsets[i] = packZ.platformOffsets();
                    }

                    // we copy pointers to device
                    auto dOutBuffers = reinterpret_cast<void **>(pm.replicatePointer(outBuffers.data(), outBuffers.size() * sizeof(void *)));
                    auto dOutTadShapes = reinterpret_cast<Nd4jLong **>(pm.replicatePointer(tadShapes.data(), tadShapes.size() * sizeof(Nd4jLong *)));
                    auto dOutTadOffsets = reinterpret_cast<Nd4jLong **>(pm.replicatePointer(tadOffsets.data(), tadOffsets.size() * sizeof(Nd4jLong *)));
                    // run kernel on device
                    dynamicPartitionTadKernel<X,Y><<<256, 256, 1024, *context->getCudaStream()>>>(input->getSpecialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), shape::length(packX.primaryShapeInfo()), indices->getSpecialBuffer(), indices->getSpecialShapeInfo(), indices->lengthOf(), dOutBuffers, dOutTadShapes, dOutTadOffsets, outSize);

                } else { // linear case
                    auto numThreads = 256;
                    auto shmemSize = numThreads * sizeof(Y) * 2 + 1024;

                    std::vector<void *> outBuffers;
                    std::vector<Nd4jLong *> outShapes;

                    for (auto v:outputList) {
                        outBuffers.emplace_back(v->getSpecialBuffer());
                        outShapes.emplace_back(v->getSpecialShapeInfo());
                    }

                    auto dOutBuffers = reinterpret_cast<void **>(pm.replicatePointer(outBuffers.data(), outBuffers.size() * sizeof(void *)));
                    auto dOutShapes = reinterpret_cast<Nd4jLong **>(pm.replicatePointer(outShapes.data(), outShapes.size() * sizeof(Nd4jLong *)));

                    dynamicPartitionScalarKernel<X,Y><<<256, numThreads, shmemSize, *context->getCudaStream()>>>(input->getSpecialBuffer(), input->getSpecialShapeInfo(), indices->getSpecialBuffer(), indices-> getSpecialShapeInfo(), dOutBuffers, dOutShapes, outSize);
                }

                pm.synchronize();
            }


            template <typename X, typename Y>
            static _CUDA_G void dynamicStitchScalarKernel(void **vx, Nd4jLong **xShapeInfos, void **vindices, Nd4jLong **iShapeInfos, int inputSize, void *vz, Nd4jLong *zShapeInfo, Nd4jLong zLength) {
                auto z = reinterpret_cast<X*>(vz);

                for (int e = blockIdx.x; e < inputSize; e += gridDim.x) {
                    auto x = reinterpret_cast<X*>(vx[e]);
                    auto indices = reinterpret_cast<Y*>(vindices[e]);

                    auto xShapeInfo = xShapeInfos[e];
                    auto iShapeInfo = iShapeInfos[e];

                    auto iLength = shape::length(iShapeInfo);

                    for (int i = threadIdx.x; i < iLength; i += blockDim.x) {
                        auto idx = indices[shape::getIndexOffset(i, iShapeInfo)];
                        if (idx >= 0 && idx < zLength)
                            z[shape::getIndexOffset(idx, zShapeInfo)] = x[shape::getIndexOffset(i, xShapeInfo)];
                    }
                }
            }

            template <typename X, typename Y>
            static _CUDA_G void dynamicStitchTadKernel(void **vx, Nd4jLong **xTadShapeInfos, Nd4jLong **xTadOffsets, void **vindices, Nd4jLong **iShapeInfos, int inputSize, void *vz, Nd4jLong *zTadShapeInfo, Nd4jLong *zTadOffsets) {
                auto bz = reinterpret_cast<X*>(vz);

                for (int e = blockIdx.x; e < inputSize; e += gridDim.x) {
                    auto indices = reinterpret_cast<Y*>(vindices[e]);
                    auto iShapeInfo = iShapeInfos[e];

                    if (shape::isEmpty(iShapeInfo))
                        continue;

                    auto iLength = shape::length(iShapeInfo);
                    auto zLength = shape::length(zTadShapeInfo);

                    auto xShapeInfo = xTadShapeInfos[e];
                    auto xLength = shape::length(xShapeInfo);

                    for (int i = 0; i < iLength; i++) {
                        auto idx = indices[shape::getIndexOffset(i, iShapeInfo)];

                        auto z = bz + zTadOffsets[idx];
                        auto x = reinterpret_cast<X*>(vx[e]) + xTadOffsets[e][i];

                        for (int f = threadIdx.x; f < zLength; f += blockDim.x) {
                            z[shape::getIndexOffset(f, zTadShapeInfo)] = x[shape::getIndexOffset(f, xShapeInfo)];
                        }

                        __syncthreads();
                    }
                }
            }

            template <typename X, typename Y>
            static int _dynamicStitchFunctor(nd4j::LaunchContext * context, std::vector<NDArray*> const& inputs, std::vector<NDArray*> const& indices, NDArray* output){

                int inputSize = inputs.size();

                PointersManager pm(context, "dynamicStitch");

                if (output->isVector()) {
                    std::vector<void *> inputBuffers(inputSize);
                    std::vector<Nd4jLong *> inputShapes(inputSize);
                    std::vector<void *> indicesBuffers(inputSize);
                    std::vector<Nd4jLong *> indicesShapes(inputSize);

                    for (int e = 0; e < inputSize; e++) {
                        inputBuffers[e] = inputs.at(e)->getSpecialBuffer();
                        indicesBuffers[e] = indices.at(e)->getSpecialBuffer();

                        inputShapes[e] = inputs.at(e)->getSpecialShapeInfo();
                        indicesShapes[e] = indices.at(e)->getSpecialShapeInfo();
                    }

                    // copying pointers to buffers to device
                    auto dInputBuffers = reinterpret_cast<void **>(pm.replicatePointer(inputBuffers.data(), inputSize * sizeof(void *)));
                    auto dIndicesBuffers = reinterpret_cast<void **>(pm.replicatePointer(indicesBuffers.data(), inputSize * sizeof(void *)));
                    auto dInputShapes = reinterpret_cast<Nd4jLong **>(pm.replicatePointer(inputShapes.data(), inputSize * sizeof(Nd4jLong *)));
                    auto dIndicesShapes = reinterpret_cast<Nd4jLong **>(pm.replicatePointer(indicesShapes.data(), inputSize * sizeof(Nd4jLong *)));

                    dynamicStitchScalarKernel<X,Y><<<256, 256, 1024, *context->getCudaStream()>>>(dInputBuffers, dInputShapes, dIndicesBuffers, dIndicesShapes, inputSize, output->specialBuffer(), output->specialShapeInfo(), output->lengthOf());
                } else {
                    std::vector<int> restDims(output->rankOf() - 1);
                    for (int i = restDims.size(); i > 0;  i--)
                        restDims[restDims.size() - i] = output->rankOf() - i;

                    auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), restDims);

                    std::vector<void *> inputBuffers(inputSize);
                    std::vector<Nd4jLong *> inputTadShapes(inputSize);
                    std::vector<Nd4jLong *> inputTadOffsets(inputSize);

                    std::vector<void *> indicesBuffers(inputSize);
                    std::vector<Nd4jLong *> indicesShapes(inputSize);

                    for (int e = 0; e < inputSize; e++) {
                        std::vector<int> sourceDims(inputs[e]->rankOf() - indices[e]->rankOf());
                        for (int i = sourceDims.size(); i > 0;  i--)
                            sourceDims[sourceDims.size() - i] = inputs[e]->rankOf() - i;

                        auto packX = ConstantTadHelper::getInstance()->tadForDimensions(inputs[e]->getShapeInfo(), sourceDims);

                        indicesBuffers[e] = indices[e]->getSpecialBuffer();
                        indicesShapes[e] = indices[e]->getSpecialShapeInfo();

                        inputBuffers[e] = inputs[e]->getSpecialBuffer();
                        inputTadShapes[e] = packX.platformShapeInfo();
                        inputTadOffsets[e] = packX.platformOffsets();
                    }

                    // copying pointers to buffers to device
                    auto dInputBuffers = reinterpret_cast<void **>(pm.replicatePointer(inputBuffers.data(), inputSize * sizeof(void *)));
                    auto dInputTadShapes = reinterpret_cast<Nd4jLong **>(pm.replicatePointer(inputTadShapes.data(), inputSize * sizeof(Nd4jLong *)));
                    auto dInputTadOffsets = reinterpret_cast<Nd4jLong **>(pm.replicatePointer(inputTadOffsets.data(), inputSize * sizeof(Nd4jLong *)));

                    auto dIndicesBuffers = reinterpret_cast<void **>(pm.replicatePointer(indicesBuffers.data(), inputSize * sizeof(void *)));
                    auto dIndicesShapes = reinterpret_cast<Nd4jLong **>(pm.replicatePointer(indicesShapes.data(), inputSize * sizeof(Nd4jLong *)));

                    dynamicStitchTadKernel<X,Y><<<256, 256, 1024, *context->getCudaStream()>>>(dInputBuffers, dInputTadShapes, dInputTadOffsets, dIndicesBuffers, dIndicesShapes, inputSize, output->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets());
                }

                pm.synchronize();

                return Status::OK();
            }

            template <typename T>
            static void _dynamicPartitionFunctorBP(NDArray const* input, NDArray const* indices, std::vector<NDArray*> const& inputGradientList, std::vector<NDArray*>& outputList) {

            }

            void dynamicPartitionFunctor(nd4j::LaunchContext * context, NDArray const* input, NDArray const* indices, std::vector<NDArray*>& outputList) {
                auto xType = input->dataType();
                auto yType = indices->dataType();

                NDArray::prepareSpecialUse({}, {indices, input});

                BUILD_DOUBLE_SELECTOR(xType, yType, _dynamicPartitionFunctor, (context, input, indices, outputList), NUMERIC_TYPES, INDEXING_TYPES);

                NDArray::registerSpecialUse({}, {indices, input});

                // TODO: it would be nice to have NDArray::registerSpecialUse signature that accepts something else beyond initializer_list
                for (auto v:outputList) {
                    v->tickWriteDevice();
                }
            }

            template <typename T>
            static int _dynamicStitchFunctorBP(std::vector<NDArray*> const& inputs, std::vector<NDArray*> const& indices, NDArray const* gradInput, std::vector<NDArray*>& outputList){
                throw std::runtime_error("Not umplemented yet");
            }

            int dynamicStitchFunctor(nd4j::LaunchContext * context, std::vector<NDArray*> const& inputs, std::vector<NDArray*> const& indices, NDArray* output){
                auto xType = inputs.at(0)->dataType();
                auto yType = indices.at(0)->dataType();

                for (auto v:indices) {
                    v->syncToDevice();
                    v->tickReadDevice();
                }

                for (auto v:inputs) {
                    v->syncToDevice();
                    v->tickReadDevice();
                }

                NDArray::prepareSpecialUse({output}, {});


                BUILD_DOUBLE_SELECTOR(xType, yType, _dynamicStitchFunctor, (context, inputs, indices, output), NUMERIC_TYPES, INDEXING_TYPES);

                NDArray::registerSpecialUse({output}, {});

                return Status::OK();
            }

            int dynamicStitchFunctorBP(nd4j::LaunchContext * context, std::vector<NDArray*> const& inputs, std::vector<NDArray*> const& indices, NDArray const* gradInput, std::vector<NDArray*>& outputList) {
                auto xType = inputs.at(0)->dataType();

                BUILD_SINGLE_SELECTOR(xType, return _dynamicStitchFunctorBP, (inputs, indices, gradInput, outputList), NUMERIC_TYPES);
            }

            void dynamicPartitionFunctorBP(nd4j::LaunchContext * context, NDArray const* input, NDArray const* indices, std::vector<NDArray*> const& inputGradientList, std::vector<NDArray*>& outputList) {
                auto xType = input->dataType();

                BUILD_SINGLE_SELECTOR(xType, _dynamicPartitionFunctorBP, (input, indices, inputGradientList, outputList), NUMERIC_TYPES);
            }

        }
    }
}

