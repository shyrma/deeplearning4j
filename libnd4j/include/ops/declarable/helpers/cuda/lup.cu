#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/top_k.h>
#include <MmulHelper.h>
#include <NDArrayFactory.h>
#include <Status.h>
#include <ConstantTadHelper.h>
#include <ShapeUtils.h>

#include <hipsolver.h>
#include <cuda_exception.h>

namespace nd4j {
namespace ops {
namespace helpers {

// ------------------------------------------------------------------------------------------------------------------ //
//  invert the second diagonal for lower diagonal matrix
    template<typename T>
    static __global__ void
    invertKernelLow(void *invertedBuf, Nd4jLong *invertedShape, void *inputBuf, Nd4jLong *inputShape, Nd4jLong n) {
        T* inverted = reinterpret_cast<T *>(invertedBuf);
        T* input = reinterpret_cast<T*>(inputBuf);

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start + 1; i < n; i += step) {
            Nd4jLong pos[] = {i, i - 1};
            Nd4jLong posX[] = {i, i};
            Nd4jLong posY[] = {i - 1, i - 1};
            auto xIndex = shape::getOffset(inputShape, pos);
            auto dxIndex = shape::getOffset(inputShape, posX);
            auto dyIndex = shape::getOffset(inputShape, posY);
            auto zIndex = shape::getOffset(invertedShape, pos);
            // invert lower triangular matrix
            inverted[zIndex] = -input[xIndex] / (input[dxIndex] * input[dyIndex]);
//            math::atomics::nd4j_atomicAdd(&inverted[zIndex], - input[xIndex] * inverted[iIndex] / input[dIndex]);
        }
    }
// ------------------------------------------------------------------------------------------------------------------ //
// invert diagonal vals to upper diagonal matrix
    template<typename T>
    static __global__ void
    upvertKernel(void *invertedBuf, Nd4jLong *invertedShape, void *inputBuf, Nd4jLong *inputShape, Nd4jLong n) {
        T *inverted = reinterpret_cast<T *>(invertedBuf);
        T *input = reinterpret_cast<T *>(inputBuf);

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start; i < n; i += step) {
            Nd4jLong pos[] = {i, i};
            auto xIndex = shape::getOffset(inputShape, pos);
            auto zIndex = shape::getOffset(invertedShape, pos);
//            math::atomics::nd4j_atomicDiv(&inverted[zIndex], input[xIndex]);
            // invert diagonal elements
            inverted[zIndex] /= input[xIndex];
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
//  invert upper second diagonal
    template<typename T>
    static __global__ void
    upvertKernelUp(void *invertedBuf, Nd4jLong *invertedShape, void *inputBuf, Nd4jLong *inputShape, Nd4jLong n) {

        __shared__ T* inverted;
        __shared__ T* input;
        if (threadIdx.x == 0) {
            inverted = reinterpret_cast<T *>(invertedBuf);
            input = reinterpret_cast<T *>(inputBuf);
        }
        __syncthreads();

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start; i < n - 1; i += step) {
            Nd4jLong pos[] = {i, i + 1};
            Nd4jLong posX[] = {i + 1, i + 1};
            auto xIndex = shape::getOffset(inputShape, pos);
            auto iIndex = shape::getOffset(invertedShape, posX);
            auto zIndex = shape::getOffset(invertedShape, pos);
            // invert upper matrix
            math::atomics::nd4j_atomicAdd(&inverted[zIndex], -input[xIndex] * inverted[iIndex]); // / input[yIndex]);
            //inputMatrix->t<T>(i, i + 1) * invertedMatrix->t<T>(i + 1, i + 1) / inputMatrix->t<T>(i, i)
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    template<typename T>
    static __global__ void
    invertLowKernel(void *invertedBuf, Nd4jLong *invertedShape, void *inputBuf, Nd4jLong *inputShape, Nd4jLong n) {
        T *inverted = reinterpret_cast<T *>(invertedBuf);
        T *input = reinterpret_cast<T *>(inputBuf);

        for (int i = blockIdx.x + 2; i < n; i += gridDim.x) {
            for (int j = i - 2; j >= 0; --j)
                for (int k = threadIdx.x; k < i; k += blockDim.x) {
                    Nd4jLong posZ[] = {i, j};
                    Nd4jLong posY[] = {k, j};
                    Nd4jLong posX[] = {i, k};
                    Nd4jLong posD[] = {i, i};

                    auto xIndex = shape::getOffset(inputShape, posX);
                    auto yIndex = shape::getOffset(invertedShape, posY);
                    auto dIndex = shape::getOffset(inputShape, posD);
                    auto zIndex = shape::getOffset(invertedShape, posZ);
                    // invert non-diagonal elements
                    math::atomics::nd4j_atomicAdd(&inverted[zIndex], -inverted[yIndex] * input[xIndex] / input[dIndex]);
                }
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
// Invertion of upper triangular matrix non-diagonal elements when main and second diagonals already processed
    template<typename T>
    static __global__ void
    invertUpKernel(void *invertedBuf, Nd4jLong *invertedShape, void *inputBuf, Nd4jLong *inputShape, Nd4jLong n) {
        __shared__ T* inverted;
        __shared__ T* input;

        if (threadIdx.x == 0) {
            inverted = reinterpret_cast<T *>(invertedBuf);;
            input = reinterpret_cast<T *>(inputBuf);
        }
        __syncthreads();

        for (int i = (int)n - blockIdx.x - 2; i >= 0; i -= gridDim.x) {
            for (int j = i + 2; j < (int)n; j++)
                for (int k = i + threadIdx.x; k < (int)n; k += blockDim.x) {
                    Nd4jLong posZ[] = {i, j};
                    Nd4jLong posY[] = {k, j};
                    Nd4jLong posX[] = {i, k};
                    // inversion with Joardan Gauss transformation
                    auto xIndex = shape::getOffset(inputShape, posX);
                    auto yIndex = shape::getOffset(invertedShape, posY);
                    auto zIndex = shape::getOffset(invertedShape, posZ);
                    // invert upper non-diagonal elements
                    math::atomics::nd4j_atomicAdd(&inverted[zIndex], -inverted[yIndex] * input[xIndex]);
                }
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
// procedure to invert lower-triangular matrix.
// In current case lower triangular matrix has main diagonal with general values
//
    template<typename T>
    static void invertLowerMatrix_(LaunchContext *context, NDArray *inputMatrix, NDArray *invertedMatrix) {
        int n = inputMatrix->rows();
        invertedMatrix->setIdentity();

        if (inputMatrix->isIdentityMatrix()) return;

        auto stream = context->getCudaStream();

        // invert lower matrix
        // invert main diagonal
        upvertKernel<T><<<1, n, 512, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
        // invert the second diagonal
        invertKernelLow<T><<<1, n, 512, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
        // invert non-diagonal elements
        invertLowKernel<T><<<n, n, 512, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
    }

// ------------------------------------------------------------------------------------------------------------------ //
// caller for invert lower matrix routine
    void invertLowerMatrix(LaunchContext *context, NDArray *inputMatrix, NDArray *invertedMatrix) {
        NDArray::prepareSpecialUse({invertedMatrix}, {inputMatrix});
        BUILD_SINGLE_SELECTOR(inputMatrix->dataType(), invertLowerMatrix_, (context, inputMatrix, invertedMatrix), FLOAT_NATIVE);
        NDArray::registerSpecialUse({invertedMatrix}, {inputMatrix});
    }

// ------------------------------------------------------------------------------------------------------------------ //
// procedure to invert upper-triangular matrix.
// In current case upper triangular matrix has main diagonal with all ones on it.
    template<typename T>
    static void invertUpperMatrix_(LaunchContext *context, NDArray* inputMatrix, NDArray* invertedMatrix) {
        int n = inputMatrix->rows();
        invertedMatrix->setIdentity();
        auto stream = context->getCudaStream();
        if (inputMatrix->isIdentityMatrix()) { // the inverse for I is I
            return;
        }

        // invert upper matrix
        // invert the second diagonal
        upvertKernelUp<T><<<1, n, 512, *stream >>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(),
                inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);

        // invert other elements
        invertUpKernel<T><<<n, n, 512, *stream >>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(),inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
    }

// ------------------------------------------------------------------------------------------------------------------ //
//  invertion of upper triangular matrix - runner routine
    void invertUpperMatrix(LaunchContext *context, NDArray *inputMatrix, NDArray *invertedMatrix) {
        NDArray::prepareSpecialUse({invertedMatrix}, {inputMatrix});
        BUILD_SINGLE_SELECTOR(invertedMatrix->dataType(), invertUpperMatrix_, (context, inputMatrix, invertedMatrix), FLOAT_NATIVE);
        NDArray::prepareSpecialUse({invertedMatrix}, {inputMatrix});
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // determinant kernel - accumulation product of all values on the main diagonal
    template<typename T>
    static __global__ void determinantKernel(T *compound, T *result, Nd4jLong len) {
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        for (auto i = start; i < len; i += step) {
            auto pos = i * len + i; //shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), di, 2);
            // multiply all diagonal elements
            math::atomics::nd4j_atomicMul(&result[0], compound[pos]);
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // determinant logarithm - accumulation sum of all logarithm values on the main diagonal. All in logarithic values
    // should be positive
    template<typename T>
    static __global__ void determinantLogKernel(T *compound, T *result, Nd4jLong len) {
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        for (auto i = start; i < len; i += step) {
            auto pos = i * len + i; //shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), di, 2);
            // sum logs of all diagonal elements
            math::atomics::nd4j_atomicAdd(result, math::nd4j_log<T,T>(math::nd4j_abs(compound[pos])));
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // kernel to copy matrix with given shape to compound tensor with given pos
    // output - a N-D tensor buffer with rank not less than 2, input - 2D square n x n matrix with n = rowLen
    template<typename T, typename F>
    static __global__ void
    fillMatrix(void *output, Nd4jLong *outShape, void *input, Nd4jLong *inputShape, Nd4jLong pos, Nd4jLong rowLen) {
        __shared__ F *matrix;
        __shared__ T *inputBuf;
        __shared__ Nd4jLong inputLen;
        __shared__ Nd4jLong n2;

        if (threadIdx.x == 0) {
            matrix = reinterpret_cast<F*>(output);
            inputBuf = reinterpret_cast<T*>(input);
            inputLen = shape::length(inputShape);
            n2 = rowLen * rowLen;
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int k = pos + start, j = start; j < n2; k += step, j += step) {
            auto xIndex = shape::getIndexOffset(k, inputShape);
            matrix[j] = (F) inputBuf[xIndex];
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
// same as above, but without type conversion
    template<typename T>
    static __global__ void
    returnMatrix(void *output, Nd4jLong *outputShape, void *input, Nd4jLong *inputShape, Nd4jLong pos, Nd4jLong rowLen) {
        __shared__ T* matrix;
        __shared__ T* outputBuf;
        __shared__ Nd4jLong outputLen;
        __shared__ Nd4jLong n2;

        if (threadIdx.x == 0) {
            matrix = reinterpret_cast<T *>(input);
            outputBuf = reinterpret_cast<T *>(output);
            outputLen = shape::length(inputShape);
            n2 = rowLen * rowLen;
        }
        __syncthreads();
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int k = pos + start, j = start; j < n2; k += step, j += step) {
            auto zIndex = shape::getIndexOffset(k, outputShape);
            outputBuf[zIndex] = matrix[j];
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // fill up permutaion matrix kernel. Permutation matrix filled with zeros and ones
    template<typename F>
    static __global__ void fillUpPermutation(void *output, Nd4jLong *shape, int *source, int rowNum) {
        F *permutation = reinterpret_cast<F *>(output);

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        for (auto i = start; i < rowNum; i += step) {
            int val = source[i] - 1;
            Nd4jLong posF[] = {i, val};
            auto pos = shape::getOffset(shape, posF);
            permutation[pos] = F(1.f);
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // LUP decomposition runner - using CUBLAS SOLVER
    // if permutation is given, then using LUP decomposition, LU decomposition otherwise
    // L - lower triangular, U - upper triangular, P - permutation matricies
    // PA = LU
    //
    // input - A matrix nxn
    // compound - C matrix L + U - I, or main diagonal and lower - L matrix, from the 2nd diagonal - U matrix
    template<typename T>
    static void lup_(LaunchContext *context, NDArray *input, NDArray *compound, NDArray *permutation) {
        auto stream = context->getCudaStream();
        auto n = input->rows();
        hipsolverHandle_t cusolverH = nullptr;
        // create solver handle
        hipsolverStatus_t status = hipsolverDnCreate(&cusolverH);
        if (HIPSOLVER_STATUS_SUCCESS != status) {
            throw cuda_exception::build("Cannot create cuSolver handle", status);
        }
        // set solver stream
        status = hipsolverSetStream(cusolverH, *stream);
        if (HIPSOLVER_STATUS_SUCCESS != status) {
            throw cuda_exception::build("Cannot set up stream for cuda solver", status);
        }
        int lwork = 0;
        int *d_info = nullptr;
        // allocate memory for permutation vector
        auto err = hipMalloc((void **) &d_info, sizeof(int));
        if (err) {
            throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver info buffer", err);
        }

        DataType dtype = input->dataType();
        switch (dtype) { // there are two implementations with cublas for LUP decomposition - double and float

            case DataType::DOUBLE: {
                double *d_work = nullptr;
                // compute internal buffer size
                double *matrix = reinterpret_cast<double *>(input->specialBuffer());
                status = hipsolverDnDgetrf_bufferSize(
                        cusolverH,
                        n,
                        n,
                        matrix,
                        n,
                        &lwork);
                if (HIPSOLVER_STATUS_SUCCESS != status) {
                    throw cuda_exception::build("helpers::lup_: Cannot create cuSolver handle", status);
                }

                err = hipMalloc((void **) &d_work, sizeof(float) * lwork);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver data buffer",
                                                err);
                }

                if (permutation == nullptr)
                    status = hipsolverDnDgetrf(
                            cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            nullptr,
                            d_info);
                else {
                    NDArray permutVector('c', {n}, nd4j::DataType::INT32, context);
                    int *permutationBuf = reinterpret_cast<int *>(permutVector.specialBuffer());
                    status = hipsolverDnDgetrf(
                            cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            permutationBuf,
                            d_info);
                    fillUpPermutation<double> << < n, n, 1024, *stream >> >
                                                              (permutation->specialBuffer(), permutation->specialShapeInfo(), permutationBuf, n);
                    permutation->tickWriteDevice();
                }
                err = hipFree(d_work);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver data buffer",
                                                err);
                }
            }
                break;
            case DataType::FLOAT32: {
                float *matrix = reinterpret_cast<float*>(input->specialBuffer());
                float *d_work = nullptr;

                status = hipsolverDnSgetrf_bufferSize(
                        cusolverH,
                        n,
                        n,
                        matrix,
                        n,
                        &lwork);
                if (HIPSOLVER_STATUS_SUCCESS != status) {
                    throw cuda_exception::build("helpers::lup_: Cannot create cuSolver handle", status);
                }

                err = hipMalloc((void **) &d_work, sizeof(float) * lwork);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver data buffer",
                                                err);
                }

                if (permutation == nullptr)
                    status = hipsolverDnSgetrf(
                            cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            nullptr,
                            d_info);
                else {
                    NDArray permutVector('c', {n}, nd4j::DataType::INT32, context);
                    int *permutationBuf = reinterpret_cast<int *>(permutVector.specialBuffer());
                    status = hipsolverDnSgetrf(
                            cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            permutationBuf,
                            d_info);
                    fillUpPermutation<T> <<< n, n, 128, *stream >> >
                                                         (permutation->specialBuffer(), permutation->specialShapeInfo(), permutationBuf, n);
                    permutation->tickWriteDevice();
                }
                err = hipFree(d_work);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver data buffer",
                                                err);
                }

            }
        }
        if (HIPSOLVER_STATUS_SUCCESS != status) {
            throw cuda_exception::build("helpers::lup_: Cannot make LU decomposition", status);
        }
        err = hipFree(d_info);
        if (err) {
            throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver info buffer", err);
        }
        hipsolverDnDestroy(cusolverH);
//        NDArray::registerSpecialUse({input}, {input});
        input->tickWriteDevice();
    }
// ------------------------------------------------------------------------------------------------------------------ //

    BUILD_SINGLE_TEMPLATE(template void lup_,(LaunchContext * context, NDArray * input, NDArray * output, NDArray * permutation), FLOAT_NATIVE);

// ------------------------------------------------------------------------------------------------------------------ //
    template<typename T>
    static int determinant_(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
        Nd4jLong n = input->sizeAt(-1);
        Nd4jLong n2 = n * n;
        std::vector<int> dims();
        auto packX = ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), {input->rankOf() - 2, input->rankOf() - 1});
        //auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(output->shapeInfo(), {output->rankOf() - 1});
//        DataType dtype = input->dataType();
//        if (dtype != DataType::DOUBLE)
//            dtype = DataType::FLOAT32;
        auto matrix = NDArrayFactory::create(input->ordering(), {n, n}, DataTypeUtils::fromT<T>(), context); //, block.getWorkspace());
        auto det = NDArrayFactory::create<T>(1);
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input});
        dim3 launchDims(256, 256, 1024);
        output->assign(1.f);
        for (int e = 0; e < output->lengthOf(); e++) {
            Nd4jLong pos = e * n2;
//            if (matrix.dataType() == input->dataType())
            fillMatrix<T, T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);
//            else
//                fillMatrix<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);

//            if (matrix.dataType() == input->dataType())
            lup_<T>(context, &matrix, nullptr, nullptr);
//            else
//                lup_<float>(context, &matrix, nullptr, nullptr);
            auto offset = shape::getIndexOffset(e, output->shapeInfo());
            auto inputBuf = reinterpret_cast<T *>(matrix.specialBuffer());
            auto outputBuf = reinterpret_cast<T *>(output->specialBuffer()) + offset;
//            if (matrix.dataType() == input->dataType())
            determinantKernel<T> << < launchDims.x, launchDims.y, launchDims.z, *stream >> >
                                                                                (inputBuf, outputBuf, n);
//            else
//                determinantKernel<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream >>> (inputBuf, outputBuf, n);
        }
        NDArray::registerSpecialUse({output}, {input});

        return Status::OK();
    }

        int determinant(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            BUILD_SINGLE_SELECTOR(input->dataType(), return determinant_, (context, input, output), FLOAT_NATIVE);
            NDArray::registerSpecialUse({output}, {input});
        }

        template<typename T>
        int logAbsDeterminant_(LaunchContext *context, NDArray *input, NDArray *output) {
            Nd4jLong n = input->sizeAt(-1);
            Nd4jLong n2 = n * n;
            std::vector<int> dims();
            auto packX = ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), {input->rankOf() - 2, input->rankOf() - 1});
            //auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(output->shapeInfo(), {output->rankOf() - 1});
            DataType dtype = input->dataType();
            if (dtype != DataType::DOUBLE)
                dtype = DataType::FLOAT32;

            auto matrix = NDArrayFactory::create(input->ordering(), {n, n}, dtype, context); //, block.getWorkspace());
            auto det = NDArrayFactory::create<T>(1);
            auto stream = context->getCudaStream();
            NDArray::prepareSpecialUse({output}, {input});
            dim3 launchDims(256, 256, 1024);
            output->assign(0.f);
            for (int e = 0; e < output->lengthOf(); e++) {
                Nd4jLong pos = e * n2;
//            if (matrix.dataType() == input->dataType())
                fillMatrix<T, T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);
//            else
//                fillMatrix<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);

//            if (matrix.dataType() == input->dataType())
                lup_<T>(context, &matrix, nullptr, nullptr);
//            else
//                lup_<float>(context, &matrix, nullptr, nullptr);
                auto offset = shape::getIndexOffset(e, output->shapeInfo());
                auto inputBuf = reinterpret_cast<T *>(matrix.specialBuffer());
                auto outputBuf = reinterpret_cast<T *>(output->specialBuffer()) + offset;
//            if (matrix.dataType() == input->dataType())
                determinantLogKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(inputBuf, outputBuf, n);
//            else
//                determinantLogKernel<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream >>> (inputBuf, outputBuf, n);
            }
            NDArray::registerSpecialUse({output}, {input});

            return Status::OK();

            return ND4J_STATUS_OK;
        }

        int logAbsDeterminant(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            BUILD_SINGLE_SELECTOR(input->dataType(), return logAbsDeterminant_, (context, input, output), FLOAT_NATIVE);
            NDArray::registerSpecialUse({output}, {input});
        }

        template<typename T>
        static __global__ void
        fillLowerUpperKernel(void *lowerBuf, Nd4jLong *lowerShape, void *upperBuf, Nd4jLong *upperShape,
                             void *matrixBuf, Nd4jLong *matrixShape, Nd4jLong n) {

            __shared__ T *lowerMatrix;
            __shared__ T *upperMatrix;
            __shared__ T *matrix;

            if (threadIdx.x == 0) {
                lowerMatrix = reinterpret_cast<T *>(lowerBuf);
                upperMatrix = reinterpret_cast<T *>(upperBuf);
                matrix = reinterpret_cast<T *>(matrixBuf);
            }
            __syncthreads();

            for (int k = blockIdx.x; k < n; k += gridDim.x) {  // and then put all values under main diagonal on to it
                for (int j = threadIdx.x; j < n; j += blockDim.x) {
                    Nd4jLong posX[] = {k, j};
                    Nd4jLong posD[] = {j, j};
                    auto xPos = shape::getOffset(lowerShape, posX);
                    auto yPos = shape::getOffset(upperShape, posX);
                    auto iPos = shape::getOffset(matrixShape, posX);
                    auto dPos = shape::getOffset(matrixShape, posD);
                    if (k >= j)
                        lowerMatrix[xPos] = matrix[iPos];//(k, j);
                    else
                        upperMatrix[yPos] = matrix[iPos]; //k, j);
                }
            }
        }

        template<typename T>
        static int inverse_(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
            auto n = input->sizeAt(-1);
            auto n2 = n * n;
            auto dtype = DataTypeUtils::fromT<T>(); //input->dataType();
//            if (dtype != DataType::DOUBLE)
//                dtype = DataType::FLOAT32;
            NDArray matrix = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray upper = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray lower = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray compound = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray permutation = NDArrayFactory::create('c', {n, n}, dtype, context);
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(),
                                                                                  {input->rankOf() - 2,
                                                                                   input->rankOf() - 1});
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(),
                                                                                  {output->rankOf() - 2,
                                                                                   output->rankOf() - 1});
            auto stream = context->getCudaStream();

            for (auto i = 0LL; i < packX.numberOfTads(); i++) {
                fillMatrix<T, T><<<1, n2, 1024, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), i * n2, n);
                matrix.tickWriteDevice();
                compound.assign(matrix);
                lup_<T>(context, &compound, nullptr, nullptr);
                fillLowerUpperKernel<T><<<n, n, 1024, *stream>>>(lower.specialBuffer(), lower.specialShapeInfo(), upper.specialBuffer(), upper.specialShapeInfo(), compound.specialBuffer(), compound.specialShapeInfo(), n);
                matrix.assign(0);
                invertUpperMatrix(context, &upper, &matrix); // U^{-1}
                matrix.tickWriteDevice();
//                matrix.printIndexedBuffer("Upper Inverted");
                compound.assign(0);
                invertLowerMatrix(context, &lower, &compound); // L{-1}
                compound.tickWriteDevice();
//                compound.printIndexedBuffer("Lower Inverted");
//                matrix.tickWriteDevice();
//                compound.tickWriteDevice();
                nd4j::MmulHelper::mmul(&matrix, &compound, &upper, 1.0, 0.0);
                upper.tickWriteDevice();
//                upper.printIndexedBuffer("Full inverted");
                returnMatrix<T> <<<1, n2, 1024, *stream>>>(output->specialBuffer(), output->specialShapeInfo(), upper.specialBuffer(), upper.specialShapeInfo(), i * n2, n);
            }
            return Status::OK();
        }

        int inverse(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            BUILD_SINGLE_SELECTOR(input->dataType(), return inverse_, (context, input, output), FLOAT_NATIVE);
            NDArray::registerSpecialUse({output}, {input});
        }

        bool checkCholeskyInput(nd4j::LaunchContext *context, NDArray const *input) {
            return true;
        }

        template<typename F>
        __global__ void fillBatchKernel(F **dArrayBatch, F *buf, Nd4jLong *offsets, Nd4jLong batchSize) {
            auto start = blockIdx.x * blockDim.x + threadIdx.x;
            auto step = blockDim.x * gridDim.x;

            for (auto i = start; i < batchSize; i += step) {
                dArrayBatch[i] = buf + offsets[i];
            }
        }

        template<typename F>
        __global__ void
        adjustResultsKernel(F *dArray, Nd4jLong *shape, Nd4jLong *offsets, Nd4jLong batchSize, Nd4jLong n) {
            //auto i = blockIdx.x * blockDim.x + threadIdx.x;
            Nd4jLong *shapeOf = shape::shapeOf(shape);
            Nd4jLong *strideOf = shape::stride(shape);

            for (auto i = blockIdx.x; i < batchSize; i += gridDim.x) {
                auto current = dArray + offsets[i];
                for (auto r = threadIdx.x; r < n; r += blockDim.x) {
                    for (auto c = r + 1; c < n; c++) {
                        Nd4jLong posRC[] = {r, c};
                        auto pos = r * n + c; //shape::getOffset(0, shapeOf, strideOf, posRC, 2);
                        current[pos] = 0.;
                    }
                }
            }
        }

        template<typename F>
        int cholesky__(LaunchContext *context, NDArray *input, NDArray *output, bool inplace) {
            if (!inplace)
                output->assign(input);
            std::unique_ptr<NDArray> tempOutput(output->dup());
            hipsolverHandle_t handle = nullptr;
            auto n = input->sizeAt(-1);
            auto n2 = n * n;
            NDArray::prepareSpecialUse({output}, {input});
            auto status = hipsolverDnCreate(&handle);
            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("helpers::cholesky_: Cannot create solver handle", status);
            }
            F **dArrayBatch = nullptr;
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempOutput->getShapeInfo(),
                                                                                  {tempOutput->rankOf() - 2,
                                                                                   tempOutput->rankOf() - 1});
            const Nd4jLong batchSize = packX.numberOfTads();
            int *dInfoArray = nullptr;
            auto err = hipMalloc((void **) &dArrayBatch, sizeof(F *) * batchSize);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver batch data buffer",
                                            err);
            }
            err = hipMalloc((void **) &dInfoArray, sizeof(int) * batchSize);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver errors buffer", err);
            }
            auto stream = context->getCudaStream();
            fillBatchKernel<F> << < 1, batchSize, 128, *stream >> >
                                                       (dArrayBatch, reinterpret_cast<F *>(tempOutput->specialBuffer()), packX.specialOffsets(), batchSize);

            status = hipsolverSetStream(handle, *stream);
            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("helpers::cholesky_: Cannot set stream to solver handle", status);
            }
            const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
            if (input->dataType() == DataType::DOUBLE)
                status = hipsolverDnDpotrfBatched(
                        handle,
                        uplo,
                        n,
                        (double **) dArrayBatch,
                        n,
                        dInfoArray,
                        batchSize);
            else
                status = hipsolverDnSpotrfBatched(
                        handle,
                        uplo,
                        n,
                        (float **) dArrayBatch,
                        n,
                        dInfoArray,
                        batchSize);

            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("helpers::cholesky_: Cholesky factorization failed for batch", status);
            }
            adjustResultsKernel<F> << < batchSize, n2, 128, *stream >> >
                                                            (reinterpret_cast<F *>(tempOutput->specialBuffer()), packX.specialShapeInfo(), packX.specialOffsets(), batchSize, n);

            err = hipFree(dArrayBatch);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot deallocate memory for solver batch data buffer",
                                            err);
            }
            err = hipFree(dInfoArray);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver errors buffer", err);
            }

            if (!inplace)
                output->assign(tempOutput.get());
            else
                input->assign(tempOutput.get());

            NDArray::registerSpecialUse({output}, {input});
            return Status::OK();
        }

//    template <typename T>
        int cholesky_(LaunchContext *context, NDArray *input, NDArray *output, bool inplace) {
            NDArray::prepareSpecialUse({output}, {input});
            if (input->dataType() == DataType::DOUBLE)
                cholesky__<double>(context, input, output, inplace);
            else if (input->dataType() == DataType::FLOAT32)
                cholesky__<float>(context, input, output, inplace);
            else {
                std::unique_ptr<NDArray> tempOutput(
                        NDArrayFactory::create_('c', input->getShapeAsVector(), DataType::FLOAT32, context));
                tempOutput->assign(input);
                cholesky__<float>(context, tempOutput.get(), tempOutput.get(), true);
                output->assign(tempOutput.get());
            }
            NDArray::registerSpecialUse({output}, {input});
            return Status::OK();
        }

        int cholesky(nd4j::LaunchContext *context, NDArray *input, NDArray *output, bool inplace) {
//        BUILD_SINGLE_SELECTOR(input->dataType(), return cholesky_, (context, input, output, inplace), FLOAT_TYPES);
            return cholesky_(context, input, output, inplace);
        }
//    BUILD_SINGLE_TEMPLATE(template int cholesky_, (LaunchContext* context, NDArray* input, NDArray* output, bool inplace), FLOAT_TYPES);
        BUILD_SINGLE_TEMPLATE(template int inverse_, (nd4j::LaunchContext * context, NDArray * input, NDArray * output),
                              FLOAT_NATIVE);

        template<typename T>
        __global__ void
        logDetKernel(T *inputBuf, Nd4jLong *inputShape, Nd4jLong batchNum, Nd4jLong *tadShape, Nd4jLong *tadOffsets,
                     T *outputBuf, Nd4jLong *outputShape) {

            __shared__ int n;
            if (threadIdx.x == 0) {
                n = shape::sizeAt(inputShape, -1); // * shape::sizeAt(inputShape, -1);
            }
            __syncthreads();

            T *output = outputBuf;
            T *input = inputBuf;

            for (auto i = blockIdx.x; i < batchNum; i += gridDim.x) {
                T *current = input + tadOffsets[i];

                auto zIndex = shape::getIndexOffset(i, outputShape);
                for (auto e = threadIdx.x; e < n; e += blockDim.x) {
                    Nd4jLong diag[] = {e, e};
                    auto xIndex = shape::getOffset(tadShape, diag);
                    math::atomics::nd4j_atomicAdd(&output[zIndex],math::nd4j_log<T, T>(current[xIndex] * current[xIndex]));
                }
            }
        }

        template<typename T>
        int logdetFunctor_(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            auto n2 = input->sizeAt(-1) * input->sizeAt(-2);
            auto stream = context->getCudaStream();
            std::unique_ptr<NDArray> tempOutput(input->dup());
//        auto inputs = tempOutput->allTensorsAlongDimension({input->rankOf() - 2, input->rankOf() - 1});
//        for (Nd4jLong e = 0; e < packX.numberOfTads(); e++) {
//            auto subArray = inputs->at(e);
//            cholesky(context, subArray, subArray, true);
//        }
//        delete inputs;
            cholesky(context, input, tempOutput.get(), false);
            tempOutput->syncToHost();
            tempOutput->printIndexedBuffer("Cholesky res!!!");
            auto outputBuf = reinterpret_cast<T*>(output->specialBuffer()); // + e * n2; // + e * n2;
            auto inputBuf = reinterpret_cast<T*>(tempOutput->specialBuffer());
            output->assign(0);
            output->syncToDevice();
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempOutput->getShapeInfo(),
                                                                                  {input->rankOf() - 2,
                                                                                   input->rankOf() - 1});
            logDetKernel<T> << < packX.numberOfTads(), n2, 128, *stream >> >
                                                                (inputBuf, tempOutput->specialShapeInfo(), packX.numberOfTads(), packX.specialShapeInfo(), packX.specialOffsets(), outputBuf, output->specialShapeInfo());
//        }
            NDArray::registerSpecialUse({output}, {input});
            //delete tempOutput;
            return Status::OK();
        }

        int logdetFunctor(nd4j::LaunchContext *context, NDArray *input, NDArray *output) {
            BUILD_SINGLE_SELECTOR(output->dataType(), logdetFunctor_, (context, input, output), FLOAT_NATIVE);
        }

        BUILD_SINGLE_TEMPLATE(template int logdetFunctor_,
                              (nd4j::LaunchContext * context, NDArray * input, NDArray * output), FLOAT_NATIVE);
    }
}
}
