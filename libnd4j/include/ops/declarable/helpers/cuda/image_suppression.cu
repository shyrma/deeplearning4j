#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/image_suppression.h>
#include <NDArrayFactory.h>
#include <NativeOps.h>
#include <cuda_exception.h>

namespace nd4j {
namespace ops {
namespace helpers {
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// needToSuppressWithThreshold - predicate for suppression
//      boxes - boxes tensor buffer
//      boxesShape boxes tensor shape
//      previousIndex - index for current pos value
//      nextIndex - index for neighbor pos value
//      threshold - threashold value to suppress
//
//      return value: true, if threshold is overcome, false otherwise
//
    template <typename T>
    static __device__ bool needToSuppressWithThreshold(T* boxes, Nd4jLong* boxesShape, int previousIndex, int nextIndex, T threshold) {
        Nd4jLong previous0[] = {previousIndex, 0};
        Nd4jLong previous1[] = {previousIndex, 1};
        Nd4jLong previous2[] = {previousIndex, 2};
        Nd4jLong previous3[] = {previousIndex, 3};
        Nd4jLong next0[] = {nextIndex, 0};
        Nd4jLong next1[] = {nextIndex, 1};
        Nd4jLong next2[] = {nextIndex, 2};
        Nd4jLong next3[] = {nextIndex, 3};

        // we have rectangle with given max values. Compute vexes of rectangle first

        T minYPrev = nd4j::math::nd4j_min(boxes[shape::getOffset(boxesShape, previous0)], boxes[shape::getOffset(boxesShape, previous2)]);
        T minXPrev = nd4j::math::nd4j_min(boxes[shape::getOffset(boxesShape, previous1)], boxes[shape::getOffset(boxesShape, previous3)]);
        T maxYPrev = nd4j::math::nd4j_max(boxes[shape::getOffset(boxesShape, previous0)], boxes[shape::getOffset(boxesShape, previous2)]);
        T maxXPrev = nd4j::math::nd4j_max(boxes[shape::getOffset(boxesShape, previous1)], boxes[shape::getOffset(boxesShape, previous3)]);
        T minYNext = nd4j::math::nd4j_min(boxes[shape::getOffset(boxesShape, next0)],     boxes[shape::getOffset(boxesShape, next2)]);
        T minXNext = nd4j::math::nd4j_min(boxes[shape::getOffset(boxesShape, next1)],     boxes[shape::getOffset(boxesShape, next3)]);
        T maxYNext = nd4j::math::nd4j_max(boxes[shape::getOffset(boxesShape, next0)],     boxes[shape::getOffset(boxesShape, next2)]);
        T maxXNext = nd4j::math::nd4j_max(boxes[shape::getOffset(boxesShape, next1)],     boxes[shape::getOffset(boxesShape, next3)]);

        // compute areas for comparation
        T areaPrev = (maxYPrev - minYPrev) * (maxXPrev - minXPrev);
        T areaNext = (maxYNext - minYNext) * (maxXNext - minXNext);

        // of course, areas should be positive
        if (areaNext <= T(0.f) || areaPrev <= T(0.f)) return false;

        // compute intersection of rectangles
        T minIntersectionY = nd4j::math::nd4j_max(minYPrev, minYNext);
        T minIntersectionX = nd4j::math::nd4j_max(minXPrev, minXNext);
        T maxIntersectionY = nd4j::math::nd4j_min(maxYPrev, maxYNext);
        T maxIntersectionX = nd4j::math::nd4j_min(maxXPrev, maxXNext);
        T intersectionArea =
                nd4j::math::nd4j_max(T(maxIntersectionY - minIntersectionY), T(0.0f)) *
                nd4j::math::nd4j_max(T(maxIntersectionX - minIntersectionX), T(0.0f));
        T intersectionValue = intersectionArea / (areaPrev + areaNext - intersectionArea);
        // final check
        return intersectionValue > threshold;
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// shouldSelectKernel - compute status for all selected rectangles (boxes)
//
// we compute boolean flag as shared uint32 and return it on final only for the first thread
//
    template <typename T, typename I>
    static __global__ void shouldSelectKernel(T* boxesBuf, Nd4jLong* boxesShape, I* indexBuf, I* selectedIndicesData, double threshold, int numSelected, int i, bool* shouldSelect) {
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;
        __shared__ unsigned int shouldSelectShared;
        if (threadIdx.x == 0) {
            shouldSelectShared = (unsigned int)shouldSelect[0];
        }
        __syncthreads();
        for (int j = numSelected - 1 - tid; j >= 0; j -= step) {
            if (shouldSelectShared) {
                if (needToSuppressWithThreshold(boxesBuf, boxesShape, indexBuf[i],
                                                                  indexBuf[selectedIndicesData[j]], T(threshold)))
                    atomicCAS(&shouldSelectShared, 1, 0); // exchange only when need to suppress
            }
        }
        __syncthreads();

        // final move: collect result
        if (threadIdx.x == 0) {
            *shouldSelect = shouldSelectShared > 0;
        }
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// indices - type depended, indicesLong - type defined (only 64bit integers)
//
    template <typename I>
    static __global__ void copyIndices(void* indices,  void* indicesLong, Nd4jLong len) {
        I* indexBuf = reinterpret_cast<I*>(indices);
        Nd4jLong* srcBuf = reinterpret_cast<Nd4jLong*>(indicesLong);;

        auto tid = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (auto i = tid; i < len; i += step)
            indexBuf[i] = (I)srcBuf[i];
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// nonMaxSuppressionV2 algorithm - given from TF NonMaxSuppressionV2 implementation
//
    template <typename T, typename I>
    static void nonMaxSuppressionV2_(nd4j::LaunchContext* context, NDArray* boxes, NDArray* scales, int maxSize, double threshold, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {boxes, scales});
        std::unique_ptr<NDArray> indices(NDArrayFactory::create_<I>('c', {scales->lengthOf()})); // - 1, scales->lengthOf()); //, scales->getContext());
        indices->linspace(0);
        indices->syncToDevice(); // linspace only on CPU, so sync to Device as well

        NDArray scores(*scales);
        Nd4jPointer extras[2] = {nullptr, stream};

        sortByValue(extras, indices->buffer(), indices->shapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), scores.buffer(), scores.shapeInfo(), scores.specialBuffer(), scores.specialShapeInfo(), true);

        auto indexBuf = reinterpret_cast<I*>(indices->specialBuffer());

        NDArray selectedIndices = NDArrayFactory::create<I>('c', {output->lengthOf()});
        int numSelected = 0;
        int numBoxes = boxes->sizeAt(0);
        auto boxesBuf = reinterpret_cast<T*>(boxes->specialBuffer());

        auto selectedIndicesData = reinterpret_cast<I*>(selectedIndices.specialBuffer());
        auto outputBuf = reinterpret_cast<I*>(output->specialBuffer());

        bool* shouldSelectD;
        auto err = hipMalloc(&shouldSelectD, sizeof(bool));
        if (err) {
            throw cuda_exception::build("helpers::nonMaxSuppressionV2: Cannot allocate memory for bool flag", err);
        }
        for (I i = 0; i < boxes->sizeAt(0); ++i) {
            bool shouldSelect = numSelected < output->lengthOf();
            if (shouldSelect) {
                err = hipMemcpy(shouldSelectD, &shouldSelect, sizeof(bool), hipMemcpyHostToDevice);
                if (err) {
                    throw cuda_exception::build("helpers::nonMaxSuppressionV2: Cannot set up bool flag to device", err);
                }

                shouldSelectKernel<T,I><<<128, 256, 1024, *stream>>>(boxesBuf, boxes->specialShapeInfo(), indexBuf, selectedIndicesData, threshold, numSelected, i, shouldSelectD);
                err = hipMemcpy(&shouldSelect, shouldSelectD, sizeof(bool), hipMemcpyDeviceToHost);
                if (err) {
                    throw cuda_exception::build("helpers::nonMaxSuppressionV2: Cannot set up bool flag to host", err);
                }
            }

            if (shouldSelect) {
                hipMemcpy(reinterpret_cast<I*>(output->specialBuffer()) + numSelected, indexBuf + i, sizeof(I), hipMemcpyDeviceToDevice);
                hipMemcpy(selectedIndicesData + numSelected, &i, sizeof(I), hipMemcpyHostToDevice);
                numSelected++;
            }
        }

        err = hipFree(shouldSelectD);
        if (err) {
            throw cuda_exception::build("helpers::nonMaxSuppressionV2: Cannot deallocate memory for bool flag", err);
        }

    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    void nonMaxSuppressionV2(nd4j::LaunchContext * context, NDArray* boxes, NDArray* scales, int maxSize, double threshold, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(boxes->dataType(), output->dataType(), nonMaxSuppressionV2_, (context, boxes, scales, maxSize, threshold, output), FLOAT_TYPES, INDEXING_TYPES);
    }

}
}
}
