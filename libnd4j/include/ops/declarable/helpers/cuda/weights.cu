#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/weights.h>

namespace nd4j {
namespace ops {
namespace helpers {



    template <typename T>
    static __device__ void adjustWeightsKernelD(void* inputBuffer,   Nd4jLong* inputShape,
                                               void* weightsBuffer, Nd4jLong* weightsShape,
                                               void* outputBuffer,  Nd4jLong inputLength,
                                               Nd4jLong outputLength, int val) {
    //    typedef Nd4jLong T;
        auto tid = threadIdx.x;
        //int threadCount = gridDim.x * blockDim.x;
        __shared__ T* outputPart;
        __shared__ Nd4jLong offset;
        //for (int e = 0; e < inputLength; e++) {
        for (Nd4jLong e = tid; e < inputLength; e += blockDim.x) {

            Nd4jLong xOffset = shape::getIndexOffset(e, inputShape);
            int current = *(reinterpret_cast<int*>(inputBuffer) + xOffset);
            if (current == val) {
                //printf("%lld\n", xOffset);
                //Nd4jLong zOffset = shape::getIndexOffset(val, outputShape);
                if (weightsBuffer != nullptr) {
                    Nd4jLong yOffset = shape::getIndexOffset(e, weightsShape);
                    //atomicAdd();
                    //*reinterpret_cast<int *>(outputBuffer) +=  reinterpret_cast<int *>(weightsBuffer)[yOffset];
                    nd4j::math::atomics::nd4j_atomicAdd(reinterpret_cast<T *>(outputBuffer), reinterpret_cast<T *>(weightsBuffer)[yOffset]); //output->p(val, output->e<T>(val) + 1);
//                    atomicAdd(reinterpret_cast<int *>(outputBuffer), reinterpret_cast<int *>(weightsBuffer)[yOffset]); //output->p(val, output->e<T>(val) + 1);
                }
                else {
                    //*reinterpret_cast<int *>(outputBuffer) += int(1);
                    //printf("outputBuffer[0] = %d\n", static_cast<int>(*(reinterpret_cast<T *>(outputBuffer))));
                    nd4j::math::atomics::nd4j_atomicAdd(reinterpret_cast<T *>(outputBuffer), T(1)); //output->p(val, output->e<T>(val) + 1);
//                    atomicAdd(reinterpret_cast<int *>(outputBuffer), int(1)); //output->p(val, output->e<T>(val) + 1);
                    //            printf("outputBuffer[%ld] = %d\n", zOffset, static_cast<int>(*(reinterpret_cast<T *>(outputBuffer) + zOffset)));
                }
                //printf("xOffset is %ld, zOffset is %ld\n", xOffset, zOffset);
            }
        }
//        if (threadIdx.x + offset < outputLength)
//            reinterpret_cast<T *>(outputBuffer)[threadIdx.x + offset] = outputPart[threadIdx.x];
    }

        template <typename T>
    static __global__ void adjustWeightsKernel(void* inputBuffer,   Nd4jLong* inputShape,
                                               void* weightsBuffer, Nd4jLong* weightsShape,
                                               void* outputBuffer,  Nd4jLong* outputShape,
                                               int minLength, int maxLength) {

        //auto tid = blockIdx.x * blockDim.x + threadIdx.x; // * blockDim.x; // + threadIdx.x;
        int threadCount = gridDim.x * blockDim.x;
        Nd4jLong inputLength = shape::length(inputShape);

        Nd4jLong outputLength = shape::length(outputShape);
        Nd4jLong borderLen = 1;

        for (Nd4jLong e = blockIdx.x; e < outputLength; e += threadCount) {
        //if (blockIdx.x < outputLength) {
            //if (e + threadCount < outputLength) {
            Nd4jLong zOffset = shape::getIndexOffset(e, outputShape);
            //printf("%d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x);
            //Nd4jLong borderLen = 1;
            T* outputBufferZ = reinterpret_cast<T*>(outputBuffer) + zOffset;
            adjustWeightsKernelD<T>(inputBuffer, inputShape, weightsBuffer, weightsShape, (void*)outputBufferZ,
                                 inputLength, outputLength, (int)zOffset);

        }
    }

    template <typename T>
    static void adjustWeights_(nd4j::LaunchContext * context, NDArray* input, NDArray* weights, NDArray* output, int minLength, int maxLength) {
//        for (int e = 0; e < input->lengthOf(); e++) {
//            int val = input->e<int>(e);
//            if (val < maxLength) {
//                if (weights != nullptr)
//                    output->p(val, output->e<T>(val) + weights->e<T>(e));
//                else
//                    output->p(val, output->e<T>(val) + 1);
//            }
//        }
        dim3 launchDims(256, 512, 8192);
        auto stream = context->getCudaStream();
        adjustWeightsKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(input->specialBuffer(),
                input->getSpecialShapeInfo(), weights?weights->specialBuffer():nullptr, weights?weights->getSpecialShapeInfo():nullptr,
                output->specialBuffer(), output->specialShapeInfo(), minLength, maxLength);
    }

    void adjustWeights(nd4j::LaunchContext * context, NDArray* input, NDArray* weights, NDArray* output, int minLength, int maxLength) {
        BUILD_SINGLE_SELECTOR(output->dataType(), adjustWeights_, (context, input, weights, output, minLength, maxLength), GENERIC_NUMERIC_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void adjustWeights_, (nd4j::LaunchContext * context, NDArray* input, NDArray* weights, NDArray* output, int minLength, int maxLength), GENERIC_NUMERIC_TYPES);
}
}
}