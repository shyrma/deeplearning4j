#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma, created on 25.02.2018
//


#include<ops/declarable/helpers/batchnorm.h>
#include <helpers/ShapeUtils.h>
#include <OmpLaunchHelper.h>
#include <ConstantTadHelper.h>
#include <PointersManager.h>

namespace nd4j 	  {
namespace ops 	  {
namespace helpers {


//////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void batchnormCuda(const void* vx, const Nd4jLong* xShapeInfo,
									const void* vMean, const Nd4jLong* meanShapeInfo,
									const void* vVariance, const Nd4jLong* varianceShapeInfo,
									const void* vGamma, const Nd4jLong* gammaShapeInfo,
									const void* vBeta, const Nd4jLong* betaShapeInfo,
										  void* vz, const Nd4jLong* zShapeInfo,
									const Nd4jLong* xTadShapeInfo, const Nd4jLong* xTadOffsets,
									const Nd4jLong* zTadShapeInfo, const Nd4jLong* zTadOffsets,
									const T epsilon) {

	const auto x    	= reinterpret_cast<const T*>(vx);
          auto z        = reinterpret_cast<T*>(vz);
	const auto mean 	= reinterpret_cast<const T*>(vMean);
	const auto variance = reinterpret_cast<const T*>(vVariance);
	const auto gamma    = reinterpret_cast<const T*>(vGamma);
	const auto beta     = reinterpret_cast<const T*>(vBeta);

    // maxRank = xRank = zRank, minRank = meanRank = varianceRank = gammaRank = betaRank
    __shared__ Nd4jLong minLen, tadLen, totalThreads;

    if (threadIdx.x == 0) {
        totalThreads = gridDim.x * blockDim.x;

        minLen = shape::length(meanShapeInfo);
        tadLen = shape::length(xShapeInfo) / minLen;
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (uint i = tid; i < minLen; i += totalThreads) {

		const auto meanOffset     = shape::getIndexOffset(i, meanShapeInfo);
    	const auto varianceOffset = shape::getIndexOffset(i, varianceShapeInfo);

    	T sigmaInvGam = 1. / nd4j::math::nd4j_sqrt<T, T>(variance[varianceOffset] + epsilon);

    	if(gamma != nullptr)
    		sigmaInvGam *= gamma[shape::getIndexOffset(i, gammaShapeInfo)];

		auto betaOffset = 0;
    	if(beta != nullptr)
    		betaOffset = shape::getIndexOffset(i, betaShapeInfo);

    	const auto xTad = x + xTadOffsets[i];
    		  auto zTad = z + zTadOffsets[i];

    	for (uint j = 0; j < tadLen; ++j) {

    		const auto xTadOffset = shape::getIndexOffset(j, xTadShapeInfo);
    		const auto zTadOffset = shape::getIndexOffset(j, zTadShapeInfo);

    		zTad[zTadOffset] = (xTad[xTadOffset] - mean[meanOffset]) * sigmaInvGam;

    		if(beta != nullptr)
				zTad[zTadOffset] += beta[betaOffset];
    	}
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void batchnormCuda2(const void* vx, const Nd4jLong* xShapeInfo,
                                    const void* vMean, const Nd4jLong* meanShapeInfo,
                                    const void* vVariance, const Nd4jLong* varianceShapeInfo,
                                    const void* vGamma, const Nd4jLong* gammaShapeInfo,
                                    const void* vBeta, const Nd4jLong* betaShapeInfo,
                                          void* vz, const Nd4jLong* zShapeInfo,
                                    const int numDims, const int* dims,
                                    const T epsilon) {

    const auto x        = reinterpret_cast<const T*>(vx);
          auto z        = reinterpret_cast<T*>(vz);
    const auto mean     = reinterpret_cast<const T*>(vMean);
    const auto variance = reinterpret_cast<const T*>(vVariance);
    const auto gamma    = reinterpret_cast<const T*>(vGamma);
    const auto beta     = reinterpret_cast<const T*>(vBeta);

    __shared__ int xRank, minRank;       // xRank == zRank. minRank = meanRank = varianceRank = gammaRank = betaRank
    __shared__ Nd4jLong xLen, totalThreads, *sharedMem; // xLen = zLen


    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem    = reinterpret_cast<Nd4jLong*>(shmem);
        totalThreads = gridDim.x * blockDim.x;

        xLen    = shape::length(xShapeInfo);
        xRank   = shape::rank(xShapeInfo);
        minRank = shape::rank(meanShapeInfo);
    }
    __syncthreads();

    auto coords = sharedMem + threadIdx.x * xRank;
    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (uint i = tid; i < xLen; i += totalThreads) {

        shape::index2coords(i, xShapeInfo, coords);

        const auto xOffset = shape::getOffset(xShapeInfo, coords);
        const auto zOffset = shape::getOffset(zShapeInfo, coords);

        if(minRank == xRank) {
            for (uint i = 0, j = 0; i < xRank; ++i) {
                if(j < numDims && i != dims[j])
                    coords[i] = 0;
                else
                    ++j;
            }
        }
        else    // minRank = numDims = 1 in this case
            coords[0] = coords[dims[0]];

        const auto meanOffset     = shape::getOffset(meanShapeInfo, coords);
        const auto varianceOffset = shape::getOffset(varianceShapeInfo, coords);

        T sigmaInvGam = 1. / nd4j::math::nd4j_sqrt<T, T>(variance[varianceOffset] + epsilon);

        if(gamma != nullptr) {
            const auto gammaOffset = shape::getOffset(gammaShapeInfo, coords);
            sigmaInvGam *= gamma[gammaOffset];
        }

        z[zOffset] = (x[xOffset] - mean[meanOffset]) * sigmaInvGam;

        if(beta != nullptr) {
            const auto betaOffset = shape::getOffset(betaShapeInfo, coords);
            z[zOffset] += beta[betaOffset];
        }
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void batchnormCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
											const void* vx, const Nd4jLong* xShapeInfo,
                                           	const void* vMean, const Nd4jLong* meanShapeInfo,
											const void* vVariance, const Nd4jLong* varianceShapeInfo,
											const void* vGamma, const Nd4jLong* gammaShapeInfo,
											const void* vBeta, const Nd4jLong* betaShapeInfo,
												  void* vz, const Nd4jLong* zShapeInfo,
											const Nd4jLong* xTadShapeInfo, const Nd4jLong* xTadOffsets,
											const Nd4jLong* zTadShapeInfo, const Nd4jLong* zTadOffsets,
											const double epsilon) {

    batchnormCuda<T><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(vx, xShapeInfo, vMean, meanShapeInfo, vVariance, varianceShapeInfo, vGamma, gammaShapeInfo, vBeta, betaShapeInfo, vz, zShapeInfo, xTadShapeInfo, xTadOffsets, zTadShapeInfo, zTadOffsets, static_cast<T>(epsilon));
}

///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void batchnormCudaLauncher2(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                            const void* vx, const Nd4jLong* xShapeInfo,
                                            const void* vMean, const Nd4jLong* meanShapeInfo,
                                            const void* vVariance, const Nd4jLong* varianceShapeInfo,
                                            const void* vGamma, const Nd4jLong* gammaShapeInfo,
                                            const void* vBeta, const Nd4jLong* betaShapeInfo,
                                                  void* vz, const Nd4jLong* zShapeInfo,
                                            const int numDims, const int* dims,
                                            const double epsilon) {

    batchnormCuda2<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vMean, meanShapeInfo, vVariance, varianceShapeInfo, vGamma, gammaShapeInfo, vBeta, betaShapeInfo, vz, zShapeInfo, numDims, dims, static_cast<T>(epsilon));
}

//////////////////////////////////////////////////////////////////////////
void batchnorm(const NDArray* input, const NDArray* mean, const NDArray* variance, const NDArray* gamma, const NDArray* beta, NDArray* output, const std::vector<int>& axes, const double epsilon) {

	std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(input->rankOf(), axes);

	auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimsToExclude);
    auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->shapeInfo(), dimsToExclude);

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (mean->lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    PointersManager manager(input->getContext(), "batchnorm");

    NDArray::prepareSpecialUse({output}, {input, mean, variance, gamma, beta});
    BUILD_SINGLE_SELECTOR(input->dataType(), batchnormCudaLauncher, (blocksPerGrid, threadsPerBlock, input->getContext()->getCudaStream(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), mean->getSpecialBuffer(), mean->getSpecialShapeInfo(), variance->getSpecialBuffer(), variance->getSpecialShapeInfo(), gamma ? gamma->getSpecialBuffer() : nullptr, gamma ? gamma->getSpecialShapeInfo() : nullptr, beta ? beta->getSpecialBuffer() : nullptr, beta ? beta->getSpecialShapeInfo() : nullptr, output->specialBuffer(), output->specialShapeInfo(), packX.platformShapeInfo(), packX.platformOffsets(), packZ.platformShapeInfo(), packZ.platformOffsets(), epsilon), FLOAT_TYPES);
    NDArray::registerSpecialUse({output}, {input, mean, variance, gamma, beta});

    manager.synchronize();


    // const int threadsPerBlock = MAX_NUM_THREADS / 4;
    // const int blocksPerGrid = (input->lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    // const int sharedMem = sizeof(Nd4jLong) * threadsPerBlock * input->rankOf() + 128;

    // PointersManager manager(input->getContext(), "batchnorm");

    // const int* dims = reinterpret_cast<int*>(manager.replicatePointer(axes.data(), axes.size() * sizeof(int)));

    // NDArray::prepareSpecialUse({output}, {input, mean, variance, gamma, beta});
    // BUILD_SINGLE_SELECTOR(input->dataType(), batchnormCudaLauncher2, (blocksPerGrid, threadsPerBlock, sharedMem, input->getContext()->getCudaStream(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), mean->getSpecialBuffer(), mean->getSpecialShapeInfo(), variance->getSpecialBuffer(), variance->getSpecialShapeInfo(), gamma ? gamma->getSpecialBuffer() : nullptr, gamma ? gamma->getSpecialShapeInfo() : nullptr, beta ? beta->getSpecialBuffer() : nullptr, beta ? beta->getSpecialShapeInfo() : nullptr, output->specialBuffer(), output->specialShapeInfo(), axes.size(), dims, epsilon), FLOAT_TYPES);
    // NDArray::registerSpecialUse({output}, {input, mean, variance, gamma, beta});

    // manager.synchronize();
}


}
}
}

