#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>
#include <NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j {
namespace ops {
namespace helpers {
    // -------------------------------------------------------------------------------------------------------------- //
    // Segment ops linear kernels
    // -------------------------------------------------------------------------------------------------------------- //
    template<typename T, typename I>
    static __global__ void
    segmentSumLinearKernel(void *input, Nd4jLong *inputShape, int *starts, int *lengths, Nd4jLong numOfClasses,
                           void *output, Nd4jLong *outputShape) {
        __shared__
        T *val;
        __shared__
        Nd4jLong xLen, zLen, segment, zIndex;
        __shared__
        T *x;
        __shared__
        T *z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T *>(input);
            z = reinterpret_cast<T *>(output);

            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);


            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape)];
            }

        }
        __syncthreads();

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape);
            nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template<typename T, typename I>
    static __global__ void
    unsortedSegmentSumLinearKernel(void *input, Nd4jLong *inputShape, void *indices, Nd4jLong *indicesShape,
                                   int *starts, int *lengths, Nd4jLong numOfClasses, void *output,
                                   Nd4jLong *outputShape) {
        __shared__
        T *val;
        __shared__
        Nd4jLong xLen, zLen, segment, zIndex;
        __shared__
        T *x;
        __shared__
        T *z;
        __shared__
        I *y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = blockIdx.x;
            x = reinterpret_cast<T *>(input);
            z = reinterpret_cast<T *>(output);
            y = reinterpret_cast<I *>(indices);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            zIndex = shape::getIndexOffset(segment, outputShape);
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape)];
            else
                z[zIndex] = 0; //DataTypeUtils::max<T>();
        }
        __syncthreads();

        if (lengths[segment] > 0)
            for (auto e = threadIdx.x; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape);
                auto yIndex = shape::getIndexOffset(e, indicesShape);
                if (y[yIndex] == segment && e != starts[segment]) {
                    nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // SegmentSum kernel
    template <typename T, typename I>
    static __global__ void segmentSumTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads, Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, zIndex, total;
        __shared__ T* z;
        __shared__ int start, finish;

        if (threadIdx.x == 0) {
            auto segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads);
                    auto zIndex = shape::getIndexOffset(e, outputTads);
                    nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads);
                    auto zIndex = shape::getIndexOffset(e, outputTads);
                    if (lengths[indices[idx]])
                        nd4j::math::atomics::nd4j_atomicAdd(&z[zIndex], x[xIndex]);
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void segmentSumFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            segmentSumLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            segmentSumTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void segmentSumFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        output->nullify();
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentSumFunctor_, (context, input, indices, output), NUMERIC_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static void unsortedSegmentSumFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), (numOfClasses + 1) * 64);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            unsortedSegmentSumLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(0);
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentSumTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void unsortedSegmentSumFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        output->nullify();
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentSumFunctor_, (context, input, indices, numOfClasses, output),
                              NUMERIC_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});

    }

    // -------------------------------------------------------------------------------------------------------------- //
    // Backpropagate ops
    // -------------------------------------------------------------------------------------------------------------- //
    // Sorted sum backpropagate
    template <typename T, typename I>
    static __global__ void segmentSumBPLinearKernel(void* inputBuf, Nd4jLong* inputShape, void* eps, Nd4jLong* epsShape,
                                                    void* indicesBuf, Nd4jLong* indicesShape, void* outputBuf, Nd4jLong* outputShape) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape);
            auto xOffset = shape::getIndexOffset(e, inputShape);
            auto yOffset = shape::getIndexOffset(e, indicesShape);
            auto classIndex = y[yOffset];
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

            z[zOffset] = gradOut[gradOffsetO];
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void segmentSumBPTadKernel(void* inputBuf, Nd4jLong* inputShape, void* eps, Nd4jLong* epsShape,
                                                 void* indicesBuf, Nd4jLong* indicesShape, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* inputTad,
                                                 Nd4jLong* inputOffsets, Nd4jLong* gradOutTad, Nd4jLong* gradOutOffsets, Nd4jLong* outTad, Nd4jLong* outOffsets) {
        __shared__ T* x;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
            auto yIndex = shape::getIndexOffset(i, indicesShape);
            auto segment = y[yIndex];
            T* currentOut = z + outOffsets[i];
            T* outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                currentOut[e] = outGrad[e];
            }
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    int segmentSumFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentSumBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentSumBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //

    int segmentSumFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentSumFunctorBP_, (context, input,
                indices, gradOut, output), FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }

    template <typename T, typename I>
    static int unsortedSegmentSumFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            Nd4jLong loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentSumBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentSumBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }
    // -------------------------------------------------------------------------------------------------------------- //
    int unsortedSegmentSumFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentSumFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }

}
}
}