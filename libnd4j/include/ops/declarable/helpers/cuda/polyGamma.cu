#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 26.04.2019
//

#include<ops/declarable/helpers/polyGamma.h>
#include<ops/declarable/helpers/zeta.h>
#include <NDArrayFactory.h>

namespace nd4j {
namespace ops {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void polyGammaCuda(const void *vn, const Nd4jLong *nShapeInfo,
                                	 const void *vx, const Nd4jLong *xShapeInfo,
                                     	   void *vz, const Nd4jLong *zShapeInfo) {

    const auto n = reinterpret_cast<const T*>(vn);
    const auto x = reinterpret_cast<const T*>(vx);
          auto z = reinterpret_cast<T*>(vz);

    __shared__ Nd4jLong len;

    if (threadIdx.x == 0)
        len = shape::length(nShapeInfo);
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    const auto totalThreads = gridDim.x * blockDim.x;

    for (int i = tid; i < len; i += totalThreads) {

        const auto nOffset = shape::getIndexOffset(i, nShapeInfo);
        const auto xOffset = shape::getIndexOffset(i, xShapeInfo);
        const auto zOffset = shape::getIndexOffset(i, zShapeInfo);

        const T nVal = n[nOffset];

        int sign = (static_cast<int>(nVal) + 1) % 2  ?  -1 : 1;

        T factorial = 1;
        if(nVal != 0 && nVal != 1)
        	for(int i = 2; i <= nVal; ++i)
				factorial *= i;

        z[zOffset] = sign * factorial * zetaScalar<T>(nVal + 1, x[xOffset]);
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
static void polyGammaCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void *vn, const Nd4jLong *nShapeInfo, const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo) {

    polyGammaCuda<T><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(vn, nShapeInfo, vx, xShapeInfo, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
void polyGamma(nd4j::LaunchContext * context, const NDArray& n, const NDArray& x, NDArray& z) {

    NDArray::prepareSpecialUse({&z}, {&n, &x});

    int threadsPerBlock = MAX_NUM_THREADS;
    int blocksPerGrid = (z.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    BUILD_SINGLE_SELECTOR(n.dataType(), polyGammaCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), n.getSpecialBuffer(), n.getSpecialShapeInfo(), x.getSpecialBuffer(), x.getSpecialShapeInfo(), z.getSpecialBuffer(), z.getSpecialShapeInfo()), FLOAT_TYPES);

    NDArray::registerSpecialUse({&z}, {&n, &x});
}

BUILD_SINGLE_TEMPLATE(template void polyGammaCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream, const void *vn, const Nd4jLong *nShapeInfo, const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo), FLOAT_TYPES);

}
}
}

