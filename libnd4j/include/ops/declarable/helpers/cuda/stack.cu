#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by Yurii Shyrma on 02.01.2018
//

#include <ops/declarable/helpers/stack.h>
#include <helpers/ShapeUtils.h>
#include <array/ResultSet.h>
#include <cuda_exception.h>
#include <TAD.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j {
namespace ops {
namespace helpers {


	template <typename T>
	static __global__ void stackKernel(void** inputList, void** inputShapeList, int inputListLength, Nd4jLong arrLen, void* vz, const Nd4jLong* zShapeInfo, Nd4jLong* tadShape, Nd4jLong *tadOffsets) {

		T* z = reinterpret_cast<T*>(vz);

		if(tadShape == nullptr) {	// scalar case

			for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < inputListLength; i += gridDim.x * blockDim.x)
				z[shape::getIndexOffset(i, zShapeInfo)] = reinterpret_cast<T*>(inputList[i])[0];
		}
		else {

			for (int t = blockIdx.x; t < inputListLength; t += gridDim.x) {

	            auto tZ = z + tadOffsets[t];
			    auto tX = reinterpret_cast<T*>(inputList[t]);
			    auto xShapeInfo = reinterpret_cast<Nd4jLong*>(inputShapeList[t]);

			    for (int e = threadIdx.x; e < arrLen; e += blockDim.x)
			        tZ[shape::getIndexOffset(e, tadShape)] = tX[shape::getIndexOffset(e, xShapeInfo)];
			}
		}
	}

	///////////////////////////////////////////////////////////////////
	template <typename T>
	static void stack_(nd4j::LaunchContext * context, const std::vector<const NDArray*>& inArrs, NDArray* outArr, const int dim) {

		const bool scalarCase = inArrs[0]->isScalar();

		const int threadsPerBlock = MAX_NUM_THREADS / 2;
		const int blocksPerGrid = scalarCase ? (outArr->lengthOf() + threadsPerBlock - 1) / threadsPerBlock : inArrs.size();

		NDArray::prepareSpecialUse({outArr}, {});

		// FIXME: !!!
		for (auto v:inArrs)
		    NDArray::prepareSpecialUse({}, {v});

		std::vector<void const*> inputList(inArrs.size());
		std::vector<Nd4jLong const*> inputShapeList(inArrs.size());

		for (size_t i = 0; i < inputList.size(); ++i) {
			inputList[i] = inArrs[i]->getSpecialBuffer();
			inputShapeList[i] = inArrs[i]->getSpecialShapeInfo();
		}

        PointersManager manager(context, "helpers::stack");
        auto dInBuffers = (void **) manager.replicatePointer(inputList.data(), inputList.size() * sizeof(Nd4jLong*));
        auto dInShapeInfo = (void **) manager.replicatePointer(inputShapeList.data(), inputShapeList.size() * sizeof(Nd4jLong*));

        if(scalarCase) {
        	stackKernel<T><<<blocksPerGrid, threadsPerBlock, 1024, *context->getCudaStream()>>>((void**)dInBuffers, (void**)dInShapeInfo, inputList.size(), inArrs[0]->lengthOf(), outArr->specialBuffer(), outArr->getSpecialShapeInfo(), nullptr, nullptr);
        }
        else {
        	std::vector<int> axis = ShapeUtils::evalDimsToExclude(outArr->rankOf(), {dim});
        	auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(outArr->getShapeInfo(), axis);
			stackKernel<T><<<blocksPerGrid, threadsPerBlock, 1024, *context->getCudaStream()>>>((void**)dInBuffers, (void**)dInShapeInfo, inputList.size(), inArrs[0]->lengthOf(), outArr->specialBuffer(), nullptr, packZ.specialShapeInfo(), packZ.specialOffsets());
        }
        manager.synchronize();

        NDArray::registerSpecialUse({outArr}, {});

        // FIXME: !!!
        for (auto v:inArrs)
            NDArray::registerSpecialUse({}, {v});
	}

	void stack(nd4j::LaunchContext * context, const std::vector<const NDArray*>& inArrs, NDArray* outArr, const int dim) {
		BUILD_SINGLE_SELECTOR(outArr->dataType(), stack_, (context, inArrs, outArr, dim), LIBND4J_TYPES);
	}

	BUILD_SINGLE_TEMPLATE(template void stack_ , (nd4j::LaunchContext * context, const std::vector<const NDArray*>& inArrs, NDArray* outArr, const int dim), LIBND4J_TYPES);

}
}
}

