#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// implementation of operations for Simple Recurrent Unit: arXiv:1709.02755v2 [cs.CL] 12 Sep 2017
//
//  @author Yurii Shyrma, created on 05.12.2017
//

#include<ops/declarable/helpers/sru.h>
#include <NDArrayFactory.h>
#include <PointersManager.h>
#include <MmulHelper.h>

namespace nd4j    {
namespace ops     {
namespace helpers {

    //////////////////////////////////////////////////////////////////////////
    static FORCEINLINE NDArray activation(const NDArray& arr) {
        // return (const_cast<NDArray<T>&>(arr)).template transform<simdOps::Tanh<T>>();
        auto result = NDArray(&arr, false, arr.getContext());
        (const_cast<NDArray&>(arr)).applyTransform(transform::Tanh, &result);
        return result;
    }


    //////////////////////////////////////////////////////////////////////////
    static FORCEINLINE NDArray sigmoid(const NDArray& arr) {
        return (const_cast<NDArray&>(arr)).transform(transform::Sigmoid);
    }


//////////////////////////////////////////////////////////////////////////
void sruCell(nd4j::LaunchContext * context, const NDArray* x, const NDArray* c0, const NDArray* w, const NDArray* b, NDArray* h, NDArray* c) {

    // x   input [bS x inSize], bS - batch size, inSize - number of features
    // c0  previous cell state c  [bS x inSize], that is at previous time step t-1
    // w   weights [inSize x 3*inSize]
    // b   biases [2*inSize]

    // h   current cell output [bS x inSize], that is at current time step t
    // c   current cell state  [bS x inSize], that is at current time step t

    const int inSize = x->sizeAt(1);           // inSize - number of features

    auto z = mmul(*x, *w);               //  [bS x 3*inSize]

    // forget gate = sigmoid(x*Wf + bf)
    auto f = sigmoid(z({0,0, inSize,   2*inSize}) + (*b)({0, inSize}));

    // reset gate = sigmoid(x*Wr + br)
    auto r = sigmoid(z({0,0, 2*inSize, 3*inSize}) + (*b)({inSize, 2*inSize}));

    // ◦ means element-wise product or so called Hadamard product
    // current sell state = f◦c0 + (1 - f)◦(x*Wc)
    c->assign(f * (*c0) + (1.f - f) * z({0, 0 ,0, inSize}) );
    // *c = f*(*c0 - z({},{0, inSize})) + z({{},{0, inSize}});

    // current cell output = r◦activation(c) + (1 - r)◦x
    h->assign( r * activation(*c) + (1.f - r) * (*x) );
    // *h = r * (activation<T>(c) - *x) + *x;
}

//////////////////////////////////////////////////////////////////////////
void sruTimeLoop(nd4j::LaunchContext * context, const NDArray* x, const NDArray* c0, const NDArray* w, const NDArray* b, NDArray* h, NDArray* c) {

    // x   input [bS x inSize x time]
    // c0  initial cell state  (at time step = 0) [bS x inSize],
    // w   weights, [3*inSize x inSize]
    // b   biases,  [2*inSize]

    // h   cell outputs [bS x inSize x time]
    // c   cell states  [bS x inSize x time]

    auto wT = w->transpose();                             // [3*inSize x inSize] -> [inSize x 3*inSize]

    const int time  = x->sizeAt(2);

    NDArray ct_1(*c0);

    // loop through time steps
    for (int t = 0; t < time; ++t) {

        auto xt = (*x)({0,0, 0,0, t,t+1});
        auto ht = (*h)({0,0, 0,0, t,t+1});
        auto ct = (*c)({0,0, 0,0, t,t+1});

        helpers::sruCell(context, &xt, &ct_1, &wT, b,  &ht, &ct);
        ct_1.assign(ct);
    }
}


//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void sruBICuda(const void* vx,    const Nd4jLong* xShapeInfo,
                                 const void* vwi,   const Nd4jLong* wiShapeInfo,
                                 const void* vb,    const Nd4jLong* bShapeInfo,
                                 const void* vc0,   const Nd4jLong* c0ShapeInfo,
                                 const void* vmask, const Nd4jLong* maskShapeInfo,
                                       void* vht,   const Nd4jLong* htShapeInfo,
                                       void* vct,   const Nd4jLong* ctShapeInfo) {
    // inputs:
    // x     [time, bS, 2*K]
    // wi    [time, bS, 6*K], wi = mmul(x, weights);
    // b     [4*K]
    // c0    [bS, 2*K]
    // mask  [bS, 2*K], optional

    // outputs
    // ht  [time, bS, 2*K]
    // ct  [time, bS, 2*K]

    const auto x    = reinterpret_cast<const T*>(vx);
    const auto wi   = reinterpret_cast<const T*>(vwi);
    const auto b    = reinterpret_cast<const T*>(vb);
    const auto c0   = reinterpret_cast<const T*>(vc0);
    const auto mask = reinterpret_cast<const T*>(vmask);
          auto ht   = reinterpret_cast<T*>(vht);
          auto ct   = reinterpret_cast<T*>(vct);

    const int rank = 3;

    __shared__ int time, K;
    __shared__ Nd4jLong len, totalThreads, *sharedMem;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        time = xShapeInfo[1];
        K    = xShapeInfo[3] / 2;
        len  = xShapeInfo[2] * xShapeInfo[3];           // 2*K*bS

        totalThreads = gridDim.x * blockDim.x;
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    Nd4jLong* coords = sharedMem + threadIdx.x * rank;

    if(tid >= len)
        return;

    shape::index2coords(tid, rank - 1, xShapeInfo + 2, coords + 1);    // loop through last two dimensions of x : {bS, 2*K}

    const auto maskOffst = mask ? shape::getOffset(maskShapeInfo, coords + 1) : 0;
    const auto c0Offset  = shape::getOffset(c0ShapeInfo, coords + 1);
    const auto bFOffset  = shape::getOffset(bShapeInfo, coords + 2);
    const auto bROffset  = bFOffset + 2 * K * bShapeInfo[2];    // 2*K*b_stride

    const T maskVal = mask ? mask[maskOffst] : static_cast<T>(1);
    const T bF      = b[bFOffset];
    const T bR      = b[bROffset];
          T c0Val   = c0[c0Offset];

    const bool flip = coords[2] >= K;

    if(flip)
        coords[0] = time - 1;
    else
        coords[0] = 0;

    auto xOffset  = shape::getOffset(xShapeInfo, coords);
    auto htOffset = shape::getOffset(htShapeInfo, coords);
    auto ctOffset = shape::getOffset(ctShapeInfo, coords);

    coords[2] *= 3;
    auto wiOffset0 = shape::getOffset(wiShapeInfo, coords);
    auto wiOffset1 = wiOffset0 + wiShapeInfo[rank + 3];   // add last stride
    auto wiOffset2 = wiOffset1 + wiShapeInfo[rank + 3];   // add last stride

    // time loop
    for (uint t = 0; t < time; ++t) {

        // evaluate sigmoids
        T ft = (1.f)/(1.f + nd4j::math::nd4j_exp<T, T>(-(wi[wiOffset1] + bF)));
        T rt = (1.f)/(1.f + nd4j::math::nd4j_exp<T, T>(-(wi[wiOffset2] + bR)));

        c0Val = (c0Val - wi[wiOffset0]) * ft + wi[wiOffset0];
        ct[ctOffset] = c0Val;
        T val  = nd4j::math::nd4j_tanh<T, T>(c0Val);
        T xVal = x[xOffset];
        ht[htOffset] = (val * maskVal - xVal) * rt + xVal;

        if(flip) {
            xOffset   -= xShapeInfo[rank + 1];      // first stride, corresponds to time step
            htOffset  -= htShapeInfo[rank + 1];
            ctOffset  -= htShapeInfo[rank + 1];
            wiOffset0 -= wiShapeInfo[rank + 1];
            wiOffset1 -= wiShapeInfo[rank + 1];
            wiOffset2 -= wiShapeInfo[rank + 1];
        }
        else {
            xOffset   += xShapeInfo[rank + 1];      // first stride, corresponds to time step
            htOffset  += htShapeInfo[rank + 1];
            ctOffset  += htShapeInfo[rank + 1];
            wiOffset0 += wiShapeInfo[rank + 1];
            wiOffset1 += wiShapeInfo[rank + 1];
            wiOffset2 += wiShapeInfo[rank + 1];
        }
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void sruBICudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                              const void* vx,    const Nd4jLong* xShapeInfo,
                              const void* vwi,   const Nd4jLong* wiShapeInfo,
                              const void* vb,    const Nd4jLong* bShapeInfo,
                              const void* vc0,   const Nd4jLong* c0ShapeInfo,
                              const void* vmask, const Nd4jLong* maskShapeInfo,
                                    void* vht,   const Nd4jLong* htShapeInfo,
                                    void* vct,   const Nd4jLong* ctShapeInfo) {

    sruBICuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vwi, wiShapeInfo, vb, bShapeInfo, vc0, c0ShapeInfo, vmask, maskShapeInfo, vht, htShapeInfo, vct, ctShapeInfo);
}

//////////////////////////////////////////////////////////////////////////
void sruBI(nd4j::LaunchContext * context, NDArray* x, const NDArray* w, const NDArray* b, const NDArray* c0, const NDArray* mask, NDArray* ht, NDArray* ct) {

    //  x = x * mask
    if(mask)
        x->applyBroadcast(broadcast::Multiply, {1, 2}, mask, x, nullptr);             // apply mask

    // U = x * w
    NDArray wi = mmul(*x, *w); //  U [time x bS x 6*K]

    PointersManager manager(context, "sru_bi");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (x->sizeAt(1) * x->sizeAt(2) + threadsPerBlock - 1) / threadsPerBlock;      // loop through last two dimensions of x array -> bS, 2*K
    const int sharedMem = threadsPerBlock * sizeof(Nd4jLong) * x->rankOf() + 128;

    NDArray::prepareSpecialUse({ht, ct}, {x, &wi, b, c0, mask});
    BUILD_SINGLE_SELECTOR(x->dataType(), sruBICudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), x->getSpecialBuffer(), x->getSpecialShapeInfo(), wi.getSpecialBuffer(), wi.getSpecialShapeInfo(), b->getSpecialBuffer(), b->getSpecialShapeInfo(), c0->getSpecialBuffer(), c0->getSpecialShapeInfo(), mask ? mask->getSpecialBuffer() : nullptr, mask ? mask->getSpecialShapeInfo() : nullptr, ht->specialBuffer(), ht->specialShapeInfo(), ct->specialBuffer(), ct->specialShapeInfo()), FLOAT_TYPES);
    NDArray::registerSpecialUse({ht, ct}, {x, &wi, b, c0, mask});

    manager.synchronize();
}









































//////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ static void sruBIBPCuda(const void* vx,       const Nd4jLong* xShapeInfo,
                                   const void* vwi,      const Nd4jLong* wiShapeInfo,
                                   const void* vb,       const Nd4jLong* bShapeInfo,
                                   const void* vc0,      const Nd4jLong* c0ShapeInfo,
                                   const void* vmask,    const Nd4jLong* maskShapeInfo,
                                   const void* vct,      const Nd4jLong* ctShapeInfo,
                                   const void* vgradHt,  const Nd4jLong* gradHtShapeInfo,
                                   const void* vgradCt,  const Nd4jLong* gradCtShapeInfo,
                                         void* vgradI,   const Nd4jLong* gradIShapeInfo,
                                         void* vgradWi,  const Nd4jLong* gradWiShapeInfo,
                                         void* vgradB,   const Nd4jLong* gradBShapeInfo,
                                         void* vgradC0,  const Nd4jLong* gradC0ShapeInfo) {
    // inputs:
    // x      [time, bS, 2*K]
    // wi     [time, bS, 6*K], wi = mmul(x, weights);
    // b      [4*K]
    // c0     [bS, 2*K]
    // mask   [bS, 2*K], optional
    // ct     [time, bS, 2*K]
    // gradHt [time, bS, 2*K]
    // gradCt [bS, 2*K]

    // outputs
    // gradI   [time, bS, 2*K]
    // gradWi  [time, 2*K, 6*K]
    // gradB   [bS, 4*K]
    // gradC0  [bS, 2*K]

    const auto x      = reinterpret_cast<const T*>(vx);
    const auto wi     = reinterpret_cast<const T*>(vwi);
    const auto b      = reinterpret_cast<const T*>(vb);
    const auto c0     = reinterpret_cast<const T*>(vc0);
    const auto mask   = reinterpret_cast<const T*>(vmask);
    const auto ct     = reinterpret_cast<const T*>(vct);
    const auto gradHt = reinterpret_cast<const T*>(vgradHt);
    const auto gradCt = reinterpret_cast<const T*>(vgradCt);

          auto gradI  = reinterpret_cast<T*>(vgradI);
          auto gradWi = reinterpret_cast<T*>(vgradWi);
          auto gradB  = reinterpret_cast<T*>(vgradB);
          auto gradC0 = reinterpret_cast<T*>(vgradC0);

    const int rank = 3;

    __shared__ int time, K;
    __shared__ Nd4jLong len, totalThreads, *sharedMem;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        time = xShapeInfo[1];
        K    = xShapeInfo[3] / 2;
        len  = xShapeInfo[2] * xShapeInfo[3];           // 2*K*bS

        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    Nd4jLong* coords = sharedMem + threadIdx.x * rank;

    if(tid >= len)
        return;

    shape::index2coords(tid, rank - 1, xShapeInfo + 2, coords + 1);    // loop through last two dimensions of x : {bS, 2*K}

    const auto maskOffst    = mask ? shape::getOffset(maskShapeInfo, coords + 1) : 0;
    const auto c0Offset     = shape::getOffset(c0ShapeInfo, coords + 1);
    const auto gradCtOffset = shape::getOffset(gradCtShapeInfo, coords + 1);
    const auto gradC0Offset = shape::getOffset(gradC0ShapeInfo, coords + 1);
    const auto bFOffset     = shape::getOffset(bShapeInfo, coords + 2);
    const auto bROffset     = bFOffset + 2 * K * bShapeInfo[2];         // 2*K*b_stride
    // const auto gradBFOffset = shape::getOffset(gradBShapeInfo, coords + 1);
    const auto gradBFOffset = coords[1] * gradBShapeInfo[3] / 2 + coords[2] * gradBShapeInfo[4];
    const auto gradBROffset = gradBFOffset + gradBShapeInfo[3];

    const bool flip = coords[2] >= K;

    if(flip)
        coords[0] = 0;
    else
        coords[0] = time - 1;

    auto xOffset      = shape::getOffset(xShapeInfo, coords);
    auto ctOffset     = shape::getOffset(ctShapeInfo, coords);
    auto gradIOffset  = shape::getOffset(gradIShapeInfo, coords);
    auto gradHtOffset = shape::getOffset(gradHtShapeInfo, coords);

    coords[2] *= 3;
    auto gradWiOffset0 = shape::getOffset(gradWiShapeInfo, coords);
    auto gradWiOffset1 = gradWiOffset0 + gradWiShapeInfo[rank + 3];   // add last stride
    auto gradWiOffset2 = gradWiOffset1 + gradWiShapeInfo[rank + 3];   // add last stride
    auto wiOffset0     = shape::getOffset(wiShapeInfo, coords);
    auto wiOffset1     = wiOffset0 + wiShapeInfo[rank + 3];   // add last stride
    auto wiOffset2     = wiOffset1 + wiShapeInfo[rank + 3];   // add last stride

    const T xVal      = x[xOffset];
    const T maskVal   = mask ? mask[maskOffst] : static_cast<T>(1);
    const T c0Val     = c0[c0Offset];
    const T bF        = b[bFOffset];
    const T bR        = b[bROffset];
          T gradCtVal = gradCt[gradCtOffset];
          T gbF       = 0.f;
          T gbR       = 0.f;

    // time loop
    for (uint t = 0; t < time; ++t) {

        // evaluate sigmoids
        T ft = (1.f)/(1.f + nd4j::math::nd4j_exp<T, T>(-(wi[wiOffset1] + bF)));
        T rt = (1.f)/(1.f + nd4j::math::nd4j_exp<T, T>(-(wi[wiOffset2] + bR)));

        T val = nd4j::math::nd4j_tanh<T,T>(ct[ctOffset]);

        T prevVal;
        if(t < time-1)
            prevVal = ct[ctOffset += flip ? ctShapeInfo[rank + 1] : -ctShapeInfo[rank + 1]];
        else
            prevVal = c0Val;

        // grad wrt input
        gradI[gradIOffset] = gradHt[gradHtOffset] - gradHt[gradHtOffset] * rt ;

        // grad wrt rt, wiR and bR
        T grt = gradHt[gradHtOffset] * (val * maskVal - x[xOffset]) * (rt - rt * rt);
        gradWi[gradWiOffset2] = grt;
        gbR += grt;

        // grad wrt state
        T gradC0Val = gradHt[gradHtOffset] * maskVal * (rt - rt * val * val) + gradCtVal;

        // grad wrt wi0
        gradWi[gradWiOffset0] = gradC0Val - gradC0Val * ft;

        // grad wrt ft, wi1, and bF
        T gft = gradC0Val * (prevVal - wi[wiOffset0]) * (ft - ft * ft);
        gradWi[gradWiOffset1] = gft;
        gbF += gft;

        // grad wrt c_previous
        gradCtVal = gradC0Val * ft;

        if(flip) {
            xOffset       += xShapeInfo[rank + 1];      // first stride, corresponds to time step
            gradHtOffset  += gradHtShapeInfo[rank + 1];
            gradIOffset   += gradIShapeInfo[rank + 1];
            wiOffset0     += wiShapeInfo[rank + 1];
            wiOffset1     += wiShapeInfo[rank + 1];
            wiOffset2     += wiShapeInfo[rank + 1];
            gradWiOffset0 += gradWiShapeInfo[rank + 1];
            gradWiOffset1 += gradWiShapeInfo[rank + 1];
            gradWiOffset2 += gradWiShapeInfo[rank + 1];
        }
        else {
            xOffset       -= xShapeInfo[rank + 1];      // first stride, corresponds to time step
            gradHtOffset  -= gradHtShapeInfo[rank + 1];
            gradIOffset   -= gradIShapeInfo[rank + 1];
            wiOffset0     -= wiShapeInfo[rank + 1];
            wiOffset1     -= wiShapeInfo[rank + 1];
            wiOffset2     -= wiShapeInfo[rank + 1];
            gradWiOffset0 -= gradWiShapeInfo[rank + 1];
            gradWiOffset1 -= gradWiShapeInfo[rank + 1];
            gradWiOffset2 -= gradWiShapeInfo[rank + 1];
        }
    }

    gradB[gradBFOffset]  = gbF;
    gradB[gradBROffset]  = gbR;
    gradC0[gradC0Offset] = gradCtVal;
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void sruBIBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const void* vx,       const Nd4jLong* xShapeInfo,
                                const void* vwi,      const Nd4jLong* wiShapeInfo,
                                const void* vb,       const Nd4jLong* bShapeInfo,
                                const void* vc0,      const Nd4jLong* c0ShapeInfo,
                                const void* vmask,    const Nd4jLong* maskShapeInfo,
                                const void* vct,      const Nd4jLong* ctShapeInfo,
                                const void* vgradHt,  const Nd4jLong* gradHtShapeInfo,
                                const void* vgradCt,  const Nd4jLong* gradCtShapeInfo,
                                      void* vgradI,   const Nd4jLong* gradIShapeInfo,
                                      void* vgradWi,  const Nd4jLong* gradWiShapeInfo,
                                      void* vgradB,   const Nd4jLong* gradBShapeInfo,
                                      void* vgradC0,  const Nd4jLong* gradC0ShapeInfo) {

    sruBIBPCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vwi, wiShapeInfo, vb, bShapeInfo, vc0, c0ShapeInfo, vmask, maskShapeInfo, vct, ctShapeInfo, vgradHt, gradHtShapeInfo, vgradCt, gradCtShapeInfo, vgradI, gradIShapeInfo, vgradWi, gradWiShapeInfo, vgradB, gradBShapeInfo, vgradC0, gradC0ShapeInfo);
}
BUILD_SINGLE_TEMPLATE(template void sruBIBPCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const void* vx, const Nd4jLong* xShapeInfo, const void* vwi, const Nd4jLong* wiShapeInfo, const void* vb, const Nd4jLong* bShapeInfo, const void* vc0, const Nd4jLong* c0ShapeInfo, const void* vmask, const Nd4jLong* maskShapeInfo, const void* vct, const Nd4jLong* ctShapeInfo, const void* vgradHt, const Nd4jLong* gradHtShapeInfo, const void* vgradCt, const Nd4jLong* gradCtShapeInfo, void* vgradI, const Nd4jLong* gradIShapeInfo, void* vgradWi, const Nd4jLong* gradWiShapeInfo, void* vgradB, const Nd4jLong* gradBShapeInfo, void* vgradC0, const Nd4jLong* gradC0ShapeInfo), FLOAT_TYPES);

//////////////////////////////////////////////////////////////////////////
void sruBIBP(nd4j::LaunchContext* context, NDArray* x, const NDArray* w, const NDArray* b, const NDArray* c0, const NDArray* ct,
                                          const NDArray* gradCt, const NDArray* gradHt, const NDArray* mask,
                                          NDArray* gradI, NDArray* gradW, NDArray* gradB, NDArray* gradC0) {

    //  x = x * mask
    if(mask)
        x->applyBroadcast(broadcast::Multiply, {1, 2}, mask, x, nullptr);             // apply mask

    // U = x * w
    NDArray wi = mmul(*x, *w); //  U [time x bS x 6*K]

    const int time = x->sizeAt(0);
    const int bS   = x->sizeAt(1);
    const int K    = x->sizeAt(2) / 2;

    NDArray gradBias(x->ordering(), {bS, 4*K}, x->dataType(), context);
    NDArray gradWi  (x->ordering(), {time, bS, 6*K}, x->dataType(), context);

    PointersManager manager(context, "sru_bi_bp");

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (x->sizeAt(1) * x->sizeAt(2) + threadsPerBlock - 1) / threadsPerBlock;      // loop through last two dimensions of x array -> bS, 2*K
    const int sharedMem = threadsPerBlock * sizeof(Nd4jLong) * x->rankOf() + 128;

    NDArray::prepareSpecialUse({gradI, &gradWi, &gradBias, gradC0}, {x, &wi, b, c0, ct, gradCt, gradHt, mask});
    BUILD_SINGLE_SELECTOR(x->dataType(), sruBIBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), x->getSpecialBuffer(), x->getSpecialShapeInfo(), wi.getSpecialBuffer(), wi.getSpecialShapeInfo(), b->getSpecialBuffer(), b->getSpecialShapeInfo(), c0->getSpecialBuffer(), c0->getSpecialShapeInfo(), mask ? mask->getSpecialBuffer() : nullptr, mask ? mask->getSpecialShapeInfo() : nullptr, ct->getSpecialBuffer(), ct->getSpecialShapeInfo(), gradHt->getSpecialBuffer(), gradHt->getSpecialShapeInfo(), gradCt->getSpecialBuffer(), gradCt->getSpecialShapeInfo(), gradI->specialBuffer(), gradI->specialShapeInfo(), gradWi.specialBuffer(), gradWi.specialShapeInfo(), gradBias.specialBuffer(), gradBias.specialShapeInfo(), gradC0->specialBuffer(), gradC0->specialShapeInfo()), FLOAT_TYPES);
    NDArray::registerSpecialUse({gradI, &gradWi, &gradBias, gradC0}, {x, &wi, b, c0, ct, gradCt, gradHt, mask});

    manager.synchronize();

    // gradB
    gradBias.reduceAlongDimension(reduce::Sum, gradB, {0});    // [4*K]

    // gradW
    x->permutei({0, 2, 1});                                    // [time, bS, 2*K] -> [time, 2*K,  bS]
    MmulHelper::mmul(x, &gradWi, gradW, 1., 0.);               // [time, 2*K, bS] x [time, bS , 6*K] = [time, 2*K, 6*K]
}


}
}
}