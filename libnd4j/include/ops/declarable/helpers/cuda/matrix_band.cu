#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author George A. Shulinok <sgazeos@gmail.com>
//
#include <ops/declarable/helpers/matrix_band.h>
#include <TAD.h>
#include <cuda_exception.h>
#include <ShapeUtils.h>
#include <helpers/ConstantTadHelper.h>

namespace nd4j {
namespace ops {
namespace helpers {
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// matrix band kernel
//
// inputBuffer - buffer of input tensor
// inputShape - shape of input tensor
// outputBuffer - buffer of output tensor
// outputShape - shape of output tensor
// lowerBand - lower band of matrix
// upperBand - upper band of matrix
// tadOnlyInputShapeInfo - TAD shape for input
// tadInputOffsets - TAD offsets for input
// tadOnlyOutputShapeInfo - TAD output shape
// tadOutputOffsets - TAD output offsets
// numTads - number of subarrays
// inputLength - input subarray length
//
    template <typename T>
    static __global__ void matrixBandKernel(void* inputBuffer, Nd4jLong* inputShape,
            void* outputBuffer, Nd4jLong* outputShape, Nd4jLong lowerBand, Nd4jLong upperBand, Nd4jLong* tadOnlyInputShapeInfo,  Nd4jLong* tadInputOffsets,
                                            Nd4jLong* tadOnlyOutputShapeInfo, Nd4jLong* tadOutputOffsets, Nd4jLong numTads, Nd4jLong inputLength) {
        int totalThreads = blockDim.x;
        Nd4jLong rows = shape::sizeAt(inputShape, -2);
        Nd4jLong cols = shape::sizeAt(inputShape, -1);
        for (Nd4jLong e = blockIdx.x; e < numTads; e += gridDim.x) {
            auto yOffset = tadInputOffsets[e];
            auto xOffset = tadOutputOffsets[e];
            for (Nd4jLong i = blockIdx.y; i < rows; i += gridDim.y) {
                for (Nd4jLong j = threadIdx.x; j < cols; j += totalThreads) {
                    Nd4jLong coords[2] = {i, j};
                    Nd4jLong tadOffsetOut = shape::getOffset(tadOnlyOutputShapeInfo, coords);
                    Nd4jLong tadOffsetIn = shape::getOffset(tadOnlyInputShapeInfo, coords);

                    if (i >= j) { // check lower diagonals
                        if (lowerBand > 0) {
                            if ((i - j) > lowerBand)
                                *(reinterpret_cast<T *>(outputBuffer) + xOffset + tadOffsetOut) = T(0);
                            else
                                *(reinterpret_cast<T *>(outputBuffer) + xOffset + tadOffsetOut) = *(
                                        reinterpret_cast<T const *>(inputBuffer) + yOffset + tadOffsetIn);
                        }
                    } else if (j > i) {
                        if (upperBand > 0)
                            if ((j - i) > upperBand)
                                *(reinterpret_cast<T *>(outputBuffer) + xOffset + tadOffsetOut) = T(0);
                            else
                                *(reinterpret_cast<T *>(outputBuffer) + xOffset + tadOffsetOut) = *(
                                        reinterpret_cast<T const *>(inputBuffer) + yOffset + tadOffsetIn);
                    }
                }
            }
        }

    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// matrixBandPart_ - main algorithm caller
//
    template <typename T>
    void matrixBandPart_(nd4j::LaunchContext * context, NDArray* input, NDArray* output, Nd4jLong lowerBand, Nd4jLong upperBand) {
        dim3 launchDims(256, 512, 8192);
        auto stream = context->getCudaStream();

        std::vector<int> lastDims({input->rankOf() - 2, input->rankOf() - 1});
        std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(input->rankOf(), lastDims);

        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), lastDims);
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), lastDims);

        const Nd4jLong numTads = packX.numberOfTads();

        NDArray::prepareSpecialUse({output}, {input});
        matrixBandKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(input->getSpecialBuffer(),
                input->getSpecialShapeInfo(), output->getSpecialBuffer(), output->getSpecialShapeInfo(),
                lowerBand, upperBand, packX.specialShapeInfo(), packX.specialOffsets(), packZ.specialShapeInfo(), packZ.specialOffsets(), numTads, input->lengthOf());
        NDArray::registerSpecialUse({output}, {input});
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void matrixBandPart(nd4j::LaunchContext * context, NDArray* input, NDArray* output, Nd4jLong lowerBand, Nd4jLong upperBand) {
        BUILD_SINGLE_SELECTOR(input->dataType(), matrixBandPart_, (context, input, output, lowerBand, upperBand), FLOAT_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void matrixBandPart_, (nd4j::LaunchContext * context, NDArray* input, NDArray* output, Nd4jLong lowerBand, Nd4jLong upperBand), FLOAT_TYPES);
}
}
}

