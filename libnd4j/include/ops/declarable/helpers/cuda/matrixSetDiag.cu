#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include "ResultSet.h"
#include <ops/declarable/helpers/matrixSetDiag.h>
#include <PointersManager.h>

namespace nd4j    {
namespace ops     {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void matrixSetDiagCuda(const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const bool zeroPad) {

    // x - input,    shape [A,B,C]
    // y - diagonal, shape [A,B]
    // z - output,   shape [A,B,C]
    // input and output are the same array (x == z) when zeroPad = true

    const auto x = reinterpret_cast<const T*>(vx);
    const auto y = reinterpret_cast<const T*>(vy);
          auto z = reinterpret_cast<T*>(vz);

    __shared__ int xRank;       // xRank = zRank, xRank = yRank + 1
    __shared__ Nd4jLong xLen, *sharedMem;   // xLen = zLen
    __shared__ bool areSameOffsets;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        areSameOffsets = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);    // shapes are definitely the same, but strides might not

        xRank = shape::rank(xShapeInfo);
        xLen  = shape::length(xShapeInfo);
    }

    __syncthreads();

    auto coords = sharedMem + threadIdx.x * xRank;               // we provide (xRank * sizeof(Nd4jLong) * threadIdx.x) amount of shared memory per each thread
    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < xLen; i += gridDim.x * blockDim.x) {

        shape::index2coords(i, xShapeInfo, coords);

        const auto xOffset = shape::getOffset(xShapeInfo, coords);
        const auto zOffset = areSameOffsets ? xOffset : shape::getOffset(zShapeInfo, coords);

        // condition to be on diagonal of innermost matrix
        if(coords[xRank - 2] == coords[xRank - 1])
            z[zOffset] = y[shape::getOffset(yShapeInfo, coords)];
        else
            z[zOffset] = zeroPad ? static_cast<T>(0) : x[xOffset];
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
static void matrixSetDiagCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const bool zeroPad) {

    matrixSetDiagCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, zeroPad);
}

///////////////////////////////////////////////////////////////////
void matrixSetDiag(nd4j::LaunchContext* context, const NDArray& input, const NDArray& diagonal, NDArray& output, const bool zeroPad) {

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = threadsPerBlock * sizeof(Nd4jLong) * input.rankOf() + 128;

    PointersManager manager(context, "matrixSetDiag");

    NDArray::prepareSpecialUse({&output}, {&input, &diagonal});
    BUILD_SINGLE_SELECTOR(input.dataType(), matrixSetDiagCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), diagonal.getSpecialBuffer(), diagonal.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), zeroPad), LIBND4J_TYPES);
    NDArray::registerSpecialUse({&output}, {&input, &diagonal});

    manager.synchronize();
}

}
}
}