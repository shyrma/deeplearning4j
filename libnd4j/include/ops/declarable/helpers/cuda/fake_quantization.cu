#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/fake_quantization.h>
#include <NDArrayFactory.h>

namespace nd4j {
namespace ops {
namespace helpers {
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// fakeQuantWithMinMaxVars_
// input - input tensor
// min - min scalar tensor
// max - max scalar tensor
// numBits - (default 16bit)
// narrowed - shrink is true
// output - output tensor
//
    template <typename T>
    void fakeQuantWithMinMaxVars_(NDArray* input, NDArray* min, NDArray* max, int numBits, bool narrowed, NDArray* output) {
        int lowIntBound = narrowed?1:0;
        int upperIntBound = 1 << numBits - 1;
        min->syncToHost();
        max->syncToHost();
        const float quant_min_float = static_cast<float>(lowIntBound);
        const float quant_max_float = static_cast<float>(upperIntBound);
        T scale = (max->t<T>(0) - min->t<T>(0)) / (quant_max_float - quant_min_float);
        const T zero_point_from_min = quant_min_float - min->t<T>(0) / scale;

        const uint16_t nudged_zero_point = [zero_point_from_min, lowIntBound,
                                        quant_min_float, upperIntBound,
                                        quant_max_float] {
            if (zero_point_from_min < quant_min_float) {
                return static_cast<uint16_t>(lowIntBound);
            }
            if (zero_point_from_min > quant_max_float) {
                return static_cast<uint16_t>(upperIntBound);
            }
            return static_cast<uint16_t>(roundf(zero_point_from_min));
        }();

        auto nudged_min = (quant_min_float - nudged_zero_point) * (scale);
        auto nudged_max = (quant_max_float - nudged_zero_point) * (scale);

        auto wiseMax = LAMBDA_T(x, nudged_min) {
            if (x < nudged_min) {
                return nudged_min;
            }
            return x;
        };

        auto wiseMin = LAMBDA_T(x, nudged_max) {
            if (x > nudged_max) {
                return nudged_max;
            }
            return x;
        };

        auto scaleTensor(*input);
        auto clamped(*input);
        scaleTensor.assign(scale);
        input->applyLambda(wiseMin, &clamped);

        clamped.applyLambda(wiseMax, output);
        *output -= nudged_min;

        (*output) /= scaleTensor;
        (*output) += T(0.5f);
        output->applyTransform(transform::Floor, nullptr, nullptr);
        (*output) *= scaleTensor;
        (*output) += nudged_min;
    }

    void fakeQuantWithMinMaxVars(NDArray* input, NDArray* min, NDArray* max, int numBits, bool narrowed, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), fakeQuantWithMinMaxVars_, (input, min, max, numBits, narrowed, output), FLOAT_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void fakeQuantWithMinMaxVars_, (NDArray* input, NDArray* min, NDArray* max, int numBits, bool narrowed, NDArray* output), FLOAT_TYPES);

}
}
}
