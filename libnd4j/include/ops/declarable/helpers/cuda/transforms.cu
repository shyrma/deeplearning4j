#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j 	  {
namespace ops 	  {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void invertPermutationCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo) {

    const T* x = reinterpret_cast<const T*>(vx);
          T* z = reinterpret_cast<T*>(vz);

    __shared__ Nd4jLong len, totalThreads;

    if (threadIdx.x == 0) {

        len  = shape::length(xShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < len; i += totalThreads) {

        const auto xOffset = shape::getIndexOffset(i, xShapeInfo);
        const Nd4jLong index = x[xOffset];
        const auto zOffset = shape::getIndexOffset(index, zShapeInfo);
        z[zOffset] = i;
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void invertPermutationCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                                   const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo) {

    invertPermutationCuda<T><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(vx, xShapeInfo, vz, zShapeInfo);
}

////////////////////////////////////////////////////////////////////////
void invertPermutation(nd4j::LaunchContext* context, const NDArray& input, NDArray& output) {

    const int threadsPerBlock = MAX_NUM_THREADS;
    const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    PointersManager manager(context, "invertPermutation");

    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_SINGLE_SELECTOR(input.dataType(), invertPermutationCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo()), LIBND4J_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

    manager.synchronize();
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void traceCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint diagLen) {

    const auto x = reinterpret_cast<const T*>(vx);
          auto z = reinterpret_cast<T*>(vz);

    __shared__ T* sharedMem;
    __shared__ int xRank, zRank;        // xRank = zRank + 2
    __shared__ Nd4jLong xLen, zLen, *coordsMem;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<T*>(shmem);
        coordsMem = reinterpret_cast<Nd4jLong*>(shmem + blockDim.x * sizeof(T));

        xRank = shape::rank(xShapeInfo);
        zRank = shape::rank(zShapeInfo);
        xLen = shape::length(xShapeInfo);
        zLen = shape::length(zShapeInfo);   // corresponds to number of matrices

    }
    __syncthreads();

    Nd4jLong* coords = coordsMem + threadIdx.x * xRank;

    for (uint m = blockIdx.x; m < zLen; m += gridDim.x) {   // one block per each element of z, that is per each matrix

        shape::index2coords(m, zShapeInfo, coords);
        const auto zOffset = shape::getOffset(zShapeInfo, coords);

        sharedMem[threadIdx.x] = 0;

          for (uint i = threadIdx.x; i < diagLen; i += blockDim.x) {

            coords[zRank] = coords[zRank + 1] = i;
            const auto xOffset = shape::getOffset(xShapeInfo, coords);
            sharedMem[threadIdx.x] += x[xOffset];
          }

          __syncthreads();

        // aggregate sum
        for (Nd4jLong activeThreads = blockDim.x / 2; activeThreads > 0; activeThreads /= 2) {
            if (threadIdx.x < activeThreads)
                sharedMem[threadIdx.x] += sharedMem[threadIdx.x + activeThreads];
            __syncthreads();
        }

        if (threadIdx.x == 0)
            z[zOffset] = *sharedMem;
        __syncthreads();
    }

}

///////////////////////////////////////////////////////////////////
template<typename T>
static void traceCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                              const void *vx, const Nd4jLong *xShapeInfo,
                                    void *vz, const Nd4jLong *zShapeInfo,
                                    const uint diagLen) {

    traceCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, diagLen);
}


///////////////////////////////////////////////////////////////////
void trace(nd4j::LaunchContext* context, const NDArray& input, NDArray& output) {

    PointersManager manager(context, "trace");

    const uint diagLen = input.sizeAt(-1) < input.sizeAt(-2) ? input.sizeAt(-1) : input.sizeAt(-2);
    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = threadsPerBlock * (sizeof(Nd4jLong) * input.rankOf() + input.sizeOfT()) + 128;

    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_SINGLE_SELECTOR(input.dataType(), traceCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), diagLen), LIBND4J_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

    manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void triuBPCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int diag) {

    // x and z have same shapes
    const auto x = reinterpret_cast<const T*>(vx);  // gradO
          auto z = reinterpret_cast<T*>(vz);        // gradI

    __shared__ int rank, areSameOffsets;                // xRank = zRank
    __shared__ Nd4jLong len, totalThreads, *sharedMem;  // xLen = zLen

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);
        areSameOffsets = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        rank = shape::rank(xShapeInfo);
        len  = shape::length(zShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    auto coords = sharedMem + threadIdx.x * rank;

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < len; i += totalThreads) {

        shape::index2coords(i, zShapeInfo, coords);

        const auto zOffset = shape::getOffset(zShapeInfo, coords);

        if((coords[rank - 2] + diag > coords[rank - 1]))    // row + diag > col
            z[zOffset] = 0;
        else
            z[zOffset] = x[areSameOffsets ? zOffset : shape::getOffset(xShapeInfo, coords)];
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
static void triuBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const int diag) {

    triuBPCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, diag);
}

///////////////////////////////////////////////////////////////////
void triuBP(nd4j::LaunchContext* context, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int diagonal) {

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (gradO.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = threadsPerBlock * sizeof(Nd4jLong) * gradO.rankOf() + 128;

    PointersManager manager(context, "triuBP");

    NDArray::prepareSpecialUse({&gradI}, {&gradO});
    BUILD_SINGLE_SELECTOR(gradI.dataType(), triuBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), gradO.getSpecialBuffer(), gradO.getSpecialShapeInfo(), gradI.specialBuffer(), gradI.specialShapeInfo(), diagonal), LIBND4J_TYPES);
    NDArray::registerSpecialUse({&gradI}, {&gradO});

    manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void tileBPCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, Nd4jLong* globMem) {

    // x and z have same shapes
    const auto x = reinterpret_cast<const T*>(vx);  // gradO
          auto z = reinterpret_cast<T*>(vz);        // gradI

    __shared__ int xRank, zRank;                // xRank >= zRank
    __shared__ Nd4jLong numOfXOffsets, zLen, totalThreads, *sharedMem;  // xLen >= zLen

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        xRank = shape::rank(zShapeInfo);
        zLen  = shape::length(zShapeInfo);
        numOfXOffsets = shape::length(xShapeInfo) / zLen;

        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    auto memBuff  = sharedMem + threadIdx.x * 2 * xRank;
    auto xOffsets = globMem + tid * numOfXOffsets;

    for (Nd4jLong i = tid; i < zLen; i += totalThreads) {

        const auto zOffset = shape::getIndexOffset(i, zShapeInfo);

        shape::outerArrayOffsets(xOffsets, i, xShapeInfo, zShapeInfo, memBuff);

        z[zOffset] = x[xOffsets[0]];                    // first offset
        for (Nd4jLong j = 1; j < numOfXOffsets; ++j)    // rest offsets
            z[zOffset] += x[xOffsets[j]];
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
static void tileBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, Nd4jLong* globMem) {

    tileBPCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, globMem);
}


//////////////////////////////////////////////////////////////////////////
void tileBP(nd4j::LaunchContext * context, const NDArray& gradO /*input*/, NDArray& gradI /*output*/, const std::vector<Nd4jLong> reps) {

    NDArray memBuff('c', gradO.getShapeAsVector(), nd4j::DataType::INT64, context);        // empty auxiliary array for storing device memory which will be used in kernel calculations

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (gradI.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = threadsPerBlock * sizeof(Nd4jLong) * 2 * gradO.rankOf() + 128;

    PointersManager manager(context, "tileBP");

    NDArray::prepareSpecialUse({&gradI}, {&gradO, &memBuff});
    BUILD_SINGLE_SELECTOR(gradI.dataType(), tileBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), gradO.getSpecialBuffer(), gradO.getSpecialShapeInfo(), gradI.specialBuffer(), gradI.specialShapeInfo(), reinterpret_cast<Nd4jLong*>(memBuff.specialBuffer())), FLOAT_TYPES);
    NDArray::registerSpecialUse({&gradI}, {&gradO, &memBuff});

    manager.synchronize();
}

//////////////////////////////////////////////////////////////////////////
// x - input, y - gradO, z - gradI
template<typename X, typename Z>
__global__ static void clipByNormBPWholeArrCuda(const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo, void* vz, const Nd4jLong* zShapeInfo, void* vreducBuff, const Z clipNormVal) {

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid >= shape::length(zShapeInfo))
        return;

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Z*>(vy);
          auto z = reinterpret_cast<Z*>(vz);

    auto reducBuff = reinterpret_cast<Z*>(vreducBuff);
    uint* count    = reinterpret_cast<uint*>(vreducBuff) + 16384;

    __shared__ Z* shMem;
    __shared__ Nd4jLong len;
    __shared__ bool amIinLastBlock;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        shMem = reinterpret_cast<Z*>(shmem);

        len = shape::length(zShapeInfo);   // xLen = yLen = zLen
    }
    __syncthreads();

    // fill shared memory with array elements
    const auto xVal = x[shape::getIndexOffset(tid, xShapeInfo)];
    const auto yVal = y[shape::getIndexOffset(tid, yShapeInfo)];

    shMem[2*threadIdx.x]     = static_cast<Z>(xVal * xVal);   // for norm
    shMem[2*threadIdx.x + 1] = static_cast<Z>(xVal * yVal);   // for input * gradO

    __syncthreads();

    // accumulate sum per block
    for (int activeThreads = blockDim.x / 2; activeThreads > 0; activeThreads /= 2) {

        if (threadIdx.x < activeThreads && tid + activeThreads < len) {

            shMem[2*threadIdx.x]     += shMem[2*(threadIdx.x + activeThreads)];
            shMem[2*threadIdx.x + 1] += shMem[2*(threadIdx.x + activeThreads) + 1];
        }
        __syncthreads();
    }

    // store accumulated sums in reduction buffer (reducBuff)
    if (threadIdx.x == 0) {

        reducBuff[2*blockIdx.x]     = shMem[0];
        reducBuff[2*blockIdx.x + 1] = shMem[1];

        __threadfence();

        amIinLastBlock = gridDim.x == 1 || (atomicInc(count, gridDim.x) == gridDim.x - 1);
    }
    __syncthreads();

    // shared memory of last block is used for final summation of values stored in reduction buffer
    if (amIinLastBlock) {

        for (int i = threadIdx.x; i < gridDim.x; i += blockDim.x) {

            shMem[2*threadIdx.x]     = (i == threadIdx.x ) ? reducBuff[2*i]     : reducBuff[2*i]     + shMem[2*threadIdx.x];
            shMem[2*threadIdx.x + 1] = (i == threadIdx.x ) ? reducBuff[2*i + 1] : reducBuff[2*i + 1] + shMem[2*threadIdx.x + 1];
        }
        __syncthreads();

        // accumulate sum
        for (int activeThreads = blockDim.x / 2; activeThreads > 0; activeThreads /= 2) {

            if (threadIdx.x < activeThreads && threadIdx.x + activeThreads < gridDim.x) {
                shMem[2*threadIdx.x]     += shMem[2*(threadIdx.x + activeThreads)];
                shMem[2*threadIdx.x + 1] += shMem[2*(threadIdx.x + activeThreads) + 1];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {

            reducBuff[0] = math::nd4j_sqrt<Z,Z>(shMem[0]);
            reducBuff[1] = shMem[1];
            count = 0;
        }
    }
}

//////////////////////////////////////////////////////////////////////////
// x - input, y - gradO, z - gradI
template<typename X, typename Z>
__global__ static void clipByNormBPCalcGradCuda(const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo, void* vz, const Nd4jLong* zShapeInfo, void* vreducBuff, const Z clipNormVal) {

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    const Nd4jLong len = shape::length(zShapeInfo);     // xLen = yLen = zLen

    if(tid >= len)
        return;

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Z*>(vy);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ Z norm, sumOfProd;

    if (threadIdx.x == 0) {

        norm = reinterpret_cast<Z*>(vreducBuff)[0];
        sumOfProd = reinterpret_cast<Z*>(vreducBuff)[1];
    }
    __syncthreads();

    const auto yOffset = shape::getIndexOffset(tid, yShapeInfo);
    const auto zOffset = shape::getIndexOffset(tid, zShapeInfo);

   if(norm > clipNormVal) {

        const auto xOffset = shape::getIndexOffset(tid, xShapeInfo);

        const Z factor1 = static_cast<Z>(1) / norm;             // 1 / norm
        const Z factor2 = factor1 / (norm * norm);              // 1 / (norm * norm * norm)

        z[zOffset] = clipNormVal * (factor1 * y[yOffset] - factor2 * sumOfProd * x[xOffset]);
    }
    else {
        z[zOffset] = y[yOffset];
    }
}

//////////////////////////////////////////////////////////////////////////
// x - input, y - gradO, z - gradI
template<typename X, typename Z>
__global__ static void clipByNormBPTadsCuda(const void* vx, const Nd4jLong* xTadShapeInfo, const Nd4jLong* xTadOffsets, const void* vy, const Nd4jLong* yTadShapeInfo, const Nd4jLong* yTadOffsets, void* vz, const Nd4jLong* zTadShapeInfo, const Nd4jLong* zTadOffsets, const Z clipNormVal) {

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Z*>(vy);
          auto z = reinterpret_cast<Z*>(vz);

    __shared__ Z* shMem;
    __shared__ Nd4jLong tadLen;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        shMem = reinterpret_cast<Z*>(shmem);
        tadLen = shape::length(zTadShapeInfo);                  // xTadLen = yTadLen = zTadLen
    }
    __syncthreads();

    const auto* xTad = x + xTadOffsets[blockIdx.x];
    const auto* yTad = y + yTadOffsets[blockIdx.x];
          auto* zTad = z + zTadOffsets[blockIdx.x];

    // *** FIRST STAGE - ACCUMULATE REQUIRED SUMS *** //

    Z norm = 0;
    Z sumOfProd = 0;

    for (uint i = threadIdx.x; i < tadLen; i += blockDim.x) {

        const auto xOffset = shape::getIndexOffset(i, xTadShapeInfo);
        const auto yOffset = shape::getIndexOffset(i, yTadShapeInfo);

        shMem[2*threadIdx.x]     = static_cast<Z>(xTad[xOffset] * xTad[xOffset]);   // for norm
        shMem[2*threadIdx.x + 1] = static_cast<Z>(xTad[xOffset] * yTad[yOffset]);   // for input * gradO

        __syncthreads();

        // accumulate sum per block
        for (uint activeThreads = blockDim.x / 2; activeThreads > 0; activeThreads /= 2) {

            if (threadIdx.x < activeThreads && i + activeThreads < tadLen) {

                shMem[2*threadIdx.x]     += shMem[2*(threadIdx.x + activeThreads)];
                shMem[2*threadIdx.x + 1] += shMem[2*(threadIdx.x + activeThreads) + 1];
            }
            __syncthreads();
        }

        norm      += shMem[0];
        sumOfProd += shMem[1];
    }

    // *** SECOND STAGE - GRADIENT CALCULATION *** //

    norm = math::nd4j_sqrt<Z,Z>(norm);

    for (uint i = threadIdx.x; i < tadLen; i += blockDim.x) {

        const auto yOffset = shape::getIndexOffset(i, yTadShapeInfo);
        const auto zOffset = shape::getIndexOffset(i, zTadShapeInfo);

        if(norm > clipNormVal) {

            const auto xOffset = shape::getIndexOffset(i, xTadShapeInfo);

            const Z factor1 = static_cast<Z>(1) / norm;             // 1 / norm
            const Z factor2 = factor1 / (norm * norm);              // 1 / (norm * norm * norm)

            zTad[zOffset] = clipNormVal * (factor1 * yTad[yOffset] - factor2 * sumOfProd * xTad[xOffset]);
        }
        else {
            zTad[zOffset] = yTad[yOffset];
        }
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename X, typename Z>
static void clipByNormBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                    const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                    const void* vy, const Nd4jLong* yShapeInfo, const Nd4jLong* yTadOffsets,
                                          void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zTadOffsets,
                                    void* vreducBuff, const double clipNormVal) {

    if(xTadOffsets == nullptr) {  // means whole array
        clipByNormBPWholeArrCuda<X,Z><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, vreducBuff, static_cast<Z>(clipNormVal));
        clipByNormBPCalcGradCuda<X,Z><<<blocksPerGrid, threadsPerBlock, 256,       *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, vreducBuff, static_cast<Z>(clipNormVal));
    }
    else                        // means tads using
        clipByNormBPTadsCuda<X,Z><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, xTadOffsets, vy, yShapeInfo, yTadOffsets, vz, zShapeInfo, zTadOffsets, static_cast<Z>(clipNormVal));
}
BUILD_DOUBLE_TEMPLATE(template void clipByNormBPCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const void *vx, const Nd4jLong *xShapeInfo, const Nd4jLong* xTadOffsets, const void *vy, const Nd4jLong *yShapeInfo, const Nd4jLong* yTadOffsets, void *vz, const Nd4jLong *zShapeInfo, const Nd4jLong* zTadOffsets, void* vreducBuff, const double clipNormVal), FLOAT_TYPES, FLOAT_TYPES);

//////////////////////////////////////////////////////////////////////////
void clipByNormBP(nd4j::LaunchContext* context, const NDArray& input, const NDArray& gradO, NDArray& gradI /*output*/, const std::vector<int>& dimensions, const NDArray& clipNorm) {

    PointersManager manager(context, "clipByNormBP");

    const double clipNormVal = clipNorm.e<double>(0);

    const auto xType = input.dataType();
    const auto zType = gradI.dataType();

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int sharedMem = threadsPerBlock * 2 * input.sizeOfT() + 128;

    NDArray::prepareSpecialUse({&gradI}, {&input, &gradO});


    if(dimensions.empty() || dimensions.size() == input.rankOf()) {  // means whole array

        const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
        BUILD_DOUBLE_SELECTOR(xType, zType, clipByNormBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), nullptr, gradO.getSpecialBuffer(), gradO.getSpecialShapeInfo(), nullptr, gradI.getSpecialBuffer(), gradI.getSpecialShapeInfo(), nullptr, context->getReductionPointer(), clipNormVal), FLOAT_TYPES, FLOAT_TYPES);
    }
    else {  // means tads using

        auto packX = ConstantTadHelper::getInstance()->tadForDimensions(input.getShapeInfo(), dimensions);
        auto packY = ConstantTadHelper::getInstance()->tadForDimensions(gradO.getShapeInfo(), dimensions);
        auto packZ = ConstantTadHelper::getInstance()->tadForDimensions(gradI.getShapeInfo(), dimensions);

        const int blocksPerGrid = packX.numberOfTads();
        BUILD_DOUBLE_SELECTOR(xType, zType, clipByNormBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.getSpecialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), gradO.getSpecialBuffer(), packY.platformShapeInfo(), packY.platformOffsets(), gradI.getSpecialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), nullptr, clipNormVal), FLOAT_TYPES, FLOAT_TYPES);
    }

    NDArray::registerSpecialUse({&gradI}, {&input, &gradO});

    manager.synchronize();
}

    template <typename T>
    static __global__ void swapShuffleKernel(T* input, Nd4jLong* shape, Nd4jLong firstDim, nd4j::graph::RandomGenerator* rng) {
        auto tid = blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = firstDim - 1 - tid - threadIdx.x; i > 0; i -= step) {
            int r = rng->relativeInt(i) % i;
            if (i != r) {
                const auto iOffset = shape::getIndexOffset(i, shape);
                const auto rOffset = shape::getIndexOffset(r, shape);
                T e0 = input[iOffset];
                T e1 = input[rOffset];
                //math::nd4j_swap<T>(input(i), input(r));
                input[iOffset] = e1;
                input[rOffset] = e0;
            }
        }
    }
    template <typename T>
    static __global__ void fillShuffleKernel(T* input, Nd4jLong* inputShape, T* output, Nd4jLong* outputShape, Nd4jLong firstDim, int* indices, nd4j::graph::RandomGenerator* rng) {

//        PRAGMA_OMP_PARALLEL_FOR_IF((firstDim-1) > Environment::getInstance()->tadThreshold())
        auto tid = blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for(int i = firstDim - 1 - tid - threadIdx.x; i > 0; i -= step) {
            int r = rng->relativeInt(i) % i;
            output[shape::getIndexOffset(i, outputShape)] = input[shape::getIndexOffset(indices[r], inputShape)];
            if(i != r) {
                output[shape::getIndexOffset(r, outputShape)] = input[shape::getIndexOffset(indices[i], inputShape)];
//                output.p(r, input.e<T>(indices[i]));
//                math::nd4j_swap<int>(indices[i], indices[r]);
                atomicExch(&indices[i], indices[r]);
            }
        }

    }
    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    void randomShuffle_(nd4j::LaunchContext * context, NDArray& input, NDArray& output, nd4j::graph::RandomGenerator& rng, const bool isInplace) {

        // check edge cases first
        int temp;
        const int firstDim = input.sizeAt(0);
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({&output}, {&input});
        if(input.lengthOf() == 1 || firstDim == 1) {
            if(!isInplace)
                output.assign(input);
        }
        else if (input.isVector() || shape::isLikeVector(input.getShapeInfo(), temp)) {

            // apply Fisher-Yates shuffle
            nd4j::graph::RandomGenerator* dRandom = nullptr;
            hipMalloc(&dRandom, sizeof(nd4j::graph::RandomGenerator));
            hipMemcpy(dRandom, &rng, sizeof(nd4j::graph::RandomGenerator), hipMemcpyHostToDevice);
            T* inputBuf = reinterpret_cast<T*>(input.specialBuffer());
            if(isInplace) {
                swapShuffleKernel<T><<<128, 256, 1024, *stream>>>(inputBuf, input.specialShapeInfo(), firstDim, dRandom);
            }
            else {
                std::vector<int> indices(firstDim);
                std::iota(indices.begin(), indices.end(), 0);
                hipMemcpy(output.specialBuffer(), input.specialBuffer(), sizeof(T), hipMemcpyDeviceToDevice);
                //output.p<T>(Nd4jLong(0), input.e<T>(0));
                PointersManager pointersManager(context, "helper::randomShuffle_");
                int* indicesDev = reinterpret_cast<int*>(pointersManager.replicatePointer(indices.data(), indices.size() * sizeof(int)));
                T* outputBuf = reinterpret_cast<T*>(output.specialBuffer());
                fillShuffleKernel<T><<<128, 256, 1024, *stream>>>(inputBuf, input.specialShapeInfo(), outputBuf, output.specialShapeInfo(), firstDim, indicesDev, dRandom);
                pointersManager.synchronize();
            }
//            rng.rewindH(firstDim - 1);
            hipFree(dRandom);
        }
        else {

            // evaluate sub-arrays list of input array through all dimensions excluding first one
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input.rankOf(), {0});
            auto subArrsListIn = input.allTensorsAlongDimension(dimensions);

            // apply Fisher-Yates shuffle
            if(isInplace) {
                PRAGMA_OMP_PARALLEL_FOR_IF((firstDim-1) > Environment::getInstance()->elementwiseThreshold())
                for(int i = firstDim - 1; i > 0; --i) {
                    int r = rng.relativeInt(i) % i;

                    if(i != r)
                        subArrsListIn->at(i)->swapUnsafe(*subArrsListIn->at(r));
                }
            }
            else {
                // evaluate sub-arrays list of output array through all dimensions excluding first one
                auto subArrsListOut = output.allTensorsAlongDimension(dimensions);
                std::vector<int> indices(firstDim);
                std::iota(indices.begin(), indices.end(), 0);
                bool isZeroShuffled = false;
                PRAGMA_OMP_PARALLEL_FOR_IF((firstDim-1) > Environment::getInstance()->tadThreshold())
                for(int i = firstDim - 1; i > 0; --i) {
                    int r = rng.relativeInt(i) % i;
                    subArrsListOut->at(i)->assign(subArrsListIn->at(indices[r]));
                    if(r == 0)
                        isZeroShuffled = true;

                    if(i != r) {
                        subArrsListOut->at(r)->assign(subArrsListIn->at(indices[i]));
                        math::nd4j_swap<int>(indices[i], indices[r]);
                    }
                }
                if(!isZeroShuffled)
                    subArrsListOut->at(0)->assign(subArrsListIn->at(0));
                delete subArrsListOut;
            }
            rng.rewindH(firstDim-1);
            delete subArrsListIn;
        }
        NDArray::registerSpecialUse({&output}, {&input});

    }

    void randomShuffle(nd4j::LaunchContext * context, NDArray& input, NDArray& output, nd4j::graph::RandomGenerator& rng, const bool isInplace) {
        BUILD_SINGLE_SELECTOR(input.dataType(), randomShuffle_, (context, input, output, rng, isInplace), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void randomShuffle_, (nd4j::LaunchContext * context, NDArray& input, NDArray& output, nd4j::graph::RandomGenerator& rng, const bool isInplace), LIBND4J_TYPES);


    //////////////////////////////////////////////////////////////////////////
    void eye(nd4j::LaunchContext * context, NDArray& output) {

        output.setIdentity();
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void clipByNormInplaceKernel(Nd4jLong numOfSubArrs, T* inputBuffer, Nd4jLong* shape, Nd4jLong* inputOffsets, T* norm2Buf, Nd4jLong* norm2shape, T clipNorm) {
        for (int arr = blockIdx.x; arr < numOfSubArrs; arr += gridDim.x) {
            __shared__ T* z;
            __shared__ Nd4jLong len;
            if (threadIdx.x == 0) {
                len = shape::length(shape);
                z = inputBuffer + inputOffsets[arr];
            }
            __syncthreads();
            for (int j = threadIdx.x; j < len; j+= blockDim.x) {
                auto xIndex = shape::getIndexOffset(j, shape);

                if(norm2Buf[arr] > clipNorm)
                z[xIndex] *= clipNorm / norm2Buf[arr]; // case with ews = 1 and ordering is 'c'
            }
        }
    }
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void clipByNormKernel(Nd4jLong numOfSubArrs, T* inputBuffer, Nd4jLong* shape, Nd4jLong* inputOffsets, T* outputBuffer, Nd4jLong* outputShape, Nd4jLong* outputOffsets, T* norm2Buf, Nd4jLong* norm2shape, T clipNorm) {

        for (Nd4jLong arr = blockIdx.x; arr < numOfSubArrs; arr += gridDim.x) {
            __shared__ T* x, *z;
            __shared__ Nd4jLong lenZ;
            __shared__ T norm2;

            if (threadIdx.x == 0) {
                x = inputBuffer + inputOffsets[arr];
                z = outputBuffer + outputOffsets[arr];
                lenZ = shape::length(outputShape);
                norm2 = norm2Buf[shape::getIndexOffset(arr, norm2shape)];
            }
            __syncthreads();
            for (Nd4jLong j = threadIdx.x; j < lenZ; j+= blockDim.x) {
                auto xIndex = shape::getIndexOffset(j, shape);
                auto zIndex = shape::getIndexOffset(j, outputShape);
                if(norm2 > clipNorm) {
                    z[zIndex] = x[xIndex] * clipNorm / norm2; // case with ews = 1 and ordering is 'c'
                } else {
                    z[zIndex] = x[xIndex];
                }
                //printf("%lld: %lf %lf\n", j, z[zIndex], x[xIndex]);
            }
            __syncthreads();
        }
    }

    //////////////////////////////////////////////////////////////////////////
    template<typename T>
    static void clipByNorm_(nd4j::LaunchContext * context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, NDArray const& clipNormA, const bool isInplace) {
        const int rank = input.rankOf();
        auto norm2 = input.reduceAlongDims(reduce::Norm2, dimensions);
        clipNormA.syncToHost();
        //norm2.printBuffer("Norm2");
        T const clipNorm = clipNormA.e<T>(0);
        //clipNormA.printBuffer("ClipNorm");
        auto stream = context->getCudaStream();
        if (isInplace) {
            if(norm2.lengthOf() == 1) {
                norm2.syncToHost();
                T norm2Val = norm2.e<T>(0);
                if(norm2Val > clipNorm)
                    input *= clipNorm / norm2Val;
            }
            else {

                std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(rank, dimensions);
                const Nd4jLong numOfSubArrs = ShapeUtils::getNumOfSubArrs(input.getShapeInfo(), dimsToExclude);
                auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input.getShapeInfo(), dimensions);
                //auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output.getShapeInfo(), dimsToExclude);
                T* inputBuffer = reinterpret_cast<T*>(input.specialBuffer());
                T* norm2buf = reinterpret_cast<T*>(norm2.specialBuffer());

                clipByNormInplaceKernel<T><<<256, 512, 1024, *stream>>>(numOfSubArrs, inputBuffer, packX.specialShapeInfo(), packX.specialOffsets(), norm2buf, norm2.specialShapeInfo(), clipNorm);
            }
        }
        else {

            if(norm2.lengthOf() == 1) {
                norm2.syncToHost();
                T norm2Val = norm2.e<T>(0);

                if(norm2Val > clipNorm)
                    output.assign( input * (clipNorm / norm2Val));
                else
                    output.assign( input );
            }
            else {

                std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(rank, dimensions);
                const Nd4jLong numOfSubArrs = ShapeUtils::getNumOfSubArrs(input.getShapeInfo(), dimsToExclude);
                auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input.getShapeInfo(), dimensions);
                auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output.getShapeInfo(), dimensions);
                T* inputBuffer = reinterpret_cast<T*>(input.specialBuffer());
                T* norm2buf = reinterpret_cast<T*>(norm2.specialBuffer());
                T* outputBuffer = reinterpret_cast<T*>(output.specialBuffer());

                clipByNormKernel<T><<<256, 512, 1024, *stream>>>(numOfSubArrs, inputBuffer, packX.specialShapeInfo(), packX.specialOffsets(), outputBuffer, packZ.specialShapeInfo(), packZ.specialOffsets(), norm2buf, norm2.specialShapeInfo(), clipNorm);
            }
        }
    }

    void clipByNorm(nd4j::LaunchContext * context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace) {
        BUILD_SINGLE_SELECTOR(output.dataType(), clipByNorm_, (context, input, output, dimensions, clipNorm, isInplace), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByNorm_, (nd4j::LaunchContext * context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace), FLOAT_TYPES);

    template <typename T>
    void clipByGlobalNorm_(nd4j::LaunchContext * context, std::vector<NDArray*> const& inputs, double clipNorm, nd4j::memory::Workspace* workspace, std::vector<NDArray*>& outputs, bool isInplace) {
        NDArray globalNorm = NDArrayFactory::create<T>(0, inputs[0]->getContext()); //sqrt(sum([l2norm(t)**2 for t in t_list]))

        for (auto i = 0; i < inputs.size(); i++) {
            auto input = inputs[i];
            auto l2norm = input->reduceNumber(reduce::Norm2);
            globalNorm += l2norm * l2norm;
        }

        globalNorm.applyTransform(transform::Sqrt, nullptr, nullptr);// = nd4j::math::nd4j_sqrt(globalNorm);
        outputs[inputs.size()]->p(0, globalNorm);
        globalNorm.syncToHost();
        const T factor = clipNorm / globalNorm.e<T>(0);

        for (size_t e = 0; e < inputs.size(); e++) {
            // all-reduce
            auto input = inputs[e];
            auto output = outputs[e];

            if (globalNorm.e<double>(0) <= clipNorm) {
                output->assign(input);
            }
            else {

                auto lambda = LAMBDA_T(_x, factor) { return _x * factor; };
                input->applyLambda(lambda, output);
            }
        }
    }

    void clipByGlobalNorm(nd4j::LaunchContext * context, std::vector<NDArray*> const& inputs, double clipNorm, nd4j::memory::Workspace* workspace, std::vector<NDArray*>& outputs, bool isInplace) {
        BUILD_SINGLE_SELECTOR(outputs[0]->dataType(), clipByGlobalNorm_, (context, inputs, clipNorm, workspace, outputs, isInplace), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByGlobalNorm_, (nd4j::LaunchContext * context, std::vector<NDArray*> const& inputs, double clipNorm, nd4j::memory::Workspace* workspace, std::vector<NDArray*>& outputs, bool isInplace), FLOAT_TYPES);


    //////////////////////////////////////////////////////////////////////////
    template<typename T>
    static void clipByAveraged_(nd4j::LaunchContext * context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace) {
        auto cn = clipNorm.e<T>(0);
        if (dimensions.size() == 0) {
            // all-reduce
            T n2 = input.reduceNumber(reduce::Norm2).e<T>(0) / input.lengthOf();
            if (n2 <= cn) {
                if (!isInplace)
                    output.assign(input);
            }
            else {
                const T factor = cn / n2;
                //auto lambda = LAMBDA_T(_x, factor) { return _x * factor; };
                //input.applyLambda<T>(lambda, &output);
                output.assign(input * factor);
            }
        }
        else {
            // along dimension
            auto norm2 = input.reduceAlongDims(reduce::Norm2, dimensions, false);
            if (!isInplace)
                output.assign(input);
            auto tads = output.allTensorsAlongDimension(dimensions);
            auto outTads = output.allTensorsAlongDimension(dimensions);
            // TODO: make this CUDA-compliant somehow
            for (int e = 0; e < tads->size(); e++) {
                T n2 = norm2.e<T>(e) / tads->at(e)->lengthOf();
                const T factor = cn / n2;
                if (n2 > cn) {
                    //auto lambda = LAMBDA_T(_x, factor) {return _x * factor;};
                    tads->at(e)->applyScalar(scalar::Multiply, factor, outTads->at(e));//applyLambda<T>(lambda, &output);
                }
            }
            delete tads;
            delete outTads;
        }
    }

    void clipByAveraged(nd4j::LaunchContext * context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace) {
        BUILD_SINGLE_SELECTOR(input.dataType(), clipByAveraged_, (context, input, output, dimensions, clipNorm, isInplace), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByAveraged_, (nd4j::LaunchContext * context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace), FLOAT_TYPES);

/*
    if (d1 > params[1])
    return params[1];
    else if (d1 < params[0])
    return params[0];
    else return d1;
*/
    template <typename T>
    static void __global__ clipByValueKernel(void* input, Nd4jLong* inputShape, void* output, Nd4jLong* outputShape, double leftBound, double rightBound) {
        __shared__ T* outputBuf;
        __shared__ T* inputBuf;
        __shared__ Nd4jLong length;
        __shared__ bool linearBuffers;
        if (threadIdx.x == 0) {
            outputBuf = reinterpret_cast<T *>(output);
            inputBuf = reinterpret_cast<T *>(input);
            length = shape::length(inputShape);
            linearBuffers = shape::elementWiseStride(inputShape) == shape::elementWiseStride(outputShape) && shape::elementWiseStride(inputShape) == 1;
        }
        __syncthreads();
        const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;

        for (Nd4jLong e = tid; e < length; e += step) {
            if (linearBuffers) {
                if (inputBuf[e] > rightBound) outputBuf[e] = (T) rightBound;
                else if (inputBuf[e] < leftBound) outputBuf[e] = (T) leftBound;
                else outputBuf[e] = inputBuf[e];
            }
            else {
                auto inputOffset = shape::getIndexOffset(e, inputShape);
                auto outputOffset = shape::getIndexOffset(e, outputShape);
                if (inputBuf[inputOffset] > rightBound) outputBuf[outputOffset] = (T) rightBound;
                else if (inputBuf[inputOffset] < leftBound) outputBuf[outputOffset] = (T) leftBound;
                else outputBuf[outputOffset] = inputBuf[outputOffset];
            }
        }
    }

    template <typename T>
    static void clipByValue_(nd4j::LaunchContext * context, NDArray& input, double leftBound, double rightBound, NDArray& output) {
        auto stream = context->getCudaStream();
        if (!input.isActualOnDeviceSide())
            input.syncToDevice();
        NDArray::prepareSpecialUse({&output}, {&input});
        clipByValueKernel<T><<<256, 512, 8192, *stream>>>(input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), leftBound, rightBound);
        NDArray::registerSpecialUse({&output}, {&input});
    }

    void clipByValue(nd4j::LaunchContext * context, NDArray& input, double leftBound, double rightBound, NDArray& output) {
        BUILD_SINGLE_SELECTOR(input.dataType(), clipByValue_, (context, input, leftBound, rightBound, output), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByValue_, (nd4j::LaunchContext * context, NDArray& input, double leftBound, double rightBound, NDArray& output);, FLOAT_TYPES);

}
}
}

