#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 31.08.2018
//

#include <ops/declarable/helpers/histogramFixedWidth.h>
#include <cuda_exception.h>
#include <PointersManager.h>

namespace nd4j    {
namespace ops     {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename X, typename Z>
__global__ static void histogramFixedWidthCuda( const void* vx, const Nd4jLong* xShapeInfo,
                                                      void* vz, const Nd4jLong* zShapeInfo,
                                                const X leftEdge, const X rightEdge) {

    const auto x  = reinterpret_cast<const X*>(vx);
    auto z = reinterpret_cast<Z*>(vz);

    __shared__ Nd4jLong xLen, zLen, totalThreads, nbins;
    __shared__ X binWidth, secondEdge, lastButOneEdge;

    if (threadIdx.x == 0) {

        xLen  = shape::length(xShapeInfo);
        nbins = shape::length(zShapeInfo);          // nbins = zLen
        totalThreads = gridDim.x * blockDim.x;

        binWidth       = (rightEdge - leftEdge ) / nbins;
        secondEdge     = leftEdge + binWidth;
        lastButOneEdge = rightEdge - binWidth;
    }

    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < xLen; i += totalThreads) {

        const X value = x[shape::getIndexOffset(i, xShapeInfo)];

        Nd4jLong zIndex;

        if(value < secondEdge)
            zIndex = 0;
        else if(value >= lastButOneEdge)
            zIndex = nbins - 1;
        else
            zIndex = static_cast<Nd4jLong>((value - leftEdge) / binWidth);

        nd4j::math::atomics::nd4j_atomicAdd<Z>(&z[shape::getIndexOffset(zIndex, zShapeInfo)], 1);
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Z>
__host__ static void histogramFixedWidthCudaLauncher(const hipStream_t *stream, const NDArray& input, const NDArray& range, NDArray& output) {

    const X leftEdge  = range.e<X>(0);
    const X rightEdge = range.e<X>(1);

    histogramFixedWidthCuda<X, Z><<<256, 256, 1024, *stream>>>(input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), leftEdge, rightEdge);
}

////////////////////////////////////////////////////////////////////////
void histogramFixedWidth(nd4j::LaunchContext* context, const NDArray& input, const NDArray& range, NDArray& output) {

    // firstly initialize output with zeros
    output.nullify();

    PointersManager manager(context, "histogramFixedWidth");

    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_DOUBLE_SELECTOR(input.dataType(), output.dataType(), histogramFixedWidthCudaLauncher, (context->getCudaStream(), input, range, output), LIBND4J_TYPES, INDEXING_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

    manager.synchronize();
}


//     template <typename T>
//     __global__ static void copyBuffers(Nd4jLong* destination, void const* source, Nd4jLong* sourceShape, Nd4jLong bufferLength) {
//         const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
//         const auto step = gridDim.x * blockDim.x;
//         for (int t = tid; t < bufferLength; t += step) {
//             destination[t] = reinterpret_cast<T const*>(source)[shape::getIndexOffset(t, sourceShape)];
//         }
//     }

//     template <typename T>
//     __global__ static void returnBuffers(void* destination, Nd4jLong const* source, Nd4jLong* destinationShape, Nd4jLong bufferLength) {
//         const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
//         const auto step = gridDim.x * blockDim.x;
//         for (int t = tid; t < bufferLength; t += step) {
//             reinterpret_cast<T*>(destination)[shape::getIndexOffset(t, destinationShape)] = source[t];
//         }
//     }

//     template <typename T>
//     static __global__ void histogramFixedWidthKernel(void* outputBuffer, Nd4jLong outputLength, void const* inputBuffer, Nd4jLong* inputShape, Nd4jLong inputLength, double const leftEdge, double binWidth, double secondEdge, double lastButOneEdge) {

//         __shared__ T const* x;
//         __shared__ Nd4jLong* z; // output buffer

//         if (threadIdx.x == 0) {
//             z = reinterpret_cast<Nd4jLong*>(outputBuffer);
//             x = reinterpret_cast<T const*>(inputBuffer);
//         }
//         __syncthreads();
//         auto tid = blockIdx.x * gridDim.x + threadIdx.x;
//         auto step = blockDim.x * gridDim.x;

//         for(auto i = tid; i < inputLength; i += step) {

//             const T value = x[shape::getIndexOffset(i, inputShape)];
//             Nd4jLong currInd = static_cast<Nd4jLong>((value - leftEdge) / binWidth);

//             if(value < secondEdge)
//                 currInd = 0;
//             else if(value >= lastButOneEdge)
//                 currInd = outputLength - 1;
//             nd4j::math::atomics::nd4j_atomicAdd(&z[currInd], 1LL);
//         }
//     }


//     template <typename T>
//     void histogramFixedWidth_(nd4j::LaunchContext * context, const NDArray& input, const NDArray& range, NDArray& output) {
//         const int nbins = output.lengthOf();
//         auto stream = context->getCudaStream();
//         // firstly initialize output with zeros
//         //if(output.ews() == 1)
//         //    memset(output.buffer(), 0, nbins * output.sizeOfT());
//         //else
//         output.assign(0);
//         if (!input.isActualOnDeviceSide())
//             input.syncToDevice();

//         const double leftEdge  = range.e<double>(0);
//         const double rightEdge = range.e<double>(1);

//         const double binWidth       = (rightEdge - leftEdge ) / nbins;
//         const double secondEdge     = leftEdge + binWidth;
//         double lastButOneEdge = rightEdge - binWidth;
//         Nd4jLong* outputBuffer;
//         hipError_t err = hipMalloc(&outputBuffer, output.lengthOf() * sizeof(Nd4jLong));
//         if (err != 0)
//             throw cuda_exception::build("helpers::histogramFixedWidth: Cannot allocate memory for output", err);
//         copyBuffers<Nd4jLong ><<<256, 512, 8192, *stream>>>(outputBuffer, output.getSpecialBuffer(), output.getSpecialShapeInfo(), output.lengthOf());
//         histogramFixedWidthKernel<T><<<256, 512, 8192, *stream>>>(outputBuffer, output.lengthOf(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), input.lengthOf(), leftEdge, binWidth, secondEdge, lastButOneEdge);
//         returnBuffers<Nd4jLong><<<256, 512, 8192, *stream>>>(output.specialBuffer(), outputBuffer, output.specialShapeInfo(), output.lengthOf());
//         //cudaSyncStream(*stream);
//         err = hipFree(outputBuffer);
//         if (err != 0)
//             throw cuda_exception::build("helpers::histogramFixedWidth: Cannot deallocate memory for output buffer", err);
//         output.tickWriteDevice();
// //#pragma omp parallel for schedule(guided)
// //        for(Nd4jLong i = 0; i < input.lengthOf(); ++i) {
// //
// //            const T value = input.e<T>(i);
// //
// //            if(value < secondEdge)
// //#pragma omp critical
// //                output.p<Nd4jLong>(0, output.e<Nd4jLong>(0) + 1);
// //            else if(value >= lastButOneEdge)
// //#pragma omp critical
// //                output.p<Nd4jLong>(nbins-1, output.e<Nd4jLong>(nbins-1) + 1);
// //            else {
// //                Nd4jLong currInd = static_cast<Nd4jLong>((value - leftEdge) / binWidth);
// //#pragma omp critical
// //                output.p<Nd4jLong>(currInd, output.e<Nd4jLong>(currInd) + 1);
// //            }
// //        }
//     }

//     void histogramFixedWidth(nd4j::LaunchContext * context, const NDArray& input, const NDArray& range, NDArray& output) {
//         BUILD_SINGLE_SELECTOR(input.dataType(), histogramFixedWidth_, (context, input, range, output), LIBND4J_TYPES);
//     }
//     BUILD_SINGLE_TEMPLATE(template void histogramFixedWidth_, (nd4j::LaunchContext * context, const NDArray& input, const NDArray& range, NDArray& output), LIBND4J_TYPES);

}
}
}