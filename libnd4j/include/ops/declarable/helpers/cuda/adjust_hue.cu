#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/adjust_hue.h>
#include <helpers/ConstantTadHelper.h>
#include <PointersManager.h>

namespace nd4j    {
namespace ops     {
namespace helpers {


///////////////////////////////////////////////////////////////////
template <typename T>
static void _CUDA_G adjustHueCuda(const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                        void* vz, const Nd4jLong *zShapeInfo, const Nd4jLong* zTadOffsets,
                                        const Nd4jLong numOfTads, const T delta, const int dimC) {

    const T* x = reinterpret_cast<const T*>(vx);
          T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank;
    __shared__ Nd4jLong xDimCstride, zDimCstride;

    if (threadIdx.x == 0) {
        rank = shape::rank(xShapeInfo);
        xDimCstride = shape::stride(xShapeInfo)[dimC];
        zDimCstride = shape::stride(zShapeInfo)[dimC];
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < numOfTads; i += gridDim.x * blockDim.x) {

        const T* xTad = x + xTadOffsets[i];
              T* zTad = z + zTadOffsets[i];

        T h, s, v;

        rgbToHsv<T>(xTad[0], xTad[xDimCstride], xTad[2 * xDimCstride], h, s, v);

        h += delta * 360;
        if(h > 360)
            h -= 360;
        else if(h < 0)
            h += 360;

        hsvToRgb<T>(h, s, v, zTad[0], zTad[zDimCstride], zTad[2 * zDimCstride]);
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
static _CUDA_H void adjustHueCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                          const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                                void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zTadOffsets,
                                          const Nd4jLong numOfTads, const NDArray* deltaScalarArr, const int dimC) {

    adjustHueCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(vx, xShapeInfo, xTadOffsets, vz, zShapeInfo, zTadOffsets, numOfTads, deltaScalarArr->e<T>(0), dimC);
}

////////////////////////////////////////////////////////////////////////
void adjustHue(nd4j::LaunchContext* context, const NDArray *input, const NDArray* deltaScalarArr, NDArray *output, const int dimC) {

    auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(),  {dimC});
    auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), {dimC});

    const Nd4jLong numOfTads = packX.numberOfTads();

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (numOfTads + threadsPerBlock - 1) / threadsPerBlock;

    PointersManager manager(context, "adjustHue");

    NDArray::prepareSpecialUse({output}, {input, deltaScalarArr});
    BUILD_SINGLE_SELECTOR(input->dataType(), adjustHueCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), packX.platformOffsets(), output->specialBuffer(), output->specialShapeInfo(), packZ.platformOffsets(), numOfTads, deltaScalarArr, dimC), FLOAT_TYPES);
    NDArray::registerSpecialUse({output}, {input, deltaScalarArr});

    manager.synchronize();
}


/*
template <typename T>
static void _CUDA_G adjustHueSingleNHWCKernel(void *xBuffer, Nd4jLong *xShapeInfo,  void *zBuffer, Nd4jLong *zShapeInfo, Nd4jLong tuples, float delta) {
    int numChannels = 3;
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;

    auto bIn = reinterpret_cast<T*>(xBuffer);
    auto bOut = reinterpret_cast<T*>(zBuffer);
    static const int kChannelRange = 6;

    for (Nd4jLong e = tid; e < tuples; e += blockDim.x * gridDim.x) {
        auto i = bIn + e * numChannels;
        auto o = bOut + e * numChannels;

        T h, v_min, v_max;
        helpers::rgb_to_hv(i[0], i[1], i[2], &h, &v_min, &v_max);

        h += delta * kChannelRange;
        while (h < (T) 0.)
            h += (T) kChannelRange;

        while (h >= (T) kChannelRange)
            h -= (T) kChannelRange;

        helpers::hv_to_rgb(h, v_min, v_max, o, o + 1, o + 2);
    }
}

template <typename T>
static void _CUDA_G adjustHueSingleNCHWKernel(void *xBuffer, Nd4jLong *xTadShapeInfo, Nd4jLong *xOffsets, void *zBuffer, Nd4jLong *zTadShapeInfo, Nd4jLong *zOffsets, Nd4jLong tadLength, Nd4jLong tuples, float delta) {
    int numChannels = 3;
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;
    static const int kChannelRange = 6;

    auto bufferR = reinterpret_cast<T *>(xBuffer) + xOffsets[0];
    auto bufferG = reinterpret_cast<T *>(xBuffer) + xOffsets[1];
    auto bufferB = reinterpret_cast<T *>(xBuffer) + xOffsets[2];

    auto outputR = reinterpret_cast<T *>(zBuffer) + zOffsets[0];
    auto outputG = reinterpret_cast<T *>(zBuffer) + zOffsets[1];
    auto outputB = reinterpret_cast<T *>(zBuffer) + zOffsets[2];


    for (Nd4jLong e = tid; e < tuples; e += blockDim.x * gridDim.x) {
        auto _ri = bufferR + shape::getIndexOffset(e, xTadShapeInfo);
        auto _gi = bufferG + shape::getIndexOffset(e, xTadShapeInfo);
        auto _bi = bufferB + shape::getIndexOffset(e, xTadShapeInfo);

        auto _ro = outputR + shape::getIndexOffset(e, xTadShapeInfo);
        auto _go = outputG + shape::getIndexOffset(e, xTadShapeInfo);
        auto _bo = outputB + shape::getIndexOffset(e, xTadShapeInfo);

        T h, v_min, v_max;
        helpers::rgb_to_hv(_ri[0], _gi[0], _bi[0], &h, &v_min, &v_max);

        h += delta * kChannelRange;
        while (h < (T) 0)
            h += (T) kChannelRange;

        while (h >= (T) kChannelRange)
            h -= (T) kChannelRange;

        helpers::hv_to_rgb(h, v_min, v_max, _ro, _go, _bo);
    }
}

template <typename T>
static void _adjust_hue_single(nd4j::LaunchContext * context, NDArray *array, NDArray *output, float delta, bool isNHWC) {
    // numChannels is always 3
    auto tuples = array->lengthOf() / 3;
    if (isNHWC) {
        adjustHueSingleNHWCKernel<T><<<256, 256, 1024, *context->getCudaStream()>>>(array->specialBuffer(), array->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), tuples, delta);
    } else {
        // TODO: check this one
        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(array->getShapeInfo(), {1, 2});
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), {1, 2});

        auto tadLength = shape::length(packX.primaryShapeInfo());

        adjustHueSingleNCHWKernel<T><<<256, 256, 1024, *context->getCudaStream()>>>(array->specialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), output->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), tadLength, tuples, delta);
    }
}


template <typename T>
static void _adjust_hue_batch(nd4j::LaunchContext * context, NDArray *array, NDArray *output, float delta, bool isNHWC) {
    auto xType = array->dataType();

    // numChannels is always 3
    auto tuples = array->lengthOf() / 3;

    if (isNHWC) {
        // in case of nhwc batch, we don't really care about examples: it's still bunch of RGB values
        BUILD_SINGLE_SELECTOR(xType, _adjust_hue_single, (context, array, output, delta, isNHWC);, FLOAT_TYPES);
    } else {
        // TODO: check this one
        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(array->getShapeInfo(), {0, 2, 3});
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), {0, 2, 3});

        auto tadLength = shape::length(packX.primaryShapeInfo());

        adjustHueSingleNCHWKernel<T><<<256, 256, 1024, *context->getCudaStream()>>>(array->specialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), output->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), tadLength, tuples, delta);
    }
}

void _adjust_hue(nd4j::LaunchContext * context, NDArray *array, NDArray *output, NDArray* delta, bool isNHWC) {
    auto xType = array->dataType();

    float d = delta->e<float>(0);
    if (array->rankOf() == 4) {
        BUILD_SINGLE_SELECTOR(xType, _adjust_hue_batch, (context, array, output, d, isNHWC);, FLOAT_TYPES);
    } else {
        BUILD_SINGLE_SELECTOR(xType, _adjust_hue_single, (context, array, output, d, isNHWC);, FLOAT_TYPES);
    }
}

*/
}
}
}
