#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>
#include <NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j {
namespace ops {
namespace helpers {
    // -------------------------------------------------------------------------------------------------------------- //
    // Segment Prod ops linear kernels
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static __global__ void segmentProdLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();
//         auto tid = threadIdx.x + blockIdx.x * blockDim.x;
//         auto step = blockDim.x * gridDim.x;

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape);
            nd4j::math::atomics::nd4j_atomicMul(&val[segment], x[xIndex]);
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            z[zIndex] = val[segment];
        }
        __syncthreads();
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void unsortedSegmentProdLinearKernel(void* input, Nd4jLong* inputShape, void* indices, Nd4jLong* indicesShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
//            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x;// / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

//            if (segment < numOfClasses) {
            zIndex = shape::getIndexOffset(segment, outputShape);
            //start = starts[segment];
            //finish = start + lengths[segment];
            if (lengths[segment] > 0)
                z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape)];
            else
                z[zIndex] = 0; //DataTypeUtils::max<T>();
//                val[segment] = z[zIndex];
//            }

        }
        __syncthreads();
        if (lengths[segment] > 0)
            for (auto e = threadIdx.x; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape);
                auto yIndex = shape::getIndexOffset(e, indicesShape);
                if (y[yIndex] == segment && e != starts[segment]) {
                    nd4j::math::atomics::nd4j_atomicMul(&z[zIndex], x[xIndex]);
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // SegmentProd kernel
    template <typename T, typename I>
    static __global__ void segmentProdTadKernel(void* inputBuf, Nd4jLong* inputShape, Nd4jLong* inputTads, Nd4jLong* inputTadOffsets, I* indices, int* starts, int* lengths, Nd4jLong numOfClasses, void* outputBuf, Nd4jLong* outputShape, Nd4jLong* outputTads, Nd4jLong* outputTadOffsets) {
        __shared__ T* val;
        __shared__ Nd4jLong len, segment, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            segment = indices[blockIdx.x]; // / threadsPerSegment;
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads);
                    auto zIndex = shape::getIndexOffset(e, outputTads);
                    nd4j::math::atomics::nd4j_atomicMul(&z[zIndex], x[xIndex]);
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads);
                    auto zIndex = shape::getIndexOffset(e, outputTads);
                    if (lengths[segment] > 0)
                        nd4j::math::atomics::nd4j_atomicMul(&z[zIndex], x[xIndex]);
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void segmentProdFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});
        output->assign(1);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            segmentProdLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            segmentProdTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void segmentProdFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), segmentProdFunctor_, (context, input, indices, output), NUMERIC_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static void unsortedSegmentProdFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses});
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses});
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(nd4j::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            unsortedSegmentProdLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(1);
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentProdTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void unsortedSegmentProdFunctor(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentProdFunctor_, (context, input, indices, numOfClasses, output),
                              NUMERIC_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void segmentProdBPLinearKernel(void* inputBuf, Nd4jLong* inputShape, void* forwardOutput,
                                                     Nd4jLong* forwardShape, void* eps, Nd4jLong* epsShape, void* indicesBuf, Nd4jLong* indicesShape,
                                                     void* outputBuf, Nd4jLong* outputShape) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            gradIn = reinterpret_cast<T*>(forwardOutput);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape);
            auto xOffset = shape::getIndexOffset(e, inputShape);
            auto yOffset = shape::getIndexOffset(e, indicesShape);
            auto classIndex = y[yOffset];
            auto gradOffsetI = shape::getIndexOffset(classIndex, forwardShape);
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

            z[zOffset] = gradOut[gradOffsetO]  * gradIn[gradOffsetI] / x[xOffset];
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static __global__ void segmentProdBPTadKernel(void* inputBuf, Nd4jLong* inputShape, void* forwardOutput,
                                                  Nd4jLong* forwardShape, void* eps, Nd4jLong* epsShape, void* indicesBuf, Nd4jLong* indicesShape,
                                                  void* outputBuf, Nd4jLong* outputShape,Nd4jLong* inputTad,
                                                  Nd4jLong* inputOffsets, Nd4jLong* gradInTad, Nd4jLong* gradInOffsets,
                                                  Nd4jLong* gradOutTad, Nd4jLong* gradOutOffsets, Nd4jLong* outTad,
                                                  Nd4jLong* outOffsets) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ Nd4jLong xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<T*>(eps);
            gradIn = reinterpret_cast<T*>(forwardOutput);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
            auto yIndex = shape::getIndexOffset(i, indicesShape);
            auto segment = y[yIndex];
            T* current = x + inputOffsets[i];
            T* currentOut = z + outOffsets[i];
            T* in = gradIn + gradInOffsets[segment];
            T* outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                currentOut[e] = outGrad[e] * in[e] / current[e];
            }
        }

    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    int segmentProdFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        segmentProdFunctor_<T, I>(context, input, indices, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            Nd4jLong loopSize = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentProdBPLinearKernel<T,I><<<gradOut->lengthOf(), loopSize, 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            auto packGradIn = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempRes.getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradInTads = packGradIn.specialShapeInfo();
            Nd4jLong* gradInTadOffsets = packGradIn.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentProdBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }

    // -------------------------------------------------------------------------------------------------------------- //

    int segmentProdFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentProdFunctorBP_, (context, input,
                indices, gradOut, output), FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }

    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static int unsortedSegmentProdFunctorBP_(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();

        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        unsortedSegmentProdFunctor_<T, I>(context, input, indices, numOfClasses, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            Nd4jLong loopSize = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<Nd4jLong>(loop_size - 1);
            segmentProdBPLinearKernel<T,I><<<gradOut->lengthOf(), loopSize, 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), dimensions);
            auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), dimensions);
            auto packGradIn = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(tempRes.getShapeInfo(), dimensions);
            auto packGradOut = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(gradOut->getShapeInfo(), dimensions);
            Nd4jLong* inputTads = packX.specialShapeInfo();
            Nd4jLong* inputTadOffsets = packX.specialOffsets();
            Nd4jLong* outputTads = packZ.specialShapeInfo();
            Nd4jLong* outputTadOffsets = packZ.specialOffsets();
            Nd4jLong* gradInTads = packGradIn.specialShapeInfo();
            Nd4jLong* gradInTadOffsets = packGradIn.specialOffsets();
            Nd4jLong* gradOutTads = packGradOut.specialShapeInfo();
            Nd4jLong* gradOutTadOffsets = packGradOut.specialOffsets();

            segmentProdBPTadKernel<T,I><<<indices->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return Status::OK();
    }

    // -------------------------------------------------------------------------------------------------------------- //
    int unsortedSegmentProdFunctorBP(nd4j::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentProdFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), FLOAT_TYPES, INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }

    // -------------------------------------------------------------------------------------------------------------- //

}
}
}
