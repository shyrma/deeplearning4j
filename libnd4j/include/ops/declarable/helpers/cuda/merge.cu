#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j {
    namespace ops {
        namespace helpers {
            //////////////////////////////////////////////////////////////////////////
            template <typename T, typename Z>
            static __global__ void global_mergeMaxIndex_(void **inArrs, void **inShapes, const int numArrays, void *voutput, Nd4jLong *outputShape, Nd4jLong length) {
                auto output = reinterpret_cast<Z*>(voutput);

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
                const auto step = gridDim.x * blockDim.x;

                for (Nd4jLong e = tid; e < length; e += step) {
                    T mVal = -DataTypeUtils::max<T>();
                    Z mIdx(0);

                    for (int i = 0; i < numArrays; i++) {
                        auto x = reinterpret_cast<T*>(inArrs[i]);
                        auto xShape = reinterpret_cast<Nd4jLong *>(inShapes[i]);
                        auto val = x[shape::getIndexOffset(e, xShape)];;
                        if (mVal < val) {
                            mIdx = static_cast<Z>(i);
                            mVal = val;
                        }
                    }
                    __syncthreads();

                    output[shape::getIndexOffset(e, outputShape)] = mIdx;
                }
            }

            template <typename T, typename Z>
            static void mergeMaxIndex_(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
                std::vector<void *> inBuffers(inArrs.size());
                std::vector<void *> inShapes(inArrs.size());

                for (int e = 0; e < inArrs.size(); e++) {
                    inBuffers[e] = inArrs[e]->getSpecialBuffer();
                    inShapes[e] = inArrs[e]->getSpecialShapeInfo();
                }

                PointersManager manager(context, "mergeMaxIndex");

                auto pInBuffers = reinterpret_cast<void **>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void *)));
                auto pInShapes = reinterpret_cast<void **>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void *)));
                auto length = output.lengthOf();

                global_mergeMaxIndex_<T,Z><<<512, 512, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, (int) inArrs.size(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), length);

                manager.synchronize();
            }

            void mergeMaxIndex(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
                NDArray::prepareSpecialUse({&output}, {});
                for (auto v:inArrs)
                    v->syncToDevice();

                BUILD_DOUBLE_SELECTOR(inArrs[0]->dataType(), output.dataType(), mergeMaxIndex_, (context, inArrs, output), LIBND4J_TYPES, INDEXING_TYPES);

                NDArray::registerSpecialUse({&output}, {});
            }


            //////////////////////////////////////////////////////////////////////////
            template <typename T>
            static __global__ void global_mergeMax_(void **inArrs, void **inShapes, const int numArrays, void *voutput, Nd4jLong *outputShape, Nd4jLong length) {
                auto output = reinterpret_cast<T*>(voutput);

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
                const auto step = gridDim.x * blockDim.x;

                for (Nd4jLong e = tid; e < length; e += step) {
                    T mVal = -DataTypeUtils::max<T>();

                    for (int i = 0; i < numArrays; i++) {
                        auto x = reinterpret_cast<T*>(inArrs[i]);
                        auto xShape = reinterpret_cast<Nd4jLong *>(inShapes[i]);
                        auto val = x[shape::getIndexOffset(e, xShape)];;
                        if (mVal < val)
                            mVal = val;
                    }
                    __syncthreads();

                    output[shape::getIndexOffset(e, outputShape)] = mVal;
                }
            }

            template<typename T>
            static void mergeMax_(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
                std::vector<void *> inBuffers(inArrs.size());
                std::vector<void *> inShapes(inArrs.size());

                for (int e = 0; e < inArrs.size(); e++) {
                    inBuffers[e] = inArrs[e]->getSpecialBuffer();
                    inShapes[e] = inArrs[e]->getSpecialShapeInfo();
                }

                PointersManager manager(context, "mergeMax");

                auto pInBuffers = reinterpret_cast<void **>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void *)));
                auto pInShapes = reinterpret_cast<void **>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void *)));
                auto length = output.lengthOf();

                global_mergeMax_<T><<<512, 512, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, (int) inArrs.size(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), length);

                manager.synchronize();
            }

            void mergeMax(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
                NDArray::prepareSpecialUse({&output}, {});
                for (auto v:inArrs)
                    v->syncToDevice();

                BUILD_SINGLE_SELECTOR(output.dataType(), mergeMax_, (context, inArrs, output), LIBND4J_TYPES);
                NDArray::registerSpecialUse({&output}, {});
            }

            //////////////////////////////////////////////////////////////////////////
            template <typename T>
            static __global__ void global_mergeAvg_(void **inArrs, void **inShapes, const int numArrays, void *voutput, Nd4jLong *outputShape, Nd4jLong length) {
                auto output = reinterpret_cast<T*>(voutput);

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
                const auto step = gridDim.x * blockDim.x;

                for (Nd4jLong e = tid; e < length; e += step) {
                    T sum(0.0f);

                    for (int i = 0; i < numArrays; i++) {
                        auto x = reinterpret_cast<T*>(inArrs[i]);
                        auto xShape = reinterpret_cast<Nd4jLong *>(inShapes[i]);

                        sum += x[shape::getIndexOffset(e, xShape)];
                    }

                    output[shape::getIndexOffset(e, outputShape)] = sum / numArrays;
                }
            }

            template<typename T>
            static void mergeAvg_(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
                std::vector<void *> inBuffers(inArrs.size());
                std::vector<void *> inShapes(inArrs.size());

                for (int e = 0; e < inArrs.size(); e++) {
                    inBuffers[e] = inArrs[e]->getSpecialBuffer();
                    inShapes[e] = inArrs[e]->getSpecialShapeInfo();
                }

                PointersManager manager(context, "mergeAvg");

                auto pInBuffers = reinterpret_cast<void **>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void *)));
                auto pInShapes = reinterpret_cast<void **>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void *)));
                auto length = output.lengthOf();

                global_mergeAvg_<T><<<512, 512, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, (int) inArrs.size(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), length);

                manager.synchronize();
            }

            void mergeAvg(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
                NDArray::prepareSpecialUse({&output}, {});
                for (auto v:inArrs)
                    v->syncToDevice();

                BUILD_SINGLE_SELECTOR(output.dataType(), mergeAvg_, (context, inArrs, output), FLOAT_TYPES);

                NDArray::registerSpecialUse({&output}, {});
            }

            //////////////////////////////////////////////////////////////////////////
            template <typename T>
            static __global__ void global_mergeAdd_(void **inArrs, void **inShapes, const int numArrays, void *voutput, Nd4jLong *outputShape, Nd4jLong length) {
                auto output = reinterpret_cast<T*>(voutput);

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
                const auto step = gridDim.x * blockDim.x;

                for (Nd4jLong e = tid; e < length; e += step) {
                    T sum(0.0f);

                    for (int i = 0; i < numArrays; i++) {
                        auto x = reinterpret_cast<T*>(inArrs[i]);
                        auto xShape = reinterpret_cast<Nd4jLong *>(inShapes[i]);

                        sum += x[shape::getIndexOffset(e, xShape)];
                    }

                    output[shape::getIndexOffset(e, outputShape)] = sum;
                }
            }

            template<typename T>
            static void mergeAdd_(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
                std::vector<void *> inBuffers(inArrs.size());
                std::vector<void *> inShapes(inArrs.size());

                for (int e = 0; e < inArrs.size(); e++) {
                    inBuffers[e] = inArrs[e]->getSpecialBuffer();
                    inShapes[e] = inArrs[e]->getSpecialShapeInfo();
                }

                PointersManager manager(context, "mergeAdd");

                auto pInBuffers = reinterpret_cast<void **>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void *)));
                auto pInShapes = reinterpret_cast<void **>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void *)));
                auto length = output.lengthOf();

                global_mergeAdd_<T><<<512, 512, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, (int) inArrs.size(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), length);

                manager.synchronize();
            }
            BUILD_SINGLE_TEMPLATE(template void mergeAdd_, (nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output), NUMERIC_TYPES);

            void mergeAdd(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output) {
                NDArray::prepareSpecialUse({&output}, {});
                for (auto v:inArrs)
                    v->syncToDevice();

                BUILD_SINGLE_SELECTOR(output.dataType(), mergeAdd_, (context, inArrs, output), NUMERIC_TYPES);

                NDArray::registerSpecialUse({&output}, {});
            }
        }
    }
}