#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/sequence_mask.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename I, typename B>
    static __global__ void sequenceMaskKernel(void* inputBuf, Nd4jLong* inputShape, void* outputBuf, Nd4jLong* outputShape, int maxIndex) {

        __shared__ I* input;
        __shared__ B* output;
        __shared__ Nd4jLong inputLen, outputLen;
        if (threadIdx.x == 0) {
            input = reinterpret_cast<I*>(inputBuf);
            output = reinterpret_cast<B*>(outputBuf);
            inputLen = shape::length(inputShape);
            outputLen = shape::length(outputShape);
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < maxIndex; i += gridDim.x)
            for(auto k = threadIdx.x; k < inputLen; k += blockDim.x)
                if (i < input[shape::getIndexOffset(k, inputShape)])
                    output[shape::getIndexOffset(k * maxIndex + i, outputShape)] = B(true);

    }

    template <typename I, typename B>
    static void sequenceMask_(LaunchContext* context, NDArray* input, NDArray* output, int maxIndex) {
        dim3 launchDims(maxIndex, input->lengthOf(), 128);
        NDArray::prepareSpecialUse({output}, {input});
        auto stream = context->getCudaStream();
        sequenceMaskKernel<I, B><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), maxIndex);
        NDArray::registerSpecialUse({output}, {input});
    }

    void sequenceMask(nd4j::LaunchContext * context, NDArray* input, NDArray* output, int maxIndex) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), output->dataType(), sequenceMask_, (context, input, output, maxIndex), INTEGER_TYPES, BOOL_TYPES);
    }

    BUILD_DOUBLE_TEMPLATE(template void sequenceMask_, (nd4j::LaunchContext* context, NDArray* input, NDArray* output, int maxIndex), INTEGER_TYPES, BOOL_TYPES);
}
}
}