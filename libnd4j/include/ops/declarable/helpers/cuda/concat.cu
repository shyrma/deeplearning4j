#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j    {
namespace ops     {
namespace helpers {


///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void concatCuda(void* pVx,  void* pxShapeInfo, void* vz, Nd4jLong* zShapeInfo, const int axis) {

    T* z = reinterpret_cast<T*>(vz);
    __shared__ Nd4jLong zLen, totalThreads, *sharedMem;
    __shared__ int rank;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        zLen = shape::length(zShapeInfo);
        rank = shape::rank(zShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid >= zLen)
        return;

    auto coords = sharedMem + threadIdx.x * rank;

    shape::index2coords(tid, zShapeInfo, coords);

    const auto zOffset = shape::getOffset(zShapeInfo, coords);

    int inArrIdx = 0;
    Nd4jLong *xShapeInfo = reinterpret_cast<Nd4jLong**>(pxShapeInfo)[inArrIdx];

    while(coords[axis] >= xShapeInfo[axis + 1]) {
        coords[axis] -= xShapeInfo[axis + 1];
        xShapeInfo = reinterpret_cast<Nd4jLong**>(pxShapeInfo)[++inArrIdx];
    }

    const auto* x      = reinterpret_cast<T*>(reinterpret_cast<void**>(pVx)[inArrIdx]);
    const auto xOffset = shape::getOffset(xShapeInfo, coords);

    z[zOffset] = x[xOffset];
}

///////////////////////////////////////////////////////////////////
template<typename T>
__host__ static void concatCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                        void* pVx, void* pxShapeInfo, void* vz, Nd4jLong* zShapeInfo, const int axis) {

    concatCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(pVx, pxShapeInfo, vz, zShapeInfo, axis);
}
BUILD_SINGLE_TEMPLATE(template void concatCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, void* pVx, void* pxShapeInfo, void* vz, Nd4jLong* zShapeInfo, const int axis), LIBND4J_TYPES);

//////////////////////////////////////////////////////////////////////////
void concat(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray& output, const int axis) {

    const int threadsPerBlock = MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = threadsPerBlock * sizeof(Nd4jLong) * output.rankOf() + 128;

    const int numOfArrs = inArrs.size();

    for(int i = 0; i < numOfArrs; ++i)
        inArrs[i]->syncToDevice();

    output.syncToDevice();

    // prepare arrays of pointers on buffers and shapes
    std::vector<void*> hInBuffers(numOfArrs);
    std::vector<Nd4jLong*> hInShapeInfo(numOfArrs);

    for(int i = 0; i < numOfArrs; ++i) {
        hInBuffers[i]   = inArrs[i]->getSpecialBuffer();
        hInShapeInfo[i] = inArrs[i]->getSpecialShapeInfo();
    }

    PointersManager manager(context, "helpers::concat");

    void* dInBuffers   = manager.replicatePointer(hInBuffers.data(),    hInBuffers.size() * sizeof(void*));
    void* dInShapeInfo = manager.replicatePointer(hInShapeInfo.data(),  hInShapeInfo.size() * sizeof(Nd4jLong*));

    BUILD_SINGLE_SELECTOR(inArrs[0]->dataType(), concatCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), dInBuffers, dInShapeInfo, output.specialBuffer(), output.specialShapeInfo(), axis), LIBND4J_TYPES);

    manager.synchronize();

    for(int i = 0; i < numOfArrs; ++i)
        inArrs[i]->tickReadDevice();

    output.tickWriteDevice();
}

}
}
}