#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/adjust_saturation.h>
#include <ops/declarable/helpers/adjust_hue.h>
#include <helpers/ConstantTadHelper.h>
#include <PointersManager.h>


namespace nd4j    {
namespace ops     {
namespace helpers {


///////////////////////////////////////////////////////////////////
template <typename T>
static void _CUDA_G adjustSaturationCuda(const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                               void* vz, const Nd4jLong *zShapeInfo, const Nd4jLong* zTadOffsets,
                                        const Nd4jLong numOfTads, const T factor, const int dimC) {

    const T* x = reinterpret_cast<const T*>(vx);
          T* z = reinterpret_cast<T*>(vz);

    __shared__ int rank;
    __shared__ Nd4jLong xDimCstride, zDimCstride;

    if (threadIdx.x == 0) {
        rank = shape::rank(xShapeInfo);
        xDimCstride = shape::stride(xShapeInfo)[dimC];
        zDimCstride = shape::stride(zShapeInfo)[dimC];
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (Nd4jLong i = tid; i < numOfTads; i += gridDim.x * blockDim.x) {

        const T* xTad = x + xTadOffsets[i];
              T* zTad = z + zTadOffsets[i];

        T h, s, v;

        rgbToHsv<T>(xTad[0], xTad[xDimCstride], xTad[2 * xDimCstride], h, s, v);

        s *= factor;
        if(s > 1.f)
            s = 1.f;
        else if(s < 0.f)
            s = 0.f;

        hsvToRgb<T>(h, s, v, zTad[0], zTad[zDimCstride], zTad[2 * zDimCstride]);
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
static _CUDA_H void adjustSaturationCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t *stream,
                                          const void* vx, const Nd4jLong* xShapeInfo, const Nd4jLong* xTadOffsets,
                                                void* vz, const Nd4jLong* zShapeInfo, const Nd4jLong* zTadOffsets,
                                          const Nd4jLong numOfTads, const NDArray* factorScalarArr, const int dimC) {

    adjustSaturationCuda<T><<<blocksPerGrid, threadsPerBlock, 256, *stream>>>(vx, xShapeInfo, xTadOffsets, vz, zShapeInfo, zTadOffsets, numOfTads, factorScalarArr->e<T>(0), dimC);
}

////////////////////////////////////////////////////////////////////////
void adjustSaturation(nd4j::LaunchContext* context, const NDArray *input, const NDArray* factorScalarArr, NDArray *output, const int dimC) {

    auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(),  {dimC});
    auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), {dimC});

    const Nd4jLong numOfTads = packX.numberOfTads();

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (numOfTads + threadsPerBlock - 1) / threadsPerBlock;

    PointersManager manager(context, "adjustSaturation");

    NDArray::prepareSpecialUse({output}, {input, factorScalarArr});
    BUILD_SINGLE_SELECTOR(input->dataType(), adjustSaturationCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), packX.platformOffsets(), output->specialBuffer(), output->specialShapeInfo(), packZ.platformOffsets(), numOfTads, factorScalarArr, dimC), FLOAT_TYPES);
    NDArray::registerSpecialUse({output}, {input, factorScalarArr});

    manager.synchronize();
}

/*
template <typename T>
static void _CUDA_G adjustSaturationSingleNHWCKernel(void *xBuffer, Nd4jLong *xShapeInfo,  void *zBuffer, Nd4jLong *zShapeInfo, Nd4jLong tuples, float delta) {
    int numChannels = 3;
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;

    auto bIn = reinterpret_cast<T*>(xBuffer);
    auto bOut = reinterpret_cast<T*>(zBuffer);
    static const int kChannelRange = 6;

    for (Nd4jLong e = tid; e < tuples; e += blockDim.x * gridDim.x) {
        auto i = bIn + e * numChannels;
        auto o = bOut + e * numChannels;

        T h, s, v;
        // Convert the RGB color to Hue/V-range.
        helpers::rgb_to_hsv(i[0], i[1], i[2], &h, &s, &v);
        s = nd4j::math::nd4j_min<T>((T) 1.0f, nd4j::math::nd4j_max<T>((T) 0.0f, s * delta));

        // Convert the hue and v-range back into RGB.
        helpers::hsv_to_rgb(h, s, v, o, o + 1, o + 2);
    }
}

template <typename T>
static void _CUDA_G adjustSaturationSingleNCHWKernel(void *xBuffer, Nd4jLong *xTadShapeInfo, Nd4jLong *xOffsets, void *zBuffer, Nd4jLong *zTadShapeInfo, Nd4jLong *zOffsets, Nd4jLong tadLength, Nd4jLong tuples, float delta) {
    int numChannels = 3;
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;
    static const int kChannelRange = 6;

    auto bufferR = reinterpret_cast<T *>(xBuffer) + xOffsets[0];
    auto bufferG = reinterpret_cast<T *>(xBuffer) + xOffsets[1];
    auto bufferB = reinterpret_cast<T *>(xBuffer) + xOffsets[2];

    auto outputR = reinterpret_cast<T *>(zBuffer) + zOffsets[0];
    auto outputG = reinterpret_cast<T *>(zBuffer) + zOffsets[1];
    auto outputB = reinterpret_cast<T *>(zBuffer) + zOffsets[2];

    for (Nd4jLong e = tid; e < tuples; e += blockDim.x * gridDim.x) {
        auto _ri = bufferR + shape::getIndexOffset(e, xTadShapeInfo);
        auto _gi = bufferG + shape::getIndexOffset(e, xTadShapeInfo);
        auto _bi = bufferB + shape::getIndexOffset(e, xTadShapeInfo);

        auto _ro = outputR + shape::getIndexOffset(e, xTadShapeInfo);
        auto _go = outputG + shape::getIndexOffset(e, xTadShapeInfo);
        auto _bo = outputB + shape::getIndexOffset(e, xTadShapeInfo);

        T h, s, v;
        // Convert the RGB color to Hue/V-range.
        helpers::rgb_to_hsv(_ri[0], _gi[0], _bi[0], &h, &s, &v);
        s = nd4j::math::nd4j_min<T>((T) 1.0f, nd4j::math::nd4j_max<T>((T) 0.0f, s * delta));
        // Convert the hue and v-range back into RGB.
        helpers::hsv_to_rgb(h, s, v, _ro, _go, _bo);
    }
}

template <typename T>
static void _adjust_saturation_single(nd4j::LaunchContext * context, NDArray *array, NDArray *output, float delta, bool isNHWC) {
    // numChannels is always 3
    auto tuples = array->lengthOf() / 3;

    if (isNHWC) {
        adjustSaturationSingleNHWCKernel<T><<<256, 256, 1024, *context->getCudaStream()>>>(array->specialBuffer(), array->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), tuples, delta);
    } else {
        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(array->getShapeInfo(), {1, 2});
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), {1, 2});

        auto tadLength = shape::length(packX.primaryShapeInfo());

        adjustSaturationSingleNCHWKernel<T><<<256, 256, 1024, *context->getCudaStream()>>>(array->specialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), output->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), tadLength, tuples, delta);
    }
}

template <typename T>
static void _adjust_saturation_batch(nd4j::LaunchContext * context, NDArray *array, NDArray *output, float delta, bool isNHWC) {
    auto xType = array->dataType();

    // numChannels is always 3
    auto tuples = array->lengthOf() / 3;

    if (isNHWC) {
        // in case of nhwc batch, we don't really care about examples: it's still bunch of RGB values
        BUILD_SINGLE_SELECTOR(xType, _adjust_saturation_single, (context, array, output, delta, isNHWC);, FLOAT_TYPES);
    } else {
        // TODO: check this one
        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(array->getShapeInfo(), {0, 2, 3});
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), {0, 2, 3});

        auto tadLength = shape::length(packX.primaryShapeInfo());

        adjustSaturationSingleNCHWKernel<T><<<256, 256, 1024, *context->getCudaStream()>>>(array->specialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), output->specialBuffer(), packZ.platformShapeInfo(), packZ.platformOffsets(), tadLength, tuples, delta);
    }
}

void adjust_saturation(nd4j::LaunchContext * context, NDArray *array, NDArray *output, NDArray* delta, bool isNHWC) {
    auto xType = array->dataType();

    float d = delta->e<float>(0);
    if (array->rankOf() == 4) {
        BUILD_SINGLE_SELECTOR(xType, _adjust_saturation_batch, (context, array, output, d, isNHWC);, FLOAT_TYPES);
    } else {
        BUILD_SINGLE_SELECTOR(xType, _adjust_saturation_single, (context, array, output, d, isNHWC);, FLOAT_TYPES);
    }
}
*/

}
}
}
