#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include <ops/declarable/helpers/helpers.h>
#include <ops/declarable/helpers/hamming.h>

namespace nd4j {
    namespace ops {
        namespace helpers {
            template <typename X, typename Z>
            static _CUDA_G void _hammingKernel(void *vx, Nd4jLong *xShapeInfo, void *vy, Nd4jLong *yShapeInfo, void *vz, void *reductionBuffer, Nd4jLong length) {
                auto x = reinterpret_cast<X*>(vx);
                auto y = reinterpret_cast<X*>(vy);
                auto z = reinterpret_cast<Z*>(vz);

                __shared__ Nd4jLong *shared;

                if (threadIdx.x == 0) {
                    extern __shared__ unsigned char shmem[];
                    shared = reinterpret_cast<Nd4jLong*>(shmem);
                }
                __syncthreads();

                // we want to nullify temporary memory before accumulating intermediate results
                shared[threadIdx.x] = 0;

                auto tid = threadIdx.x + blockIdx.x * blockDim.x;
                for (Nd4jLong e = tid; e < length; e += blockDim.x * gridDim.x) {
                    auto _x = static_cast<unsigned long long>(x[shape::getIndexOffset(e, xShapeInfo)]);
                    auto _y = static_cast<unsigned long long>(y[shape::getIndexOffset(e, yShapeInfo)]);

                    // we save intermediate result into shared memory
                    shared[threadIdx.x] += __popcll(_x ^ _y);
                }
                __syncthreads();

                // now we accumulate values
                auto numItems = nd4j::math::nd4j_min<Nd4jLong>(blockDim.x, length);
                auto floorPow2 = numItems;
                if (floorPow2 & (floorPow2 - 1)) {

                    while (floorPow2 & (floorPow2 - 1))
                        floorPow2 &= floorPow2 - 1;

                    if (threadIdx.x >= floorPow2)
                        shared[threadIdx.x - floorPow2] = shared[threadIdx.x - floorPow2] + shared[threadIdx.x];

                    __syncthreads();
                }
                __syncthreads();

                for (Nd4jLong activeThreads = floorPow2 >> 1; activeThreads; activeThreads >>= 1) {
                    if (threadIdx.x < activeThreads && threadIdx.x + activeThreads < numItems)
                        shared[threadIdx.x] = shared[threadIdx.x] + shared[threadIdx.x + activeThreads];

                    __syncthreads();
                }
                __syncthreads();

                // FIXME: do we really want atomicAdd on global memory here
                // and store them to output
                if (threadIdx.x == 0 && shared[0] > 0)
                    nd4j::math::atomics::nd4j_atomicAdd<Z>(&z[0], static_cast<Z>(shared[threadIdx.x]));
            }

            template <typename X, typename Z>
            static void _hamming(LaunchContext *context, NDArray &x, NDArray &y, NDArray &z) {
                _hammingKernel<X, Z><<<256, 256, 256 * sizeof(Nd4jLong) + 256, *context->getCudaStream()>>>(x.specialBuffer(), x.specialShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), z.specialBuffer(), nullptr, x.lengthOf());
            }

            void hamming(LaunchContext *context, NDArray &x, NDArray &y, NDArray &output) {
                NDArray::prepareSpecialUse({&output}, {&x, &y});
                BUILD_DOUBLE_SELECTOR(x.dataType(), output.dataType(), _hamming, (context, x, y, output), INTEGER_TYPES, INDEXING_TYPES);
                NDArray::registerSpecialUse({&output}, {&x, &y});
            }
        }
    }
}