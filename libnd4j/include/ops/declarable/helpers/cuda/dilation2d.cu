#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/dilation2d.h>
#include <array/DataTypeUtils.h>
#include <PointersManager.h>

namespace nd4j    {
namespace ops 	  {
namespace helpers {

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
__global__ static void dilation2dCuda(const void* vx, const Nd4jLong* xShapeInfo,
									  const void* vy, const Nd4jLong* yShapeInfo,
									  		void* vz, const Nd4jLong* zShapeInfo,
									  const int sH, const int sW,
									  const int pH, const int pW,
									  const int dH, const int dW) {

	// x [bS, iH, iW, iC]
	// y [kH, kW, iC]
    // z [bS, oH, oW, iC]

    const X* x = reinterpret_cast<const X*>(vx);
    const X* y = reinterpret_cast<const X*>(vy);
          Z* z = reinterpret_cast<Z*>(vz);

    __shared__ int xzRank, yRank;
    __shared__ uint iH, iW, kH, kW;
    __shared__ Nd4jLong *sharedMem, zLen;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        zLen = shape::length(zShapeInfo);

        xzRank = shape::rank(xShapeInfo);
        yRank  = shape::rank(yShapeInfo);

        iH = xShapeInfo[2];
        iW = xShapeInfo[3];

        kH = yShapeInfo[1];
        kW = yShapeInfo[2];
    }
    __syncthreads();

    const auto zInd = threadIdx.x + blockIdx.x * blockDim.x;

    if(zInd >= zLen)
        return;

    auto xzCoords = sharedMem + threadIdx.x * (xzRank + yRank);
    auto yCoords  = xzCoords + xzRank;

    shape::index2coords(zInd, zShapeInfo, xzCoords);

    const auto zOffset = shape::getOffset(zShapeInfo, xzCoords);

    yCoords[2] = xzCoords[3];		// iC coordinate is same for x, y and z

    const auto oh = xzCoords[1];
    const auto ow = xzCoords[2];

    X max = -DataTypeUtils::max<X>();

	for (yCoords[0] = 0; yCoords[0] < kH; ++yCoords[0]) {
    	xzCoords[1] = oh * sH - pH + yCoords[0] * dH;
        if (xzCoords[1] < 0 || xzCoords[1] >= iH) continue;

        for (yCoords[1] = 0; yCoords[1] < kW; ++yCoords[1]) {
        	xzCoords[2] = ow * sW - pW + yCoords[1] * dW;
            if(xzCoords[2] < 0 || xzCoords[2] >= iW) continue;

            const X val = x[shape::getOffset(xShapeInfo, xzCoords)] + y[shape::getOffset(yShapeInfo, yCoords)];
            if (val > max)
            	max = val;
		}
	}

	z[zOffset] = static_cast<Z>(max);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Z>
static void dilation2dCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                   const void* vx, const Nd4jLong* xShapeInfo,
                                   const void* vy, const Nd4jLong* yShapeInfo,
                                         void* vz, const Nd4jLong* zShapeInfo,
                                   const int sH, const int sW,
								   const int pH, const int pW,
								   const int dH, const int dW) {

    dilation2dCuda<X,Z><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, sH, sW, pH, pW, dH, dW);
}

void dilation2d(nd4j::LaunchContext* context, NDArray *input, NDArray *weights, NDArray *output, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW) {

   	PointersManager manager(context, "dilation2d");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (output->lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = (weights->rankOf() + output->rankOf()) * sizeof(Nd4jLong) * threadsPerBlock  + 128;

    NDArray::prepareSpecialUse({output}, {input, weights});
    BUILD_SINGLE_SELECTOR_TWICE(input->dataType(), dilation2dCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), weights->getSpecialBuffer(), weights->getSpecialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), sH, sW, pH, pW, dH, dW), FLOAT_TYPES);
    NDArray::registerSpecialUse({output}, {input, weights});

    manager.synchronize();
}


}
}
}
