#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma, created on 16.04.2018
//

#include <ops/declarable/helpers/reverse.h>
#include <helpers/ShapeUtils.h>
#include <array/ResultSet.h>
#include <TAD.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>


namespace nd4j    {
namespace ops     {
namespace helpers {

    template <typename T>
    static __global__ void reverseArrayKernel(void* input, Nd4jLong *inputShape, void* output, Nd4jLong *outputShape, Nd4jLong numOfElemsToReverse) {
        const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;
        __shared__ int linearStatus;
        __shared__ T* inputArr;
        __shared__ T* outputArr;
        __shared__ char inputOrder, outputOrder;

        if (threadIdx.x == 0) {
            linearStatus = (shape::elementWiseStride(inputShape) == shape::elementWiseStride(outputShape)) && (inputOrder == outputOrder)? shape::elementWiseStride(inputShape):0;

            char inputOrder = shape::order(inputShape);
            char outputOrder = shape::order(outputShape);
            inputArr = reinterpret_cast<T*>(input);
            outputArr = reinterpret_cast<T*>(output);
        }
        __syncthreads();

        auto odd = numOfElemsToReverse % 2 != 0;
        auto limit = numOfElemsToReverse / 2;

        for (Nd4jLong e = tid; e < limit; e += step) {
            // we're calculating offsets within input array
            auto fOffset = shape::getIndexOffset(e, inputShape);
            auto lOffset = shape::getIndexOffset(numOfElemsToReverse - e - 1, inputShape);

            // now we're storing input values
            auto v1 = inputArr[fOffset];
            auto v2 = inputArr[lOffset];

            // now we're calculating offsets within output array
            auto zfOffset = shape::getIndexOffset(e, outputShape);
            auto zlOffset = shape::getIndexOffset(numOfElemsToReverse - e - 1, outputShape);

            // and saving values to output arrays
            outputArr[zfOffset] = v2;
            outputArr[zlOffset] = v1;
        }

        // in case of odd array we'll have to move middle value
        if (odd && tid == 0) {
            auto xOffset = shape::getIndexOffset(limit, inputShape);
            auto zOffset = shape::getIndexOffset(limit, outputShape);

            outputArr[zOffset] = inputArr[xOffset];
        }
    }

    template<typename T>
    static void reverseArray(nd4j::LaunchContext * context, NDArray* input, NDArray* output, Nd4jLong numOfElemsToReverse) {
        auto stream = context->getCudaStream();
        Nd4jLong numOfReverse = numOfElemsToReverse;
        if (numOfElemsToReverse == 0)
            numOfReverse = input->lengthOf();

        reverseArrayKernel<T><<<256, 512, 8192, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), numOfReverse);
    }


    ///////////////////////////////////////////////////////////////////
    template <typename T>
    static void reverseSequence_(nd4j::LaunchContext * context, const NDArray* input, const NDArray* seqLengths, NDArray* output, int seqDim, const int batchDim){
        int posOfNonUnityDim = -1;
        seqLengths->syncToHost();
        auto stream = context->getCudaStream();

        if(input->isVector() || shape::isLikeVector(input->getShapeInfo(), posOfNonUnityDim) || seqLengths->lengthOf() == 1) {
            int numOfElemsToReverse = seqLengths->e<int>(0);
            if((seqDim == 0 && input->sizeAt(0) == 1) || (batchDim == posOfNonUnityDim))
                output->assign(input);
            else
                reverseArrayKernel<T><<<256, 512, 8192, *stream>>>(input->getSpecialBuffer(), input->getSpecialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), numOfElemsToReverse);//helpers::reverseArray<T>(context, const_cast<NDArray*>(input), output, numOfElemsToReverse);
        }
        else {

            if(seqDim > batchDim)
                --seqDim;

            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {batchDim});

            auto inSubArrsSet  = input->allTensorsAlongDimension(dimensions);
            auto outSubArrsSet = output->allTensorsAlongDimension(dimensions);

            for(int i = 0; i < inSubArrsSet->size(); ++i) {

                int numOfElemsToReverse = seqLengths->e<int>(i);

                if(numOfElemsToReverse == 0 || numOfElemsToReverse == 1) {
                    outSubArrsSet->at(i)->assign(inSubArrsSet->at(i));
                }
                else {
                    auto inInnerSet  = inSubArrsSet->at(i)->allTensorsAlongDimension({seqDim});
                    auto outInnerSet = outSubArrsSet->at(i)->allTensorsAlongDimension({seqDim});
                    for(int j = 0; j < inInnerSet->size(); ++j)
                        reverseArray<T>(context, inInnerSet->at(j), outInnerSet->at(j), numOfElemsToReverse);

                    delete inInnerSet;
                    delete outInnerSet;
                }
            }
            delete inSubArrsSet;
            delete outSubArrsSet;
        }

    }

    void reverseSequence(nd4j::LaunchContext * context, const NDArray* input, const NDArray* seqLengths, NDArray* output, int seqDim, const int batchDim) {
        NDArray::prepareSpecialUse({output}, {input, seqLengths});

        // if op isn't inplace - copy original data into output array
        if (output->getSpecialBuffer() != input->getSpecialBuffer())
            output->assign(input);

        BUILD_SINGLE_SELECTOR(input->dataType(), reverseSequence_, (context, input, seqLengths, output, seqDim, batchDim), LIBND4J_TYPES);
        NDArray::registerSpecialUse({output}, {input, seqLengths});
    }

    //////////////////////////////////////////////////////////////////////////
    void reverse(nd4j::LaunchContext * context, const NDArray* input, NDArray* output, const std::vector<int>* intArgs, bool isBackProp) {
        // we need to reverse axis only if that's new op
        std::vector<int> dimensions = isBackProp ? ShapeUtils::evalDimsToExclude(input->rankOf(), *intArgs) : *intArgs;
        std::vector<int> axis = ShapeUtils::evalDimsToExclude(input->rankOf(), dimensions);
        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), axis);
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), axis);

        auto listOut = output->allTensorsAlongDimension(dimensions);
        auto listIn  = input->allTensorsAlongDimension(dimensions);

        NDArray *subArrIn, *subArrOut;

        NDArray::prepareSpecialUse({output}, {input});
        for(int i = 0; i < listIn->size(); ++i) {               // listIn->size() = listOut->size()
            subArrIn   = listIn->at(i);
            subArrOut  = listOut->at(i);
            BUILD_SINGLE_SELECTOR(input->dataType(), reverseArray, (context, subArrIn, subArrOut, 0), LIBND4J_TYPES);
        }
        //BUILD_SINGLE_SELECTOR(input->dataType(), reverseArray, (context, const_cast<NDArray*>(input), output, (int)0), LIBND4J_TYPES);
        NDArray::registerSpecialUse({output}, {input});
        delete listOut;
        delete listIn;
    }

BUILD_SINGLE_TEMPLATE(template void reverseArray, (nd4j::LaunchContext * context, NDArray *inArr, NDArray *outArr, Nd4jLong numOfElemsToReverse), LIBND4J_TYPES);

}
}
}

