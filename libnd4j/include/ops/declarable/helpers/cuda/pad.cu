#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j {
    namespace ops {
        namespace helpers {
///////////////////////////////////////////////////////////////////
// x - input, y - paddings, z - output
            template<typename X, typename Y>
            __global__ static void padCuda(const int mode,
                                           const void *vx, const Nd4jLong *xShapeInfo,
                                           const void *vy, const Nd4jLong *yShapeInfo,
                                           void *vz, const Nd4jLong *zShapeInfo,
                                           const void *vPadVal) {

                const X padVal = *reinterpret_cast<const X*>(vPadVal);

                const auto x = reinterpret_cast<const X*>(vx);
                const auto y = reinterpret_cast<const Y*>(vy);
                auto z = reinterpret_cast<X*>(vz);

                __shared__ int rank, rankMinusOne;
                __shared__ Nd4jLong zLen, totalThreads, *coords, *xShape, *zShape, shift1, shift2, yStride0;

                if (threadIdx.x == 0) {
                    extern __shared__ unsigned char shmem[];
                    coords    = reinterpret_cast<Nd4jLong*>(shmem);
                    zLen     = shape::length(zShapeInfo);
                    xShape   = shape::shapeOf(const_cast<Nd4jLong*>(xShapeInfo));
                    zShape   = shape::shapeOf(const_cast<Nd4jLong*>(zShapeInfo));
                    yStride0 = shape::stride(const_cast<Nd4jLong*>(yShapeInfo))[0];
                    rank     = shape::rank(xShapeInfo);
                    zLen     = shape::length(zShapeInfo);
                    rankMinusOne = rank - 1;
                    totalThreads = gridDim.x * blockDim.x;
                    shift1 = mode == 1 ? 0 : 1;         // REFLECT : SYMMETRIC
                    shift2 = mode == 1 ? 2 : 1;         // REFLECT : SYMMETRIC
                }

                __syncthreads();

                auto xzCoord = coords + threadIdx.x * rank;       // we use xzCoord storage both for x and z arrays

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

                if(mode == 0) { // CONSTANT case

                    for (Nd4jLong i = tid; i < zLen; i += totalThreads) {

                        shape::index2coords(i, zShapeInfo, xzCoord);
                        const auto zOffset = shape::getOffset(zShapeInfo, xzCoord);

                        bool within = true;
                        for(int j = rankMinusOne; j >= 0; --j) {
                            if(xShape[j] == zShape[j]) continue;
                            const auto left = y[shape::getIndexOffset(yStride0 * j, yShapeInfo)];
                            if(xzCoord[j] < left || xzCoord[j] >= left + xShape[j]) {within = false; break;}
                            else                                                    {xzCoord[j] = xzCoord[j] - left;}
                        }

                        if(within)
                            z[zOffset] = x[shape::getOffset(xShapeInfo, xzCoord)];
                        else
                            z[zOffset] = padVal;
                    }
                }
                else {  // REFLECT and SYMMETRIC cases

                    for (Nd4jLong i = tid; i < zLen; i += totalThreads) {

                        shape::index2coords(i, zShapeInfo, xzCoord);
                        const auto zOffset = shape::getOffset(zShapeInfo, xzCoord);

                        for(int j = rankMinusOne; j >= 0; --j) {

                            if(xShape[j] == zShape[j]) continue;
                            xzCoord[j] = xzCoord[j] - y[shape::getIndexOffset(yStride0 * j, yShapeInfo)];    // are ready to fill middle (within input dimension range)
                            if(xzCoord[j] < 0)               xzCoord[j] = -xzCoord[j] - shift1;                // means fill from left
                            else if(xzCoord[j] >= xShape[j]) xzCoord[j] = 2 * xShape[j] - xzCoord[j] - shift2; // means fill from right
                        }

                        const auto xOffset = shape::getOffset(xShapeInfo, xzCoord);
                        z[zOffset] = x[xOffset];
                    }
                }
            }

///////////////////////////////////////////////////////////////////
            template<typename X, typename Y>
            static void padCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                        const int mode,
                                        const void *vx, const Nd4jLong *xShapeInfo,
                                        const void *vy, const Nd4jLong *yShapeInfo,
                                        void *vz, const Nd4jLong *zShapeInfo,
                                        const void* padVal) {

                padCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(mode, vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, padVal);
            }

///////////////////////////////////////////////////////////////////
            void pad(nd4j::LaunchContext * context, const int mode, const NDArray& input, const NDArray& paddings, NDArray& output, const NDArray& padValue) {

                PointersManager manager(context, "pad");

                NDArray::prepareSpecialUse({&output}, {&input, &paddings, &padValue});

                const int threadsPerBlock = MAX_NUM_THREADS / 4;
                const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
                const int sharedMem = 8 * threadsPerBlock * output.rankOf() + 128;

                const auto xType = input.dataType();
                const auto yType = paddings.dataType();

                BUILD_DOUBLE_SELECTOR(xType, yType, padCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), mode, input.getSpecialBuffer(), input.getSpecialShapeInfo(), paddings.getSpecialBuffer(), paddings.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), padValue.getSpecialBuffer()), LIBND4J_TYPES, INDEXING_TYPES);

                NDArray::registerSpecialUse({&output}, {&input, &paddings, &padValue});
                manager.synchronize();
            }


            ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
            template <typename T>
            static __global__ void mirrorPadLinearKernel(void const* vx, Nd4jLong* xShape, void* vz, Nd4jLong* zShape, Nd4jLong leftSide, Nd4jLong leftSideCorrected, Nd4jLong xLen, Nd4jLong len, Nd4jLong zLen) {

                __shared__ T const* x;
                __shared__ T* z;
                if (threadIdx.x == 0) {
                    x = reinterpret_cast<T const*>(vx);
                    z = reinterpret_cast<T*>(vz);
                }
                __syncthreads();
                auto start = blockIdx.x * blockDim.x + threadIdx.x;
                auto step = blockDim.x * gridDim.x;

                for(int i = start; i < zLen; i+= step) {
                    auto zIndex = shape::getIndexOffset(i, zShape);
                    auto xIndex = shape::getIndexOffset(len - i, xShape);

                    if (i < leftSide)                                   // left side
                        xIndex = shape::getIndexOffset(leftSideCorrected - i, xShape);

                    else if(i >= leftSide && i < leftSide + xLen)       // middle
                        xIndex = shape::getIndexOffset(i - leftSide, xShape);

//            else                                                // right side
//                z[i] = x[len - i];
                    z[zIndex] = x[xIndex];
                }

            }

            template <typename F, typename I>
            static __global__ void mirrorPadKernel(void const* vx, Nd4jLong* xShape, void* vz, Nd4jLong* zShape, Nd4jLong outLen, void const* paddings, Nd4jLong* paddingShape, int reflBorder) {

                __shared__ F const* x;
                __shared__ I const* pads;
                __shared__ F* z;
                __shared__ Nd4jLong zRank, rank;
                __shared__ Nd4jLong* xIdx;
                if (threadIdx.x == 0) {
                    extern __shared__ unsigned char shmem[];
                    xIdx    = reinterpret_cast<Nd4jLong*>(shmem);
                    rank = shape::rank(xShape);

                    x = reinterpret_cast<F const*>(vx);//
                    pads = reinterpret_cast<I const*>(paddings);
                    z = reinterpret_cast<F*>(vz);
                }
                __syncthreads();
                auto start = threadIdx.x + blockIdx.x * blockDim.x;
                auto step = blockDim.x * gridDim.x;

                for(Nd4jLong i = start; i < outLen; i+= step) {
                    auto xzCoord = xIdx + threadIdx.x * rank;
                    //auto zxCoord = xIdx + (threadIdx.x + threadIdx.x % 2 + 1) * rank;

                    shape::index2coords(i, zShape, xzCoord);
                    auto outOffset = shape::getOffset(zShape, xzCoord);
//                auto intStep = blockDim.y * gridDim.y;
                    for(int j = 0; j < rank; j++) {

                        const Nd4jLong inLen         = shape::sizeAt(xShape, j);
                        Nd4jLong coords[2] = {j, 0};
                        auto padOffset = shape::getOffset(paddingShape, coords); // padding already has rank 2
                        const auto leftSide          = pads[padOffset];
                        const auto leftSideCorrected = leftSide - reflBorder;
                        const Nd4jLong len           = 2 * (inLen - 1) + leftSide + reflBorder;

                        if(xzCoord[j] < leftSide)                                        // left side
                            xzCoord[j] = leftSideCorrected - xzCoord[j];

                        else if(xzCoord[j] >= leftSide && xzCoord[j] < leftSide + inLen)  // middle
                            xzCoord[j] = xzCoord[j] - leftSide;

                        else if (len > xzCoord[j])                                                           // right side
                            xzCoord[j] = len - xzCoord[j];
                        else
                            xzCoord[j] = xzCoord[j] - len;
                    }

                    auto inOffset  = shape::getOffset(xShape, xzCoord);
                    z[outOffset] = x[inOffset];
                }
            }

            template<typename F, typename I>
            static void mirrorPad_(nd4j::LaunchContext * context, const NDArray& input, const NDArray& paddings, NDArray& output, const int mode) {
                // mode:  0 - REFLECT, else - SYMMETRIC
                const int reflBorder = (bool)mode ? 1 : 0;
                const int rank        = input.rankOf();
                const Nd4jLong outLen = output.lengthOf();
                auto stream = context->getCudaStream();
                NDArray::prepareSpecialUse({&output}, {&input, &paddings});

                if(rank <= 1) {

                    const Nd4jLong inLen         = input.lengthOf();
                    const auto leftSide          = paddings.e<Nd4jLong>(0);
                    const auto leftSideCorrected = leftSide - reflBorder;
                    const Nd4jLong len           = 2*(inLen-1) + leftSide + reflBorder;

                    mirrorPadLinearKernel<F><<<256, 512, 256, *stream>>>(input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), leftSide, leftSideCorrected, inLen, len, outLen);
                    nd4j::DebugHelper::checkErrorCode(stream, "helpers::mirrorPadLinearKernel(...) failed");
                }
                else {
                    mirrorPadKernel<F, I><<<256, 256, 8192, *stream>>>(input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), outLen, paddings.getSpecialBuffer(), paddings.getSpecialShapeInfo(), reflBorder);
                    nd4j::DebugHelper::checkErrorCode(stream, "helpers::mirrorPadKernel(...) failed");
                }
                NDArray::registerSpecialUse({&output}, {&input, &paddings});
            }

            void mirrorPad(nd4j::LaunchContext * context, const NDArray& input, const NDArray& paddings, NDArray& output, const int mode) {
                BUILD_DOUBLE_SELECTOR(input.dataType(), paddings.dataType(), mirrorPad_, (context, input, paddings, output, mode), LIBND4J_TYPES, INDEXING_TYPES);
            }


        }
    }
}