#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by raver119 on 19.01.18.
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <ops/declarable/helpers/s_t_b.h>
#include <PointersManager.h>

namespace nd4j    {
namespace ops     {
namespace helpers {


///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void batchToSpaceCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint cropBottom, const uint cropLeft) {

    // input [bS, H * blockSize, W * blockSize, iC]
    // output [bS, H * blockSize - cropBottom - cropTop, W * blockSize - cropLeft - cropRight, iC]

    // if (cropTop = cropBottom = cropRight = cropLeft = 0) shapes are the same
    // else:
    // oH -> [cropBottom, iH - cropTop]
    // oW -> [cropLeft,   iH - cropRight]
    // xLen >= zLen

    const auto x = reinterpret_cast<const T*>(vx);
          auto z = reinterpret_cast<T*>(vz);

    __shared__ int rank;
    __shared__ Nd4jLong zLen, *sharedMem;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        rank  = shape::rank(zShapeInfo);
        zLen  = shape::length(zShapeInfo);
    }
    __syncthreads();

    auto coords = sharedMem + threadIdx.x * rank;

    const auto i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= zLen)
        return;

    shape::index2coords(i, zShapeInfo, coords);

    const auto zOffset = shape::getOffset(zShapeInfo, coords);

    coords[1] += cropBottom;
    coords[2] += cropLeft;

    const auto xOffset = shape::getOffset(xShapeInfo, coords);

    z[zOffset] = x[xOffset];

}

///////////////////////////////////////////////////////////////////
template<typename T>
static void batchToSpaceCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint cropBottom, const uint cropLeft) {

    batchToSpaceCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, cropBottom, cropLeft);
}
BUILD_SINGLE_TEMPLATE(template void batchToSpaceCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint cropBottom, const uint cropLeft), LIBND4J_TYPES);

///////////////////////////////////////////////////////////////////
void batchToSpace(nd4j::LaunchContext* context, const NDArray& input, NDArray& output, const uint cropBottom, const uint cropTop, const uint cropLeft, const uint cropRight, const uint blockSize) {

    // [bS*blockSize*blockSize, H/blockSize, W/blockSize, iC] is rearranged/permuted to [bS, oH, oW, iC]
    // oH = H - cropTop  - cropBottom
    // oW = W - cropLeft - cropRight

    NDArray inputRearranged0 = input.reshape(input.ordering(), {blockSize, blockSize, output.sizeAt(0), input.sizeAt(1), input.sizeAt(2), input.sizeAt(3)});
    inputRearranged0.permutei({2, 3,0, 4,1, 5});

    if(input.lengthOf() == output.lengthOf()) {

        output.assign(inputRearranged0);
    }
    else {

        NDArray inputRearranged1 = inputRearranged0.reshape(input.ordering(), {output.sizeAt(0), input.sizeAt(1) * blockSize, input.sizeAt(2) * blockSize, input.sizeAt(3)});

        const int threadsPerBlock = MAX_NUM_THREADS / 2;
        const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
        const int sharedMem = threadsPerBlock * sizeof(Nd4jLong) * output.rankOf() + 128;

        PointersManager manager(context, "batchToSpace");

        NDArray::prepareSpecialUse({&output}, {&inputRearranged1});
        BUILD_SINGLE_SELECTOR(input.dataType(), batchToSpaceCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), inputRearranged1.getSpecialBuffer(), inputRearranged1.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), cropBottom, cropLeft), LIBND4J_TYPES);
        NDArray::registerSpecialUse({&output}, {&inputRearranged1});

        manager.synchronize();
    }
}



///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__global__ static void batchToSpaceNDCuda(const void* vx, const Nd4jLong* xShapeInfo,
                                          const void* vy, const Nd4jLong* yShapeInfo,
                                                void* vz, const Nd4jLong* zShapeInfo,
                                          const uint numOfSpatialDims) {

    // 4D example, numOfSpatialDims = 2
    // input [bS, H * blockShape[0], W * blockShape[1], iC]
    // output [bS, H * blockShape[0] - cropBottom - cropTop, W * blockShape[1] - cropLeft - cropRight, iC]

    // if (cropTop = cropBottom = cropRight = cropLeft = 0) shapes are the same
    // else:
    // oH -> [cropBottom, iH - cropTop]
    // oW -> [cropLeft,   iH - cropRight]
    // xLen >= zLen

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<X*>(vz);

    __shared__ int rank;
    __shared__ Nd4jLong zLen, *sharedMem;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        rank  = shape::rank(zShapeInfo);
        zLen  = shape::length(zShapeInfo);
    }

    __syncthreads();

    auto coords = sharedMem + threadIdx.x * rank;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < zLen; i += gridDim.x * blockDim.x) {

        shape::index2coords(i, zShapeInfo, coords);

        const auto zOffset = shape::getOffset(zShapeInfo, coords);

        // evaluate spatial coordinates for x
        for(uint j = 1; j <= numOfSpatialDims; ++j) {
            const auto yOffset  = (j - 1) * yShapeInfo[3];  // yRank = 2, calculate offset manually
            coords[j] += y[yOffset];                        // add crop left
        }

        const auto xOffset = shape::getOffset(xShapeInfo, coords);

        z[zOffset] = x[xOffset];
    }
}

///////////////////////////////////////////////////////////////////
template<typename X,typename Y>
static void batchToSpaceNDCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint numOfSpatialDims) {

    batchToSpaceNDCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, numOfSpatialDims);
}
BUILD_DOUBLE_TEMPLATE(template void batchToSpaceNDCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint numOfSpatialDims), LIBND4J_TYPES, INTEGER_TYPES);

//////////////////////////////////////////////////////////////////////////
void batchToSpaceND(nd4j::LaunchContext* context, const NDArray& input, const NDArray& blockShape, const NDArray& crop, NDArray& output) {

    // 4D example, numOfSpatialDims = 2 - two spatial dimensions
    // [bS*blockShape[0]*blockShape[1], iH, iW, iC] is rearranged/permuted to [bS, iH*blockShape[0] - cropTop  - cropBottom, iW*blockShape[1] - cropLeft - cropRight, iC]

    const uint rank = input.rankOf();
    const uint numOfSpatialDims = blockShape.sizeAt(0);

    //*** construct reshaping std::vector for first reshape of input array ***//

    std::vector<Nd4jLong> temp(numOfSpatialDims + rank);

    int i;
    for(i = 0; i < numOfSpatialDims; ++i)
        temp[i] = blockShape.e<Nd4jLong>(i);
    temp[i++] = output.sizeAt(0);
    for(int j = 1; j < rank; ++i, ++j)
        temp[i] = input.sizeAt(j);

    NDArray inputRearranged0 = input.reshape(input.ordering(), temp);

    //*** construct permuting std::vector for permutation of input array ***//

    temp[0] = numOfSpatialDims;

    for(i = 1; i <= numOfSpatialDims; ++i) {
        temp[2*i - 1] = numOfSpatialDims + i;
        temp[2*i]     = i - 1;
    }
    for(i = 2 * numOfSpatialDims + 1; i < temp.size(); ++i)
        temp[i] = i;

    inputRearranged0.permutei(temp);


    if(input.lengthOf() == output.lengthOf()) {

        output.assign(inputRearranged0);
    }
    else {
        //*** construct reshaping std::vector for second reshape of input array ***//

        temp.resize(rank);

        temp[0] = output.sizeAt(0);

        for(i = 1; i < rank; ++i)
            temp[i] = (i <= numOfSpatialDims) ? input.sizeAt(i) * blockShape.e<Nd4jLong>(i - 1) : input.sizeAt(i);

        NDArray inputRearranged1 = inputRearranged0.reshape(input.ordering(), temp);

        const int threadsPerBlock = MAX_NUM_THREADS / 4;
        const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
        const int sharedMem = threadsPerBlock * sizeof(Nd4jLong) * output.rankOf() + 128;

        PointersManager manager(context, "batchToSpaceND");

        NDArray::prepareSpecialUse({&output}, {&inputRearranged1, &crop});
        BUILD_DOUBLE_SELECTOR(input.dataType(), crop.dataType(), batchToSpaceNDCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), inputRearranged1.getSpecialBuffer(), inputRearranged1.getSpecialShapeInfo(), crop.getSpecialBuffer(), crop.getSpecialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), numOfSpatialDims), LIBND4J_TYPES, INTEGER_TYPES);
        NDArray::registerSpecialUse({&output}, {&inputRearranged1, &crop});

        manager.synchronize();
    }
}



///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void spaceToBatchCuda(const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint padBottom, const uint padTop, const uint padLeft, const uint padRight) {

    // input [bS, H * blockSize - padBottom - padTop, W * blockSize - padLeft - padRight, iC]
    // output [bs, H * blockSize, W * blockSize, iC]

    // if (padTop = padBottom = padRight = padLeft = 0) shapes are the same
    // else:
    // iH -> [padBottom, oH - padTop]
    // iW -> [padLeft,   oW - padRight]
    // zLen > xLen

    const auto x = reinterpret_cast<const T*>(vx);
          auto z = reinterpret_cast<T*>(vz);

    __shared__ int rank;
    __shared__ Nd4jLong zLen, *sharedMem;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        rank  = shape::rank(zShapeInfo);
        zLen  = shape::length(zShapeInfo);
    }
    __syncthreads();

    auto coords = sharedMem + threadIdx.x * rank;

    const auto i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= zLen)
        return;

    shape::index2coords(i, zShapeInfo, coords);

    const auto zOffset = shape::getOffset(zShapeInfo, coords);

    if(coords[1] >= padBottom && coords[1] < zShapeInfo[2] - padTop && coords[2] >= padLeft && coords[2] < zShapeInfo[3] - padRight) {

        coords[1] -= padBottom;
        coords[2] -= padLeft;

        const auto xOffset = shape::getOffset(xShapeInfo, coords);

        z[zOffset] = x[xOffset];
    }
    else
        z[zOffset] = 0.f;
}

///////////////////////////////////////////////////////////////////
template<typename T>
static void spaceToBatchCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint padBottom, const uint padTop, const uint padLeft, const uint padRight) {

    spaceToBatchCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vz, zShapeInfo, padBottom, padTop, padLeft, padRight);
}
BUILD_SINGLE_TEMPLATE(template void spaceToBatchCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint padBottom, const uint padTop, const uint padLeft, const uint padRight), LIBND4J_TYPES);

///////////////////////////////////////////////////////////////////
void spaceToBatch(nd4j::LaunchContext* context, const NDArray& input, NDArray& output, const uint padBottom, const uint padTop, const uint padLeft, const uint padRight, const uint blockSize) {

    // [bS, iH, iW, iC] is rearranged/permuted to [bS*blockSize*blockSize, (iH + padBottom + padTop)/blockSize, (iW + padLeft + padRight)/blockSize, iC]

    NDArray outputRearranged0 = output.reshape(output.ordering(), {blockSize, blockSize, input.sizeAt(0), output.sizeAt(1), output.sizeAt(2), input.sizeAt(3)});
    outputRearranged0.permutei({2, 3,0, 4,1, 5});

    if(input.lengthOf() == output.lengthOf()) {

        outputRearranged0.assign(input);
    }
    else {

        NDArray outputRearranged1 = outputRearranged0.reshape(output.ordering(), {input.sizeAt(0), output.sizeAt(1) * blockSize, output.sizeAt(2) * blockSize, input.sizeAt(3)});

        const int threadsPerBlock = MAX_NUM_THREADS / 2;
        const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
        const int sharedMem = threadsPerBlock * sizeof(Nd4jLong) * output.rankOf() + 128;

        PointersManager manager(context, "spaceToBatch");

        NDArray::prepareSpecialUse({&outputRearranged1}, {&input});
        BUILD_SINGLE_SELECTOR(input.dataType(), spaceToBatchCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), outputRearranged1.specialBuffer(), outputRearranged1.specialShapeInfo(), padBottom, padTop, padLeft, padRight), LIBND4J_TYPES);
        NDArray::registerSpecialUse({&outputRearranged1}, {&input});

        manager.synchronize();

        if(output.getSpecialBuffer() != outputRearranged1.getSpecialBuffer())
            outputRearranged0.assign(outputRearranged1);
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__global__ static void spaceToBatchNDCuda(const void* vx, const Nd4jLong* xShapeInfo,
                                          const void* vy, const Nd4jLong* yShapeInfo,
                                                void* vz, const Nd4jLong* zShapeInfo,
                                          const uint numOfSpatialDims) {

    // x - input, y - padding, z - output

    // 4D example
    // input [bS, H * blockShape[0] - padBottom - padTop, W * blockShape[1] - padLeft - padRight, iC]
    // output [bS, H * blockShape[0], W * blockShape[1], iC]

    // if (padTop = padBottom = padRight = padLeft = 0) shapes are the same
    // else:
    // iH -> [padBottom, oH - padTop]
    // iW -> [padLeft,   oW - padRight]
    // zLen > xLen

    const auto x = reinterpret_cast<const X*>(vx);
    const auto y = reinterpret_cast<const Y*>(vy);
          auto z = reinterpret_cast<X*>(vz);

    __shared__ int rank;    // xRank = zRank, yRank = 2;
    __shared__ Nd4jLong zLen, totalThreads, *sharedMem;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        rank  = shape::rank(zShapeInfo);
        zLen  = shape::length(zShapeInfo);
        totalThreads = gridDim.x * blockDim.x;
    }

    __syncthreads();

    auto coords = sharedMem + threadIdx.x * rank;

    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < zLen; i += totalThreads) {

        shape::index2coords(i, zShapeInfo, coords);

        const auto zOffset = shape::getOffset(zShapeInfo, coords);

        bool within = true;

        for(uint j = 1; j <= numOfSpatialDims; ++j) {

            // yRank = 2, calculate offset manually
            const auto yOffset  = (j - 1) * yShapeInfo[3];
            const auto padLeft  = y[yOffset];
            const auto padRight = y[yOffset + yShapeInfo[4]];

            within &= (coords[j] >= padLeft && coords[j] < shape::shapeOf(const_cast<Nd4jLong*>(zShapeInfo))[j] - padRight);

            if(!within)
                break;

            coords[j] -= padLeft;       // get coordinates for x
        }

        if(within)
            z[zOffset] = x[shape::getOffset(xShapeInfo, coords)];
        else
            z[zOffset] = 0.f;
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
static void spaceToBatchNDCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint numOfSpatialDims) {

    spaceToBatchNDCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, numOfSpatialDims);
}
BUILD_DOUBLE_TEMPLATE(template void spaceToBatchNDCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,  const void* vx, const Nd4jLong* xShapeInfo, const void* vy, const Nd4jLong* yShapeInfo, void* vz, const Nd4jLong* zShapeInfo, const uint numOfSpatialDims), LIBND4J_TYPES, INTEGER_TYPES);

//////////////////////////////////////////////////////////////////////////
void spaceToBatchND(nd4j::LaunchContext* context, const NDArray& input, const NDArray& blockShape, const NDArray& padding, NDArray& output ) {

    // 4D example with two spatial dimensions
    // [bS, iH, iW, iC] is rearranged/permuted to [bS*blockShape[0]*blockShape[1], (iH + padBottom + padTop)/blockShape[0], (iW + padLeft + padRight)/blockShape[1], iC]

    const uint rank = input.rankOf();

    const uint numOfSpatialDims = blockShape.sizeAt(0);

    //*** construct reshaping std::vector for first reshape of output array ***//
    std::vector<Nd4jLong> temp(numOfSpatialDims + rank);

    int i;
    for(i = 0; i < numOfSpatialDims; ++i)
        temp[i] = blockShape.e<Nd4jLong>(i);
    temp[i++] = input.sizeAt(0);
    for(int j = 1; j < rank; ++i, ++j)
        temp[i] = output.sizeAt(j);

    NDArray outputRearranged0 = output.reshape(output.ordering(), temp);

    //*** construct permuting std::vector for permutation of output array ***//

    temp[0] = numOfSpatialDims;

    for(i = 1; i <= numOfSpatialDims; ++i) {
        temp[2*i - 1] = numOfSpatialDims + i;
        temp[2*i]     = i - 1;
    }
    for(i = 2 * numOfSpatialDims + 1; i < temp.size(); ++i)
        temp[i] = i;

    outputRearranged0.permutei(temp);

    // ****** //

    if(input.lengthOf() == output.lengthOf()) {
        outputRearranged0.assign(input);
    }
    else {

        //*** construct reshaping std::vector for second reshape of output array ***//
        temp.resize(rank);

        temp[0] = input.sizeAt(0);

        for(i = 1; i < rank; ++i)
            temp[i] = (i <= numOfSpatialDims) ? output.sizeAt(i) * blockShape.e<Nd4jLong>(i - 1) : output.sizeAt(i);

        NDArray outputRearranged1 = outputRearranged0.reshape(output.ordering(), temp);

        const int threadsPerBlock = MAX_NUM_THREADS / 4;
        const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
        const int sharedMem = threadsPerBlock * sizeof(Nd4jLong) * output.rankOf() + 128;

        PointersManager manager(context, "spaceToBatchND");

        NDArray::prepareSpecialUse({&outputRearranged1}, {&input, &padding});
        BUILD_DOUBLE_SELECTOR(input.dataType(), padding.dataType(), spaceToBatchNDCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), padding.getSpecialBuffer(), padding.getSpecialShapeInfo(), outputRearranged1.specialBuffer(), outputRearranged1.specialShapeInfo(), numOfSpatialDims), LIBND4J_TYPES, INTEGER_TYPES);
        NDArray::registerSpecialUse({&outputRearranged1}, {&input, &padding});

        manager.synchronize();

        if(output.getSpecialBuffer() != outputRearranged1.getSpecialBuffer())
            outputRearranged0.assign(outputRearranged1);
    }
}


/*
    template <int N, bool B2S>
    struct SpaceToBatchHelper {
        template <typename T>
        static void run(T *ptrSpace, const Nd4jLong *space_shape, const Nd4jLong *space_strides, const Nd4jLong *block_shape, const Nd4jLong *pad_start, const Nd4jLong *block_offsets, T *ptrBatch, const Nd4jLong *batch_shape, const Nd4jLong *batch_strides) {
            for (int batch_pos = 0; batch_pos < batch_shape[0]; ++batch_pos) {
                const int space_pos = batch_pos * block_shape[0] + block_offsets[0] - pad_start[0];
                if (space_pos >= 0 && space_pos < space_shape[0]) {
                    SpaceToBatchHelper<N - 1, B2S>::run(ptrSpace + space_pos * space_strides[0], space_shape + 1, space_strides + 1, block_shape + 1, pad_start + 1, block_offsets + 1, ptrBatch, batch_shape + 1, batch_strides + 1);
                } else {
                    if (!B2S)
                        for (int i = 0; i < batch_strides[0]; i++)
                            ptrBatch[i] = (T) 0.f;
                }

                ptrBatch += batch_strides[0];
            }
        }
    };

    template <bool B2S>
    struct SpaceToBatchHelper<0, B2S> {
        template <typename T>
        static void run(T *ptrSpace, const Nd4jLong *space_shape, const Nd4jLong *space_strides, const Nd4jLong *block_shape, const Nd4jLong *pad_start, const Nd4jLong *block_offsets, T *ptrBatch, const Nd4jLong *batch_shape, const Nd4jLong *batch_strides) {
            int str = batch_strides[-1];
            for (int i = 0; i < str; i++)
                if (B2S)
                    ptrSpace[i] = ptrBatch[i];
                else
                    ptrBatch[i] = ptrSpace[i];
        }
    };

    template <typename T, int NUM_BLOCK_DIMS, bool B2S>
    void _execute(nd4j::LaunchContext * context, void *vptrSpace, const Nd4jLong *space_shape, const Nd4jLong *space_strides, const Nd4jLong *block_shape, const Nd4jLong *pad_start, const Nd4jLong *block_offsets, void *vptrBatch, const Nd4jLong *batch_shape, const Nd4jLong *batch_strides) {
        auto ptrSpace = reinterpret_cast<T *>(vptrSpace);
        auto ptrBatch = reinterpret_cast<T *>(vptrBatch);
        SpaceToBatchHelper<NUM_BLOCK_DIMS, B2S>::run(ptrSpace, space_shape, space_strides, block_shape, pad_start, block_offsets, ptrBatch, batch_shape, batch_strides);
    };

    Nd4jStatus _batchToSpace(nd4j::LaunchContext * context, int internal_block_dims, NDArray *input, NDArray *output, std::vector<Nd4jLong> &internal_input_shape, std::vector<Nd4jLong> &internal_output_shape, Nd4jLong *block_shape, Nd4jLong *crops) {

        return Status::OK();
    }

#define STB_DIM (0, 1),\
                (1, 2),\
                (2, 3),\
                (3, 4)

#define STB_BOOL (0, false),\
                 (1, true)

    BUILD_TRIPLE_TEMPLATE(template void _execute, (nd4j::LaunchContext * context, void *ptrSpace, const Nd4jLong *space_shape, const Nd4jLong *space_strides, const Nd4jLong *block_shape, const Nd4jLong *pad_start, const Nd4jLong *block_offsets, void *ptrBatch, const Nd4jLong *batch_shape, const Nd4jLong *batch_strides), LIBND4J_TYPES, STB_DIM, STB_BOOL);

#undef STB_BOOL
#undef STB_DIM
*/

}
}
}