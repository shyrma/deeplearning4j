#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j {
    namespace ops {
        namespace helpers {
            ///////////////////////////////////////////////////////////////////
            template<typename T>
            __global__ static void scatterUpdateCuda(const int opCode, const int numOfInd,
                                                     void* vx, const Nd4jLong *xShapeInfo, const Nd4jLong *xOffsets,
                                                     void* vy, const Nd4jLong *yShapeInfo, const Nd4jLong *yOffsets,
                                                     const int* indexes) {

                __shared__ T *x, *y;
                __shared__ Nd4jLong arrLenX, arrLenY;

                for (int e = 0; e < numOfInd; e++ ) {

                    const auto xIndex = indexes[e];
                    const bool isOwner = xIndex < gridDim.x ? blockIdx.x == xIndex : blockIdx.x == xIndex % gridDim.x;

                    if (!isOwner)
                        continue;

                    if (threadIdx.x == 0) {
                        x = reinterpret_cast<T*>(vx) + xOffsets[xIndex];
                        y = reinterpret_cast<T*>(vy) + yOffsets[e];
                        arrLenX = shape::length(xShapeInfo);
                        arrLenY = shape::length(yShapeInfo);
                    }
                    __syncthreads();

                    if (arrLenX != arrLenY)
                        return;

                    for (Nd4jLong i = threadIdx.x; i < arrLenX; i += blockDim.x) {

                        const auto xOffset = shape::getIndexOffset(i, xShapeInfo);
                        const auto yOffset = shape::getIndexOffset(i, yShapeInfo);

                        switch (opCode) {
                            case 0:
                                x[xOffset] += y[yOffset];
                                break;
                            case 1:
                                x[xOffset] -= y[yOffset];
                                break;
                            case 2:
                                x[xOffset] *= y[yOffset];
                                break;
                            case 3:
                                x[xOffset] /= y[yOffset];
                                break;
                            case 4:
                                x[xOffset] = y[yOffset] - x[xOffset];
                                break;
                            case 5:
                                x[xOffset] = y[yOffset] / x[xOffset];
                                break;
                            case 6:
                                x[xOffset] = y[yOffset];
                                break;
                            default:
                                continue;
                        }
                    }
                    __syncthreads();
                }
            }

            template<typename T>
            __host__ static void scatterUpdateCudaLauncher(const hipStream_t* stream, const int opCode, const int numOfInd, void* vx, const Nd4jLong *xShapeInfo, const Nd4jLong *xOffsets, void* vy, const Nd4jLong *yShapeInfo, const Nd4jLong *yOffsets, const int* indexes) {

                scatterUpdateCuda<T><<<512, 256, MAX_NUM_THREADS, *stream>>>(opCode, numOfInd, vx, xShapeInfo, xOffsets, vy, yShapeInfo, yOffsets, indexes);
            }


//////////////////////////////////////////////////////////////////////////
            void scatterUpdate(nd4j::LaunchContext* context, NDArray& input, NDArray& updates, const std::vector<int>* intArgs) {

                const int opCode    = (*intArgs)[0];
                const int numOfDims = (*intArgs)[1];
                const int numOfInd  = (*intArgs)[2 + numOfDims];

                std::vector<int> tadDimensions(numOfDims);
                for (int e = 2; e < 2 + numOfDims; e++)
                    tadDimensions[e-2] = (*intArgs)[e];

                auto packX = ConstantTadHelper::getInstance()->tadForDimensions(input.getShapeInfo(), tadDimensions);
                auto packY = ConstantTadHelper::getInstance()->tadForDimensions(updates.getShapeInfo(), tadDimensions);

                NDArray indices(const_cast<int*>(intArgs->data()) + numOfDims + 3, 'c', {numOfInd}, nd4j::DataType::INT32, context);

                PointersManager manager(context, "scatterUpdate");

                NDArray::prepareSpecialUse({&input}, {&input, &updates, &indices});
                BUILD_SINGLE_SELECTOR(input.dataType(), scatterUpdateCudaLauncher, (context->getCudaStream(), opCode, numOfInd, input.specialBuffer(), packX.platformShapeInfo(), packX.platformOffsets(), updates.specialBuffer(), packY.platformShapeInfo(), packY.platformOffsets(), reinterpret_cast<int*>(indices.getSpecialBuffer())), LIBND4J_TYPES);
                NDArray::registerSpecialUse({&input}, {&input, &updates, &indices});

                manager.synchronize();
            }
        }
    }
}