#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/confusion.h>
#include <cuda_exception.h>
#include <TAD.h>
#include <PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    __global__ static void copyBuffers(Nd4jLong* destination, void const* source, Nd4jLong bufferLength) {
        const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;
        for (int t = tid; t < bufferLength; t += step) {
            destination[t] = static_cast<Nd4jLong>(reinterpret_cast<T const*>(source)[t]);
        }
    }

    template <typename T>
    __global__ static void confusionFunctorKernel(Nd4jLong* labelsBuffer, Nd4jLong* predictionBuffer, Nd4jLong bufferLength, void const* weightsBuffer, void* outputBuffer, Nd4jLong* tadShape, Nd4jLong* tadOffsets) {
        __shared__ int arrIdx, blocksPerArr;
        __shared__ T *z;
        __shared__ T const* w;
        __shared__ Nd4jLong *zShapeInfo, *xShapeInfo, arrLen;

        if (threadIdx.x == 0) {
            z = reinterpret_cast<T*>(outputBuffer);
            w = reinterpret_cast<T const*>(weightsBuffer);
            arrLen = shape::length(tadShape);
        }
        __syncthreads();

        const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;
        for (int t = tid; t < bufferLength; t += step) {
            auto label = labelsBuffer[t]; //->e<Nd4jLong>(j);
            auto pred = predictionBuffer[t]; //->e<Nd4jLong>(j);
            auto tZ = z + tadOffsets[label];
            T val = (weightsBuffer == nullptr ? (T)1.0f : w[t]);

            auto idx = shape::getIndexOffset(pred, tadShape);
            tZ[idx] = val;
        }
    }

    template <typename X, typename Z>
    void _confusionFunctor(nd4j::LaunchContext * context, NDArray* labels, NDArray* predictions, NDArray* weights, NDArray* output) {
        auto stream = context->getCudaStream();

        auto pack = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->shapeInfo(), 1);

        PointersManager manager(context, "helpers::confusion");

        Nd4jLong* labelsLongBuffer = labels->dataType() == nd4j::DataType::INT64?(Nd4jLong*)labels->specialBuffer():nullptr;
        Nd4jLong* predictionLongBuffer = predictions->dataType() == nd4j::DataType::INT64?(Nd4jLong*)predictions->specialBuffer():nullptr;

        if (labelsLongBuffer == nullptr) {
            auto err = hipMalloc(&labelsLongBuffer, labels->lengthOf() * sizeof(Nd4jLong));
            if (err != 0)
                throw nd4j::cuda_exception::build("Cannot allocate memory for labels long buffer", err);
            // copy with type conversion
            copyBuffers<X><<<256, 512, 1024, *stream>>>(labelsLongBuffer, labels->getSpecialBuffer(), labels->lengthOf());
        }

        if (predictionLongBuffer == nullptr) {
            auto err = hipMalloc(&predictionLongBuffer, predictions->lengthOf() * sizeof(Nd4jLong));
            if (err != 0)
                throw nd4j::cuda_exception::build("Cannot allocate memory for predictions long buffer", err);
            // copy with type conversion
            copyBuffers<X><<<256, 512, 1024, *stream>>>(predictionLongBuffer, predictions->getSpecialBuffer(), predictions->lengthOf());
        }

        auto bufferLength = labels->lengthOf();
        dim3 launchDims(32, 32, 1024);
        confusionFunctorKernel<Z><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(labelsLongBuffer, predictionLongBuffer, bufferLength, weights != nullptr? weights->getSpecialBuffer():nullptr, output->specialBuffer(), pack.specialShapeInfo(), pack.specialOffsets());

        manager.synchronize();

        if (predictionLongBuffer != predictions->getSpecialBuffer()) {
            hipError_t err = hipFree(predictionLongBuffer);
            if (err != 0)
                throw nd4j::cuda_exception::build("Cannot deallocate memory for predictions long buffer", err);
        }

        if (labelsLongBuffer != labels->getSpecialBuffer()) {
            hipError_t err = hipFree(labelsLongBuffer);
            if (err != 0)
                throw nd4j::cuda_exception::build("Cannot deallocate memory for labels long buffer", err);
        }
    }

    void confusionFunctor(nd4j::LaunchContext * context, NDArray* labels, NDArray* predictions, NDArray* weights, NDArray* output) {
        auto xType = predictions->dataType();
        auto zType = output->dataType(); // weights can be null
        NDArray::prepareSpecialUse({output}, {labels, predictions, weights});
        BUILD_DOUBLE_SELECTOR(xType, zType, _confusionFunctor, (context, labels, predictions, weights, output), INDEXING_TYPES, NUMERIC_TYPES);
        NDArray::registerSpecialUse({output}, {labels, predictions, weights});
    }
}
}
}