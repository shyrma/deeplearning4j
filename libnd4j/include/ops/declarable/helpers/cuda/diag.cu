#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by GS <sgazeos@gmail.com> on 4/6/2018.
//

#include "ResultSet.h"
#include <ops/declarable/helpers/diag.h>

namespace nd4j {
namespace ops {
namespace helpers {
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// diag functor cuda kernel
// outputBuffer - output tensor buffer
// outputShape - output tensor shape
// inputBuffer - input tensor buffer - this tensor should be placed on diagonal position of output
// inputShape - input tensor shape
// inputLength - length for input tensor
//
template <typename T>
static __global__ void diagFunctorKernel(void* outputBuffer, Nd4jLong* outputShape, void const* inputBuffer, Nd4jLong* inputShape, Nd4jLong inputLength) {
    __shared__ T *z;
    __shared__ T const* x;
    __shared__ Nd4jLong outputLength;

    if (threadIdx.x == 0) {
        z = reinterpret_cast<T*>(outputBuffer);
        x = reinterpret_cast<T const*>(inputBuffer);

        outputLength = shape::length(outputShape);
    }
    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    const auto step = gridDim.x * blockDim.x;

    for (int t = tid; t < inputLength; t += step) { // for all vals in input, put all on diagonal position to output
        z[shape::getIndexOffset(t * (inputLength + 1), outputShape)] = x[shape::getIndexOffset(t, inputShape)]; //tX];
    }

}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// diag part functor cuda kernel
// outputBuffer - output tensor buffer - linear sequence of diagonal values
// outputShape - output tensor shape
// inputBuffer - input tensor buffer - this tensor should be placed on diagonal position of output
// inputShape - input tensor shape
// outputLength - given length of output
// inputLength - given length for input tensor
//
    template <typename T>
    static __global__ void diagPartFunctorKernel(void* outputBuffer, Nd4jLong* outputShape, void const* inputBuffer, Nd4jLong* inputShape, Nd4jLong outputLength, Nd4jLong inputLength) {
        __shared__ T *z;
        __shared__ T const* x;

        if (threadIdx.x == 0) {
            z = reinterpret_cast<T*>(outputBuffer);
            x = reinterpret_cast<T const*>(inputBuffer);

        }
        __syncthreads();

        const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;
        Nd4jLong i = threadIdx.x * (outputLength + 1); // pos to diagonal value
        for (int t = tid; t < outputLength && i < inputLength; t += step) { // loop by output, but input matrix may not be square
            // put diagonal val from input onto output
            z[shape::getIndexOffset(t, outputShape)] = x[shape::getIndexOffset(i, inputShape)]; 
            i += outputLength + 1; // shift to next diagonal value
        }
    }

//////////////////////////////////////////////////////////////////////////
// Returns a batched matrix tensor with new batched diagonal values.
// for detailed explanations please take a look on web page: https://www.tensorflow.org/api_docs/python/tf/matrix_set_diag
    template <typename T>
    static void _diagFunctor(nd4j::LaunchContext * context, const NDArray* input, NDArray* output) {
        auto stream = context->getCudaStream();
        auto inputLength = input->lengthOf();
        dim3 launchDims(256, 512, 8192);
        if (!input->isActualOnDeviceSide())
            input->syncToDevice();
        diagFunctorKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(output->specialBuffer(), output->specialShapeInfo(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), inputLength);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// diagFunctor - caller for diag functor processor
    void diagFunctor(nd4j::LaunchContext * context, const NDArray* input, NDArray* output) {
        auto xType = input->dataType();

        BUILD_SINGLE_SELECTOR(xType, _diagFunctor, (context, input, output), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void _diagFunctor, (nd4j::LaunchContext * context, const NDArray* input, NDArray* output);, LIBND4J_TYPES);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// diagPartFunctor - caller for diag part functor kernel
    template <typename T>
    void _diagPartFunctor(nd4j::LaunchContext * context, NDArray const* input, NDArray* output) {
        const int outLen = output->lengthOf();
        const int inLen = input->lengthOf();
        auto stream = context->getCudaStream();

        dim3 launchDims(256, 512, 8192);
        if (!input->isActualOnDeviceSide())
            input->syncToDevice();

        diagPartFunctorKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(output->specialBuffer(), output->specialShapeInfo(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), outLen, inLen);
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// diagPartFunctor - caller for diag part functor processor
    void diagPartFunctor(nd4j::LaunchContext * context, NDArray const* input, NDArray* output) {
        auto zType = output->dataType();
        BUILD_SINGLE_SELECTOR(zType, _diagPartFunctor, (context, input, output), NUMERIC_TYPES);

    }

}
}
}