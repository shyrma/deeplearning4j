#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j {
    namespace ops {
        namespace helpers {
            ///////////////////////////////////////////////////////////////////
// x - input, y - indices, z - output
            template<typename X, typename Y>
            __global__ static void gatherNDCuda(const void *vx, const Nd4jLong *xShapeInfo,
                                                const void *vy, const Nd4jLong *yShapeInfo,
                                                void *vz, const Nd4jLong *zShapeInfo) {

                const auto x = reinterpret_cast<const X*>(vx);
                const auto y = reinterpret_cast<const Y*>(vy);
                auto z = reinterpret_cast<X*>(vz);

                __shared__ int xRank, yRank, zRank, maxRank, yLastDim;
                __shared__ Nd4jLong zLen, totalThreads, *sharedMem;

                if (threadIdx.x == 0) {
                    extern __shared__ unsigned char shmem[];
                    sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

                    xRank   = shape::rank(xShapeInfo);
                    yRank   = shape::rank(yShapeInfo);
                    zRank   = shape::rank(zShapeInfo);
                    maxRank = nd4j::math::nd4j_max<int>(yRank, nd4j::math::nd4j_max<int>(xRank, zRank));

                    zLen     = shape::length(zShapeInfo);
                    yLastDim = yShapeInfo[yRank];

                    totalThreads = gridDim.x * blockDim.x;
                }
                __syncthreads();

                auto coord = sharedMem + threadIdx.x * maxRank;

                Nd4jLong *zCoordStart, *xCoordStart;

                if(yLastDim == xRank) {
                    zCoordStart = coord;
                    xCoordStart = coord;
                }
                if(zRank >= xRank) {
                    zCoordStart = coord;
                    xCoordStart = coord + zRank - xRank;
                }
                else {
                    zCoordStart = coord + xRank - zRank;
                    xCoordStart = coord;
                }

                const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

                for (Nd4jLong i = tid; i < zLen; i += totalThreads) {

                    shape::index2coords(i, zShapeInfo, zCoordStart);

                    const auto zOffset = shape::getOffset(zShapeInfo, zCoordStart);

                    // last y coordinate
                    int coordToRestore;
                    if(yLastDim != xRank)
                        coordToRestore = static_cast<int>(zCoordStart[yRank - 1]);

                    zCoordStart[yRank - 1] = 0; // last y coordinate
                    const auto yOffset = shape::getOffset(yShapeInfo, zCoordStart);

                    //restore z coordinate
                    if(yLastDim != xRank)
                        zCoordStart[yRank - 1] = coordToRestore;

                    // construct coordinates for x
                    for(uint j = 0; j < yLastDim; ++j)
                        xCoordStart[j] = y[yOffset + j * yShapeInfo[2 * yRank]];   // last stride

                    const auto xOffset = shape::getOffset(xShapeInfo, xCoordStart);

                    z[zOffset] = x[xOffset];
                    printf("z[%lld] = x[%lld] = %f\n", zOffset, xOffset, (float) z[zOffset]);
                }
            }

///////////////////////////////////////////////////////////////////
            template<typename X, typename Y>
            static void gatherNDCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                             const void *vx, const Nd4jLong *xShapeInfo,
                                             const void *vy, const Nd4jLong *yShapeInfo,
                                             void *vz, const Nd4jLong *zShapeInfo) {

                gatherNDCuda<X,Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo);
            }

///////////////////////////////////////////////////////////////////
            void gatherND(nd4j::LaunchContext * context, NDArray& input, NDArray& indices, NDArray& output) {

                const int maxRank = nd4j::math::nd4j_max<int>(indices.rankOf(), nd4j::math::nd4j_max<int>(input.rankOf(), output.rankOf()));

                const int threadsPerBlock = 256;
                const int blocksPerGrid = (output.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
                const int sharedMem = 8 * threadsPerBlock * maxRank + 128;

                const auto xType = input.dataType();
                const auto yType = indices.dataType();

                PointersManager manager(context, "gatherND");

                NDArray::prepareSpecialUse({&output}, {&input, &indices});
                BUILD_DOUBLE_SELECTOR(xType, yType, gatherNDCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), indices.getSpecialBuffer(), indices.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo()), LIBND4J_TYPES, INDEXING_TYPES);
                NDArray::registerSpecialUse({&output}, {&input, &indices});

                manager.synchronize();
            }
        }
    }
}