/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/flatten.h>
#include <helpers/PointersManager.h>

namespace nd4j {
    namespace ops {
        namespace helpers {
            template <typename T>
            void _CUDA_G flattenKernel(void **xBuffers, Nd4jLong **xShapeInfos, Nd4jLong *offsets, Nd4jLong numInputs, void *zBuffer, Nd4jLong *zShapeInfo, char order) {

                Nd4jLong xCoord[MAX_RANK];

                // each block of threads works on 1 input array
                for (Nd4jLong e = blockIdx.x; e < numInputs; e += gridDim.x) {
                    auto z = reinterpret_cast<T*>(zBuffer) + offsets[e];

                    auto xBuffer = reinterpret_cast<T*>(xBuffers[e]);
                    auto xShapeInfo = xShapeInfos[e];
                    auto xLength = shape::length(xShapeInfo);

                    // each element of this input array has own place within common output array
                    for (uint i = threadIdx.x; i < xLength; i += blockDim.x)
                        z[i] = xBuffer[getIndexOffsetOrdered(i, xShapeInfo, order)];
                }
            }

            template <typename T>
            void flatten_(nd4j::LaunchContext *context, std::vector<NDArray*> &inputs, NDArray *output, char order) {
                PointersManager pm(context, "flatten");

                std::vector<void*> hdBuffers(inputs.size());
                std::vector<Nd4jLong> hOffsets(inputs.size());
                std::vector<Nd4jLong *> hdShapes(inputs.size());
                Nd4jLong cOffset = 0;

                // calculating offsets in output
                for (int e = 0; e < inputs.size(); e++) {
                    hOffsets[e] = cOffset;
                    cOffset += inputs[e]->lengthOf();

                    hdBuffers[e] = inputs[e]->specialBuffer();
                    hdShapes[e] = inputs[e]->specialShapeInfo();
                }

                // copying pointers to device
                auto dBuffers = (void **) pm.replicatePointer(hdBuffers.data(), inputs.size() * sizeof(void*));
                auto dShapes = (Nd4jLong **)pm.replicatePointer(hdShapes.data(), inputs.size() * sizeof(Nd4jLong*));
                auto dOffsets = (Nd4jLong *) pm.replicatePointer(hOffsets.data(), inputs.size() * sizeof(Nd4jLong));


                flattenKernel<T><<<256, 512, 8192, *context->getCudaStream()>>>(dBuffers, dShapes, dOffsets, inputs.size(), output->getSpecialBuffer(), output->getSpecialShapeInfo(), order);

                pm.synchronize();
            }

            void flatten(nd4j::LaunchContext *context, std::vector<NDArray*> &inputs, NDArray *output, char order) {
                // FIXME: we want NDArrayFactory::prepareSpecialUse here eventually
                for (auto v:inputs)
                    v->syncToDevice();

                BUILD_SINGLE_SELECTOR(output->dataType(), flatten_, (context, inputs, output, order), LIBND4J_TYPES);
                NDArray::registerSpecialUse({output}, {});
            }
        }
    }
}