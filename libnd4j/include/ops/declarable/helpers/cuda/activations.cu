#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 19.04.2018
// @author raver119@gmail.com
//

#include <op_boilerplate.h>
#include <ops/declarable/helpers/activations.h>
#include <ShapeUtils.h>
#include <numeric>
#include <PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace nd4j    {
namespace ops     {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__global__ void preluCuda(const void *vx, const Nd4jLong *xShapeInfo,
		   			 	  const void *vy, const Nd4jLong *yShapeInfo,
						        void *vz) {

	const auto x = reinterpret_cast<const X*>(vx);
	const auto y = reinterpret_cast<const Y*>(vy);
		  auto z = reinterpret_cast<X*>(vz);

	__shared__ Nd4jLong xzLen, totalThreads, *sharedMem;
	__shared__ int xzRank, yRank;

	if (threadIdx.x == 0) {
		extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

		xzLen = shape::length(xShapeInfo);
		totalThreads = gridDim.x * blockDim.x;

		xzRank = shape::rank(xShapeInfo);
		yRank  = shape::rank(yShapeInfo);
	}
	__syncthreads();

	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	Nd4jLong* coords = sharedMem + threadIdx.x * xzRank;

	for (int i = tid; i < xzLen; i += totalThreads) {

    	shape::index2coords(i, xShapeInfo, coords);

		const auto xzOffset = shape::getOffset(xShapeInfo, coords);

		const auto xVal = x[xzOffset];

		if(xVal < 0) {

			for (uint j = 0; j < yRank; ++j)
				if(yShapeInfo[j + 1] == 1)
					coords[j + 1] = 0;

			z[xzOffset] = xVal * y[shape::getOffset(yShapeInfo, coords + 1)];
		}
		else
			z[xzOffset] = xVal;
	}
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
linkage void preluCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const void *vx, const Nd4jLong *xShapeInfo, const void *vy, const Nd4jLong *yShapeInfo, void *vz) {

	preluCuda<X, Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz);
}

///////////////////////////////////////////////////////////////////
void prelu(nd4j::LaunchContext * context, const NDArray& input, const NDArray& alpha, NDArray& output) {

	PointersManager manager(context, "prelu");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = input.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;

	const auto xType = input.dataType();
	const auto yType = alpha.dataType();

	NDArray::prepareSpecialUse({&output}, {&input, &alpha});
	BUILD_SINGLE_SELECTOR_TWICE(xType, preluCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), alpha.getSpecialBuffer(), alpha.getSpecialShapeInfo(), output.getSpecialBuffer()), FLOAT_TYPES);
	NDArray::registerSpecialUse({&output}, {&input, &alpha});

	manager.synchronize();
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__global__ linkage void preluBPCuda(const void *vIn,    const Nd4jLong *inShapeInfo,
								   const void *vAlpha, const Nd4jLong *alphaShapeInfo,
								   const void *vdLdO,  const Nd4jLong *dLdOShapeInfo,
										 void *vdLdI,  const Nd4jLong *dLdIShapeInfo,
										 void *vdLdA,  const Nd4jLong *dLdAShapeInfo) {

	const auto in    = reinterpret_cast<const X*>(vIn);
	const auto alpha = reinterpret_cast<const Y*>(vAlpha);
	const auto dLdO  = reinterpret_cast<const Y*>(vdLdO);
		  auto dLdI  = reinterpret_cast<Y*>(vdLdI);
		  auto dLdA  = reinterpret_cast<Y*>(vdLdA);

	__shared__ Nd4jLong inLen, totalThreads, *sharedMem;
	__shared__ int inRank, alphaRank;

	if (threadIdx.x == 0) {
		extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

		inLen = shape::length(inShapeInfo);
		totalThreads = gridDim.x * blockDim.x;

		inRank     = shape::rank(inShapeInfo);
		alphaRank  = shape::rank(alphaShapeInfo);
	}
	__syncthreads();

	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	Nd4jLong* coords = sharedMem + threadIdx.x * inRank;

	for (int i = tid; i < inLen; i += totalThreads) {

    	shape::index2coords(i, inShapeInfo, coords);

		const auto inOffset   = shape::getOffset(inShapeInfo, coords);
		const auto dLdOOffset = shape::getOffset(dLdOShapeInfo, coords);
		const auto dLdIOffset = shape::getOffset(dLdIShapeInfo, coords);

		const auto xVal = in[inOffset];
		const auto grO  = dLdO[dLdOOffset];

		if(xVal < 0) {

			for (uint j = 0; j < alphaRank; ++j)
				if(alphaShapeInfo[j + 1] == 1)
					coords[j + 1] = 0;

			const auto alphaOffset = shape::getOffset(alphaShapeInfo, coords + 1);
			const auto dLdAOffset  = shape::getOffset(dLdAShapeInfo, coords + 1);

			dLdI[dLdIOffset] =  grO * alpha[alphaOffset];

			nd4j::math::atomics::nd4j_atomicAdd<Y>(&dLdA[dLdAOffset], static_cast<Y>(grO * xVal));
		}
		else
			dLdI[dLdIOffset] = grO;
	}
}

//////////////////////////////////////////////////////////////////////////
template<typename X, typename Y>
__host__ linkage void preluBPCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream, const void *vIn, const Nd4jLong *inShapeInfo, const void *vAlpha, const Nd4jLong *alphaShapeInfo, const void *vdLdO,  const Nd4jLong *dLdOShapeInfo, void *vdLdI,  const Nd4jLong *dLdIShapeInfo, void *vdLdA,  const Nd4jLong *dLdAShapeInfo) {

	preluBPCuda<X, Y><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vIn, inShapeInfo, vAlpha, alphaShapeInfo, vdLdO, dLdOShapeInfo, vdLdI, dLdIShapeInfo, vdLdA, dLdAShapeInfo);
}

//////////////////////////////////////////////////////////////////////////
void preluBP(nd4j::LaunchContext* context, const NDArray& input, const NDArray& alpha, const NDArray& dLdO, NDArray& dLdI, NDArray& dLdA) {

	dLdA.nullify();

	PointersManager manager(context, "preluBP");

    const int threadsPerBlock = MAX_NUM_THREADS / 2;
    const int blocksPerGrid = (input.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = input.rankOf() * sizeof(Nd4jLong) * threadsPerBlock + 128;

	const auto xType = input.dataType();
	const auto zType = alpha.dataType();

	NDArray::prepareSpecialUse({&dLdI, &dLdA}, {&input, &alpha, &dLdO});
	BUILD_SINGLE_SELECTOR_TWICE(xType, preluBPCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), alpha.getSpecialBuffer(), alpha.getSpecialShapeInfo(), dLdO.getSpecialBuffer(),  dLdO.getSpecialShapeInfo(), dLdI.getSpecialBuffer(), dLdI.getSpecialShapeInfo(), dLdA.getSpecialBuffer(), dLdA.getSpecialShapeInfo()), FLOAT_TYPES);
	NDArray::registerSpecialUse({&dLdI, &dLdA}, {&input, &alpha, &dLdO});

	manager.synchronize();
}


///////////////////////////////////////////////////////////////////
template<typename T>
__device__ void softMaxForVectorCuda(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo) {

	// logic of this kernel is based on assumption gridDim = 1

	const auto x = reinterpret_cast<const T*>(vx);
		  auto z = reinterpret_cast<T*>(vz);

	__shared__ Nd4jLong  len;
	__shared__ int numOfIters;
	__shared__ T* shmem;

	if (threadIdx.x == 0) {
		extern __shared__ char shared[];
		shmem = reinterpret_cast<T*>(shared);
		len = shape::length(xShapeInfo);
		numOfIters = (len + blockDim.x - 1) / blockDim.x;   // ceil (len / blockDim.x)
	}
	__syncthreads();

	T temp = -DataTypeUtils::max<T>();	// set start value to compare with at first iteration, FIXME: what if T is unsigned ??

	// ************ evaluate max element in input array x ************ //
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong xOffset = shape::getIndexOffset(elemIdx, xShapeInfo);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? x[xOffset] : nd4j::math::nd4j_max<T>(x[xOffset], temp);	// take into account max element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = -DataTypeUtils::max<T>();	// FIXME: what if T is unsigned ??

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] = nd4j::math::nd4j_max<T>(shmem[threadIdx.x], shmem[threadIdx.x + s]);
			__syncthreads();
		}

		temp = shmem[0];	// save max value calculated at current iteration
	}

	const T max = temp;
	temp = 0;

	// ************ evaluate value of exp(x[offset] - max) per each element, store it to shared memory shmem ************ //
	// at the same evaluate sum of exponents, sum will be stored in shmem[0]
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong xOffset = shape::getIndexOffset(elemIdx, xShapeInfo);
			const Nd4jLong zOffset = shape::getIndexOffset(elemIdx, zShapeInfo);
			z[zOffset] = nd4j::math::nd4j_exp<T, T>(x[xOffset] - max);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? z[zOffset] : (z[zOffset] + temp); // take into account sum element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = 0;

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] += shmem[threadIdx.x + s];
			__syncthreads();
		}

		temp = shmem[0];	// save sum calculated at current iteration
	}

	// ************ evaluate z[offset] / sum  ************ //
	for (int i = 0; i < numOfIters; ++i) {
		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx >= len) continue;
		const Nd4jLong zOffset = shape::getIndexOffset(elemIdx, zShapeInfo);
		z[zOffset] /= shmem[0];
	}
}

template<typename T>
__global__ void softMaxForVectorCudaGlobal(const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo) {

	softMaxForVectorCuda<T>(vx, xShapeInfo, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
template <typename T>
linkage void softMaxForVectorCudaLauncher(const hipStream_t* stream, const void *vx, const Nd4jLong *xShapeInfo, void *vz, const Nd4jLong *zShapeInfo) {

	softMaxForVectorCudaGlobal<T><<<1, MAX_NUM_THREADS / 4 , (MAX_NUM_THREADS / 4) * sizeof(T) + 512, *stream>>>(vx, xShapeInfo, vz, zShapeInfo);
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void softMaxCuda(const void* vx, const Nd4jLong *xTadShapeInfo, const Nd4jLong *xOffsets,
                                         void* vz, const Nd4jLong *zTadShapeInfo, const Nd4jLong *zOffsets) {

    const auto x = reinterpret_cast<const T*>(vx);
          auto z = reinterpret_cast<T*>(vz);

    const auto* xTad = x + xOffsets[blockIdx.x];
          auto* zTad = z + zOffsets[blockIdx.x];

    softMaxForVectorCuda<T>(xTad, xTadShapeInfo, zTad, zTadShapeInfo);
}

///////////////////////////////////////////////////////////////////
template<typename T>
static void softMaxCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const void* vx, const Nd4jLong *xTadShapeInfo, const Nd4jLong *xOffsets,
                                	  void* vz, const Nd4jLong *zTadShapeInfo, const Nd4jLong *zOffsets) {

    softMaxCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vx, xTadShapeInfo, xOffsets, vz, zTadShapeInfo, zOffsets);
}


//////////////////////////////////////////////////////////////////////////
void softmax(nd4j::LaunchContext * context, const NDArray& input, NDArray& output, const int dimension) {

	if(!input.isActualOnDeviceSide()) input.syncToDevice();
	const int rank = input.rankOf();

	PointersManager manager(context, "helpers::softmax");

	if(input.isVector()) {

		if(rank == 1 || input.sizeAt(dimension) != 1) {
			NDArray::prepareSpecialUse({&output}, {&input});
			BUILD_SINGLE_SELECTOR(input.dataType(), softMaxForVectorCudaLauncher, (context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer(), output.getSpecialShapeInfo()), FLOAT_TYPES);
			NDArray::registerSpecialUse({&output}, {&input});
		}
		else
			output = 1.;
	}
	else {

		auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input.getShapeInfo(), {dimension});
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output.getShapeInfo(), {dimension});

        const int threadsPerBlock = MAX_NUM_THREADS / 4;
        const int blocksPerGrid = packZ.numberOfTads();
        const int sharedMem = input.sizeOfT() * threadsPerBlock + 512;

        NDArray::prepareSpecialUse({&output}, {&input});
    	BUILD_SINGLE_SELECTOR(input.dataType(), softMaxCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, context->getCudaStream(), input.getSpecialBuffer(), packX.specialShapeInfo(), packX.specialOffsets(), output.specialBuffer(), packZ.specialShapeInfo(), packZ.specialOffsets()), FLOAT_TYPES);
    	NDArray::registerSpecialUse({&output}, {&input});

		// auto maxAlongDim = const_cast<NDArray&>(input).reduceAlongDims(reduce::Max, {dimension}, true);
		// (input - maxAlongDim).applyTransform(transform::Exp, &output); // output contains exponents temporarily
		// auto sumAlongDim = output.reduceAlongDims(reduce::Sum, {dimension}, true);
		// output /= sumAlongDim;
		// input.tickReadDevice();
	}


	manager.synchronize();

	output.tickWriteDevice();
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__  void logSoftMaxForVectorCuda(const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	// logic of this kernel is based on assumption gridDim = 1

	const auto x = reinterpret_cast<const T*>(vx);
		  auto z = reinterpret_cast<T*>(vz);

	__shared__ Nd4jLong  len;
	__shared__ int numOfIters;
	__shared__ T* shmem;

	if (threadIdx.x == 0) {
		extern __shared__ char shared[];
		shmem = reinterpret_cast<T*>(shared);
		len = shape::length(xzShapeInfo);
		numOfIters = (len + blockDim.x - 1) / blockDim.x;   // ceil (len / blockDim.x)
	}
	__syncthreads();

	T temp = -DataTypeUtils::max<T>();	// set start value to compare with at first iteration, FIXME: what if T is unsigned ??

	// ************ evaluate max element in input array x ************ //
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? x[offset] : nd4j::math::nd4j_max<T>(x[offset], temp);	// take into account max element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = -DataTypeUtils::max<T>();	// FIXME: what if T is unsigned ??

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] = nd4j::math::nd4j_max<T>(shmem[threadIdx.x], shmem[threadIdx.x + s]);
			__syncthreads();
		}

		temp = shmem[0];	// save max value calculated at current iteration
	}

	const T max = temp;
	temp = 0;

	// ************ evaluate value of exp(x[offset] - max) per each element, store it to shared memory shmem ************ //
	// at the same time evaluate sum of exponents, sum will be stored in shmem[0]
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
			z[offset] = nd4j::math::nd4j_exp<T, T>(x[offset] - max);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? z[offset] : (z[offset] + temp); // take into account sum element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = 0;

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] += shmem[threadIdx.x + s];
			__syncthreads();
		}

		temp = shmem[0];	// save sum calculated at current iteration
	}

	// ************ evaluate log(z[offset] / sum)  ************ //
	for (int i = 0; i < numOfIters; ++i) {
		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx >= len) continue;
		const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
		z[offset] = nd4j::math::nd4j_log<T,T>(z[offset] / shmem[0]);
	}
}

///////////////////////////////////////////////////////////////////
template <typename T>
linkage void logSoftMaxForVectorCudaLauncher(const hipStream_t* stream, const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	logSoftMaxForVectorCuda<T><<<1, MAX_NUM_THREADS, MAX_NUM_THREADS * sizeof(T) + 512, *stream>>>(vx, xzShapeInfo, vz);
}

//////////////////////////////////////////////////////////////////////////
void logSoftmax(nd4j::LaunchContext * context, const NDArray& input, NDArray& output, const int dimension) {

	if(!input.isActualOnDeviceSide()) input.syncToDevice();
	const int rank = input.rankOf();

	if(input.isVector()) {

		if(rank == 1 || input.sizeAt(dimension) != 1) {
			BUILD_SINGLE_SELECTOR(input.dataType(), logSoftMaxForVectorCudaLauncher, (context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer()), FLOAT_TYPES);
			input.tickReadDevice();
		}
		else
			output = 0.;
	}
	else {

		auto maxAlongDim = const_cast<NDArray&>(input).reduceAlongDims(reduce::Max, {dimension}, true);
		(input - maxAlongDim).applyTransform(transform::Exp, &output); // output contains exponents temporarily
		auto sumAlongDim = output.reduceAlongDims(reduce::Sum, {dimension}, true);
		output /= sumAlongDim;
		output.applyTransform(transform::Log);
		input.tickReadDevice();
	}

	PointersManager manager(context, "helpers::logSoftmax");
	manager.synchronize();

	output.tickWriteDevice();
}

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ linkage void softMaxDerivForVectorCuda(const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	// logic of this kernel is based on assumption gridDim = 1

	const auto x = reinterpret_cast<const T*>(vx);
		  auto z = reinterpret_cast<T*>(vz);

	__shared__ Nd4jLong  len;
	__shared__ int numOfIters;
	__shared__ T* shmem;

	if (threadIdx.x == 0) {
		extern __shared__ char shared[];
		shmem = reinterpret_cast<T*>(shared);
		len = shape::length(xzShapeInfo);
		numOfIters = (len + blockDim.x - 1) / blockDim.x;   // ceil (len / blockDim.x)
	}
	__syncthreads();

	T temp = -DataTypeUtils::max<T>();	// set start value to compare with at first iteration, FIXME: what if T is unsigned ??

	// ************ evaluate max element in input array x ************ //
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? x[offset] : nd4j::math::nd4j_max<T>(x[offset], temp);	// take into account max element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = -DataTypeUtils::max<T>();	// FIXME: what if T is unsigned ??

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] = nd4j::math::nd4j_max<T>(shmem[threadIdx.x], shmem[threadIdx.x + s]);
			__syncthreads();
		}

		temp = shmem[0];	// save max value calculated at current iteration
	}

	const T max = temp;
	temp = 0;

	// ************ evaluate value of exp(x[offset] - max) per each element, store it to shared memory shmem ************ //
	// at the same evaluate sum of exponents, sum will be stored in shmem[0]
	for (int i = 0; i < numOfIters; ++i) {

		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx < len) {
			const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
			z[offset] = nd4j::math::nd4j_exp<T, T>(x[offset] - max);
			shmem[threadIdx.x] = (threadIdx.x != 0) ? z[offset] : (z[offset] + temp); // take into account sum element evaluated on previous iteration and stored in temp
		}
		else
			shmem[threadIdx.x] = 0;

		__syncthreads();

		for (int s = blockDim.x / 2; s > 0; s /= 2) {
			if(threadIdx.x < s)
				shmem[threadIdx.x] += shmem[threadIdx.x + s];
			__syncthreads();
		}

		temp = shmem[0];	// save sum calculated at current iteration
	}

	// ************ evaluate (z[offset] / sum) and derivative z[offset] = z[offset] * (1 - z[offset]) ************ //
	for (int i = 0; i < numOfIters; ++i) {
		const Nd4jLong elemIdx = i * blockDim.x + threadIdx.x;
		if(elemIdx >= len) continue;
		const Nd4jLong offset = shape::getIndexOffset(elemIdx, xzShapeInfo);
		z[offset] /= shmem[0];
		z[offset] *= (1.f - z[offset]);		// derivative
	}
}

///////////////////////////////////////////////////////////////////
template <typename T>
linkage void softMaxDerivForVectorCudaLauncher(const hipStream_t* stream, const void *vx, const Nd4jLong *xzShapeInfo, void *vz) {

	softMaxDerivForVectorCuda<T><<<1, MAX_NUM_THREADS, MAX_NUM_THREADS * sizeof(T) + 512, *stream>>>(vx, xzShapeInfo, vz);
}

///////////////////////////////////////////////////////////////////
void softmaxDerivative(nd4j::LaunchContext * context, const NDArray& input, NDArray& output, const int dimension) {

	if(!input.isActualOnDeviceSide()) input.syncToDevice();
	const int rank = input.rankOf();
	int temp;

	if(shape::isCommonVector(input.getShapeInfo(), temp)) {

		BUILD_SINGLE_SELECTOR(input.dataType(), softMaxDerivForVectorCudaLauncher, (context->getCudaStream(), input.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialBuffer()), FLOAT_TYPES);
		input.tickReadDevice();
	}
	else {

		auto maxAlongDim = const_cast<NDArray&>(input).reduceAlongDims(reduce::Max, {dimension}, true);
		(input - maxAlongDim).applyTransform(transform::Exp, &output); // output contains exponents temporarily
		auto sumAlongDim = output.reduceAlongDims(reduce::Sum, {dimension}, true);
		output /= sumAlongDim;
		output *= (1.f - output);	// derivative
		input.tickReadDevice();
	}

	PointersManager manager(context, "helpers::softmaxDerivative");
	manager.synchronize();

	output.tickWriteDevice();
}


	template <typename T>
	linkage void thresholdRelu_(NDArray const& input, double threshold, NDArray& output) {
		auto routine = LAMBDA_T(_x, threshold) {
			return _x > (T)threshold ? _x: (T)0.f;
		};
		const_cast<NDArray&>(input).applyLambda(routine, &output);
	}

	void thresholdRelu(nd4j::LaunchContext * context, NDArray const& input, double threshold, NDArray& output) {
		BUILD_SINGLE_SELECTOR(input.dataType(), thresholdRelu_, (input, threshold, output), FLOAT_TYPES);
	}

	template <typename T>
	linkage void thresholdReluDerivative_(NDArray* input, double theta, NDArray* dLdO, NDArray* output) {
        auto derivative = LAMBDA_TT(_x, grO, theta) {if (_x > theta) return grO; else return static_cast<T>(0); };

        input->applyPairwiseLambda(dLdO, derivative, output);
	}

	void thresholdReluDerivative(nd4j::LaunchContext * context, NDArray* input, double threshold, NDArray* dLdO, NDArray* output) {
		BUILD_SINGLE_SELECTOR(input->dataType(), thresholdReluDerivative_, (input, threshold, dLdO, output), FLOAT_TYPES);
	}

}
}
}

