#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <svd.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ShapeUtils.h>

namespace nd4j    {
namespace ops     {
namespace helpers {


// FIXME -> we should optimize these helpers for the case when input matrices have c order (perform transpositions appropriately)

template <typename T>
__global__ static void inverseColumnSignCuda(void* vu, const Nd4jLong* uShapeInfo, void* vv, const Nd4jLong* vShapeInfo) {

    T* u = reinterpret_cast<T*>(vu);
    T* v = reinterpret_cast<T*>(vv);

    __shared__ int rank, uLastButOneColumn, vLastButOneColumn;    // uRank = vRank
    __shared__ Nd4jLong uLen, vLen;
    __shared__ Nd4jLong *sharedMem;

    if (threadIdx.x == 0) {

        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<Nd4jLong*>(shmem);

        rank = shape::rank(uShapeInfo);
        uLen = shape::length(uShapeInfo);
        vLen = shape::length(vShapeInfo);

        uLastButOneColumn = uShapeInfo[rank]     - 2;
        vLastButOneColumn = vShapeInfo[rank - 1] - 2;
    }

    __syncthreads();

    const auto ind = threadIdx.x + blockIdx.x * blockDim.x;

    auto coords = sharedMem + threadIdx.x * rank;

    // u
    for (Nd4jLong i = ind; i < uLen; i += gridDim.x * blockDim.x) {

        shape::index2coords(i, uShapeInfo, coords);

        if(coords[rank - 1] == 0 || coords[rank - 1] == uLastButOneColumn)   // do not change sign in first and last but one columns
            continue;

        const auto uOffset = shape::getOffset(uShapeInfo, coords);

        u[uOffset] = -u[uOffset];
    }

    // v
    for (Nd4jLong i = ind; i < vLen; i += gridDim.x * blockDim.x) {

        shape::index2coords(i, vShapeInfo, coords);

        if(coords[rank - 2] == 0 || coords[rank - 2] == vLastButOneColumn)   // do not change sign in first and last but one columns
            continue;

        const auto vOffset = shape::getOffset(vShapeInfo, coords);

        v[vOffset] = -v[vOffset];
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void inverseColumnSignCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                         void* vu, const Nd4jLong* uShapeInfo,
                                         void* vv, const Nd4jLong* vShapeInfo) {

    inverseColumnSignCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(vu, uShapeInfo, vv, vShapeInfo);
}
BUILD_SINGLE_TEMPLATE(template void inverseColumnSignCudaLauncher, (const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t* stream, void* vu, const Nd4jLong* uShapeInfo, void* vv, const Nd4jLong* vShapeInfo), FLOAT_TYPES);

//////////////////////////////////////////////////////////////////////////
static void svdQR(nd4j::LaunchContext* context, const NDArray* A, NDArray* S, NDArray* U, NDArray* VT, const bool fullUV, const bool calcUV) {

    // since cusa api hipsolverDnDgesvd/hipsolverDnSgesvd have following constrain on input matrix A: A_rows >= A_columns && A_order = 'f'
    // we make this function to have deal with 2 valid cases only:
    // 1) A_rows >= A_columns and A_corder = 'f'
    // 2) A_rows <= A_columns and A_corder = 'c'    - int this case perform transposition to get f order
    // if 1) or 2) are not met then throw exception

    // A  [m, n]
    // S  [n]
    // U  [m, m] or [m, n] if fullUV = false and m > n
    // VT [n, n] or [m, n] if fullUV = false and m < n

    if(A->rankOf() != 2)
        throw std::runtime_error("svdQR: rank of A array is not equal 2 !");

    auto m = A->sizeAt(0);
    auto n = A->sizeAt(1);
    const int minDim = m < n ? m : n;
    const char orderA = A->ordering();

    if(m < n)
        throw std::runtime_error("svdQR: due to cuda api input constrains given shape of A array are not valid !");

    if(ShapeUtils::shapeAsString({minDim}) != ShapeUtils::shapeAsString(S))
        throw std::runtime_error("svdQR: wrong shape of S array !");

    if(calcUV) {

        if(fullUV && ShapeUtils::shapeAsString({m,m}) != ShapeUtils::shapeAsString(U))
            throw std::runtime_error("svdQR: wrong shape of U array !");
        else if(!fullUV && ShapeUtils::shapeAsString({m,minDim}) != ShapeUtils::shapeAsString(U))
            throw std::runtime_error("svdQR: wrong shape of U array !");

        if(fullUV && ShapeUtils::shapeAsString({n,n}) != ShapeUtils::shapeAsString(VT))
            throw std::runtime_error("svdQR: wrong shape of VT array !");
        else if(!fullUV && ShapeUtils::shapeAsString({minDim,n}) != ShapeUtils::shapeAsString(VT))
            throw std::runtime_error("svdQR: wrong shape of VT array !");
    }

    NDArray* pA  = const_cast<NDArray*>(A);
    NDArray* pS  = S;
    NDArray* pU  = U;
    NDArray* pVT = VT;

    std::vector<NDArray*> toDelete;

    if(pA->ews() != 1 || pA->ordering() == 'c') {
        pA = A->dup('f');
        toDelete.push_back(pA);
    }

    if(S->ews() != 1) {
        pS = S->dup('f');
        toDelete.push_back(pS);
    }

    if(calcUV) {

        if(pU->ews() != 1 || pU->ordering() == 'c') {
            pU = U->dup('f');
            toDelete.push_back(pU);
        }

        if(pVT->ews() != 1 || pVT->ordering() == 'c') {
            pVT = VT->dup('f');
            toDelete.push_back(pVT);
        }
    }

    // create cusolverDn handle
    hipsolverHandle_t handle = nullptr;
    hipsolverStatus_t status = hipsolverDnCreate(&handle);
    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdQR: cuda failed !", status);

    // stream
    status = hipsolverSetStream(handle, *context->getCudaStream());
    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdQR: cuda failed !", status);

    // query working space of SVD
    int lwork = 0;
    if(A->dataType() == DataType::DOUBLE)
        status = hipsolverDnDgesvd_bufferSize(handle, m, n, &lwork);
    else if(A->dataType() == DataType::FLOAT32)
        status = hipsolverDnSgesvd_bufferSize(handle, m, n, &lwork);
    else
        throw std::invalid_argument("svdQR: given data type is unsupported !");

    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdQR: cuda failed !", status);

    // allocate memory for dWork
    void* dWork = nullptr;
    hipError_t status2 = hipMalloc((void**)&dWork , A->sizeOfT() * lwork);
    if(status2 != hipSuccess)
        throw cuda_exception::build("svdQR: cuda failed !", status2);

    signed char jobu, jobvt;

    if(calcUV) {
        if(fullUV)
            jobu = jobvt = 'A';
        else
            jobu = jobvt = 'S';
    }
    else {
        jobu = jobvt = 'N';
    }

    int *devInfo = nullptr;
    void* rWork = nullptr;

    int lda(m), ldu, ldvt;

    if(calcUV) {
        ldu  = pU->sizeAt(0);
        ldvt = pVT->sizeAt(0);
    }

    PointersManager manager(context, "svdQR");

    NDArray::prepareSpecialUse({pS, pU, pVT}, {pA});

    // choose appropriate cuda gemm api depending on data types
    if(A->dataType() == DataType::DOUBLE) {
        status = hipsolverDnDgesvd(handle, jobu, jobvt, m, n, reinterpret_cast<double*>(pA->getSpecialBuffer()), lda, reinterpret_cast<double*>(pS->getSpecialBuffer()), calcUV ? reinterpret_cast<double*>(pU->getSpecialBuffer()) : nullptr, ldu, calcUV ? reinterpret_cast<double*>(pVT->getSpecialBuffer()) : nullptr, ldvt, reinterpret_cast<double*>(dWork), lwork, reinterpret_cast<double*>(rWork), devInfo);
    }
    else if(A->dataType() == DataType::FLOAT32) {
        status = hipsolverDnSgesvd(handle, jobu, jobvt, m, n, reinterpret_cast<float*>(pA->getSpecialBuffer()), lda, reinterpret_cast<float*>(pS->getSpecialBuffer()), calcUV ? reinterpret_cast<float*>(pU->getSpecialBuffer()) : nullptr, ldu, calcUV ? reinterpret_cast<float*>(pVT->getSpecialBuffer()) : nullptr, ldvt, reinterpret_cast<float*>(dWork), lwork, reinterpret_cast<float*>(rWork), devInfo);
    }
    else
        throw std::invalid_argument("svdQR: given data type is unsupported !");

    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdQR: cuda failed !", status);

    manager.synchronize();

    NDArray::registerSpecialUse({pS, pU, pVT}, {pA});

    S->assign(pS);

    if(calcUV) {
        U->assign(pU);
        VT->assign(pVT);
    }

    for (int i = toDelete.size() - 1; i >= 0; --i)
        delete toDelete[i];

    if (devInfo)
        hipFree(devInfo);
    if (dWork )
        hipFree(dWork);
    if (rWork)
        hipFree(rWork);

    if(handle)
        hipsolverDnDestroy(handle);

    // hipDeviceReset();
}

//////////////////////////////////////////////////////////////////////////
static void svdJcb(nd4j::LaunchContext* context, const NDArray* A, NDArray* S, NDArray* U, NDArray* V, const bool fullUV, const bool calcUV) {

    // A [m, n]
    // S [n]
    // U [m, m] or [m, n] if fullUV = false and m > n
    // V [n, n] or [n, m] if fullUV = false and m < n

    if(A->rankOf() != 2)
        throw std::runtime_error("svdJcb: rank of A array is not equal 2 !");

    auto m = A->sizeAt(0);
    auto n = A->sizeAt(1);
    const int minDim = m < n ? m : n;

    if(ShapeUtils::shapeAsString({minDim}) != ShapeUtils::shapeAsString(S))
        throw std::runtime_error("svdJcb: wrong shape of S array !");

    if(calcUV) {

        if(fullUV && ShapeUtils::shapeAsString({m,m}) != ShapeUtils::shapeAsString(U))
            throw std::runtime_error("svdJcb: wrong shape of U array !");
        else if(!fullUV && ShapeUtils::shapeAsString({m,minDim}) != ShapeUtils::shapeAsString(U))
            throw std::runtime_error("svdJcb: wrong shape of U array !");

        if(fullUV && ShapeUtils::shapeAsString({n,n}) != ShapeUtils::shapeAsString(V))
            throw std::runtime_error("svdJcb: wrong shape of V array !");
        else if(!fullUV && ShapeUtils::shapeAsString({n,minDim}) != ShapeUtils::shapeAsString(V))
            throw std::runtime_error("svdJcb: wrong shape of V array !");
    }

    NDArray* pA = const_cast<NDArray*>(A);
    NDArray* pS = S;
    NDArray* pU = U;
    NDArray* pV = V;

    std::vector<NDArray*> toDelete;

    if(pA->ews() != 1 || pA->ordering() == 'c') {
        pA = A->dup('f');
        toDelete.push_back(pA);
    }

    if(S->ews() != 1) {
        pS = S->dup('f');
        toDelete.push_back(pS);
    }

    if(calcUV) {

        if(pU->ews() != 1 || pU->ordering() == 'c') {
            pU = U->dup('f');
            toDelete.push_back(pU);
        }

        if(pV->ews() != 1 || pV->ordering() == 'c') {
            pV = V->dup('f');
            toDelete.push_back(pV);
        }
    }

    // create cusolverDn handle
    hipsolverHandle_t handle = nullptr;
    hipsolverStatus_t status = hipsolverDnCreate(&handle);
    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdJcb: cuda failed !", status);

    // stream
    status = hipsolverSetStream(handle, *context->getCudaStream());
    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdJcb: cuda failed !", status);

    // set parameters
    hipsolverGesvdjInfo_t gesvdjParams = nullptr;
    status = hipsolverDnCreateGesvdjInfo(&gesvdjParams);
    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdJcb: cuda failed !", status);
     status = hipsolverDnXgesvdjSetTolerance(gesvdjParams, 1.e-7);   // tolerance
     if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdJcb: cuda failed !", status);
    status = hipsolverDnXgesvdjSetMaxSweeps(gesvdjParams, 15);      // max_sweeps
    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdJcb: cuda failed !", status);

    int *devInfo = nullptr;
    const hipsolverEigMode_t jobz = calcUV ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;
    const int econ = !fullUV;

    int lda(m), ldu(m), ldv(m);

    if(calcUV) {
        ldu = pU->sizeAt(0);
        ldv = pV->sizeAt(0);
    }

    // query working space of SVD
    int lwork = 0;
    if(A->dataType() == DataType::DOUBLE)
        status = hipsolverDnDgesvdj_bufferSize(handle, jobz, econ, m, n, reinterpret_cast<double*>(pA->getSpecialBuffer()), lda, reinterpret_cast<double*>(pS->getSpecialBuffer()), calcUV ? reinterpret_cast<double*>(pU->getSpecialBuffer()) : nullptr, ldu, calcUV ? reinterpret_cast<double*>(pV->getSpecialBuffer()) : nullptr, ldv, &lwork, gesvdjParams);
    else if(A->dataType() == DataType::FLOAT32)
        status = hipsolverDnSgesvdj_bufferSize(handle, jobz, econ, m, n, reinterpret_cast<float*>(pA->getSpecialBuffer()), lda, reinterpret_cast<float*>(pS->getSpecialBuffer()), calcUV ? reinterpret_cast<float*>(pU->getSpecialBuffer()) : nullptr, ldu, calcUV ? reinterpret_cast<float*>(pV->getSpecialBuffer()) : nullptr, ldv, &lwork, gesvdjParams);
    else
        throw std::invalid_argument("svdJcb: given data type is unsupported !");

    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdJcb: cuda failed !", status);

    // allocate memory dWork
    void* dWork = nullptr;
    auto status2 = hipMalloc((void**)&dWork , A->sizeOfT() * lwork);
    if(status2 != hipSuccess)
        throw cuda_exception::build("svdJcb: cuda failed !", status2);

    PointersManager manager(context, "svdJcb");

    NDArray::prepareSpecialUse({pS, pU, pV}, {pA});

    // choose appropriate cuda gemm api depending on data types
    if(A->dataType() == DataType::DOUBLE) {
        status = hipsolverDnDgesvdj(handle, jobz, econ, m, n, reinterpret_cast<double*>(pA->getSpecialBuffer()), lda, reinterpret_cast<double*>(pS->getSpecialBuffer()), calcUV ? reinterpret_cast<double*>(pU->getSpecialBuffer()) : nullptr, ldu, calcUV ? reinterpret_cast<double*>(pV->getSpecialBuffer()) : nullptr, ldv, reinterpret_cast<double*>(dWork), lwork, devInfo, gesvdjParams);
    }
    else if(A->dataType() == DataType::FLOAT32) {
        status = hipsolverDnSgesvdj(handle, jobz, econ, m, n, reinterpret_cast<float*>(pA->getSpecialBuffer()), lda, reinterpret_cast<float*>(pS->getSpecialBuffer()), calcUV ? reinterpret_cast<float*>(pU->getSpecialBuffer()) : nullptr, ldu, calcUV ? reinterpret_cast<float*>(pV->getSpecialBuffer()) : nullptr, ldv, reinterpret_cast<float*>(dWork), lwork, devInfo, gesvdjParams);
    }
    else
        throw std::invalid_argument("svdJcb: given data type is unsupported !");

    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdJcb: cuda failed !", status);

    manager.synchronize();

    NDArray::registerSpecialUse({pS, pU, pV}, {pA});

    S->assign(pS);

    if(calcUV) {
        U->assign(pU);
        V->assign(pV);
    }

    for (int i = toDelete.size() - 1; i >= 0; --i)
        delete toDelete[i];

    if (devInfo)
        hipFree(devInfo);
    if (dWork )
        hipFree(dWork);
    if(handle)
        hipsolverDnDestroy(handle);
    if(gesvdjParams)
        hipsolverDnDestroyGesvdjInfo(gesvdjParams);

    // hipDeviceReset();
}

//////////////////////////////////////////////////////////////////////////
static void svdBatched(nd4j::LaunchContext* context, const NDArray* A, NDArray* S, NDArray* U, NDArray* V, const bool fullUV, const bool calcUV) {

    // A [..., m, n]
    // S [..., n]
    // U [..., m, m] or [..., m, n] if fullUV = false and m > n
    // V [..., n, n] or [..., n, m] if fullUV = false and m < n

    auto m = A->sizeAt(-2);
    auto n = A->sizeAt(-1);
    const int minDim = m < n ? m : n;
    const Nd4jLong bS = A->lengthOf() / (m * n);

    if(m > 32 || n > 32)
        throw std::runtime_error("svdBatched: numbers of rows and columns should be <= 32 !");

    if(minDim != S->sizeAt(-1))
        throw std::runtime_error("svdBatched: wrong shape of S array !");

    if(calcUV) {

        if(U->sizeAt(-2) != m)
            throw std::runtime_error("svdBatched: wrong shape of U array !");
        if(U->sizeAt(-1) != (fullUV ? m : minDim))
            throw std::runtime_error("svdBatched: wrong shape of U array !");
        if(U->lengthOf() / (U->sizeAt(-2) * U->sizeAt(-1)) != bS)
            throw std::runtime_error("svdBatched: wrong shape of U array !");

        if(V->sizeAt(-2) != n)
              throw std::runtime_error("svdBatched: wrong shape of V array !");
        if(V->sizeAt(-1) != (fullUV ? n : minDim))
            throw std::runtime_error("svdBatched: wrong shape of V array !");
        if(V->lengthOf() / (V->sizeAt(-2) * V->sizeAt(-1)) != bS)
            throw std::runtime_error("svdBatched: wrong shape of V array !");
    }

    NDArray* pA = const_cast<NDArray*>(A);
    NDArray* pS = S;
    NDArray* pU = U;
    NDArray* pV = V;

    std::vector<NDArray*> toDelete;

    if(pA->ews() != 1 || pA->ordering() == 'c') {
        pA = A->dup('f');
        toDelete.push_back(pA);
    }

    if(S->ews() != 1) {
        pS = S->dup('f');
        toDelete.push_back(pS);
    }

    if(calcUV) {

        if(pU->ews() != 1 || pU->ordering() == 'c') {
            pU = U->dup('f');
            toDelete.push_back(pU);
        }

        if(pV->ews() != 1 || pV->ordering() == 'c') {
            pV = V->dup('f');
            toDelete.push_back(pV);
        }
    }

    // create cusolverDn handle
    hipsolverHandle_t handle = nullptr;
    hipsolverStatus_t status = hipsolverDnCreate(&handle);
    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdBatched: cuda failed !", status);

    // stream
    status = hipsolverSetStream(handle, *context->getCudaStream());
    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdBatched: cuda failed !", status);

    // set parameters
    hipsolverGesvdjInfo_t gesvdjParams = nullptr;
    status = hipsolverDnCreateGesvdjInfo(&gesvdjParams);
    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdBatched: cuda failed !", status);
     status = hipsolverDnXgesvdjSetTolerance(gesvdjParams, 1.e-7);   // tolerance
     if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdBatched: cuda failed !", status);
    status = hipsolverDnXgesvdjSetMaxSweeps(gesvdjParams, 15);      // max_sweeps
    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdBatched: cuda failed !", status);

    // devInfo
    int *devInfo = nullptr;
    auto status2 = hipMalloc((void**)&devInfo, sizeof(int) * bS);
    if(status2 != hipSuccess)
        throw cuda_exception::build("svdBatched: cuda failed !", status2);
    status2 = hipDeviceSynchronize();
    if(status2 != hipSuccess)
        throw cuda_exception::build("svdJcb: cuda failed !", status2);

    const hipsolverEigMode_t jobz = calcUV ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;

    int lda(m), ldu, ldv;

    if(calcUV) {
        ldu = pU->sizeAt(-2);
        ldv = pV->sizeAt(-2);
    }

    // Ak (i,j) = A[i + 5*j + 25*k]

    // query working space of SVD
    int lwork = 0;
    if(A->dataType() == DataType::DOUBLE)
        status = hipsolverDnDgesvdjBatched_bufferSize(handle, jobz, m, n, reinterpret_cast<double*>(pA->getSpecialBuffer()), lda, reinterpret_cast<double*>(pS->getSpecialBuffer()), calcUV ? reinterpret_cast<double*>(pU->getSpecialBuffer()) : nullptr, ldu, calcUV ? reinterpret_cast<double*>(pV->getSpecialBuffer()) : nullptr, ldv, &lwork, gesvdjParams, bS);
    else if(A->dataType() == DataType::FLOAT32)
        status = hipsolverDnSgesvdjBatched_bufferSize(handle, jobz, m, n, reinterpret_cast<float*>(pA->getSpecialBuffer()), lda, reinterpret_cast<float*>(pS->getSpecialBuffer()), calcUV ? reinterpret_cast<float*>(pU->getSpecialBuffer()) : nullptr, ldu, calcUV ? reinterpret_cast<float*>(pV->getSpecialBuffer()) : nullptr, ldv, &lwork, gesvdjParams, bS);
    else
        throw std::invalid_argument("svdBatched: given data type is unsupported !");

    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdBatched: cuda failed !", status);

    // allocate memory dWork
    void* dWork = nullptr;
    status2 = hipMalloc((void**)&dWork , A->sizeOfT() * lwork);
    if(status2 != hipSuccess)
        throw cuda_exception::build("svdBatched: cuda failed !", status2);
    status2 = hipDeviceSynchronize();
    if(status2 != hipSuccess)
        throw cuda_exception::build("svdBatched: cuda failed !", status2);

    PointersManager manager(context, "svdBatched");

    NDArray::prepareSpecialUse({pS, pU, pV}, {pA});

    // choose appropriate cuda gemm api depending on data types
    if(A->dataType() == DataType::DOUBLE) {
        status = hipsolverDnDgesvdjBatched(handle, jobz, m, n, reinterpret_cast<double*>(pA->getSpecialBuffer()), lda, reinterpret_cast<double*>(pS->getSpecialBuffer()), calcUV ? reinterpret_cast<double*>(pU->getSpecialBuffer()) : nullptr, ldu, calcUV ? reinterpret_cast<double*>(pV->getSpecialBuffer()) : nullptr, ldv, reinterpret_cast<double*>(dWork), lwork, devInfo, gesvdjParams, bS);
    }
    else if(A->dataType() == DataType::FLOAT32) {
        status = hipsolverDnSgesvdjBatched(handle, jobz, m, n, reinterpret_cast<float*>(pA->getSpecialBuffer()), lda, reinterpret_cast<float*>(pS->getSpecialBuffer()), calcUV ? reinterpret_cast<float*>(pU->getSpecialBuffer()) : nullptr, ldu, calcUV ? reinterpret_cast<float*>(pV->getSpecialBuffer()) : nullptr, ldv, reinterpret_cast<float*>(dWork), lwork, devInfo, gesvdjParams, bS);
    }
    else
        throw std::invalid_argument("svdBatched: given data type is unsupported !");

    if(status != HIPSOLVER_STATUS_SUCCESS)
        throw cuda_exception::build("svdBatched: cuda failed !", status);

    manager.synchronize();

    NDArray::registerSpecialUse({pS, pU, pV}, {pA});

    S->assign(pS);

    if(calcUV) {
        U->assign(pU);
        V->assign(pV);
    }

    for (int i = toDelete.size() - 1; i >= 0; --i)
        delete toDelete[i];

    if (devInfo)
        hipFree(devInfo);
    if (dWork )
        hipFree(dWork);
    if(handle)
        hipsolverDnDestroy(handle);
    if(gesvdjParams)
        hipsolverDnDestroyGesvdjInfo(gesvdjParams);

    // hipDeviceReset();
}

////////////////////////////////////////////////////////////////////
void svd(nd4j::LaunchContext* context, const NDArray* x, const std::vector<NDArray*>& outArrs, const bool fullUV, const bool calcUV, const int switchNum) {

    NDArray* S = outArrs[0];
    NDArray* U = outArrs[1];
    // NDArray VT = outArrs[2]->transpose();
    NDArray* V = outArrs[2];

    NDArray::prepareSpecialUse({S, U, V}, {x});

    if(x->rankOf() == 2) {
        // svdQR(context, x, S, U, VT, fullUV, calcUV);
        svdJcb(context, x, S, U, V, fullUV, calcUV);
    }
    else {

        // svdBatched(context, *x, *S, *U, *V, fullUV, calcUV);

        ResultSet *tadsU(nullptr), *tadsV(nullptr);

        auto tadsX = x->allTensorsAlongDimension({x->rankOf() - 2, x->rankOf() - 1});
        auto tadsS = S->allTensorsAlongDimension({S->rankOf() - 1});

        if(calcUV) {
            tadsU = U->allTensorsAlongDimension({U->rankOf() - 2, U->rankOf() - 1});
            tadsV = V->allTensorsAlongDimension({V->rankOf() - 2, V->rankOf() - 1});
        }

        for (int i = 0; i < tadsX->size(); ++i)
            svdJcb(context, tadsX->at(i), tadsS->at(i), calcUV ? tadsU->at(i) : nullptr, calcUV ? tadsV->at(i) : nullptr, fullUV, calcUV);

        delete tadsX;
        delete tadsS;

        if(calcUV) {
            delete tadsU;
            delete tadsV;
        }
    }

    NDArray::registerSpecialUse({S, U, V}, {x});
}


}
}
}